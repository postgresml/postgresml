#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Simple example of DeviceSelect::Unique().
 *
 * Selects the first element from each run of identical values from a sequence
 * of int keys.
 *
 * To compile using the command line:
 *   nvcc -arch=sm_XX example_device_select_unique.cu -I../.. -lcudart -O3
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>

#include <cub/util_allocator.cuh>
#include <cub/device/device_select.cuh>

#include "../../test/test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose = false;  // Whether to display input/output to console
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory


//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------


/**
 * Initialize problem, setting runs of random length chosen from [1..max_segment]
 */
void Initialize(
    int     *h_in,
    int     num_items,
    int     max_segment)
{
    int key = 0;
    int i = 0;
    while (i < num_items)
    {
        // Randomly select number of repeating occurrences uniformly from [1..max_segment]
        unsigned short max_short = (unsigned short) -1;
        unsigned short repeat;
        RandomBits(repeat);
        repeat = (unsigned short) ((float(repeat) * (float(max_segment) / float(max_short))));
        repeat = CUB_MAX(1, repeat);

        int j = i;
        while (j < CUB_MIN(i + repeat, num_items))
        {
            h_in[j] = key;
            j++;
        }

        i = j;
        key++;
    }

    if (g_verbose)
    {
        printf("Input:\n");
        DisplayResults(h_in, num_items);
        printf("\n\n");
    }
}


/**
 * Solve unique problem
 */
int Solve(
    int         *h_in,
    int         *h_reference,
    int         num_items)
{
    int num_selected = 0;
    if (num_items > 0)
    {
        h_reference[num_selected] = h_in[0];
        num_selected++;
    }

    for (int i = 1; i < num_items; ++i)
    {
        if (h_in[i] != h_in[i - 1])
        {
            h_reference[num_selected] = h_in[i];
            num_selected++;
        }
    }

    return num_selected;
}


//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    int num_items           = 150;
    int max_segment         = 40;       // Maximum segment length

    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("n", num_items);
    args.GetCmdLineArgument("maxseg", max_segment);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--n=<input items> "
            "[--device=<device-id>] "
            "[--maxseg=<max segment length>]"
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Allocate host arrays
    int*  h_in        = new int[num_items];
    int*  h_reference = new int[num_items];

    // Initialize problem and solution
    Initialize(h_in, num_items, max_segment);
    int num_selected = Solve(h_in, h_reference, num_items);

    printf("hipcub::DeviceSelect::Unique %d items (%d-byte elements), %d selected (avg run length %d)\n",
        num_items, (int) sizeof(int), num_selected, num_items / num_selected);
    fflush(stdout);

    // Allocate problem device arrays
    int *d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(int) * num_items));

    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(int) * num_items, hipMemcpyHostToDevice));

    // Allocate device output array and num selected
    int     *d_out            = NULL;
    int     *d_num_selected_out   = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(int) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_num_selected_out, sizeof(int)));

    // Allocate temporary storage
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;
    CubDebugExit(DeviceSelect::Unique(d_temp_storage, temp_storage_bytes, d_in, d_out, d_num_selected_out, num_items));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Run
    CubDebugExit(DeviceSelect::Unique(d_temp_storage, temp_storage_bytes, d_in, d_out, d_num_selected_out, num_items));

    // Check for correctness (and display results, if specified)
    int compare = CompareDeviceResults(h_reference, d_out, num_selected, true, g_verbose);
    printf("\t Data %s ", compare ? "FAIL" : "PASS");
    compare = compare | CompareDeviceResults(&num_selected, d_num_selected_out, 1, true, g_verbose);
    printf("\t Count %s ", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_reference) delete[] h_reference;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_num_selected_out) CubDebugExit(g_allocator.DeviceFree(d_num_selected_out));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

    printf("\n\n");

    return 0;
}



