#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Simple demonstration of hipcub::BlockReduce
 *
 * To compile using the command line:
 *   nvcc -arch=sm_XX example_block_reduce.cu -I../.. -lcudart -O3
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console (define before including cub.h)
#define HIPCUB_STDERR

#include <stdio.h>
#include <iostream>

#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_reduce.cuh>

#include "../../test/test_util.h"

using namespace hipcub;

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

/// Verbose output
bool g_verbose = false;

/// Timing iterations
int g_timing_iterations = 100;

/// Default grid size
int g_grid_size = 1;



//---------------------------------------------------------------------
// Kernels
//---------------------------------------------------------------------

/**
 * Simple kernel for performing a block-wide exclusive prefix sum over integers
 */
template <
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    BlockReduceAlgorithm    ALGORITHM>
__global__ void BlockSumKernel(
    int         *d_in,          // Tile of input
    int         *d_out,         // Tile aggregate
    clock_t     *d_elapsed)     // Elapsed cycle count of block reduction
{
    // Specialize BlockReduce type for our thread block
    typedef BlockReduce<int, BLOCK_THREADS, ALGORITHM> BlockReduceT;

    // Shared memory
    __shared__ typename BlockReduceT::TempStorage temp_storage;

    // Per-thread tile data
    int data[ITEMS_PER_THREAD];
    LoadDirectStriped<BLOCK_THREADS>(threadIdx.x, d_in, data);

    // Start cycle timer
    clock_t start = clock();

    // Compute sum
    int aggregate = BlockReduceT(temp_storage).Sum(data);

    // Stop cycle timer
    clock_t stop = clock();

    // Store aggregate and elapsed clocks
    if (threadIdx.x == 0)
    {
        *d_elapsed = (start > stop) ? start - stop : stop - start;
        *d_out = aggregate;
    }
}



//---------------------------------------------------------------------
// Host utilities
//---------------------------------------------------------------------

/**
 * Initialize reduction problem (and solution).
 * Returns the aggregate
 */
int Initialize(int *h_in, int num_items)
{
    int inclusive = 0;

    for (int i = 0; i < num_items; ++i)
    {
        h_in[i] = i % 17;
        inclusive += h_in[i];
    }

    return inclusive;
}


/**
 * Test thread block reduction
 */
template <
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    BlockReduceAlgorithm    ALGORITHM>
void Test()
{
    const int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

    // Allocate host arrays
    int *h_in           = new int[TILE_SIZE];
    int *h_gpu          = new int[TILE_SIZE + 1];

    // Initialize problem and reference output on host
    int h_aggregate = Initialize(h_in, TILE_SIZE);

    // Initialize device arrays
    int *d_in           = NULL;
    int *d_out          = NULL;
    clock_t *d_elapsed  = NULL;
    hipMalloc((void**)&d_in,          sizeof(int) * TILE_SIZE);
    hipMalloc((void**)&d_out,         sizeof(int) * 1);
    hipMalloc((void**)&d_elapsed,     sizeof(clock_t));

    // Display input problem data
    if (g_verbose)
    {
        printf("Input data: ");
        for (int i = 0; i < TILE_SIZE; i++)
            printf("%d, ", h_in[i]);
        printf("\n\n");
    }

    // Kernel props
    int max_sm_occupancy;
    CubDebugExit(MaxSmOccupancy(max_sm_occupancy, BlockSumKernel<BLOCK_THREADS, ITEMS_PER_THREAD, ALGORITHM>, BLOCK_THREADS));

    // Copy problem to device
    hipMemcpy(d_in, h_in, sizeof(int) * TILE_SIZE, hipMemcpyHostToDevice);

    printf("BlockReduce algorithm %s on %d items (%d timing iterations, %d blocks, %d threads, %d items per thread, %d SM occupancy):\n",
        (ALGORITHM == BLOCK_REDUCE_RAKING) ? "BLOCK_REDUCE_RAKING" : "BLOCK_REDUCE_WARP_REDUCTIONS",
        TILE_SIZE, g_timing_iterations, g_grid_size, BLOCK_THREADS, ITEMS_PER_THREAD, max_sm_occupancy);

    // Run aggregate/prefix kernel
    BlockSumKernel<BLOCK_THREADS, ITEMS_PER_THREAD, ALGORITHM><<<g_grid_size, BLOCK_THREADS>>>(
        d_in,
        d_out,
        d_elapsed);

    // Check total aggregate
    printf("\tAggregate: ");
    int compare = CompareDeviceResults(&h_aggregate, d_out, 1, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Run this several times and average the performance results
    GpuTimer    timer;
    float       elapsed_millis          = 0.0;
    clock_t     elapsed_clocks          = 0;

    for (int i = 0; i < g_timing_iterations; ++i)
    {
        // Copy problem to device
        hipMemcpy(d_in, h_in, sizeof(int) * TILE_SIZE, hipMemcpyHostToDevice);

        timer.Start();

        // Run aggregate/prefix kernel
        BlockSumKernel<BLOCK_THREADS, ITEMS_PER_THREAD, ALGORITHM><<<g_grid_size, BLOCK_THREADS>>>(
            d_in,
            d_out,
            d_elapsed);

        timer.Stop();
        elapsed_millis += timer.ElapsedMillis();

        // Copy clocks from device
        clock_t clocks;
        CubDebugExit(hipMemcpy(&clocks, d_elapsed, sizeof(clock_t), hipMemcpyDeviceToHost));
        elapsed_clocks += clocks;

    }

    // Check for kernel errors and STDIO from the kernel, if any
    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());

    // Display timing results
    float avg_millis            = elapsed_millis / g_timing_iterations;
    float avg_items_per_sec     = float(TILE_SIZE * g_grid_size) / avg_millis / 1000.0f;
    float avg_clocks            = float(elapsed_clocks) / g_timing_iterations;
    float avg_clocks_per_item   = avg_clocks / TILE_SIZE;

    printf("\tAverage BlockReduce::Sum clocks: %.3f\n", avg_clocks);
    printf("\tAverage BlockReduce::Sum clocks per item: %.3f\n", avg_clocks_per_item);
    printf("\tAverage kernel millis: %.4f\n", avg_millis);
    printf("\tAverage million items / sec: %.4f\n", avg_items_per_sec);

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_gpu) delete[] h_gpu;
    if (d_in) hipFree(d_in);
    if (d_out) hipFree(d_out);
    if (d_elapsed) hipFree(d_elapsed);
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("i", g_timing_iterations);
    args.GetCmdLineArgument("grid-size", g_grid_size);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--i=<timing iterations>] "
            "[--grid-size=<grid size>] "
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Run tests
    Test<1024, 1, BLOCK_REDUCE_RAKING>();
    Test<512, 2, BLOCK_REDUCE_RAKING>();
    Test<256, 4, BLOCK_REDUCE_RAKING>();
    Test<128, 8, BLOCK_REDUCE_RAKING>();
    Test<64, 16, BLOCK_REDUCE_RAKING>();
    Test<32, 32, BLOCK_REDUCE_RAKING>();
    Test<16, 64, BLOCK_REDUCE_RAKING>();

    printf("-------------\n");

    Test<1024, 1, BLOCK_REDUCE_WARP_REDUCTIONS>();
    Test<512, 2, BLOCK_REDUCE_WARP_REDUCTIONS>();
    Test<256, 4, BLOCK_REDUCE_WARP_REDUCTIONS>();
    Test<128, 8, BLOCK_REDUCE_WARP_REDUCTIONS>();
    Test<64, 16, BLOCK_REDUCE_WARP_REDUCTIONS>();
    Test<32, 32, BLOCK_REDUCE_WARP_REDUCTIONS>();
    Test<16, 64, BLOCK_REDUCE_WARP_REDUCTIONS>();

    return 0;
}

