#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of iterator utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iterator>
#include <stdio.h>
#include <typeinfo>

#include <cub/iterator/arg_index_input_iterator.cuh>
#include <cub/iterator/cache_modified_input_iterator.cuh>
#include <cub/iterator/cache_modified_output_iterator.cuh>
#include <cub/iterator/constant_input_iterator.cuh>
#include <cub/iterator/counting_input_iterator.cuh>
#include <cub/iterator/tex_obj_input_iterator.cuh>
#include <cub/iterator/tex_ref_input_iterator.cuh>
#include <cub/iterator/transform_input_iterator.cuh>

#include <cub/util_type.cuh>
#include <cub/util_allocator.cuh>

#include "test_util.h"

#include <thrust/device_ptr.h>
#include <thrust/copy.h>

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose = false;
CachingDeviceAllocator  g_allocator(true);

// Dispatch types
enum Backend
{
    CUB,        // CUB method
    THRUST,     // Thrust method
    CDP,        // GPU-based (dynamic parallelism) dispatch to CUB method
};


template <typename T>
struct TransformOp
{
    // Increment transform
    __host__ __device__ __forceinline__ T operator()(T input) const
    {
        T addend;
        InitValue(INTEGER_SEED, addend, 1);
        return input + addend;
    }
};

struct SelectOp
{
    template <typename T>
    __host__ __device__ __forceinline__ bool operator()(T input)
    {
        return true;
    }
};


//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------

/**
 * Test random access input iterator
 */
template <
    typename InputIteratorT,
    typename T>
__global__ void Kernel(
    InputIteratorT    d_in,
    T                 *d_out,
    InputIteratorT    *d_itrs)
{
    d_out[0] = *d_in;               // Value at offset 0
    d_out[1] = d_in[100];           // Value at offset 100
    d_out[2] = *(d_in + 1000);      // Value at offset 1000
    d_out[3] = *(d_in + 10000);     // Value at offset 10000

    d_in++;
    d_out[4] = d_in[0];             // Value at offset 1

    d_in += 20;
    d_out[5] = d_in[0];             // Value at offset 21
    d_itrs[0] = d_in;               // Iterator at offset 21

    d_in -= 10;
    d_out[6] = d_in[0];             // Value at offset 11;

    d_in -= 11;
    d_out[7] = d_in[0];             // Value at offset 0
    d_itrs[1] = d_in;               // Iterator at offset 0
}



//---------------------------------------------------------------------
// Host testing subroutines
//---------------------------------------------------------------------


/**
 * Run iterator test on device
 */
template <
    typename        InputIteratorT,
    typename        T,
    int             TEST_VALUES>
void Test(
    InputIteratorT  d_in,
    T               (&h_reference)[TEST_VALUES])
{
    // Allocate device arrays
    T                 *d_out    = NULL;
    InputIteratorT    *d_itrs   = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out,     sizeof(T) * TEST_VALUES));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_itrs,    sizeof(InputIteratorT) * 2));

    int compare;

    // Run unguarded kernel
    Kernel<<<1, 1>>>(d_in, d_out, d_itrs);

    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());

    // Check results
    compare = CompareDeviceResults(h_reference, d_out, TEST_VALUES, g_verbose, g_verbose);
    printf("\tValues: %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Check iterator at offset 21
    InputIteratorT h_itr = d_in + 21;
    compare = CompareDeviceResults(&h_itr, d_itrs, 1, g_verbose, g_verbose);
    printf("\tIterators: %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Check iterator at offset 0
    compare = CompareDeviceResults(&d_in, d_itrs + 1, 1, g_verbose, g_verbose);
    printf("\tIterators: %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Cleanup
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_itrs) CubDebugExit(g_allocator.DeviceFree(d_itrs));
}


/**
 * Test constant iterator
 */
template <typename T>
void TestConstant(T base)
{
    printf("\nTesting constant iterator on type %s (base: %lld)\n", typeid(T).name(), (unsigned long long) (base)); fflush(stdout);

    //
    // Test iterator manipulation in kernel
    //

    T h_reference[8] = {base, base, base, base, base, base, base, base};
    ConstantInputIterator<T> d_itr(base);
    Test(d_itr, h_reference);

#if (THRUST_VERSION >= 100700)  // Thrust 1.7 or newer

    //
    // Test with thrust::copy_if()
    //

    int copy_items  = 100;
    T   *h_copy     = new T[copy_items];
    T   *d_copy     = NULL;

    for (int i = 0; i < copy_items; ++i)
        h_copy[i] = d_itr[i];

    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_copy, sizeof(T) * copy_items));
    thrust::device_ptr<T> d_copy_wrapper(d_copy);

    thrust::copy_if(d_itr, d_itr + copy_items, d_copy_wrapper, SelectOp());

    int compare = CompareDeviceResults(h_copy, d_copy, copy_items, g_verbose, g_verbose);
    printf("\tthrust::copy_if(): %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    if (h_copy) delete[] h_copy;
    if (d_copy) CubDebugExit(g_allocator.DeviceFree(d_copy));

#endif // THRUST_VERSION
}


/**
 * Test counting iterator
 */
template <typename T>
void TestCounting(T base)
{
    printf("\nTesting counting iterator on type %s (base: %d) \n", typeid(T).name(), int(base)); fflush(stdout);

    //
    // Test iterator manipulation in kernel
    //

    // Initialize reference data
    T h_reference[8];
    h_reference[0] = base + 0;          // Value at offset 0
    h_reference[1] = base + 100;        // Value at offset 100
    h_reference[2] = base + 1000;       // Value at offset 1000
    h_reference[3] = base + 10000;      // Value at offset 10000
    h_reference[4] = base + 1;          // Value at offset 1
    h_reference[5] = base + 21;         // Value at offset 21
    h_reference[6] = base + 11;         // Value at offset 11
    h_reference[7] = base + 0;          // Value at offset 0;

    CountingInputIterator<T> d_itr(base);
    Test(d_itr, h_reference);

#if (THRUST_VERSION >= 100700)  // Thrust 1.7 or newer

    //
    // Test with thrust::copy_if()
    //

    unsigned long long  max_items   = ((1ull << ((sizeof(T) * 8) - 1)) - 1);
    size_t  copy_items              = (size_t) CUB_MIN(max_items - base, 100);     // potential issue with differencing overflows when T is a smaller type than can handle the offset
    T                   *h_copy     = new T[copy_items];
    T                   *d_copy     = NULL;

    for (unsigned long long i = 0; i < copy_items; ++i)
        h_copy[i] = d_itr[i];

    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_copy, sizeof(T) * copy_items));
    thrust::device_ptr<T> d_copy_wrapper(d_copy);
    thrust::copy_if(d_itr, d_itr + copy_items, d_copy_wrapper, SelectOp());

    int compare = CompareDeviceResults(h_copy, d_copy, copy_items, g_verbose, g_verbose);
    printf("\tthrust::copy_if(): %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    if (h_copy) delete[] h_copy;
    if (d_copy) CubDebugExit(g_allocator.DeviceFree(d_copy));

#endif // THRUST_VERSION
}


/**
 * Test modified iterator
 */
template <typename T, typename CastT>
void TestModified()
{
    printf("\nTesting cache-modified iterator on type %s\n", typeid(T).name()); fflush(stdout);

    //
    // Test iterator manipulation in kernel
    //

    constexpr int TEST_VALUES = 11000;

    T *h_data = new T[TEST_VALUES];
    for (int i = 0; i < TEST_VALUES; ++i)
    {
        RandomBits(h_data[i]);
    }

    // Allocate device arrays
    T *d_data = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_data, sizeof(T) * TEST_VALUES));
    CubDebugExit(hipMemcpy(d_data, h_data, sizeof(T) * TEST_VALUES, hipMemcpyHostToDevice));

    // Initialize reference data
    T h_reference[8];
    h_reference[0] = h_data[0];          // Value at offset 0
    h_reference[1] = h_data[100];        // Value at offset 100
    h_reference[2] = h_data[1000];       // Value at offset 1000
    h_reference[3] = h_data[10000];      // Value at offset 10000
    h_reference[4] = h_data[1];          // Value at offset 1
    h_reference[5] = h_data[21];         // Value at offset 21
    h_reference[6] = h_data[11];         // Value at offset 11
    h_reference[7] = h_data[0];          // Value at offset 0;

    Test(CacheModifiedInputIterator<LOAD_DEFAULT, T>((CastT*) d_data), h_reference);
    Test(CacheModifiedInputIterator<LOAD_CA, T>((CastT*) d_data), h_reference);
    Test(CacheModifiedInputIterator<LOAD_CG, T>((CastT*) d_data), h_reference);
    Test(CacheModifiedInputIterator<LOAD_CS, T>((CastT*) d_data), h_reference);
    Test(CacheModifiedInputIterator<LOAD_CV, T>((CastT*) d_data), h_reference);
    Test(CacheModifiedInputIterator<LOAD_LDG, T>((CastT*) d_data), h_reference);
    Test(CacheModifiedInputIterator<LOAD_VOLATILE, T>((CastT*) d_data), h_reference);

#if (THRUST_VERSION >= 100700)  // Thrust 1.7 or newer

    //
    // Test with thrust::copy_if()
    //

    T *d_copy = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_copy, sizeof(T) * TEST_VALUES));

    CacheModifiedInputIterator<LOAD_CG, T> d_in_itr((CastT*) d_data);
    CacheModifiedOutputIterator<STORE_CG, T> d_out_itr((CastT*) d_copy);

    thrust::copy_if(d_in_itr, d_in_itr + TEST_VALUES, d_out_itr, SelectOp());

    int compare = CompareDeviceResults(h_data, d_copy, TEST_VALUES, g_verbose, g_verbose);
    printf("\tthrust::copy_if(): %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Cleanup
    if (d_copy) CubDebugExit(g_allocator.DeviceFree(d_copy));

#endif // THRUST_VERSION

    if (h_data) delete[] h_data;
    if (d_data) CubDebugExit(g_allocator.DeviceFree(d_data));
}


/**
 * Test transform iterator
 */
template <typename T, typename CastT>
void TestTransform()
{
    printf("\nTesting transform iterator on type %s\n", typeid(T).name()); fflush(stdout);

    //
    // Test iterator manipulation in kernel
    //

    constexpr int TEST_VALUES = 11000;

    T *h_data = new T[TEST_VALUES];
    for (int i = 0; i < TEST_VALUES; ++i)
    {
        InitValue(INTEGER_SEED, h_data[i], i);
    }

    // Allocate device arrays
    T *d_data = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_data, sizeof(T) * TEST_VALUES));
    CubDebugExit(hipMemcpy(d_data, h_data, sizeof(T) * TEST_VALUES, hipMemcpyHostToDevice));

    TransformOp<T> op;

    // Initialize reference data
    T h_reference[8];
    h_reference[0] = op(h_data[0]);          // Value at offset 0
    h_reference[1] = op(h_data[100]);        // Value at offset 100
    h_reference[2] = op(h_data[1000]);       // Value at offset 1000
    h_reference[3] = op(h_data[10000]);      // Value at offset 10000
    h_reference[4] = op(h_data[1]);          // Value at offset 1
    h_reference[5] = op(h_data[21]);         // Value at offset 21
    h_reference[6] = op(h_data[11]);         // Value at offset 11
    h_reference[7] = op(h_data[0]);          // Value at offset 0;

    TransformInputIterator<T, TransformOp<T>, CastT*> d_itr((CastT*) d_data, op);
    Test(d_itr, h_reference);

#if (THRUST_VERSION >= 100700)  // Thrust 1.7 or newer

    //
    // Test with thrust::copy_if()
    //

    T *h_copy = new T[TEST_VALUES];
    for (int i = 0; i < TEST_VALUES; ++i)
        h_copy[i] = op(h_data[i]);

    T *d_copy = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_copy, sizeof(T) * TEST_VALUES));
    thrust::device_ptr<T> d_copy_wrapper(d_copy);

    thrust::copy_if(d_itr, d_itr + TEST_VALUES, d_copy_wrapper, SelectOp());

    int compare = CompareDeviceResults(h_copy, d_copy, TEST_VALUES, g_verbose, g_verbose);
    printf("\tthrust::copy_if(): %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Cleanup
    if (h_copy) delete[] h_copy;
    if (d_copy) CubDebugExit(g_allocator.DeviceFree(d_copy));

#endif // THRUST_VERSION

    if (h_data) delete[] h_data;
    if (d_data) CubDebugExit(g_allocator.DeviceFree(d_data));
}


/**
 * Test tex-obj texture iterator
 */
template <typename T, typename CastT>
void TestTexObj()
{
    printf("\nTesting tex-obj iterator on type %s\n", typeid(T).name()); fflush(stdout);

    //
    // Test iterator manipulation in kernel
    //

    const unsigned int TEST_VALUES          = 11000;
    const unsigned int DUMMY_OFFSET         = 500;
    const unsigned int DUMMY_TEST_VALUES    = TEST_VALUES - DUMMY_OFFSET;

    T *h_data = new T[TEST_VALUES];
    for (int i = 0; i < TEST_VALUES; ++i)
    {
        RandomBits(h_data[i]);
    }

    // Allocate device arrays
    T *d_data   = NULL;
    T *d_dummy  = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_data, sizeof(T) * TEST_VALUES));
    CubDebugExit(hipMemcpy(d_data, h_data, sizeof(T) * TEST_VALUES, hipMemcpyHostToDevice));

    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_dummy, sizeof(T) * DUMMY_TEST_VALUES));
    CubDebugExit(hipMemcpy(d_dummy, h_data + DUMMY_OFFSET, sizeof(T) * DUMMY_TEST_VALUES, hipMemcpyHostToDevice));

    // Initialize reference data
    T h_reference[8];
    h_reference[0] = h_data[0];          // Value at offset 0
    h_reference[1] = h_data[100];        // Value at offset 100
    h_reference[2] = h_data[1000];       // Value at offset 1000
    h_reference[3] = h_data[10000];      // Value at offset 10000
    h_reference[4] = h_data[1];          // Value at offset 1
    h_reference[5] = h_data[21];         // Value at offset 21
    h_reference[6] = h_data[11];         // Value at offset 11
    h_reference[7] = h_data[0];          // Value at offset 0;

    // Create and bind obj-based test iterator
    TexObjInputIterator<T> d_obj_itr;
    CubDebugExit(d_obj_itr.BindTexture((CastT*) d_data, sizeof(T) * TEST_VALUES));

    Test(d_obj_itr, h_reference);

#if (THRUST_VERSION >= 100700)  // Thrust 1.7 or newer

    //
    // Test with thrust::copy_if()
    //

    T *d_copy = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_copy, sizeof(T) * TEST_VALUES));
    thrust::device_ptr<T> d_copy_wrapper(d_copy);

    CubDebugExit(hipMemset(d_copy, 0, sizeof(T) * TEST_VALUES));
    thrust::copy_if(d_obj_itr, d_obj_itr + TEST_VALUES, d_copy_wrapper, SelectOp());

    int compare = CompareDeviceResults(h_data, d_copy, TEST_VALUES, g_verbose, g_verbose);
    printf("\tthrust::copy_if(): %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Cleanup
    CubDebugExit(d_obj_itr.UnbindTexture());

    if (d_copy) CubDebugExit(g_allocator.DeviceFree(d_copy));

#endif  // THRUST_VERSION

    if (h_data) delete[] h_data;
    if (d_data) CubDebugExit(g_allocator.DeviceFree(d_data));
    if (d_dummy) CubDebugExit(g_allocator.DeviceFree(d_dummy));
}


#if CUDART_VERSION >= 5050

/**
 * Test tex-ref texture iterator
 */
template <typename T, typename CastT>
void TestTexRef()
{
    printf("\nTesting tex-ref iterator on type %s\n", typeid(T).name()); fflush(stdout);

    //
    // Test iterator manipulation in kernel
    //

    constexpr int TEST_VALUES                   = 11000;
    constexpr unsigned int DUMMY_OFFSET         = 500;
    constexpr unsigned int DUMMY_TEST_VALUES    = TEST_VALUES - DUMMY_OFFSET;

    T *h_data = new T[TEST_VALUES];
    for (int i = 0; i < TEST_VALUES; ++i)
    {
        RandomBits(h_data[i]);
    }

    // Allocate device arrays
    T *d_data   = NULL;
    T *d_dummy  = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_data, sizeof(T) * TEST_VALUES));
    CubDebugExit(hipMemcpy(d_data, h_data, sizeof(T) * TEST_VALUES, hipMemcpyHostToDevice));

    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_dummy, sizeof(T) * DUMMY_TEST_VALUES));
    CubDebugExit(hipMemcpy(d_dummy, h_data + DUMMY_OFFSET, sizeof(T) * DUMMY_TEST_VALUES, hipMemcpyHostToDevice));

    // Initialize reference data
    T h_reference[8];
    h_reference[0] = h_data[0];          // Value at offset 0
    h_reference[1] = h_data[100];        // Value at offset 100
    h_reference[2] = h_data[1000];       // Value at offset 1000
    h_reference[3] = h_data[10000];      // Value at offset 10000
    h_reference[4] = h_data[1];          // Value at offset 1
    h_reference[5] = h_data[21];         // Value at offset 21
    h_reference[6] = h_data[11];         // Value at offset 11
    h_reference[7] = h_data[0];          // Value at offset 0;

    // Create and bind ref-based test iterator
    TexRefInputIterator<T, __LINE__> d_ref_itr;
    CubDebugExit(d_ref_itr.BindTexture((CastT*) d_data, sizeof(T) * TEST_VALUES));

    // Create and bind dummy iterator of same type to check with interferance
    TexRefInputIterator<T, __LINE__> d_ref_itr2;
    CubDebugExit(d_ref_itr2.BindTexture((CastT*) d_dummy, sizeof(T) * DUMMY_TEST_VALUES));

    Test(d_ref_itr, h_reference);

#if (THRUST_VERSION >= 100700)  // Thrust 1.7 or newer

    //
    // Test with thrust::copy_if()
    //

    T *d_copy = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_copy, sizeof(T) * TEST_VALUES));
    thrust::device_ptr<T> d_copy_wrapper(d_copy);

    CubDebugExit(hipMemset(d_copy, 0, sizeof(T) * TEST_VALUES));
    thrust::copy_if(d_ref_itr, d_ref_itr + TEST_VALUES, d_copy_wrapper, SelectOp());

    int compare = CompareDeviceResults(h_data, d_copy, TEST_VALUES, g_verbose, g_verbose);
    printf("\tthrust::copy_if(): %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    if (d_copy) CubDebugExit(g_allocator.DeviceFree(d_copy));

#endif  // THRUST_VERSION

    CubDebugExit(d_ref_itr.UnbindTexture());
    CubDebugExit(d_ref_itr2.UnbindTexture());

    if (h_data) delete[] h_data;
    if (d_data) CubDebugExit(g_allocator.DeviceFree(d_data));
    if (d_dummy) CubDebugExit(g_allocator.DeviceFree(d_dummy));
}


/**
 * Test texture transform iterator
 */
template <typename T, typename CastT>
void TestTexTransform()
{
    printf("\nTesting tex-transform iterator on type %s\n", typeid(T).name()); fflush(stdout);

    //
    // Test iterator manipulation in kernel
    //

    constexpr int TEST_VALUES = 11000;

    T *h_data = new T[TEST_VALUES];
    for (int i = 0; i < TEST_VALUES; ++i)
    {
        InitValue(INTEGER_SEED, h_data[i], i);
    }

    // Allocate device arrays
    T *d_data = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_data, sizeof(T) * TEST_VALUES));
    CubDebugExit(hipMemcpy(d_data, h_data, sizeof(T) * TEST_VALUES, hipMemcpyHostToDevice));

    TransformOp<T> op;

    // Initialize reference data
    T h_reference[8];
    h_reference[0] = op(h_data[0]);          // Value at offset 0
    h_reference[1] = op(h_data[100]);        // Value at offset 100
    h_reference[2] = op(h_data[1000]);       // Value at offset 1000
    h_reference[3] = op(h_data[10000]);      // Value at offset 10000
    h_reference[4] = op(h_data[1]);          // Value at offset 1
    h_reference[5] = op(h_data[21]);         // Value at offset 21
    h_reference[6] = op(h_data[11]);         // Value at offset 11
    h_reference[7] = op(h_data[0]);          // Value at offset 0;

    // Create and bind texture iterator
    typedef TexRefInputIterator<T, __LINE__> TextureIterator;

    TextureIterator d_tex_itr;
    CubDebugExit(d_tex_itr.BindTexture((CastT*) d_data, sizeof(T) * TEST_VALUES));

    // Create transform iterator
    TransformInputIterator<T, TransformOp<T>, TextureIterator> xform_itr(d_tex_itr, op);

    Test(xform_itr, h_reference);

#if (THRUST_VERSION >= 100700)  // Thrust 1.7 or newer

    //
    // Test with thrust::copy_if()
    //

    T *h_copy = new T[TEST_VALUES];
    for (int i = 0; i < TEST_VALUES; ++i)
        h_copy[i] = op(h_data[i]);

    T *d_copy = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_copy, sizeof(T) * TEST_VALUES));
    thrust::device_ptr<T> d_copy_wrapper(d_copy);

    thrust::copy_if(xform_itr, xform_itr + TEST_VALUES, d_copy_wrapper, SelectOp());

    int compare = CompareDeviceResults(h_copy, d_copy, TEST_VALUES, g_verbose, g_verbose);
    printf("\tthrust::copy_if(): %s\n", (compare) ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Cleanup
    if (h_copy) delete[] h_copy;
    if (d_copy) CubDebugExit(g_allocator.DeviceFree(d_copy));

#endif  // THRUST_VERSION

    CubDebugExit(d_tex_itr.UnbindTexture());
    if (h_data) delete[] h_data;
    if (d_data) CubDebugExit(g_allocator.DeviceFree(d_data));
}

#endif  // CUDART_VERSION




/**
 * Run non-integer tests
 */
template <typename T, typename CastT>
void Test(Int2Type<false> /* is_integer */)
{
    TestModified<T, CastT>();
    TestTransform<T, CastT>();

#if CUB_CDP
    // Test tex-obj iterators if CUDA dynamic parallelism enabled
    TestTexObj<T, CastT>(type_string);
#endif  // CUB_CDP

#if CUDART_VERSION >= 5050
    // Test tex-ref iterators for CUDA 5.5
    TestTexRef<T, CastT>();
    TestTexTransform<T, CastT>();
#endif  // CUDART_VERSION
}

/**
 * Run integer tests
 */
template <typename T, typename CastT>
void Test(Int2Type<true> /* is_integer */)
{
    TestConstant<T>(0);
    TestConstant<T>(99);

    TestCounting<T>(0);
    TestCounting<T>(99);

    // Run non-integer tests
    Test<T, CastT>(Int2Type<false>());
}

/**
 * Run tests
 */
template <typename T>
void Test()
{
    enum {
        IS_INTEGER = (Traits<T>::CATEGORY == SIGNED_INTEGER) || (Traits<T>::CATEGORY == UNSIGNED_INTEGER)
    };

    // Test non-const type
    Test<T, T>(Int2Type<IS_INTEGER>());

    // Test non-const type
    Test<T, const T>(Int2Type<IS_INTEGER>());
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Get ptx version
    int ptx_version = 0;
    CubDebugExit(PtxVersion(ptx_version));

    // Evaluate different data types
    Test<char>();
    Test<short>();
    Test<int>();
    Test<long>();
    Test<long long>();
    Test<float>();
    if (ptx_version > 120)                          // Don't check doubles on PTX120 or below because they're down-converted
        Test<double>();

    Test<char2>();
    Test<short2>();
    Test<int2>();
    Test<long2>();
    Test<longlong2>();
    Test<float2>();
    if (ptx_version > 120)                          // Don't check doubles on PTX120 or below because they're down-converted
        Test<double2>();

    Test<char3>();
    Test<short3>();
    Test<int3>();
    Test<long3>();
    Test<longlong3>();
    Test<float3>();
    if (ptx_version > 120)                          // Don't check doubles on PTX120 or below because they're down-converted
        Test<double3>();

    Test<char4>();
    Test<short4>();
    Test<int4>();
    Test<long4>();
    Test<longlong4>();
    Test<float4>();
    if (ptx_version > 120)                          // Don't check doubles on PTX120 or below because they're down-converted
        Test<double4>();

    Test<TestFoo>();
    Test<TestBar>();

    printf("\nTest complete\n"); fflush(stdout);

    return 0;
}



