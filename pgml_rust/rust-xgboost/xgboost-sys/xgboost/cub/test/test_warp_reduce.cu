#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of WarpReduce utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <typeinfo>

#include <cub/warp/warp_reduce.cuh>
#include <cub/util_allocator.cuh>

#include "test_util.h"

using namespace hipcub;

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose       = false;
int                     g_repeat        = 0;
CachingDeviceAllocator  g_allocator(true);


/**
 * \brief WrapperFunctor (for precluding test-specialized dispatch to *Sum variants)
 */
template<
    typename    OpT,
    int         LOGICAL_WARP_THREADS>
struct WrapperFunctor
{
    OpT op;
    int num_valid;

    inline __host__ __device__ WrapperFunctor(OpT op, int num_valid) : op(op), num_valid(num_valid) {}

    template <typename T>
    inline __host__ __device__ T operator()(const T &a, const T &b) const
    {
#if HIPCUB_ARCH != 0
        if ((hipcub::LaneId() % LOGICAL_WARP_THREADS) >= num_valid)
            hipcub::ThreadTrap();
#endif

        return op(a, b);
    }

};


//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------

/**
 * Generic reduction
 */
template <
    typename    T,
    typename    ReductionOp,
    typename    WarpReduce,
    bool        PRIMITIVE = Traits<T>::PRIMITIVE>
struct DeviceTest
{
    static __device__ __forceinline__ T Reduce(
        typename WarpReduce::TempStorage    &temp_storage,
        T                                   &data,
        ReductionOp                         &reduction_op)
    {
        return WarpReduce(temp_storage).Reduce(data, reduction_op);
    }

    static __device__ __forceinline__ T Reduce(
        typename WarpReduce::TempStorage    &temp_storage,
        T                                   &data,
        ReductionOp                         &reduction_op,
        const int                           &valid_warp_threads)
    {
        return WarpReduce(temp_storage).Reduce(data, reduction_op, valid_warp_threads);
    }

    template <typename FlagT>
    static __device__ __forceinline__ T HeadSegmentedReduce(
        typename WarpReduce::TempStorage    &temp_storage,
        T                                   &data,
        FlagT                                &flag,
        ReductionOp                         &reduction_op)
    {
        return WarpReduce(temp_storage).HeadSegmentedReduce(data, flag, reduction_op);
    }

    template <typename FlagT>
    static __device__ __forceinline__ T TailSegmentedReduce(
        typename WarpReduce::TempStorage    &temp_storage,
        T                                   &data,
        FlagT                                &flag,
        ReductionOp                         &reduction_op)
    {
        return WarpReduce(temp_storage).TailSegmentedReduce(data, flag, reduction_op);
    }

};


/**
 * Summation
 */
template <
    typename    T,
    typename    WarpReduce>
struct DeviceTest<T, Sum, WarpReduce, true>
{
    static __device__ __forceinline__ T Reduce(
        typename WarpReduce::TempStorage    &temp_storage,
        T                                   &data,
        Sum                              &reduction_op)
    {
        return WarpReduce(temp_storage).Sum(data);
    }

    static __device__ __forceinline__ T Reduce(
        typename WarpReduce::TempStorage    &temp_storage,
        T                                   &data,
        Sum                              &reduction_op,
        const int                           &valid_warp_threads)
    {
        return WarpReduce(temp_storage).Sum(data, valid_warp_threads);
    }

    template <typename FlagT>
    static __device__ __forceinline__ T HeadSegmentedReduce(
        typename WarpReduce::TempStorage    &temp_storage,
        T                                   &data,
        FlagT                                &flag,
        Sum                              &reduction_op)
    {
        return WarpReduce(temp_storage).HeadSegmentedSum(data, flag);
    }

    template <typename FlagT>
    static __device__ __forceinline__ T TailSegmentedReduce(
        typename WarpReduce::TempStorage    &temp_storage,
        T                                   &data,
        FlagT                                &flag,
        Sum                              &reduction_op)
    {
        return WarpReduce(temp_storage).TailSegmentedSum(data, flag);
    }

};


/**
 * Full-tile warp reduction kernel
 */
template <
    int         WARPS,
    int         LOGICAL_WARP_THREADS,
    typename    T,
    typename    ReductionOp>
__global__ void FullWarpReduceKernel(
    T               *d_in,
    T               *d_out,
    ReductionOp     reduction_op,
    clock_t         *d_elapsed)
{
    // Cooperative warp-reduce utility type (1 warp)
    typedef WarpReduce<T, LOGICAL_WARP_THREADS> WarpReduce;

    // Allocate temp storage in shared memory
    __shared__ typename WarpReduce::TempStorage temp_storage[WARPS];

    // Per-thread tile data
    T input = d_in[threadIdx.x];

    // Record elapsed clocks
    __threadfence_block();      // workaround to prevent clock hoisting
    clock_t start = clock();
    __threadfence_block();      // workaround to prevent clock hoisting

    // Test warp reduce
    int warp_id = threadIdx.x / LOGICAL_WARP_THREADS;

    T output = DeviceTest<T, ReductionOp, WarpReduce>::Reduce(
        temp_storage[warp_id], input, reduction_op);

    // Record elapsed clocks
    __threadfence_block();      // workaround to prevent clock hoisting
    clock_t stop = clock();
    __threadfence_block();      // workaround to prevent clock hoisting

    *d_elapsed = stop - start;

    // Store aggregate
    d_out[threadIdx.x] = (threadIdx.x % LOGICAL_WARP_THREADS == 0) ?
        output :
        input;
}

/**
 * Partially-full warp reduction kernel
 */
template <
    int         WARPS,
    int         LOGICAL_WARP_THREADS,
    typename    T,
    typename    ReductionOp>
__global__ void PartialWarpReduceKernel(
    T           *d_in,
    T           *d_out,
    ReductionOp reduction_op,
    clock_t     *d_elapsed,
    int         valid_warp_threads)
{
    // Cooperative warp-reduce utility type
    typedef WarpReduce<T, LOGICAL_WARP_THREADS> WarpReduce;

    // Allocate temp storage in shared memory
    __shared__ typename WarpReduce::TempStorage temp_storage[WARPS];

    // Per-thread tile data
    T input = d_in[threadIdx.x];

    // Record elapsed clocks
    __threadfence_block();      // workaround to prevent clock hoisting
    clock_t start = clock();
    __threadfence_block();      // workaround to prevent clock hoisting

    // Test partial-warp reduce
    int warp_id = threadIdx.x / LOGICAL_WARP_THREADS;
    T output = DeviceTest<T, ReductionOp, WarpReduce>::Reduce(
        temp_storage[warp_id], input, reduction_op, valid_warp_threads);

    // Record elapsed clocks
    __threadfence_block();      // workaround to prevent clock hoisting
    clock_t stop = clock();
    __threadfence_block();      // workaround to prevent clock hoisting

    *d_elapsed = stop - start;

    // Store aggregate
    d_out[threadIdx.x] = (threadIdx.x % LOGICAL_WARP_THREADS == 0) ?
        output :
        input;
}


/**
 * Head-based segmented warp reduction test kernel
 */
template <
    int         WARPS,
    int         LOGICAL_WARP_THREADS,
    typename    T,
    typename    FlagT,
    typename    ReductionOp>
__global__ void WarpHeadSegmentedReduceKernel(
    T           *d_in,
    FlagT        *d_head_flags,
    T           *d_out,
    ReductionOp reduction_op,
    clock_t     *d_elapsed)
{
    // Cooperative warp-reduce utility type
    typedef WarpReduce<T, LOGICAL_WARP_THREADS> WarpReduce;

    // Allocate temp storage in shared memory
    __shared__ typename WarpReduce::TempStorage temp_storage[WARPS];

    // Per-thread tile data
    T       input       = d_in[threadIdx.x];
    FlagT   head_flag   = d_head_flags[threadIdx.x];

    // Record elapsed clocks
    __threadfence_block();      // workaround to prevent clock hoisting
    clock_t start = clock();
    __threadfence_block();      // workaround to prevent clock hoisting

    // Test segmented warp reduce
    int warp_id = threadIdx.x / LOGICAL_WARP_THREADS;
    T output = DeviceTest<T, ReductionOp, WarpReduce>::HeadSegmentedReduce(
        temp_storage[warp_id], input, head_flag, reduction_op);

    // Record elapsed clocks
    __threadfence_block();      // workaround to prevent clock hoisting
    clock_t stop = clock();
    __threadfence_block();      // workaround to prevent clock hoisting

    *d_elapsed = stop - start;

    // Store aggregate
    d_out[threadIdx.x] = ((threadIdx.x % LOGICAL_WARP_THREADS == 0) || head_flag) ?
        output :
        input;
}


/**
 * Tail-based segmented warp reduction test kernel
 */
template <
    int         WARPS,
    int         LOGICAL_WARP_THREADS,
    typename    T,
    typename    FlagT,
    typename    ReductionOp>
__global__ void WarpTailSegmentedReduceKernel(
    T           *d_in,
    FlagT       *d_tail_flags,
    T           *d_out,
    ReductionOp reduction_op,
    clock_t     *d_elapsed)
{
    // Cooperative warp-reduce utility type
    typedef WarpReduce<T, LOGICAL_WARP_THREADS> WarpReduce;

    // Allocate temp storage in shared memory
    __shared__ typename WarpReduce::TempStorage temp_storage[WARPS];

    // Per-thread tile data
    T       input       = d_in[threadIdx.x];
    FlagT    tail_flag   = d_tail_flags[threadIdx.x];
    FlagT    head_flag   = (threadIdx.x == 0) ?
                            0 :
                            d_tail_flags[threadIdx.x - 1];

    // Record elapsed clocks
    __threadfence_block();      // workaround to prevent clock hoisting
    clock_t start = clock();
    __threadfence_block();      // workaround to prevent clock hoisting

    // Test segmented warp reduce
    int warp_id = threadIdx.x / LOGICAL_WARP_THREADS;
    T output = DeviceTest<T, ReductionOp, WarpReduce>::TailSegmentedReduce(
        temp_storage[warp_id], input, tail_flag, reduction_op);

    // Record elapsed clocks
    __threadfence_block();      // workaround to prevent clock hoisting
    clock_t stop = clock();
    __threadfence_block();      // workaround to prevent clock hoisting

    *d_elapsed = stop - start;

    // Store aggregate
    d_out[threadIdx.x] = ((threadIdx.x % LOGICAL_WARP_THREADS == 0) || head_flag) ?
        output :
        input;
}


//---------------------------------------------------------------------
// Host utility subroutines
//---------------------------------------------------------------------

/**
 * Initialize reduction problem (and solution)
 */
template <
    typename    T,
    typename    ReductionOp>
void Initialize(
    GenMode     gen_mode,
    int         flag_entropy,
    T           *h_in,
    int         *h_flags,
    int         warps,
    int         warp_threads,
    int         valid_warp_threads,
    ReductionOp reduction_op,
    T           *h_head_out,
    T           *h_tail_out)
{
    for (int i = 0; i < warps * warp_threads; ++i)
    {
        // Sample a value for this item
        InitValue(gen_mode, h_in[i], i);
        h_head_out[i] = h_in[i];
        h_tail_out[i] = h_in[i];

        // Sample whether or not this item will be a segment head
        char bits;
        RandomBits(bits, flag_entropy);
        h_flags[i] = bits & 0x1;
    }

    // Accumulate segments (lane 0 of each warp is implicitly a segment head)
    for (int warp = 0; warp < warps; ++warp)
    {
        int warp_offset  = warp * warp_threads;
        int item_offset = warp_offset + valid_warp_threads - 1;

        // Last item in warp
        T head_aggregate = h_in[item_offset];
        T tail_aggregate = h_in[item_offset];

        if (h_flags[item_offset])
            h_head_out[item_offset] = head_aggregate;
        item_offset--;

        // Work backwards
        while (item_offset >= warp_offset)
        {
            if (h_flags[item_offset + 1])
            {
                head_aggregate = h_in[item_offset];
            }
            else
            {
                head_aggregate = reduction_op(head_aggregate, h_in[item_offset]);
            }

            if (h_flags[item_offset])
            {
                h_head_out[item_offset] = head_aggregate;
                h_tail_out[item_offset + 1] = tail_aggregate;
                tail_aggregate = h_in[item_offset];
            }
            else
            {
                tail_aggregate = reduction_op(tail_aggregate, h_in[item_offset]);
            }

            item_offset--;
        }

        // Record last segment head_aggregate to head offset
        h_head_out[warp_offset] = head_aggregate;
        h_tail_out[warp_offset] = tail_aggregate;
    }
}


/**
 * Test warp reduction
 */
template <
    int         WARPS,
    int         LOGICAL_WARP_THREADS,
    typename    T,
    typename    ReductionOp>
void TestReduce(
    GenMode     gen_mode,
    ReductionOp reduction_op,
    int         valid_warp_threads = LOGICAL_WARP_THREADS)
{
    const int BLOCK_THREADS = LOGICAL_WARP_THREADS * WARPS;

    // Allocate host arrays
    T   *h_in           = new T[BLOCK_THREADS];
    int *h_flags        = new int[BLOCK_THREADS];
    T   *h_out          = new T[BLOCK_THREADS];
    T   *h_tail_out     = new T[BLOCK_THREADS];

    // Initialize problem
    Initialize(gen_mode, -1, h_in, h_flags, WARPS, LOGICAL_WARP_THREADS, valid_warp_threads, reduction_op, h_out, h_tail_out);

    // Initialize/clear device arrays
    T *d_in = NULL;
    T *d_out = NULL;
    clock_t *d_elapsed = NULL;

    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(T) * BLOCK_THREADS));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(T) * BLOCK_THREADS));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_elapsed, sizeof(clock_t)));
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * BLOCK_THREADS, hipMemcpyHostToDevice));
    CubDebugExit(hipMemset(d_out, 0, sizeof(T) * BLOCK_THREADS));

    if (g_verbose)
    {
        printf("Data:\n");
        for (int i = 0; i < WARPS; ++i)
            DisplayResults(h_in + (i * LOGICAL_WARP_THREADS), valid_warp_threads);
    }

    // Run kernel
    printf("\nGen-mode %d, %d warps, %d warp threads, %d valid lanes, %s (%d bytes) elements:\n",
        gen_mode,
        WARPS,
        LOGICAL_WARP_THREADS,
        valid_warp_threads,
        typeid(T).name(),
        (int) sizeof(T));
    fflush(stdout);

    if (valid_warp_threads == LOGICAL_WARP_THREADS)
    {
        // Run full-warp kernel
        FullWarpReduceKernel<WARPS, LOGICAL_WARP_THREADS><<<1, BLOCK_THREADS>>>(
            d_in,
            d_out,
            reduction_op,
            d_elapsed);
    }
    else
    {
        // Run partial-warp kernel
        PartialWarpReduceKernel<WARPS, LOGICAL_WARP_THREADS><<<1, BLOCK_THREADS>>>(
            d_in,
            d_out,
            reduction_op,
            d_elapsed,
            valid_warp_threads);
    }

    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());

    // Copy out and display results
    printf("\tReduction results: ");
    int compare = CompareDeviceResults(h_out, d_out, BLOCK_THREADS, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);
    printf("\tElapsed clocks: ");
    DisplayDeviceResults(d_elapsed, 1);

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_flags) delete[] h_flags;
    if (h_out) delete[] h_out;
    if (h_tail_out) delete[] h_tail_out;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_elapsed) CubDebugExit(g_allocator.DeviceFree(d_elapsed));
}


/**
 * Test warp segmented reduction
 */
template <
    int         WARPS,
    int         LOGICAL_WARP_THREADS,
    typename    T,
    typename    ReductionOp>
void TestSegmentedReduce(
    GenMode     gen_mode,
    int         flag_entropy,
    ReductionOp reduction_op)
{
    const int BLOCK_THREADS = LOGICAL_WARP_THREADS * WARPS;

    // Allocate host arrays
    int compare;
    T   *h_in           = new T[BLOCK_THREADS];
    int *h_flags        = new int[BLOCK_THREADS];
    T   *h_head_out     = new T[BLOCK_THREADS];
    T   *h_tail_out     = new T[BLOCK_THREADS];

    // Initialize problem
    Initialize(gen_mode, flag_entropy, h_in, h_flags, WARPS, LOGICAL_WARP_THREADS, LOGICAL_WARP_THREADS, reduction_op, h_head_out, h_tail_out);

    // Initialize/clear device arrays
    T           *d_in = NULL;
    int         *d_flags = NULL;
    T           *d_head_out = NULL;
    T           *d_tail_out = NULL;
    clock_t     *d_elapsed = NULL;

    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(T) * BLOCK_THREADS));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_flags, sizeof(int) * BLOCK_THREADS));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_head_out, sizeof(T) * BLOCK_THREADS));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_tail_out, sizeof(T) * BLOCK_THREADS));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_elapsed, sizeof(clock_t)));
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * BLOCK_THREADS, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_flags, h_flags, sizeof(int) * BLOCK_THREADS, hipMemcpyHostToDevice));
    CubDebugExit(hipMemset(d_head_out, 0, sizeof(T) * BLOCK_THREADS));
    CubDebugExit(hipMemset(d_tail_out, 0, sizeof(T) * BLOCK_THREADS));

    if (g_verbose)
    {
        printf("Data:\n");
        for (int i = 0; i < WARPS; ++i)
            DisplayResults(h_in + (i * LOGICAL_WARP_THREADS), LOGICAL_WARP_THREADS);

        printf("\nFlags:\n");
        for (int i = 0; i < WARPS; ++i)
            DisplayResults(h_flags + (i * LOGICAL_WARP_THREADS), LOGICAL_WARP_THREADS);
    }

    printf("\nGen-mode %d, head flag entropy reduction %d, %d warps, %d warp threads, %s (%d bytes) elements:\n",
        gen_mode,
        flag_entropy,
        WARPS,
        LOGICAL_WARP_THREADS,
        typeid(T).name(),
        (int) sizeof(T));
    fflush(stdout);

    // Run head-based kernel
    WarpHeadSegmentedReduceKernel<WARPS, LOGICAL_WARP_THREADS><<<1, BLOCK_THREADS>>>(
        d_in,
        d_flags,
        d_head_out,
        reduction_op,
        d_elapsed);

    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());

    // Copy out and display results
    printf("\tHead-based segmented reduction results: ");
    compare = CompareDeviceResults(h_head_out, d_head_out, BLOCK_THREADS, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);
    printf("\tElapsed clocks: ");
    DisplayDeviceResults(d_elapsed, 1);

    // Run tail-based kernel
    WarpTailSegmentedReduceKernel<WARPS, LOGICAL_WARP_THREADS><<<1, BLOCK_THREADS>>>(
        d_in,
        d_flags,
        d_tail_out,
        reduction_op,
        d_elapsed);

    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());

    // Copy out and display results
    printf("\tTail-based segmented reduction results: ");
    compare = CompareDeviceResults(h_tail_out, d_tail_out, BLOCK_THREADS, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);
    printf("\tElapsed clocks: ");
    DisplayDeviceResults(d_elapsed, 1);

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_flags) delete[] h_flags;
    if (h_head_out) delete[] h_head_out;
    if (h_tail_out) delete[] h_tail_out;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_flags) CubDebugExit(g_allocator.DeviceFree(d_flags));
    if (d_head_out) CubDebugExit(g_allocator.DeviceFree(d_head_out));
    if (d_tail_out) CubDebugExit(g_allocator.DeviceFree(d_tail_out));
    if (d_elapsed) CubDebugExit(g_allocator.DeviceFree(d_elapsed));
}


/**
 * Run battery of tests for different full and partial tile sizes
 */
template <
    int         WARPS,
    int         LOGICAL_WARP_THREADS,
    typename    T,
    typename    ReductionOp>
void Test(
    GenMode     gen_mode,
    ReductionOp reduction_op)
{
    // Partial tiles
    for (
        int valid_warp_threads = 1;
        valid_warp_threads < LOGICAL_WARP_THREADS;
        valid_warp_threads += CUB_MAX(1, LOGICAL_WARP_THREADS / 5))
    {
        // Without wrapper (to test non-excepting PTX POD-op specializations)
        TestReduce<WARPS, LOGICAL_WARP_THREADS, T>(gen_mode, reduction_op, valid_warp_threads);

        // With wrapper to ensure no ops called on OOB lanes
        WrapperFunctor<ReductionOp, LOGICAL_WARP_THREADS> wrapped_op(reduction_op, valid_warp_threads);
        TestReduce<WARPS, LOGICAL_WARP_THREADS, T>(gen_mode, wrapped_op, valid_warp_threads);
    }

    // Full tile
    TestReduce<WARPS, LOGICAL_WARP_THREADS, T>(gen_mode, reduction_op, LOGICAL_WARP_THREADS);

    // Segmented reduction with different head flags
    for (int flag_entropy = 0; flag_entropy < 10; ++flag_entropy)
    {
        TestSegmentedReduce<WARPS, LOGICAL_WARP_THREADS, T>(gen_mode, flag_entropy, reduction_op);
    }
}


/**
 * Run battery of tests for different data types and reduce ops
 */
template <
    int WARPS,
    int LOGICAL_WARP_THREADS>
void Test(GenMode gen_mode)
{
    // primitive
    Test<WARPS, LOGICAL_WARP_THREADS, char>(                gen_mode, Sum());
    Test<WARPS, LOGICAL_WARP_THREADS, short>(               gen_mode, Sum());
    Test<WARPS, LOGICAL_WARP_THREADS, int>(                 gen_mode, Sum());
    Test<WARPS, LOGICAL_WARP_THREADS, long long>(           gen_mode, Sum());

    Test<WARPS, LOGICAL_WARP_THREADS, unsigned char>(       gen_mode, Sum());
    Test<WARPS, LOGICAL_WARP_THREADS, unsigned short>(      gen_mode, Sum());
    Test<WARPS, LOGICAL_WARP_THREADS, unsigned int>(        gen_mode, Sum());
    Test<WARPS, LOGICAL_WARP_THREADS, unsigned long long>(  gen_mode, Sum());

    if (gen_mode != RANDOM)
    {
        Test<WARPS, LOGICAL_WARP_THREADS, float>(           gen_mode, Sum());
        Test<WARPS, LOGICAL_WARP_THREADS, double>(          gen_mode, Sum());
    }

    // primitive (alternative reduce op)
    Test<WARPS, LOGICAL_WARP_THREADS, unsigned char>(       gen_mode, Max());
    Test<WARPS, LOGICAL_WARP_THREADS, unsigned short>(      gen_mode, Max());
    Test<WARPS, LOGICAL_WARP_THREADS, unsigned int>(        gen_mode, Max());
    Test<WARPS, LOGICAL_WARP_THREADS, unsigned long long>(  gen_mode, Max());

    // vec-1
    Test<WARPS, LOGICAL_WARP_THREADS, uchar1>(              gen_mode, Sum());

    // vec-2
    Test<WARPS, LOGICAL_WARP_THREADS, uchar2>(              gen_mode, Sum());
    Test<WARPS, LOGICAL_WARP_THREADS, ushort2>(             gen_mode, Sum());
    Test<WARPS, LOGICAL_WARP_THREADS, uint2>(               gen_mode, Sum());
    Test<WARPS, LOGICAL_WARP_THREADS, ulonglong2>(          gen_mode, Sum());

    // vec-4
    Test<WARPS, LOGICAL_WARP_THREADS, uchar4>(              gen_mode, Sum());
    Test<WARPS, LOGICAL_WARP_THREADS, ushort4>(             gen_mode, Sum());
    Test<WARPS, LOGICAL_WARP_THREADS, uint4>(               gen_mode, Sum());
    Test<WARPS, LOGICAL_WARP_THREADS, ulonglong4>(          gen_mode, Sum());

    // complex
    Test<WARPS, LOGICAL_WARP_THREADS, TestFoo>(             gen_mode, Sum());
    Test<WARPS, LOGICAL_WARP_THREADS, TestBar>(             gen_mode, Sum());
}


/**
 * Run battery of tests for different problem generation options
 */
template <
    int WARPS,
    int LOGICAL_WARP_THREADS>
void Test()
{
    Test<WARPS, LOGICAL_WARP_THREADS>(UNIFORM);
    Test<WARPS, LOGICAL_WARP_THREADS>(INTEGER_SEED);
    Test<WARPS, LOGICAL_WARP_THREADS>(RANDOM);
}


/**
 * Run battery of tests for different number of active warps
 */
template <int LOGICAL_WARP_THREADS>
void Test()
{
    Test<1, LOGICAL_WARP_THREADS>();

    // Only power-of-two subwarps can be tiled
    if ((LOGICAL_WARP_THREADS == 32) || PowerOfTwo<LOGICAL_WARP_THREADS>::VALUE)
        Test<2, LOGICAL_WARP_THREADS>();
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("repeat", g_repeat);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--repeat=<repetitions of entire test suite>]"
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

#ifdef QUICK_TEST

    // Compile/run quick tests
    TestReduce<1, 32, int>(UNIFORM, Sum());

    TestReduce<1, 32, double>(UNIFORM, Sum());
    TestReduce<2, 16, TestBar>(UNIFORM, Sum());
    TestSegmentedReduce<1, 32, int>(UNIFORM, 1, Sum());

#else

    // Compile/run thorough tests
    for (int i = 0; i <= g_repeat; ++i)
    {
        // Test logical warp sizes
        Test<32>();
        Test<16>();
        Test<9>();
        Test<7>();
    }

#endif

    return 0;
}




