#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of BlockRadixSort utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <algorithm>
#include <iostream>

#include <cub/block/block_radix_sort.cuh>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/util_allocator.cuh>

#include "test_util.h"

using namespace hipcub;

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose = false;
CachingDeviceAllocator  g_allocator(true);


//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------


/// Specialized descending, blocked -> blocked
template <int BLOCK_THREADS, typename BlockRadixSort, int ITEMS_PER_THREAD, typename Key, typename Value>
__device__ __forceinline__ void TestBlockSort(
    typename BlockRadixSort::TempStorage &temp_storage,
    Key                         (&keys)[ITEMS_PER_THREAD],
    Value                       (&values)[ITEMS_PER_THREAD],
    Key                         *d_keys,
    Value                       *d_values,
    int                         begin_bit,
    int                         end_bit,
    clock_t                     &stop,
    Int2Type<true>              is_descending,
    Int2Type<true>              is_blocked_output)
{
    BlockRadixSort(temp_storage).SortDescending(keys, values, begin_bit, end_bit);
    stop = clock();
    StoreDirectBlocked(threadIdx.x, d_keys, keys);
    StoreDirectBlocked(threadIdx.x, d_values, values);
}

/// Specialized descending, blocked -> striped
template <int BLOCK_THREADS, typename BlockRadixSort, int ITEMS_PER_THREAD, typename Key, typename Value>
__device__ __forceinline__ void TestBlockSort(
    typename BlockRadixSort::TempStorage &temp_storage,
    Key                         (&keys)[ITEMS_PER_THREAD],
    Value                       (&values)[ITEMS_PER_THREAD],
    Key                         *d_keys,
    Value                       *d_values,
    int                         begin_bit,
    int                         end_bit,
    clock_t                     &stop,
    Int2Type<true>              is_descending,
    Int2Type<false>             is_blocked_output)
{
    BlockRadixSort(temp_storage).SortDescendingBlockedToStriped(keys, values, begin_bit, end_bit);
    stop = clock();
    StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_keys, keys);
    StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_values, values);
}

/// Specialized ascending, blocked -> blocked
template <int BLOCK_THREADS, typename BlockRadixSort, int ITEMS_PER_THREAD, typename Key, typename Value>
__device__ __forceinline__ void TestBlockSort(
    typename BlockRadixSort::TempStorage &temp_storage,
    Key                         (&keys)[ITEMS_PER_THREAD],
    Value                       (&values)[ITEMS_PER_THREAD],
    Key                         *d_keys,
    Value                       *d_values,
    int                         begin_bit,
    int                         end_bit,
    clock_t                     &stop,
    Int2Type<false>             is_descending,
    Int2Type<true>              is_blocked_output)
{
    BlockRadixSort(temp_storage).Sort(keys, values, begin_bit, end_bit);
    stop = clock();
    StoreDirectBlocked(threadIdx.x, d_keys, keys);
    StoreDirectBlocked(threadIdx.x, d_values, values);
}

/// Specialized ascending, blocked -> striped
template <int BLOCK_THREADS, typename BlockRadixSort, int ITEMS_PER_THREAD, typename Key, typename Value>
__device__ __forceinline__ void TestBlockSort(
    typename BlockRadixSort::TempStorage &temp_storage,
    Key                         (&keys)[ITEMS_PER_THREAD],
    Value                       (&values)[ITEMS_PER_THREAD],
    Key                         *d_keys,
    Value                       *d_values,
    int                         begin_bit,
    int                         end_bit,
    clock_t                     &stop,
    Int2Type<false>             is_descending,
    Int2Type<false>             is_blocked_output)
{
    BlockRadixSort(temp_storage).SortBlockedToStriped(keys, values, begin_bit, end_bit);
    stop = clock();
    StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_keys, keys);
    StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_values, values);
}



/**
 * BlockRadixSort kernel
 */
template <
    int                 BLOCK_THREADS,
    int                 ITEMS_PER_THREAD,
    int                 RADIX_BITS,
    bool                MEMOIZE_OUTER_SCAN,
    BlockScanAlgorithm  INNER_SCAN_ALGORITHM,
    hipSharedMemConfig SMEM_CONFIG,
    int                 DESCENDING,
    int                 BLOCKED_OUTPUT,
    typename            Key,
    typename            Value>
__launch_bounds__ (BLOCK_THREADS, 1)
__global__ void Kernel(
    Key                         *d_keys,
    Value                       *d_values,
    int                         begin_bit,
    int                         end_bit,
    clock_t                     *d_elapsed)
{
    // Threadblock load/store abstraction types
    typedef BlockRadixSort<
            Key,
            BLOCK_THREADS,
            ITEMS_PER_THREAD,
            Value,
            RADIX_BITS,
            MEMOIZE_OUTER_SCAN,
            INNER_SCAN_ALGORITHM,
            SMEM_CONFIG>
        BlockRadixSortT;

    // Allocate temp storage in shared memory
    __shared__ typename BlockRadixSortT::TempStorage temp_storage;

    // Items per thread
    Key     keys[ITEMS_PER_THREAD];
    Value   values[ITEMS_PER_THREAD];

    LoadDirectBlocked(threadIdx.x, d_keys, keys);
    LoadDirectBlocked(threadIdx.x, d_values, values);

    // Start cycle timer
    clock_t stop;
    clock_t start = clock();

    TestBlockSort<BLOCK_THREADS, BlockRadixSortT>(
        temp_storage, keys, values, d_keys, d_values, begin_bit, end_bit, stop, Int2Type<DESCENDING>(), Int2Type<BLOCKED_OUTPUT>());

    // Store time
    if (threadIdx.x == 0)
        *d_elapsed = (start > stop) ? start - stop : stop - start;
}



//---------------------------------------------------------------------
// Host testing subroutines
//---------------------------------------------------------------------


/**
 * Simple key-value pairing
 */
template <
    typename Key,
    typename Value,
    bool IS_FLOAT = (Traits<Key>::CATEGORY == FLOATING_POINT)>
struct Pair
{
    Key     key;
    Value   value;

    bool operator<(const Pair &b) const
    {
        return (key < b.key);
    }
};

/**
 * Simple key-value pairing (specialized for floating point types)
 */
template <typename Key, typename Value>
struct Pair<Key, Value, true>
{
    Key     key;
    Value   value;

    bool operator<(const Pair &b) const
    {
        if (key < b.key)
            return true;

        if (key > b.key)
            return false;

        // Key in unsigned bits
        typedef typename Traits<Key>::UnsignedBits UnsignedBits;

        // Return true if key is negative zero and b.key is positive zero
        UnsignedBits key_bits   = SafeBitCast<UnsignedBits>(key);
        UnsignedBits b_key_bits = SafeBitCast<UnsignedBits>(b.key);
        UnsignedBits HIGH_BIT   = Traits<Key>::HIGH_BIT;

        return ((key_bits & HIGH_BIT) != 0) && ((b_key_bits & HIGH_BIT) == 0);
    }
};


/**
 * Initialize key-value sorting problem.
 */
template <bool DESCENDING, typename Key, typename Value>
void Initialize(
    GenMode         gen_mode,
    Key             *h_keys,
    Value           *h_values,
    Key             *h_reference_keys,
    Value           *h_reference_values,
    int             num_items,
    int             entropy_reduction,
    int             begin_bit,
    int             end_bit)
{
    (void)entropy_reduction; // unused

    Pair<Key, Value> *h_pairs = new Pair<Key, Value>[num_items];

    for (int i = 0; i < num_items; ++i)
    {
        InitValue(gen_mode, h_keys[i], i);

        RandomBits(h_values[i]);

        // Mask off unwanted portions
        int num_bits = end_bit - begin_bit;
        if ((begin_bit > 0) || (end_bit < static_cast<int>(sizeof(Key) * 8)))
        {
            unsigned long long base = 0;
            memcpy(&base, &h_keys[i], sizeof(Key));
            base &= ((1ull << num_bits) - 1) << begin_bit;
            memcpy(&h_keys[i], &base, sizeof(Key));
        }

        h_pairs[i].key    = h_keys[i];
        h_pairs[i].value  = h_values[i];
    }

    if (DESCENDING) std::reverse(h_pairs, h_pairs + num_items);
    std::stable_sort(h_pairs, h_pairs + num_items);
    if (DESCENDING) std::reverse(h_pairs, h_pairs + num_items);

    for (int i = 0; i < num_items; ++i)
    {
        h_reference_keys[i]     = h_pairs[i].key;
        h_reference_values[i]   = h_pairs[i].value;
    }

    delete[] h_pairs;
}




/**
 * Test BlockRadixSort kernel
 */
template <
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    int                     RADIX_BITS,
    bool                    MEMOIZE_OUTER_SCAN,
    BlockScanAlgorithm      INNER_SCAN_ALGORITHM,
    hipSharedMemConfig     SMEM_CONFIG,
    bool                    DESCENDING,
    bool                    BLOCKED_OUTPUT,
    typename                Key,
    typename                Value>
void TestDriver(
    GenMode                 gen_mode,
    int                     entropy_reduction,
    int                     begin_bit,
    int                     end_bit)
{
    enum
    {
        TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD,
        KEYS_ONLY = Equals<Value, NullType>::VALUE,
    };

    // Allocate host arrays
    Key     *h_keys             = new Key[TILE_SIZE];
    Key     *h_reference_keys   = new Key[TILE_SIZE];
    Value   *h_values           = new Value[TILE_SIZE];
    Value   *h_reference_values = new Value[TILE_SIZE];

    // Allocate device arrays
    Key     *d_keys     = NULL;
    Value   *d_values   = NULL;
    clock_t *d_elapsed  = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys, sizeof(Key) * TILE_SIZE));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values, sizeof(Value) * TILE_SIZE));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_elapsed, sizeof(clock_t)));

    // Initialize problem and solution on host
    Initialize<DESCENDING>(gen_mode, h_keys, h_values, h_reference_keys, h_reference_values,
        TILE_SIZE, entropy_reduction, begin_bit, end_bit);

    // Copy problem to device
    CubDebugExit(hipMemcpy(d_keys, h_keys, sizeof(Key) * TILE_SIZE, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_values, h_values, sizeof(Value) * TILE_SIZE, hipMemcpyHostToDevice));

    printf("%s "
        "BLOCK_THREADS(%d) "
        "ITEMS_PER_THREAD(%d) "
        "RADIX_BITS(%d) "
        "MEMOIZE_OUTER_SCAN(%d) "
        "INNER_SCAN_ALGORITHM(%d) "
        "SMEM_CONFIG(%d) "
        "DESCENDING(%d) "
        "BLOCKED_OUTPUT(%d) "
        "sizeof(Key)(%d) "
        "sizeof(Value)(%d) "
        "gen_mode(%d), "
        "entropy_reduction(%d) "
        "begin_bit(%d) "
        "end_bit(%d), "
        "samples(%d)\n",
            ((KEYS_ONLY) ? "Keys-only" : "Key-value"),
            BLOCK_THREADS,
            ITEMS_PER_THREAD,
            RADIX_BITS,
            MEMOIZE_OUTER_SCAN,
            INNER_SCAN_ALGORITHM,
            SMEM_CONFIG,
            DESCENDING,
            BLOCKED_OUTPUT,
            (int) sizeof(Key),
            (int) sizeof(Value),
            gen_mode,
            entropy_reduction,
            begin_bit,
            end_bit,
            g_num_rand_samples);

    // Set shared memory config
    hipDeviceSetSharedMemConfig(SMEM_CONFIG);

    // Run kernel
    Kernel<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, SMEM_CONFIG, DESCENDING, BLOCKED_OUTPUT><<<1, BLOCK_THREADS>>>(
        d_keys, d_values, begin_bit, end_bit, d_elapsed);

    // Flush kernel output / errors
    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());

    // Check keys results
    printf("\tKeys: ");
    int compare = CompareDeviceResults(h_reference_keys, d_keys, TILE_SIZE, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Check value results
    if (!KEYS_ONLY)
    {
        printf("\tValues: ");
        int compare = CompareDeviceResults(h_reference_values, d_values, TILE_SIZE, g_verbose, g_verbose);
        printf("%s\n", compare ? "FAIL" : "PASS");
        AssertEquals(0, compare);
    }
    printf("\n");

    printf("\tElapsed clocks: ");
    DisplayDeviceResults(d_elapsed, 1);
    printf("\n");

    // Cleanup
    if (h_keys)             delete[] h_keys;
    if (h_reference_keys)   delete[] h_reference_keys;
    if (h_values)           delete[] h_values;
    if (h_reference_values) delete[] h_reference_values;
    if (d_keys)             CubDebugExit(g_allocator.DeviceFree(d_keys));
    if (d_values)           CubDebugExit(g_allocator.DeviceFree(d_values));
    if (d_elapsed)          CubDebugExit(g_allocator.DeviceFree(d_elapsed));
}


/**
 * Test driver (valid tile size <= MAX_SMEM_BYTES)
 */
template <
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    int                     RADIX_BITS,
    bool                    MEMOIZE_OUTER_SCAN,
    BlockScanAlgorithm      INNER_SCAN_ALGORITHM,
    hipSharedMemConfig     SMEM_CONFIG,
    bool                    DESCENDING,
    bool                    BLOCKED_OUTPUT,
    typename                Key,
    typename                Value>
void TestValid(Int2Type<true> /*fits_smem_capacity*/)
{
    // Iterate begin_bit
    for (int begin_bit = 0; begin_bit <= 1; begin_bit++)
    {
        // Iterate end bit
        for (int end_bit = begin_bit + 1;
             end_bit <= static_cast<int>(sizeof(Key) * 8);
             end_bit = end_bit * 2 + begin_bit)
        {
            // Uniform key distribution
            TestDriver<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, SMEM_CONFIG, DESCENDING, BLOCKED_OUTPUT, Key, Value>(
                UNIFORM, 0, begin_bit, end_bit);

            // Sequential key distribution
            TestDriver<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, SMEM_CONFIG, DESCENDING, BLOCKED_OUTPUT, Key, Value>(
                INTEGER_SEED, 0, begin_bit, end_bit);

            // Iterate random with entropy_reduction
            for (int entropy_reduction = 0; entropy_reduction <= 9; entropy_reduction += 3)
            {
                TestDriver<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, SMEM_CONFIG, DESCENDING, BLOCKED_OUTPUT, Key, Value>(
                    RANDOM, entropy_reduction, begin_bit, end_bit);
            }
        }
    }
}


/**
 * Test driver (invalid tile size)
 */
template <
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    int                     RADIX_BITS,
    bool                    MEMOIZE_OUTER_SCAN,
    BlockScanAlgorithm      INNER_SCAN_ALGORITHM,
    hipSharedMemConfig     SMEM_CONFIG,
    bool                    DESCENDING,
    bool                    BLOCKED_OUTPUT,
    typename                Key,
    typename                Value>
void TestValid(Int2Type<false> fits_smem_capacity)
{}


/**
 * Test ascending/descending and to-blocked/to-striped
 */
template <
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    int                     RADIX_BITS,
    bool                    MEMOIZE_OUTER_SCAN,
    BlockScanAlgorithm      INNER_SCAN_ALGORITHM,
    hipSharedMemConfig     SMEM_CONFIG,
    typename                Key,
    typename                Value>
void Test()
{
    // Check size of smem storage for the target arch to make sure it will fit
    typedef BlockRadixSort<Key, BLOCK_THREADS, ITEMS_PER_THREAD, Value, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, SMEM_CONFIG> BlockRadixSortT;

#if defined(SM100) || defined(SM110) || defined(SM130)
    Int2Type<sizeof(typename BlockRadixSortT::TempStorage) <= 16 * 1024> fits_smem_capacity;
#else
    Int2Type<(sizeof(typename BlockRadixSortT::TempStorage) <= 48 * 1024)> fits_smem_capacity;
#endif

    // Sort-ascending, to-striped
    TestValid<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, SMEM_CONFIG, true, false, Key, Value>(fits_smem_capacity);

    // Sort-descending, to-blocked
    TestValid<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, SMEM_CONFIG, false, true, Key, Value>(fits_smem_capacity);

    // Not necessary
//    TestValid<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, SMEM_CONFIG, false, false, Key, Value>(fits_smem_capacity);
//    TestValid<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, SMEM_CONFIG, true, true, Key, Value>(fits_smem_capacity);
}


/**
 * Test value type and smem config
 */
template <
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    int                     RADIX_BITS,
    bool                    MEMOIZE_OUTER_SCAN,
    BlockScanAlgorithm      INNER_SCAN_ALGORITHM,
    typename                Key>
void TestKeys()
{
    // Test keys-only sorting with both smem configs
    Test<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, hipSharedMemBankSizeFourByte, Key, NullType>();    // Keys-only (4-byte smem bank config)
#if !defined(SM100) && !defined(SM110) && !defined(SM130) && !defined(SM200)
    Test<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, hipSharedMemBankSizeEightByte, Key, NullType>();   // Keys-only (8-byte smem bank config)
#endif
}


/**
 * Test value type and smem config
 */
template <
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    int                     RADIX_BITS,
    bool                    MEMOIZE_OUTER_SCAN,
    BlockScanAlgorithm      INNER_SCAN_ALGORITHM,
    typename                Key>
void TestKeysAndPairs()
{
    // Test pairs sorting with only 4-byte configs
    Test<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, hipSharedMemBankSizeFourByte, Key, char>();        // With small-values
    Test<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, hipSharedMemBankSizeFourByte, Key, Key>();         // With same-values
    Test<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, hipSharedMemBankSizeFourByte, Key, TestFoo>();     // With large values
}


/**
 * Test key type
 */
template <
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    int                     RADIX_BITS,
    bool                    MEMOIZE_OUTER_SCAN,
    BlockScanAlgorithm      INNER_SCAN_ALGORITHM>
void Test()
{
    // Get ptx version
    int ptx_version = 0;
    CubDebugExit(PtxVersion(ptx_version));

#ifdef TEST_KEYS_ONLY

    // Test unsigned types with keys-only
    TestKeys<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, unsigned char>();
    TestKeys<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, unsigned short>();
    TestKeys<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, unsigned int>();
    TestKeys<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, unsigned long>();
    TestKeys<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, unsigned long long>();

#else

    // Test signed and fp types with paired values
    TestKeysAndPairs<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, char>();
    TestKeysAndPairs<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, short>();
    TestKeysAndPairs<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, int>();
    TestKeysAndPairs<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, long>();
    TestKeysAndPairs<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, long long>();
    TestKeysAndPairs<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, float>();
    if (ptx_version > 120)
    {
        // Don't check doubles on PTX120 or below because they're down-converted
        TestKeysAndPairs<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, INNER_SCAN_ALGORITHM, double>();
    }

#endif
}


/**
 * Test inner scan algorithm
 */
template <
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    int                     RADIX_BITS,
    bool                    MEMOIZE_OUTER_SCAN>
void Test()
{
    Test<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, BLOCK_SCAN_RAKING>();
    Test<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, MEMOIZE_OUTER_SCAN, BLOCK_SCAN_WARP_SCANS>();
}


/**
 * Test outer scan algorithm
 */
template <
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    int                     RADIX_BITS>
void Test()
{
    Test<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, true>();
    Test<BLOCK_THREADS, ITEMS_PER_THREAD, RADIX_BITS, false>();
}


/**
 * Test radix bits
 */
template <
    int BLOCK_THREADS,
    int ITEMS_PER_THREAD>
void Test()
{
    Test<BLOCK_THREADS, ITEMS_PER_THREAD, 1>();
    Test<BLOCK_THREADS, ITEMS_PER_THREAD, 2>();
    Test<BLOCK_THREADS, ITEMS_PER_THREAD, 5>();
}


/**
 * Test items per thread
 */
template <int BLOCK_THREADS>
void Test()
{
    Test<BLOCK_THREADS, 1>();
#if defined(SM100) || defined(SM110) || defined(SM130)
    // Open64 compiler can't handle the number of test cases
#else
    Test<BLOCK_THREADS, 4>();
#endif
    Test<BLOCK_THREADS, 11>();
}



/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

#ifdef QUICK_TEST

    {
        typedef float T;
        TestDriver<32, 4, 4, true, BLOCK_SCAN_WARP_SCANS, hipSharedMemBankSizeFourByte, false, false, T, NullType>(INTEGER_SEED, 0, 0, sizeof(T) * 8);
    }
/*
    // Compile/run quick tests
    typedef unsigned int T;
    TestDriver<64, 17, 4, true, BLOCK_SCAN_WARP_SCANS, hipSharedMemBankSizeFourByte, false, false, T, NullType>(RANDOM, 0, 0, sizeof(T) * 8);
    TestDriver<96, 8, 4, true, BLOCK_SCAN_WARP_SCANS, hipSharedMemBankSizeFourByte, false, false, T, NullType>(RANDOM, 0, 0, sizeof(T) * 8);
    TestDriver<128, 2, 4, true, BLOCK_SCAN_WARP_SCANS, hipSharedMemBankSizeFourByte, false, false, T, NullType>(RANDOM, 0, 0, sizeof(T) * 8);
*/

#else

    // Compile/run thorough tests
    Test<32>();
    Test<64>();
    Test<160>();


#endif  // QUICK_TEST

    return 0;
}



