#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2016, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of DeviceHistogram utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <limits>
#include <algorithm>
#include <typeinfo>

#if defined(QUICK_TEST) || defined(QUICKER_TEST)
    #include <npp.h>
#endif

#include <cub/util_allocator.cuh>
#include <cub/device/device_histogram.cuh>

#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------


// Dispatch types
enum Backend
{
    CUB,        // CUB method
    NPP,        // NPP method
    CDP,        // GPU-based (dynamic parallelism) dispatch to CUB method
};


bool                    g_verbose_input     = false;
bool                    g_verbose           = false;
int                     g_timing_iterations = 0;
int                     g_repeat            = 0;
CachingDeviceAllocator  g_allocator(true);




//---------------------------------------------------------------------
// Dispatch to NPP histogram
//---------------------------------------------------------------------

#if defined(QUICK_TEST) || defined(QUICKER_TEST)

/**
 * Dispatch to single-channel 8b NPP histo-even
 */
template <typename CounterT, typename LevelT, typename OffsetT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t DispatchEven(
    Int2Type<1>             num_channels,
    Int2Type<1>             num_active_channels,
    Int2Type<NPP>           dispatch_to,
    int                     timing_timing_iterations,
    size_t                  *d_temp_storage_bytes,
    hipError_t             *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    unsigned char       *d_samples,               ///< [in] The pointer to the multi-channel input sequence of data samples. The samples from different channels are assumed to be interleaved (e.g., an array of 32-bit pixels where each pixel consists of four RGBA 8-bit samples).
    CounterT            *d_histogram[1],          ///< [out] The pointers to the histogram counter output arrays, one for each active channel.  For channel<sub><em>i</em></sub>, the allocation length of <tt>d_histograms[i]</tt> should be <tt>num_levels[i]</tt> - 1.
    int                 num_levels[1],            ///< [in] The number of boundaries (levels) for delineating histogram samples in each active channel.  Implies that the number of bins for channel<sub><em>i</em></sub> is <tt>num_levels[i]</tt> - 1.
    LevelT              lower_level[1],           ///< [in] The lower sample value bound (inclusive) for the lowest histogram bin in each active channel.
    LevelT              upper_level[1],           ///< [in] The upper sample value bound (exclusive) for the highest histogram bin in each active channel.
    OffsetT             num_row_pixels,           ///< [in] The number of multi-channel pixels per row in the region of interest
    OffsetT             num_rows,                 ///< [in] The number of rows in the region of interest
    OffsetT             row_stride_bytes,         ///< [in] The number of bytes between starts of consecutive rows in the region of interest
    hipStream_t        stream,
    bool                debug_synchronous)
{
    typedef unsigned char SampleT;

    hipError_t error = hipSuccess;
    NppiSize oSizeROI = {
        num_row_pixels,
        num_rows
    };

    if (d_temp_storage_bytes == NULL)
    {
        int nDeviceBufferSize;
        nppiHistogramEvenGetBufferSize_8u_C1R(oSizeROI, num_levels[0] ,&nDeviceBufferSize);
        temp_storage_bytes = nDeviceBufferSize;
    }
    else
    {
        for (int i = 0; i < timing_timing_iterations; ++i)
        {
            // compute the histogram
            nppiHistogramEven_8u_C1R(
                d_samples,
                row_stride_bytes,
                oSizeROI,
                d_histogram[0],
                num_levels[0],
                lower_level[0],
                upper_level[0],
                (Npp8u*) d_temp_storage);
        }
    }

    return error;
}


/**
 * Dispatch to 3/4 8b NPP histo-even
 */
template <typename CounterT, typename LevelT, typename OffsetT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t DispatchEven(
    Int2Type<4>          num_channels,
    Int2Type<3>   num_active_channels,
    Int2Type<NPP>           dispatch_to,
    int                     timing_timing_iterations,
    size_t                  *d_temp_storage_bytes,
    hipError_t             *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    unsigned char       *d_samples,               ///< [in] The pointer to the multi-channel input sequence of data samples. The samples from different channels are assumed to be interleaved (e.g., an array of 32-bit pixels where each pixel consists of four RGBA 8-bit samples).
    CounterT            *d_histogram[3],          ///< [out] The pointers to the histogram counter output arrays, one for each active channel.  For channel<sub><em>i</em></sub>, the allocation length of <tt>d_histograms[i]</tt> should be <tt>num_levels[i]</tt> - 1.
    int                 num_levels[3],            ///< [in] The number of boundaries (levels) for delineating histogram samples in each active channel.  Implies that the number of bins for channel<sub><em>i</em></sub> is <tt>num_levels[i]</tt> - 1.
    LevelT              lower_level[3],           ///< [in] The lower sample value bound (inclusive) for the lowest histogram bin in each active channel.
    LevelT              upper_level[3],           ///< [in] The upper sample value bound (exclusive) for the highest histogram bin in each active channel.
    OffsetT             num_row_pixels,           ///< [in] The number of multi-channel pixels per row in the region of interest
    OffsetT             num_rows,                 ///< [in] The number of rows in the region of interest
    OffsetT             row_stride_bytes,         ///< [in] The number of bytes between starts of consecutive rows in the region of interest
    hipStream_t        stream,
    bool                debug_synchronous)
{
    typedef unsigned char SampleT;

    hipError_t error = hipSuccess;
    NppiSize oSizeROI = {
        num_row_pixels,
        num_rows
    };

    if (d_temp_storage_bytes == NULL)
    {
        int nDeviceBufferSize;
        nppiHistogramEvenGetBufferSize_8u_AC4R(oSizeROI, num_levels ,&nDeviceBufferSize);
        temp_storage_bytes = nDeviceBufferSize;
    }
    else
    {
        for (int i = 0; i < timing_timing_iterations; ++i)
        {
            // compute the histogram
            nppiHistogramEven_8u_AC4R(
                d_samples,
                row_stride_bytes,
                oSizeROI,
                d_histogram,
                num_levels,
                lower_level,
                upper_level,
                (Npp8u*) d_temp_storage);
        }
    }

    return error;
}


#endif // #if defined(QUICK_TEST) || defined(QUICKER_TEST)


//---------------------------------------------------------------------
// Dispatch to different DeviceHistogram entrypoints
//---------------------------------------------------------------------

/**
 * Dispatch to CUB single histogram-even entrypoint
 */
template <typename SampleIteratorT, typename CounterT, typename LevelT, typename OffsetT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t DispatchEven(
    Int2Type<1>             num_channels,
    Int2Type<1>             num_active_channels,
    Int2Type<CUB>           dispatch_to,
    int                     timing_timing_iterations,
    size_t                  *d_temp_storage_bytes,
    hipError_t             *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    SampleIteratorT     d_samples,                                  ///< [in] The pointer to the multi-channel input sequence of data samples. The samples from different channels are assumed to be interleaved (e.g., an array of 32-bit pixels where each pixel consists of four RGBA 8-bit samples).
    CounterT            *d_histogram[1],                            ///< [out] The pointers to the histogram counter output arrays, one for each active channel.  For channel<sub><em>i</em></sub>, the allocation length of <tt>d_histograms[i]</tt> should be <tt>num_levels[i]</tt> - 1.
    int                 num_levels[1],                              ///< [in] The number of boundaries (levels) for delineating histogram samples in each active channel.  Implies that the number of bins for channel<sub><em>i</em></sub> is <tt>num_levels[i]</tt> - 1.
    LevelT              lower_level[1],                             ///< [in] The lower sample value bound (inclusive) for the lowest histogram bin in each active channel.
    LevelT              upper_level[1],                             ///< [in] The upper sample value bound (exclusive) for the highest histogram bin in each active channel.
    OffsetT             num_row_pixels,                             ///< [in] The number of multi-channel pixels per row in the region of interest
    OffsetT             num_rows,                                   ///< [in] The number of rows in the region of interest
    OffsetT             row_stride_bytes,                                 ///< [in] The number of bytes between starts of consecutive rows in the region of interest
    hipStream_t        stream,
    bool                debug_synchronous)
{
    typedef typename std::iterator_traits<SampleIteratorT>::value_type SampleT;

    hipError_t error = hipSuccess;
    for (int i = 0; i < timing_timing_iterations; ++i)
    {
        error = DeviceHistogram::HistogramEven(
            d_temp_storage,
            temp_storage_bytes,
            (const SampleT *) d_samples,
            d_histogram[0],
            num_levels[0],
            lower_level[0],
            upper_level[0],
            num_row_pixels,
            num_rows,
            row_stride_bytes,
            stream,
            debug_synchronous);
    }
    return error;
}

/**
 * Dispatch to CUB multi histogram-even entrypoint
 */
template <int NUM_ACTIVE_CHANNELS, int NUM_CHANNELS, typename SampleIteratorT, typename CounterT, typename LevelT, typename OffsetT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t DispatchEven(
    Int2Type<NUM_CHANNELS>          num_channels,
    Int2Type<NUM_ACTIVE_CHANNELS>   num_active_channels,
    Int2Type<CUB>           dispatch_to,
    int                     timing_timing_iterations,
    size_t                  *d_temp_storage_bytes,
    hipError_t             *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    SampleIteratorT     d_samples,                                  ///< [in] The pointer to the multi-channel input sequence of data samples. The samples from different channels are assumed to be interleaved (e.g., an array of 32-bit pixels where each pixel consists of four RGBA 8-bit samples).
    CounterT            *d_histogram[NUM_ACTIVE_CHANNELS],          ///< [out] The pointers to the histogram counter output arrays, one for each active channel.  For channel<sub><em>i</em></sub>, the allocation length of <tt>d_histograms[i]</tt> should be <tt>num_levels[i]</tt> - 1.
    int                 num_levels[NUM_ACTIVE_CHANNELS],            ///< [in] The number of boundaries (levels) for delineating histogram samples in each active channel.  Implies that the number of bins for channel<sub><em>i</em></sub> is <tt>num_levels[i]</tt> - 1.
    LevelT              lower_level[NUM_ACTIVE_CHANNELS],           ///< [in] The lower sample value bound (inclusive) for the lowest histogram bin in each active channel.
    LevelT              upper_level[NUM_ACTIVE_CHANNELS],           ///< [in] The upper sample value bound (exclusive) for the highest histogram bin in each active channel.
    OffsetT             num_row_pixels,                             ///< [in] The number of multi-channel pixels per row in the region of interest
    OffsetT             num_rows,                                   ///< [in] The number of rows in the region of interest
    OffsetT             row_stride_bytes,                                 ///< [in] The number of bytes between starts of consecutive rows in the region of interest
    hipStream_t        stream,
    bool                debug_synchronous)
{
    typedef typename std::iterator_traits<SampleIteratorT>::value_type SampleT;

    hipError_t error = hipSuccess;
    for (int i = 0; i < timing_timing_iterations; ++i)
    {
        error = DeviceHistogram::MultiHistogramEven<NUM_CHANNELS, NUM_ACTIVE_CHANNELS>(
            d_temp_storage,
            temp_storage_bytes,
            (const SampleT *) d_samples,
            d_histogram,
            num_levels,
            lower_level,
            upper_level,
            num_row_pixels,
            num_rows,
            row_stride_bytes,
            stream,
            debug_synchronous);
    }
    return error;
}


/**
 * Dispatch to CUB single histogram-range entrypoint
 */
template <typename SampleIteratorT, typename CounterT, typename LevelT, typename OffsetT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t DispatchRange(
    Int2Type<1>             num_channels,
    Int2Type<1>             num_active_channels,
    Int2Type<CUB>           dispatch_to,
    int                     timing_timing_iterations,
    size_t                  *d_temp_storage_bytes,
    hipError_t             *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    SampleIteratorT     d_samples,                                  ///< [in] The pointer to the multi-channel input sequence of data samples. The samples from different channels are assumed to be interleaved (e.g., an array of 32-bit pixels where each pixel consists of four RGBA 8-bit samples).
    CounterT            *d_histogram[1],                            ///< [out] The pointers to the histogram counter output arrays, one for each active channel.  For channel<sub><em>i</em></sub>, the allocation length of <tt>d_histograms[i]</tt> should be <tt>num_levels[i]</tt> - 1.
    int                 num_levels[1],                              ///< [in] The number of boundaries (levels) for delineating histogram samples in each active channel.  Implies that the number of bins for channel<sub><em>i</em></sub> is <tt>num_levels[i]</tt> - 1.
    LevelT              *d_levels[1],                               ///< [in] The pointers to the arrays of boundaries (levels), one for each active channel.  Bin ranges are defined by consecutive boundary pairings: lower sample value boundaries are inclusive and upper sample value boundaries are exclusive.
    OffsetT             num_row_pixels,                             ///< [in] The number of multi-channel pixels per row in the region of interest
    OffsetT             num_rows,                                   ///< [in] The number of rows in the region of interest
    OffsetT             row_stride_bytes,                                 ///< [in] The number of bytes between starts of consecutive rows in the region of interest
    hipStream_t        stream,
    bool                debug_synchronous)
{
    typedef typename std::iterator_traits<SampleIteratorT>::value_type SampleT;

    hipError_t error = hipSuccess;
    for (int i = 0; i < timing_timing_iterations; ++i)
    {
        error = DeviceHistogram::HistogramRange(
            d_temp_storage,
            temp_storage_bytes,
            (const SampleT *) d_samples,
            d_histogram[0],
            num_levels[0],
            d_levels[0],
            num_row_pixels,
            num_rows,
            row_stride_bytes,
            stream,
            debug_synchronous);
    }
    return error;
}


/**
 * Dispatch to CUB multi histogram-range entrypoint
 */
template <int NUM_ACTIVE_CHANNELS, int NUM_CHANNELS, typename SampleIteratorT, typename CounterT, typename LevelT, typename OffsetT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t DispatchRange(
    Int2Type<NUM_CHANNELS>          num_channels,
    Int2Type<NUM_ACTIVE_CHANNELS>   num_active_channels,
    Int2Type<CUB>           dispatch_to,
    int                     timing_timing_iterations,
    size_t                  *d_temp_storage_bytes,
    hipError_t             *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    SampleIteratorT     d_samples,                                  ///< [in] The pointer to the multi-channel input sequence of data samples. The samples from different channels are assumed to be interleaved (e.g., an array of 32-bit pixels where each pixel consists of four RGBA 8-bit samples).
    CounterT            *d_histogram[NUM_ACTIVE_CHANNELS],          ///< [out] The pointers to the histogram counter output arrays, one for each active channel.  For channel<sub><em>i</em></sub>, the allocation length of <tt>d_histograms[i]</tt> should be <tt>num_levels[i]</tt> - 1.
    int                 num_levels[NUM_ACTIVE_CHANNELS],            ///< [in] The number of boundaries (levels) for delineating histogram samples in each active channel.  Implies that the number of bins for channel<sub><em>i</em></sub> is <tt>num_levels[i]</tt> - 1.
    LevelT              *d_levels[NUM_ACTIVE_CHANNELS],             ///< [in] The pointers to the arrays of boundaries (levels), one for each active channel.  Bin ranges are defined by consecutive boundary pairings: lower sample value boundaries are inclusive and upper sample value boundaries are exclusive.
    OffsetT             num_row_pixels,                             ///< [in] The number of multi-channel pixels per row in the region of interest
    OffsetT             num_rows,                                   ///< [in] The number of rows in the region of interest
    OffsetT             row_stride_bytes,                                 ///< [in] The number of bytes between starts of consecutive rows in the region of interest
    hipStream_t        stream,
    bool                debug_synchronous)
{
    typedef typename std::iterator_traits<SampleIteratorT>::value_type SampleT;

    hipError_t error = hipSuccess;
    for (int i = 0; i < timing_timing_iterations; ++i)
    {
        error = DeviceHistogram::MultiHistogramRange<NUM_CHANNELS, NUM_ACTIVE_CHANNELS>(
            d_temp_storage,
            temp_storage_bytes,
            (const SampleT *) d_samples,
            d_histogram,
            num_levels,
            d_levels,
            num_row_pixels,
            num_rows,
            row_stride_bytes,
            stream,
            debug_synchronous);
    }
    return error;
}



//---------------------------------------------------------------------
// CUDA nested-parallelism test kernel
//---------------------------------------------------------------------

/**
 * Simple wrapper kernel to invoke DeviceHistogram
 * /
template <int BINS, int NUM_CHANNELS, int NUM_ACTIVE_CHANNELS, typename SampleT, typename SampleIteratorT, typename CounterT, int ALGORITHM>
__global__ void CnpDispatchKernel(
    Int2Type<ALGORITHM> algorithm,
    int                 timing_timing_iterations,
    size_t              *d_temp_storage_bytes,
    hipError_t         *d_cdp_error,

    void*               d_temp_storage,
    size_t              temp_storage_bytes,
    SampleT             *d_samples,
    SampleIteratorT      d_sample_itr,
    ArrayWrapper<CounterT*, NUM_ACTIVE_CHANNELS> d_out_histograms,
    int                 num_samples,
    bool                debug_synchronous)
{
#ifndef CUB_CDP
    *d_cdp_error = hipErrorNotSupported;
#else
    *d_cdp_error = Dispatch<BINS, NUM_CHANNELS, NUM_ACTIVE_CHANNELS>(algorithm, Int2Type<false>(), timing_timing_iterations, d_temp_storage_bytes, d_cdp_error, d_temp_storage, temp_storage_bytes, d_samples, d_sample_itr, d_out_histograms.array, num_samples, 0, debug_synchronous);
    *d_temp_storage_bytes = temp_storage_bytes;
#endif
}


/ **
 * Dispatch to CDP kernel
 * /
template <int BINS, int NUM_CHANNELS, int NUM_ACTIVE_CHANNELS, typename SampleT, typename SampleIteratorT, typename CounterT, int ALGORITHM>
hipError_t Dispatch(
    Int2Type<ALGORITHM> algorithm,
    Int2Type<true>      use_cdp,
    int                 timing_timing_iterations,
    size_t              *d_temp_storage_bytes,
    hipError_t         *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    SampleT             *d_samples,
    SampleIteratorT      d_sample_itr,
    CounterT        *d_histograms[NUM_ACTIVE_CHANNELS],
    int                 num_samples,
    hipStream_t        stream,
    bool                debug_synchronous)
{
    // Setup array wrapper for histogram channel output (because we can't pass static arrays as kernel parameters)
    ArrayWrapper<CounterT*, NUM_ACTIVE_CHANNELS> d_histo_wrapper;
    for (int CHANNEL = 0; CHANNEL < NUM_ACTIVE_CHANNELS; ++CHANNEL)
        d_histo_wrapper.array[CHANNEL] = d_histograms[CHANNEL];

    // Invoke kernel to invoke device-side dispatch
    CnpDispatchKernel<BINS, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, SampleIteratorT, CounterT, ALGORITHM><<<1,1>>>(algorithm, timing_timing_iterations, d_temp_storage_bytes, d_cdp_error, d_temp_storage, temp_storage_bytes, d_samples, d_sample_itr, d_histo_wrapper, num_samples, debug_synchronous);

    // Copy out temp_storage_bytes
    CubDebugExit(hipMemcpy(&temp_storage_bytes, d_temp_storage_bytes, sizeof(size_t) * 1, hipMemcpyDeviceToHost));

    // Copy out error
    hipError_t retval;
    CubDebugExit(hipMemcpy(&retval, d_cdp_error, sizeof(hipError_t) * 1, hipMemcpyDeviceToHost));
    return retval;
}
*/


//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------

// Searches for bin given a list of bin-boundary levels
template <typename LevelT>
struct SearchTransform
{
    LevelT          *levels;      // Pointer to levels array
    int             num_levels;   // Number of levels in array

    // Functor for converting samples to bin-ids (num_levels is returned if sample is out of range)
    template <typename SampleT>
    int operator()(SampleT sample)
    {
        int bin = int(std::upper_bound(levels, levels + num_levels, (LevelT) sample) - levels - 1);
        if (bin < 0)
        {
            // Sample out of range
            return num_levels;
        }
        return bin;
    }
};


// Scales samples to evenly-spaced bins
template <typename LevelT>
struct ScaleTransform
{
    int    num_levels;  // Number of levels in array
    LevelT max;         // Max sample level (exclusive)
    LevelT min;         // Min sample level (inclusive)
    LevelT scale;       // Bin scaling factor

    void Init(
        int    num_levels,  // Number of levels in array
        LevelT max,         // Max sample level (exclusive)
        LevelT min,         // Min sample level (inclusive)
        LevelT scale)       // Bin scaling factor
    {
        this->num_levels = num_levels;
        this->max = max;
        this->min = min;
        this->scale = scale;
    }

    // Functor for converting samples to bin-ids  (num_levels is returned if sample is out of range)
    template <typename SampleT>
    int operator()(SampleT sample)
    {
        if ((sample < min) || (sample >= max))
        {
            // Sample out of range
            return num_levels;
        }

        return (int) ((((LevelT) sample) - min) / scale);
    }
};

// Scales samples to evenly-spaced bins
template <>
struct ScaleTransform<float>
{
    int   num_levels;  // Number of levels in array
    float max;         // Max sample level (exclusive)
    float min;         // Min sample level (inclusive)
    float scale;       // Bin scaling factor

    void Init(
        int    num_levels,  // Number of levels in array
        float max,         // Max sample level (exclusive)
        float min,         // Min sample level (inclusive)
        float scale)       // Bin scaling factor
    {
        this->num_levels = num_levels;
        this->max = max;
        this->min = min;
        this->scale = 1.0f / scale;
    }

    // Functor for converting samples to bin-ids  (num_levels is returned if sample is out of range)
    template <typename SampleT>
    int operator()(SampleT sample)
    {
        if ((sample < min) || (sample >= max))
        {
            // Sample out of range
            return num_levels;
        }

        return (int) ((((float) sample) - min) * scale);
    }
};


/**
 * Generate sample
 */
template <typename T, typename LevelT>
void Sample(T &datum, LevelT max_level, int entropy_reduction)
{
    unsigned int max = (unsigned int) -1;
    unsigned int bits;
    RandomBits(bits, entropy_reduction);
    float fraction = (float(bits) / max);

    datum = (T) (fraction * max_level);
}


/**
 * Initialize histogram problem (and solution)
 */
template <
    int             NUM_CHANNELS,
    int             NUM_ACTIVE_CHANNELS,
    typename        LevelT,
    typename        SampleT,
    typename        CounterT,
    typename        TransformOp,
    typename        OffsetT>
void Initialize(
    LevelT          max_level,
    int             entropy_reduction,
    SampleT         *h_samples,
    int             num_levels[NUM_ACTIVE_CHANNELS],        ///< [in] The number of boundaries (levels) for delineating histogram samples in each active channel.  Implies that the number of bins for channel<sub><em>i</em></sub> is <tt>num_levels[i]</tt> - 1.
    TransformOp     transform_op[NUM_ACTIVE_CHANNELS],      ///< [in] The lower sample value bound (inclusive) for the lowest histogram bin in each active channel.
    CounterT        *h_histogram[NUM_ACTIVE_CHANNELS],      ///< [out] The pointers to the histogram counter output arrays, one for each active channel.  For channel<sub><em>i</em></sub>, the allocation length of <tt>d_histograms[i]</tt> should be <tt>num_levels[i]</tt> - 1.
    OffsetT         num_row_pixels,                         ///< [in] The number of multi-channel pixels per row in the region of interest
    OffsetT         num_rows,                               ///< [in] The number of rows in the region of interest
    OffsetT         row_stride_bytes)                             ///< [in] The number of bytes between starts of consecutive rows in the region of interest
{
    printf("Initializing... "); fflush(stdout);

    // Init bins
    for (int CHANNEL = 0; CHANNEL < NUM_ACTIVE_CHANNELS; ++CHANNEL)
    {
        for (int bin = 0; bin < num_levels[CHANNEL] - 1; ++bin)
        {
            h_histogram[CHANNEL][bin] = 0;
        }
    }

    // Initialize samples
    if (g_verbose_input) printf("Samples: \n");
    for (OffsetT row = 0; row < num_rows; ++row)
    {
        for (OffsetT pixel = 0; pixel < num_row_pixels; ++pixel)
        {
            if (g_verbose_input) printf("[");
            for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
            {
                // Sample offset
                OffsetT offset = (row * (row_stride_bytes / sizeof(SampleT))) + (pixel * NUM_CHANNELS) + channel;

                // Init sample value
                Sample(h_samples[offset], max_level, entropy_reduction);
                if (g_verbose_input)
                {
                    if (channel > 0) printf(", ");
                    std::cout << CoutCast(h_samples[offset]);
                }

                // Update sample bin
                int bin = transform_op[channel](h_samples[offset]);
                if (g_verbose_input) printf(" (%d)", bin); fflush(stdout);
                if ((bin >= 0) && (bin < num_levels[channel] - 1))
                {
                    // valid bin
                    h_histogram[channel][bin]++;
                }
            }
            if (g_verbose_input) printf("]");
        }
        if (g_verbose_input) printf("\n\n");
    }

    printf("Done\n"); fflush(stdout);
}


/**
 * Test histogram-even
 */
template <
    Backend         BACKEND,
    int             NUM_CHANNELS,
    int             NUM_ACTIVE_CHANNELS,
    typename        SampleT,
    typename        CounterT,
    typename        LevelT,
    typename        OffsetT>
void TestEven(
    LevelT          max_level,
    int             entropy_reduction,
    int             num_levels[NUM_ACTIVE_CHANNELS],            ///< [in] The number of boundaries (levels) for delineating histogram samples in each active channel.  Implies that the number of bins for channel<sub><em>i</em></sub> is <tt>num_levels[i]</tt> - 1.
    LevelT          lower_level[NUM_ACTIVE_CHANNELS],           ///< [in] The lower sample value bound (inclusive) for the lowest histogram bin in each active channel.
    LevelT          upper_level[NUM_ACTIVE_CHANNELS],           ///< [in] The upper sample value bound (exclusive) for the highest histogram bin in each active channel.
    OffsetT         num_row_pixels,                             ///< [in] The number of multi-channel pixels per row in the region of interest
    OffsetT         num_rows,                                   ///< [in] The number of rows in the region of interest
    OffsetT         row_stride_bytes)                                 ///< [in] The number of bytes between starts of consecutive rows in the region of interest
{
    OffsetT total_samples = num_rows * (row_stride_bytes / sizeof(SampleT));

    printf("\n----------------------------\n");
    printf("%s hipcub::DeviceHistogramEven %d pixels (%d height, %d width, %d-byte row stride), %d %d-byte %s samples (entropy reduction %d), %s counters, %d/%d channels, max sample ",
        (BACKEND == CDP) ? "CDP CUB" : (BACKEND == NPP) ? "NPP" : "CUB",
        (int) (num_row_pixels * num_rows),
        (int) num_rows,
        (int) num_row_pixels,
        (int) row_stride_bytes,
        (int) total_samples,
        (int) sizeof(SampleT),
        typeid(SampleT).name(),
        entropy_reduction,
        typeid(CounterT).name(),
        NUM_ACTIVE_CHANNELS,
        NUM_CHANNELS);
    std::cout << CoutCast(max_level) << "\n";
    for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
        std::cout << "\n\tChannel " << channel << ": " << num_levels[channel] - 1 << " bins [" << lower_level[channel] << ", " << upper_level[channel] << ")\n";
    fflush(stdout);

    // Allocate and initialize host and device data

    typedef SampleT Foo;        // rename type to quelch gcc warnings (bug?)
    SampleT*                    h_samples = new Foo[total_samples];
    CounterT*                   h_histogram[NUM_ACTIVE_CHANNELS];
    ScaleTransform<LevelT>      transform_op[NUM_ACTIVE_CHANNELS];

    for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
    {
        int bins = num_levels[channel] - 1;
        h_histogram[channel] = new CounterT[bins];

        transform_op[channel].Init(
            num_levels[channel],
            upper_level[channel],
            lower_level[channel],
            ((upper_level[channel] - lower_level[channel]) / bins));
    }

    Initialize<NUM_CHANNELS, NUM_ACTIVE_CHANNELS>(
        max_level, entropy_reduction, h_samples, num_levels, transform_op, h_histogram, num_row_pixels, num_rows, row_stride_bytes);

    // Allocate and initialize device data

    SampleT*        d_samples = NULL;
    CounterT*       d_histogram[NUM_ACTIVE_CHANNELS];

    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_samples, sizeof(SampleT) * total_samples));
    CubDebugExit(hipMemcpy(d_samples, h_samples, sizeof(SampleT) * total_samples, hipMemcpyHostToDevice));
    for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
    {
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_histogram[channel], sizeof(CounterT) * (num_levels[channel] - 1)));
        CubDebugExit(hipMemset(d_histogram[channel], 0, sizeof(CounterT) * (num_levels[channel] - 1)));
    }

    // Allocate CDP device arrays
    size_t          *d_temp_storage_bytes = NULL;
    hipError_t     *d_cdp_error = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_temp_storage_bytes,  sizeof(size_t) * 1));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_cdp_error,           sizeof(hipError_t) * 1));

    // Allocate temporary storage
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;

    DispatchEven(
        Int2Type<NUM_CHANNELS>(), Int2Type<NUM_ACTIVE_CHANNELS>(), Int2Type<BACKEND>(), 1, d_temp_storage_bytes, d_cdp_error,
        d_temp_storage, temp_storage_bytes,
        d_samples, d_histogram, num_levels, lower_level, upper_level,
        num_row_pixels, num_rows, row_stride_bytes,
        0, true);

    // Allocate temporary storage with "canary" zones
    int 	canary_bytes 	= 256;
    char 	canary_token 	= 8;
    char* 	canary_zone 	= new char[canary_bytes];

    memset(canary_zone, canary_token, canary_bytes);
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes + (canary_bytes * 2)));
    CubDebugExit(hipMemset(d_temp_storage, canary_token, temp_storage_bytes + (canary_bytes * 2)));

    // Run warmup/correctness iteration
    DispatchEven(
        Int2Type<NUM_CHANNELS>(), Int2Type<NUM_ACTIVE_CHANNELS>(), Int2Type<BACKEND>(), 1, d_temp_storage_bytes, d_cdp_error,
        ((char *) d_temp_storage) + canary_bytes, temp_storage_bytes,
        d_samples, d_histogram, num_levels, lower_level, upper_level,
        num_row_pixels, num_rows, row_stride_bytes,
        0, true);

    // Check canary zones
    int error = CompareDeviceResults(canary_zone, (char *) d_temp_storage, canary_bytes, true, g_verbose);
    AssertEquals(0, error);
    error = CompareDeviceResults(canary_zone, ((char *) d_temp_storage) + canary_bytes + temp_storage_bytes, canary_bytes, true, g_verbose);
    AssertEquals(0, error);

    // Flush any stdout/stderr
    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());
    fflush(stdout);
    fflush(stderr);

    // Check for correctness (and display results, if specified)
    for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
    {
        int channel_error = CompareDeviceResults(h_histogram[channel], d_histogram[channel], num_levels[channel] - 1, true, g_verbose);
        printf("\tChannel %d %s", channel, channel_error ? "FAIL" : "PASS\n");
        error |= channel_error;
    }

    // Performance
    GpuTimer gpu_timer;
    gpu_timer.Start();

    DispatchEven(
        Int2Type<NUM_CHANNELS>(), Int2Type<NUM_ACTIVE_CHANNELS>(), Int2Type<BACKEND>(), g_timing_iterations, d_temp_storage_bytes, d_cdp_error,
        d_temp_storage, temp_storage_bytes,
        d_samples, d_histogram, num_levels, lower_level, upper_level,
        num_row_pixels, num_rows, row_stride_bytes,
        0, false);

    gpu_timer.Stop();
    float elapsed_millis = gpu_timer.ElapsedMillis();

    // Display performance
    if (g_timing_iterations > 0)
    {
        float avg_millis = elapsed_millis / g_timing_iterations;
        float giga_rate = float(total_samples) / avg_millis / 1000.0f / 1000.0f;
        float giga_bandwidth = giga_rate * sizeof(SampleT);
        printf("\t%.3f avg ms, %.3f billion samples/s, %.3f billion bins/s, %.3f billion pixels/s, %.3f logical GB/s",
            avg_millis,
            giga_rate,
            giga_rate * NUM_ACTIVE_CHANNELS / NUM_CHANNELS,
            giga_rate / NUM_CHANNELS,
            giga_bandwidth);
    }

    printf("\n\n");

    // Cleanup
    if (h_samples) delete[] h_samples;

    for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
    {
        if (h_histogram[channel])
            delete[] h_histogram[channel];

        if (d_histogram[channel])
            CubDebugExit(g_allocator.DeviceFree(d_histogram[channel]));
    }

    if (d_samples) CubDebugExit(g_allocator.DeviceFree(d_samples));
    if (d_temp_storage_bytes) CubDebugExit(g_allocator.DeviceFree(d_temp_storage_bytes));
    if (d_cdp_error) CubDebugExit(g_allocator.DeviceFree(d_cdp_error));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

    // Correctness asserts
    AssertEquals(0, error);
}





/**
 * Test histogram-range
 */
template <
    Backend         BACKEND,
    int             NUM_CHANNELS,
    int             NUM_ACTIVE_CHANNELS,
    typename        SampleT,
    typename        CounterT,
    typename        LevelT,
    typename        OffsetT>
void TestRange(
    LevelT          max_level,
    int             entropy_reduction,
    int             num_levels[NUM_ACTIVE_CHANNELS],            ///< [in] The number of boundaries (levels) for delineating histogram samples in each active channel.  Implies that the number of bins for channel<sub><em>i</em></sub> is <tt>num_levels[i]</tt> - 1.
    LevelT*         levels[NUM_ACTIVE_CHANNELS],                ///< [in] The lower sample value bound (inclusive) for the lowest histogram bin in each active channel.
    OffsetT         num_row_pixels,                             ///< [in] The number of multi-channel pixels per row in the region of interest
    OffsetT         num_rows,                                   ///< [in] The number of rows in the region of interest
    OffsetT         row_stride_bytes)                                 ///< [in] The number of bytes between starts of consecutive rows in the region of interest
{
    OffsetT total_samples = num_rows * (row_stride_bytes / sizeof(SampleT));

    printf("\n----------------------------\n");
    printf("%s hipcub::DeviceHistogramRange %d pixels (%d height, %d width, %d-byte row stride), %d %d-byte %s samples (entropy reduction %d), %s counters, %d/%d channels, max sample ",
        (BACKEND == CDP) ? "CDP CUB" : (BACKEND == NPP) ? "NPP" : "CUB",
        (int) (num_row_pixels * num_rows),
        (int) num_rows,
        (int) num_row_pixels,
        (int) row_stride_bytes,
        (int) total_samples,
        (int) sizeof(SampleT),
        typeid(SampleT).name(),
        entropy_reduction,
        typeid(CounterT).name(),
        NUM_ACTIVE_CHANNELS,
        NUM_CHANNELS);
    std::cout << CoutCast(max_level) << "\n";
    for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
    {
        printf("Channel %d: %d bins [", channel, num_levels[channel] - 1);
        std::cout << levels[channel][0];
        for (int level = 1; level < num_levels[channel]; ++level)
            std::cout << ", " << levels[channel][level];
        printf("]\n");
    }
    fflush(stdout);

    // Allocate and initialize host and device data
    typedef SampleT Foo;        // rename type to quelch gcc warnings (bug?)
    SampleT*                    h_samples = new Foo[total_samples];
    CounterT*                   h_histogram[NUM_ACTIVE_CHANNELS];
    SearchTransform<LevelT>     transform_op[NUM_ACTIVE_CHANNELS];

    for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
    {
        transform_op[channel].levels = levels[channel];
        transform_op[channel].num_levels = num_levels[channel];

        int bins = num_levels[channel] - 1;
        h_histogram[channel] = new CounterT[bins];
    }

    Initialize<NUM_CHANNELS, NUM_ACTIVE_CHANNELS>(
        max_level, entropy_reduction, h_samples, num_levels, transform_op, h_histogram, num_row_pixels, num_rows, row_stride_bytes);

    // Allocate and initialize device data
    SampleT*        d_samples = NULL;
    LevelT*         d_levels[NUM_ACTIVE_CHANNELS];
    CounterT*       d_histogram[NUM_ACTIVE_CHANNELS];

    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_samples, sizeof(SampleT) * total_samples));
    CubDebugExit(hipMemcpy(d_samples, h_samples, sizeof(SampleT) * total_samples, hipMemcpyHostToDevice));

    for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
    {
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_levels[channel], sizeof(LevelT) * num_levels[channel]));
        CubDebugExit(hipMemcpy(d_levels[channel], levels[channel],         sizeof(LevelT) * num_levels[channel], hipMemcpyHostToDevice));

        int bins = num_levels[channel] - 1;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_histogram[channel],  sizeof(CounterT) * bins));
        CubDebugExit(hipMemset(d_histogram[channel], 0,                        sizeof(CounterT) * bins));
    }

    // Allocate CDP device arrays
    size_t          *d_temp_storage_bytes = NULL;
    hipError_t     *d_cdp_error = NULL;

    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_temp_storage_bytes,  sizeof(size_t) * 1));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_cdp_error,           sizeof(hipError_t) * 1));

    // Allocate temporary storage
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;

    DispatchRange(
        Int2Type<NUM_CHANNELS>(), Int2Type<NUM_ACTIVE_CHANNELS>(), Int2Type<BACKEND>(), 1, d_temp_storage_bytes, d_cdp_error,
        d_temp_storage, temp_storage_bytes,
        d_samples, d_histogram, num_levels, d_levels,
        num_row_pixels, num_rows, row_stride_bytes,
        0, true);

    // Allocate temporary storage with "canary" zones
    int     canary_bytes    = 256;
    char    canary_token    = 9;
    char*   canary_zone     = new char[canary_bytes];

    memset(canary_zone, canary_token, canary_bytes);
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes + (canary_bytes * 2)));
    CubDebugExit(hipMemset(d_temp_storage, canary_token, temp_storage_bytes + (canary_bytes * 2)));

    // Run warmup/correctness iteration
    DispatchRange(
        Int2Type<NUM_CHANNELS>(), Int2Type<NUM_ACTIVE_CHANNELS>(), Int2Type<BACKEND>(), 1, d_temp_storage_bytes, d_cdp_error,
        ((char *) d_temp_storage) + canary_bytes, temp_storage_bytes,
        d_samples, d_histogram, num_levels, d_levels,
        num_row_pixels, num_rows, row_stride_bytes,
        0, true);

    // Check canary zones
    int error = CompareDeviceResults(canary_zone, (char *) d_temp_storage, canary_bytes, true, g_verbose);
    AssertEquals(0, error);
    error = CompareDeviceResults(canary_zone, ((char *) d_temp_storage) + canary_bytes + temp_storage_bytes, canary_bytes, true, g_verbose);
    AssertEquals(0, error);

    // Flush any stdout/stderr
    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());
    fflush(stdout);
    fflush(stderr);

    // Check for correctness (and display results, if specified)
    for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
    {
        int channel_error = CompareDeviceResults(h_histogram[channel], d_histogram[channel], num_levels[channel] - 1, true, g_verbose);
        printf("\tChannel %d %s", channel, channel_error ? "FAIL" : "PASS\n");
        error |= channel_error;
    }

    // Performance
    GpuTimer gpu_timer;
    gpu_timer.Start();

    DispatchRange(
        Int2Type<NUM_CHANNELS>(), Int2Type<NUM_ACTIVE_CHANNELS>(), Int2Type<BACKEND>(), g_timing_iterations, d_temp_storage_bytes, d_cdp_error,
        d_temp_storage, temp_storage_bytes,
        d_samples, d_histogram, num_levels, d_levels,
        num_row_pixels, num_rows, row_stride_bytes,
        0, false);

    gpu_timer.Stop();
    float elapsed_millis = gpu_timer.ElapsedMillis();

    // Display performance
    if (g_timing_iterations > 0)
    {
        float avg_millis = elapsed_millis / g_timing_iterations;
        float giga_rate = float(total_samples) / avg_millis / 1000.0f / 1000.0f;
        float giga_bandwidth = giga_rate * sizeof(SampleT);
        printf("\t%.3f avg ms, %.3f billion samples/s, %.3f billion bins/s, %.3f billion pixels/s, %.3f logical GB/s",
            avg_millis,
            giga_rate,
            giga_rate * NUM_ACTIVE_CHANNELS / NUM_CHANNELS,
            giga_rate / NUM_CHANNELS,
            giga_bandwidth);
    }

    printf("\n\n");

    // Cleanup
    if (h_samples) delete[] h_samples;

    for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
    {
        if (h_histogram[channel])
            delete[] h_histogram[channel];

        if (d_histogram[channel])
            CubDebugExit(g_allocator.DeviceFree(d_histogram[channel]));

        if (d_levels[channel])
            CubDebugExit(g_allocator.DeviceFree(d_levels[channel]));
    }

    if (d_samples) CubDebugExit(g_allocator.DeviceFree(d_samples));
    if (d_temp_storage_bytes) CubDebugExit(g_allocator.DeviceFree(d_temp_storage_bytes));
    if (d_cdp_error) CubDebugExit(g_allocator.DeviceFree(d_cdp_error));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

    // Correctness asserts
    AssertEquals(0, error);
}


/**
 * Test histogram-even
 */
template <
    Backend         BACKEND,
    typename        SampleT,
    int             NUM_CHANNELS,
    int             NUM_ACTIVE_CHANNELS,
    typename        CounterT,
    typename        LevelT,
    typename        OffsetT>
void TestEven(
    OffsetT         num_row_pixels,
    OffsetT         num_rows,
    OffsetT         row_stride_bytes,
    int             entropy_reduction,
    int             num_levels[NUM_ACTIVE_CHANNELS],
    LevelT          max_level,
    int             max_num_levels)
{
    LevelT lower_level[NUM_ACTIVE_CHANNELS];
    LevelT upper_level[NUM_ACTIVE_CHANNELS];

    // Find smallest level increment
    int max_bins = max_num_levels - 1;
    LevelT min_level_increment = max_level / max_bins;

    // Set upper and lower levels for each channel
    for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
    {
        int num_bins = num_levels[channel] - 1;
        lower_level[channel] = (max_level - (num_bins * min_level_increment)) / 2;
        upper_level[channel] = (max_level + (num_bins * min_level_increment)) / 2;
    }

    TestEven<BACKEND, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, SampleT, CounterT, LevelT, OffsetT>(
        max_level, entropy_reduction, num_levels, lower_level, upper_level, num_row_pixels, num_rows, row_stride_bytes);
}



/**
 * Test histogram-range
 */
template <
    Backend         BACKEND,
    typename        SampleT,
    int             NUM_CHANNELS,
    int             NUM_ACTIVE_CHANNELS,
    typename        CounterT,
    typename        LevelT,
    typename        OffsetT>
void TestRange(
    OffsetT         num_row_pixels,
    OffsetT         num_rows,
    OffsetT         row_stride_bytes,
    int             entropy_reduction,
    int             num_levels[NUM_ACTIVE_CHANNELS],
    LevelT          max_level,
    int             max_num_levels)
{
    // Find smallest level increment
    int max_bins = max_num_levels - 1;
    LevelT min_level_increment = max_level / max_bins;

    LevelT* levels[NUM_ACTIVE_CHANNELS];
    for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
    {
        levels[channel] = new LevelT[num_levels[channel]];

        int num_bins = num_levels[channel] - 1;
        LevelT lower_level = (max_level - (num_bins * min_level_increment)) / 2;

        for (int level = 0; level < num_levels[channel]; ++level)
            levels[channel][level] = lower_level + (level * min_level_increment);
    }

    TestRange<BACKEND, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, SampleT, CounterT, LevelT, OffsetT>(
        max_level, entropy_reduction, num_levels, levels, num_row_pixels, num_rows, row_stride_bytes);

    for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
        delete[] levels[channel];

}



/**
 * Test different entrypoints
 */
template <
    typename        SampleT,
    int             NUM_CHANNELS,
    int             NUM_ACTIVE_CHANNELS,
    typename        CounterT,
    typename        LevelT,
    typename        OffsetT>
void Test(
    OffsetT         num_row_pixels,
    OffsetT         num_rows,
    OffsetT         row_stride_bytes,
    int             entropy_reduction,
    int             num_levels[NUM_ACTIVE_CHANNELS],
    LevelT          max_level,
    int             max_num_levels)
{
    TestEven<CUB, SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
        num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, max_num_levels);

    TestRange<CUB, SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
        num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, max_num_levels);
}


/**
 * Test different number of levels
 */
template <
    typename        SampleT,
    int             NUM_CHANNELS,
    int             NUM_ACTIVE_CHANNELS,
    typename        CounterT,
    typename        LevelT,
    typename        OffsetT>
void Test(
    OffsetT         num_row_pixels,
    OffsetT         num_rows,
    OffsetT         row_stride_bytes,
    int             entropy_reduction,
    LevelT          max_level,
    int             max_num_levels)
{
    int num_levels[NUM_ACTIVE_CHANNELS];

    // All the same level
    for (int channel = 0; channel < NUM_ACTIVE_CHANNELS; ++channel)
    {
        num_levels[channel] = max_num_levels;
    }
    Test<SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
        num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, max_num_levels);

    // All different levels
    num_levels[0] = max_num_levels;
    for (int channel = 1; channel < NUM_ACTIVE_CHANNELS; ++channel)
    {
        num_levels[channel] = (num_levels[channel - 1] / 2) + 1;
    }
    Test<SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
        num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, max_num_levels);
}



/**
 * Test different entropy-levels
 */
template <
    typename        SampleT,
    int             NUM_CHANNELS,
    int             NUM_ACTIVE_CHANNELS,
    typename        CounterT,
    typename        LevelT,
    typename        OffsetT>
void Test(
    OffsetT         num_row_pixels,
    OffsetT         num_rows,
    OffsetT         row_stride_bytes,
    LevelT          max_level,
    int             max_num_levels)
{
    Test<SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
        num_row_pixels, num_rows, row_stride_bytes, 0,   max_level, max_num_levels);

    Test<SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
        num_row_pixels, num_rows, row_stride_bytes, -1,  max_level, max_num_levels);

    Test<SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
        num_row_pixels, num_rows, row_stride_bytes, 5,   max_level, max_num_levels);
}


/**
 * Test different row strides
 */
template <
    typename        SampleT,
    int             NUM_CHANNELS,
    int             NUM_ACTIVE_CHANNELS,
    typename        CounterT,
    typename        LevelT,
    typename        OffsetT>
void Test(
    OffsetT         num_row_pixels,
    OffsetT         num_rows,
    LevelT          max_level,
    int             max_num_levels)
{
    OffsetT row_stride_bytes = num_row_pixels * NUM_CHANNELS * sizeof(SampleT);

    // No padding
    Test<SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
        num_row_pixels, num_rows, row_stride_bytes, max_level, max_num_levels);

    // 13 samples padding
    Test<SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
        num_row_pixels, num_rows, row_stride_bytes + (13 * sizeof(SampleT)), max_level, max_num_levels);
}


/**
 * Test different problem sizes
 */
template <
    typename        SampleT,
    int             NUM_CHANNELS,
    int             NUM_ACTIVE_CHANNELS,
    typename        CounterT,
    typename        LevelT,
    typename        OffsetT>
void Test(
    LevelT          max_level,
    int             max_num_levels)
{
    // 0 images
    Test<SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
        OffsetT(1920), OffsetT(0), max_level, max_num_levels);
    Test<SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
        OffsetT(0), OffsetT(0), max_level, max_num_levels);

    // 1080 image
    Test<SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
        OffsetT(1920), OffsetT(1080), max_level, max_num_levels);

    // 720 image
    Test<SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
        OffsetT(1280), OffsetT(720), max_level, max_num_levels);

    // Sample different image sizes
    for (OffsetT rows = 1; rows < 1000000; rows *= 100)
    {
        for (OffsetT cols = 1; cols < (1000000 / rows); cols *= 100)
        {
            Test<SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
                cols, rows, max_level, max_num_levels);
        }
    }

    // Randomly select linear problem size between 1:10,000,000
    unsigned int max_int = (unsigned int) -1;
    for (int i = 0; i < 10; ++i)
    {
        unsigned int num_items;
        RandomBits(num_items);
        num_items = (unsigned int) ((double(num_items) * double(10000000)) / double(max_int));
        num_items = CUB_MAX(1, num_items);

        Test<SampleT, NUM_CHANNELS, NUM_ACTIVE_CHANNELS, CounterT, LevelT, OffsetT>(
            OffsetT(num_items), 1, max_level, max_num_levels);
    }
}



/**
 * Test different channel interleavings (valid specialiation)
 */
template <
    typename        SampleT,
    typename        CounterT,
    typename        LevelT,
    typename        OffsetT>
void TestChannels(
    LevelT          max_level,
    int             max_num_levels,
    Int2Type<true>  is_valid_tag)
{
    Test<SampleT, 1, 1, CounterT, LevelT, OffsetT>(max_level, max_num_levels);
    Test<SampleT, 4, 3, CounterT, LevelT, OffsetT>(max_level, max_num_levels);
    Test<SampleT, 3, 3, CounterT, LevelT, OffsetT>(max_level, max_num_levels);
    Test<SampleT, 4, 4, CounterT, LevelT, OffsetT>(max_level, max_num_levels);
}


/**
 * Test different channel interleavings (invalid specialiation)
 */
template <
    typename        SampleT,
    typename        CounterT,
    typename        LevelT,
    typename        OffsetT>
void TestChannels(
    LevelT          max_level,
    int             max_num_levels,
    Int2Type<false> is_valid_tag)
{}



//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------




/**
 * Main
 */
int main(int argc, char** argv)
{
    int num_row_pixels = -1;
    int entropy_reduction = 0;
    int num_rows = 1;

    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    g_verbose_input = args.CheckCmdLineFlag("v2");
    args.GetCmdLineArgument("n", num_row_pixels);

    int row_stride_pixels = num_row_pixels;

    args.GetCmdLineArgument("rows", num_rows);
    args.GetCmdLineArgument("stride", row_stride_pixels);
    args.GetCmdLineArgument("i", g_timing_iterations);
    args.GetCmdLineArgument("repeat", g_repeat);
    args.GetCmdLineArgument("entropy", entropy_reduction);

    bool compare_npp = args.CheckCmdLineFlag("npp");


    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--n=<pixels per row> "
            "[--rows=<number of rows> "
            "[--stride=<row stride in pixels> "
            "[--i=<timing iterations> "
            "[--device=<device-id>] "
            "[--repeat=<repetitions of entire test suite>]"
            "[--entropy=<entropy-reduction factor (default 0)>]"
            "[--v] "
            "[--cdp]"
            "[--npp]"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Get ptx version
    int ptx_version;
    CubDebugExit(PtxVersion(ptx_version));

    if (num_row_pixels < 0)
    {
        num_row_pixels      = 1920 * 1080;
        row_stride_pixels   = num_row_pixels;
    }

#if defined(QUICKER_TEST)

    // Compile/run quick tests
    {
        // HistogramEven: unsigned char 256 bins
        typedef unsigned char       SampleT;
        typedef int                 LevelT;

        LevelT  max_level           = 256;
        int     num_levels[1]       = {257};
        int     row_stride_bytes    = sizeof(SampleT) * row_stride_pixels * 1;

        TestEven<CUB, SampleT, 1, 1, int, LevelT, int>(num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, num_levels[0]);
        if (compare_npp)
            TestEven<NPP, SampleT, 1, 1, int, LevelT, int>(num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, num_levels[0]);
    }


#elif defined(QUICK_TEST)

    // Compile/run quick tests
    {
        // HistogramEven: unsigned char 256 bins
        typedef unsigned char       SampleT;
        typedef int                 LevelT;

        LevelT  max_level           = 256;
        int     num_levels[1]       = {257};
        int     row_stride_bytes    = sizeof(SampleT) * row_stride_pixels * 1;

        TestEven<CUB, SampleT, 1, 1, int, LevelT, int>(num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, num_levels[0]);
        if (compare_npp)
            TestEven<NPP, SampleT, 1, 1, int, LevelT, int>(num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, num_levels[0]);
    }

    {
        // HistogramEven: 4/4 multichannel Unsigned char 256 bins
        typedef unsigned char       SampleT;
        typedef int                 LevelT;

        LevelT  max_level           = 256;
        int     num_levels[4]       = {257, 257, 257, 257};
        int     row_stride_bytes    = sizeof(SampleT) * row_stride_pixels * 4;

        TestEven<CUB, SampleT, 4, 4, int, LevelT, int>(num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, num_levels[0]);
    }

    {
        // HistogramEven: 3/4 multichannel Unsigned char 256 bins
        typedef unsigned char       SampleT;
        typedef int                 LevelT;

        LevelT  max_level           = 256;
        int     num_levels[3]       = {257, 257, 257};
        int     row_stride_bytes    = sizeof(SampleT) * row_stride_pixels * 4;

        TestEven<CUB, SampleT, 4, 3, int, LevelT, int>(num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, num_levels[0]);
        if (compare_npp)
            TestEven<NPP, SampleT, 4, 3, int, LevelT, int>(num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, num_levels[0]);
    }

    {
        // HistogramEven: short [0,1024] 256 bins
        typedef unsigned short      SampleT;
        typedef unsigned short      LevelT;

        LevelT  max_level           = 1024;
        int     num_levels[1]       = {257};
        int     row_stride_bytes    = sizeof(SampleT) * row_stride_pixels * 1;

        TestEven<CUB, SampleT, 1, 1, int, LevelT, int>(num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, num_levels[0]);
    }

    {
        // HistogramEven: float [0,1.0] 256 bins
        typedef float               SampleT;
        typedef float               LevelT;

        LevelT  max_level           = 1.0;
        int     num_levels[1]       = {257};
        int     row_stride_bytes    = sizeof(SampleT) * row_stride_pixels * 1;

        TestEven<CUB, SampleT, 1, 1, int, LevelT, int>(num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, num_levels[0]);
    }

    {
        // HistogramEven: 3/4 multichannel float [0,1.0] 256 bins
        typedef float               SampleT;
        typedef float               LevelT;

         LevelT  max_level           = 1.0;
         int     num_levels[3]       = {257, 257, 257};
         int     row_stride_bytes    = sizeof(SampleT) * row_stride_pixels * 4;

         TestEven<CUB, SampleT, 4, 3, int, LevelT, int>(num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, num_levels[0]);
    }

    {
        // HistogramRange: signed char 256 bins
        typedef signed char         SampleT;
        typedef int                 LevelT;

        LevelT  max_level           = 256;
        int     num_levels[1]       = {257};
        int     row_stride_bytes    = sizeof(SampleT) * row_stride_pixels * 1;

        TestRange<CUB, SampleT, 1, 1, int, LevelT, int>(num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, num_levels[0]);
    }

    {
        // HistogramRange: 3/4 channel, unsigned char, varied bins (256, 128, 64)
        typedef unsigned char       SampleT;
        typedef int                 LevelT;

        LevelT  max_level           = 256;
        int     num_levels[3]       = {257, 129, 65};
        int     row_stride_bytes    = sizeof(SampleT) * row_stride_pixels * 4;

        TestRange<CUB, SampleT, 4, 3, int, LevelT, int>(num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, num_levels[0]);
    }

    if (ptx_version > 120)                          // Don't check doubles on PTX120 or below because they're down-converted
    {
        // HistogramEven: double [0,1.0] 64 bins
        typedef double              SampleT;
        typedef double              LevelT;

        LevelT  max_level           = 1.0;
        int     num_levels[1]       = {65};
        int     row_stride_bytes    = sizeof(SampleT) * row_stride_pixels * 1;

        TestEven<CUB, SampleT, 1, 1, int, LevelT, int>(num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, num_levels[0]);
    }

    {
        // HistogramEven: short [0,1024] 512 bins
        typedef unsigned short      SampleT;
        typedef unsigned short      LevelT;

        LevelT  max_level           = 1024;
        int     num_levels[1]       = {513};
        int     row_stride_bytes    = sizeof(SampleT) * row_stride_pixels * 1;

        TestEven<CUB, SampleT, 1, 1, int, LevelT, int>(num_row_pixels, num_rows, row_stride_bytes, entropy_reduction, num_levels, max_level, num_levels[0]);
    }

#else

    // Compile/run thorough tests
    for (int i = 0; i <= g_repeat; ++i)
    {
        TestChannels <unsigned char,    int, int,   int>(256,   256 + 1, Int2Type<true>());
        TestChannels <signed char,      int, int,   int>(256,   256 + 1, Int2Type<true>());
        TestChannels <unsigned short,   int, int,   int>(128,   128 + 1, Int2Type<true>());
        TestChannels <unsigned short,   int, int,   int>(8192,  8192 + 1, Int2Type<true>());
        TestChannels <float,            int, float, int>(1.0,   256 + 1, Int2Type<true>());

		// Test down-conversion of size_t offsets to int
        TestChannels <unsigned char,    int, int,   long long>(256, 256 + 1, Int2Type<(sizeof(size_t) != sizeof(int))>());
    }

#endif

    return 0;
}

