#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2016, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test evaluation for software global barrier throughput
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>

#include <cub/grid/grid_barrier.cuh>

#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------

/**
 * Kernel that iterates through the specified number of software global barriers
 */
__global__ void Kernel(
    GridBarrier global_barrier,
    int iterations)
{
    for (int i = 0; i < iterations; i++)
    {
        global_barrier.Sync();
    }
}


//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    hipError_t retval = hipSuccess;

    // Defaults
    int iterations = 10000;
    int block_size = 128;
    int grid_size = -1;

    // Initialize command line
    CommandLineArgs args(argc, argv);

    // Get args
    args.GetCmdLineArgument("i", iterations);
    args.GetCmdLineArgument("grid-size", grid_size);
    args.GetCmdLineArgument("block-size", block_size);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>]"
            "[--i=<iterations>]"
            "[--grid-size<grid-size>]"
            "[--block-size<block-size>]"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Get device ordinal
    int device_ordinal;
    CubDebugExit(hipGetDevice(&device_ordinal));

    // Get device SM version
    int sm_version;
    CubDebugExit(SmVersion(sm_version, device_ordinal));

    // Get SM properties
    int sm_count, max_block_threads, max_sm_occupancy;
    CubDebugExit(hipDeviceGetAttribute(&sm_count, hipDeviceAttributeMultiprocessorCount, device_ordinal));
    CubDebugExit(hipDeviceGetAttribute(&max_block_threads, hipDeviceAttributeMaxThreadsPerBlock, device_ordinal));
    CubDebugExit(MaxSmOccupancy(max_sm_occupancy, EmptyKernel<void>, 32));

    // Compute grid size and occupancy
    int occupancy = CUB_MIN((max_block_threads / block_size), max_sm_occupancy);

    if (grid_size == -1)
    {
        grid_size = occupancy * sm_count;
    }
    else
    {
        occupancy = grid_size / sm_count;
    }

    printf("Initializing software global barrier for Kernel<<<%d,%d>>> with %d occupancy\n",
        grid_size, block_size, occupancy);
    fflush(stdout);

    // Init global barrier
    GridBarrierLifetime global_barrier;
    global_barrier.Setup(grid_size);

    // Time kernel
    GpuTimer gpu_timer;
    gpu_timer.Start();
    Kernel<<<grid_size, block_size>>>(global_barrier, iterations);
    gpu_timer.Stop();

    retval = HipcubDebug(hipDeviceSynchronize());

    // Output timing results
    float avg_elapsed = gpu_timer.ElapsedMillis() / float(iterations);
    printf("%d iterations, %f total elapsed millis, %f avg elapsed millis\n",
        iterations,
        gpu_timer.ElapsedMillis(),
        avg_elapsed);

    return retval;
}
