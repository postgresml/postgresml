#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2016, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test evaluation for caching allocator of device memory
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>

#include <cub/util_allocator.cuh>
#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>]"
            "[--bytes=<timing bytes>]"
            "[--i=<timing iterations>]"
            "\n", argv[0]);
        exit(0);
    }

#if (HIPCUB_ARCH == 0)

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Get number of GPUs and current GPU
    int num_gpus;
    int initial_gpu;
    int timing_iterations           = 10000;
    int timing_bytes                = 1024 * 1024;

    if (HipcubDebug(hipGetDeviceCount(&num_gpus))) exit(1);
    if (HipcubDebug(hipGetDevice(&initial_gpu))) exit(1);
    args.GetCmdLineArgument("i", timing_iterations);
    args.GetCmdLineArgument("bytes", timing_bytes);

    // Create default allocator (caches up to 6MB in device allocations per GPU)
    CachingDeviceAllocator allocator;
    allocator.debug = true;

    printf("Running single-gpu tests...\n"); fflush(stdout);

    //
    // Test0
    //

    // Create a new stream
    hipStream_t other_stream;
    CubDebugExit(hipStreamCreate(&other_stream));

    // Allocate 999 bytes on the current gpu in stream0
    char *d_999B_stream0_a;
    char *d_999B_stream0_b;
    CubDebugExit(allocator.DeviceAllocate((void **) &d_999B_stream0_a, 999, 0));

    // Run some big kernel in stream 0
    EmptyKernel<void><<<32000, 512, 1024 * 8, 0>>>();

    // Free d_999B_stream0_a
    CubDebugExit(allocator.DeviceFree(d_999B_stream0_a));

    // Allocate another 999 bytes in stream 0
    CubDebugExit(allocator.DeviceAllocate((void **) &d_999B_stream0_b, 999, 0));

    // Check that that we have 1 live block on the initial GPU
    AssertEquals(allocator.live_blocks.size(), 1);

    // Check that that we have no cached block on the initial GPU
    AssertEquals(allocator.cached_blocks.size(), 0);

    // Run some big kernel in stream 0
    EmptyKernel<void><<<32000, 512, 1024 * 8, 0>>>();

    // Free d_999B_stream0_b
    CubDebugExit(allocator.DeviceFree(d_999B_stream0_b));

    // Allocate 999 bytes on the current gpu in other_stream
    char *d_999B_stream_other_a;
    char *d_999B_stream_other_b;
    allocator.DeviceAllocate((void **) &d_999B_stream_other_a, 999, other_stream);

    // Check that that we have 1 live blocks on the initial GPU (that we allocated a new one because d_999B_stream0_b is only available for stream 0 until it becomes idle)
    AssertEquals(allocator.live_blocks.size(), 1);

    // Check that that we have one cached block on the initial GPU
    AssertEquals(allocator.cached_blocks.size(), 1);

    // Run some big kernel in other_stream
    EmptyKernel<void><<<32000, 512, 1024 * 8, other_stream>>>();

    // Free d_999B_stream_other
    CubDebugExit(allocator.DeviceFree(d_999B_stream_other_a));

    // Check that we can now use both allocations in stream 0 after synchronizing the device
    CubDebugExit(hipDeviceSynchronize());
    CubDebugExit(allocator.DeviceAllocate((void **) &d_999B_stream0_a, 999, 0));
    CubDebugExit(allocator.DeviceAllocate((void **) &d_999B_stream0_b, 999, 0));

    // Check that that we have 2 live blocks on the initial GPU
    AssertEquals(allocator.live_blocks.size(), 2);

    // Check that that we have no cached block on the initial GPU
    AssertEquals(allocator.cached_blocks.size(), 0);

    // Free d_999B_stream0_a and d_999B_stream0_b
    CubDebugExit(allocator.DeviceFree(d_999B_stream0_a));
    CubDebugExit(allocator.DeviceFree(d_999B_stream0_b));

    // Check that we can now use both allocations in other_stream
    CubDebugExit(hipDeviceSynchronize());
    CubDebugExit(allocator.DeviceAllocate((void **) &d_999B_stream_other_a, 999, other_stream));
    CubDebugExit(allocator.DeviceAllocate((void **) &d_999B_stream_other_b, 999, other_stream));

    // Check that that we have 2 live blocks on the initial GPU
    AssertEquals(allocator.live_blocks.size(), 2);

    // Check that that we have no cached block on the initial GPU
    AssertEquals(allocator.cached_blocks.size(), 0);

    // Run some big kernel in other_stream
    EmptyKernel<void><<<32000, 512, 1024 * 8, other_stream>>>();

    // Free d_999B_stream_other_a and d_999B_stream_other_b
    CubDebugExit(allocator.DeviceFree(d_999B_stream_other_a));
    CubDebugExit(allocator.DeviceFree(d_999B_stream_other_b));

    // Check that we can now use both allocations in stream 0 after synchronizing the device and destroying the other stream
    CubDebugExit(hipDeviceSynchronize());
    CubDebugExit(hipStreamDestroy(other_stream));
    CubDebugExit(allocator.DeviceAllocate((void **) &d_999B_stream0_a, 999, 0));
    CubDebugExit(allocator.DeviceAllocate((void **) &d_999B_stream0_b, 999, 0));

    // Check that that we have 2 live blocks on the initial GPU
    AssertEquals(allocator.live_blocks.size(), 2);

    // Check that that we have no cached block on the initial GPU
    AssertEquals(allocator.cached_blocks.size(), 0);

    // Free d_999B_stream0_a and d_999B_stream0_b
    CubDebugExit(allocator.DeviceFree(d_999B_stream0_a));
    CubDebugExit(allocator.DeviceFree(d_999B_stream0_b));

    // Free all cached
    CubDebugExit(allocator.FreeAllCached());

    //
    // Test1
    //

    // Allocate 5 bytes on the current gpu
    char *d_5B;
    CubDebugExit(allocator.DeviceAllocate((void **) &d_5B, 5));

    // Check that that we have zero free bytes cached on the initial GPU
    AssertEquals(allocator.cached_bytes[initial_gpu].free, 0);

    // Check that that we have 1 live block on the initial GPU
    AssertEquals(allocator.live_blocks.size(), 1);

    //
    // Test2
    //

    // Allocate 4096 bytes on the current gpu
    char *d_4096B;
    CubDebugExit(allocator.DeviceAllocate((void **) &d_4096B, 4096));

    // Check that that we have 2 live blocks on the initial GPU
    AssertEquals(allocator.live_blocks.size(), 2);

    //
    // Test3
    //

    // DeviceFree d_5B
    CubDebugExit(allocator.DeviceFree(d_5B));

    // Check that that we have min_bin_bytes free bytes cached on the initial gpu
    AssertEquals(allocator.cached_bytes[initial_gpu].free, allocator.min_bin_bytes);

    // Check that that we have 1 live block on the initial GPU
    AssertEquals(allocator.live_blocks.size(), 1);

    // Check that that we have 1 cached block on the initial GPU
    AssertEquals(allocator.cached_blocks.size(), 1);

    //
    // Test4
    //

    // DeviceFree d_4096B
    CubDebugExit(allocator.DeviceFree(d_4096B));

    // Check that that we have the 4096 + min_bin free bytes cached on the initial gpu
    AssertEquals(allocator.cached_bytes[initial_gpu].free, allocator.min_bin_bytes + 4096);

    // Check that that we have 0 live block on the initial GPU
    AssertEquals(allocator.live_blocks.size(), 0);

    // Check that that we have 2 cached block on the initial GPU
    AssertEquals(allocator.cached_blocks.size(), 2);

    //
    // Test5
    //

    // Allocate 768 bytes on the current gpu
    char *d_768B;
    CubDebugExit(allocator.DeviceAllocate((void **) &d_768B, 768));

    // Check that that we have the min_bin free bytes cached on the initial gpu (4096 was reused)
    AssertEquals(allocator.cached_bytes[initial_gpu].free, allocator.min_bin_bytes);

    // Check that that we have 1 live block on the initial GPU
    AssertEquals(allocator.live_blocks.size(), 1);

    // Check that that we have 1 cached block on the initial GPU
    AssertEquals(allocator.cached_blocks.size(), 1);

    //
    // Test6
    //

    // Allocate max_cached_bytes on the current gpu
    char *d_max_cached;
    CubDebugExit(allocator.DeviceAllocate((void **) &d_max_cached, allocator.max_cached_bytes));

    // DeviceFree d_max_cached
    CubDebugExit(allocator.DeviceFree(d_max_cached));

    // Check that that we have the min_bin free bytes cached on the initial gpu (max cached was not returned because we went over)
    AssertEquals(allocator.cached_bytes[initial_gpu].free, allocator.min_bin_bytes);

    // Check that that we have 1 live block on the initial GPU
    AssertEquals(allocator.live_blocks.size(), 1);

    // Check that that we still have 1 cached block on the initial GPU
    AssertEquals(allocator.cached_blocks.size(), 1);

    //
    // Test7
    //

    // Free all cached blocks on all GPUs
    CubDebugExit(allocator.FreeAllCached());

    // Check that that we have 0 bytes cached on the initial GPU
    AssertEquals(allocator.cached_bytes[initial_gpu].free, 0);

    // Check that that we have 0 cached blocks across all GPUs
    AssertEquals(allocator.cached_blocks.size(), 0);

    // Check that that still we have 1 live block across all GPUs
    AssertEquals(allocator.live_blocks.size(), 1);

    //
    // Test8
    //

    // Allocate max cached bytes + 1 on the current gpu
    char *d_max_cached_plus;
    CubDebugExit(allocator.DeviceAllocate((void **) &d_max_cached_plus, allocator.max_cached_bytes + 1));

    // DeviceFree max cached bytes
    CubDebugExit(allocator.DeviceFree(d_max_cached_plus));

    // DeviceFree d_768B
    CubDebugExit(allocator.DeviceFree(d_768B));

    unsigned int power;
    size_t rounded_bytes;
    allocator.NearestPowerOf(power, rounded_bytes, allocator.bin_growth, 768);

    // Check that that we have 4096 free bytes cached on the initial gpu
    AssertEquals(allocator.cached_bytes[initial_gpu].free, rounded_bytes);

    // Check that that we have 1 cached blocks across all GPUs
    AssertEquals(allocator.cached_blocks.size(), 1);

    // Check that that still we have 0 live block across all GPUs
    AssertEquals(allocator.live_blocks.size(), 0);

#ifndef CUB_CDP
    // BUG: find out why these tests fail when one GPU is CDP compliant and the other is not

    if (num_gpus > 1)
    {
        printf("\nRunning multi-gpu tests...\n"); fflush(stdout);

        //
        // Test9
        //

        // Allocate 768 bytes on the next gpu
        int next_gpu = (initial_gpu + 1) % num_gpus;
        char *d_768B_2;
        CubDebugExit(allocator.DeviceAllocate(next_gpu, (void **) &d_768B_2, 768));

        // DeviceFree d_768B on the next gpu
        CubDebugExit(allocator.DeviceFree(next_gpu, d_768B_2));

        // Re-allocate 768 bytes on the next gpu
        CubDebugExit(allocator.DeviceAllocate(next_gpu, (void **) &d_768B_2, 768));

        // Re-free d_768B on the next gpu
        CubDebugExit(allocator.DeviceFree(next_gpu, d_768B_2));

        // Check that that we have 4096 free bytes cached on the initial gpu
        AssertEquals(allocator.cached_bytes[initial_gpu].free, rounded_bytes);

        // Check that that we have 4096 free bytes cached on the second gpu
        AssertEquals(allocator.cached_bytes[next_gpu].free, rounded_bytes);

        // Check that that we have 2 cached blocks across all GPUs
        AssertEquals(allocator.cached_blocks.size(), 2);

        // Check that that still we have 0 live block across all GPUs
        AssertEquals(allocator.live_blocks.size(), 0);
    }
#endif  // CUB_CDP

    //
    // Performance
    //

    printf("\nCPU Performance (%d timing iterations, %d bytes):\n", timing_iterations, timing_bytes);
    fflush(stdout); fflush(stderr);

    // CPU performance comparisons vs cached.  Allocate and free a 1MB block 2000 times
    CpuTimer    cpu_timer;
    char        *d_1024MB                       = NULL;
    allocator.debug                             = false;

    // Prime the caching allocator and the kernel
    CubDebugExit(allocator.DeviceAllocate((void **) &d_1024MB, timing_bytes));
    CubDebugExit(allocator.DeviceFree(d_1024MB));
    hipcub::EmptyKernel<void><<<1, 32>>>();

    // CUDA
    cpu_timer.Start();
    for (int i = 0; i < timing_iterations; ++i)
    {
        CubDebugExit(hipMalloc((void **) &d_1024MB, timing_bytes));
        CubDebugExit(hipFree(d_1024MB));
    }
    cpu_timer.Stop();
    float cuda_malloc_elapsed_millis = cpu_timer.ElapsedMillis();

    // CUB
    cpu_timer.Start();
    for (int i = 0; i < timing_iterations; ++i)
    {
        CubDebugExit(allocator.DeviceAllocate((void **) &d_1024MB, timing_bytes));
        CubDebugExit(allocator.DeviceFree(d_1024MB));
    }
    cpu_timer.Stop();
    float cub_calloc_elapsed_millis = cpu_timer.ElapsedMillis();

    printf("\t CUB CachingDeviceAllocator allocation CPU speedup: %.2f (avg hipMalloc %.4f ms vs. avg DeviceAllocate %.4f ms)\n",
        cuda_malloc_elapsed_millis / cub_calloc_elapsed_millis,
        cuda_malloc_elapsed_millis / timing_iterations,
        cub_calloc_elapsed_millis / timing_iterations);

    // GPU performance comparisons.  Allocate and free a 1MB block 2000 times
    GpuTimer gpu_timer;

    printf("\nGPU Performance (%d timing iterations, %d bytes):\n", timing_iterations, timing_bytes);
    fflush(stdout); fflush(stderr);

    // Kernel-only
    gpu_timer.Start();
    for (int i = 0; i < timing_iterations; ++i)
    {
        hipcub::EmptyKernel<void><<<1, 32>>>();
    }
    gpu_timer.Stop();
    float cuda_empty_elapsed_millis = gpu_timer.ElapsedMillis();

    // CUDA
    gpu_timer.Start();
    for (int i = 0; i < timing_iterations; ++i)
    {
        CubDebugExit(hipMalloc((void **) &d_1024MB, timing_bytes));
        hipcub::EmptyKernel<void><<<1, 32>>>();
        CubDebugExit(hipFree(d_1024MB));
    }
    gpu_timer.Stop();
    cuda_malloc_elapsed_millis = gpu_timer.ElapsedMillis() - cuda_empty_elapsed_millis;

    // CUB
    gpu_timer.Start();
    for (int i = 0; i < timing_iterations; ++i)
    {
        CubDebugExit(allocator.DeviceAllocate((void **) &d_1024MB, timing_bytes));
        hipcub::EmptyKernel<void><<<1, 32>>>();
        CubDebugExit(allocator.DeviceFree(d_1024MB));
    }
    gpu_timer.Stop();
    cub_calloc_elapsed_millis = gpu_timer.ElapsedMillis() - cuda_empty_elapsed_millis;

    printf("\t CUB CachingDeviceAllocator allocation GPU speedup: %.2f (avg hipMalloc %.4f ms vs. avg DeviceAllocate %.4f ms)\n",
        cuda_malloc_elapsed_millis / cub_calloc_elapsed_millis,
        cuda_malloc_elapsed_millis / timing_iterations,
        cub_calloc_elapsed_millis / timing_iterations);


#endif

    printf("Success\n");

    return 0;
}

