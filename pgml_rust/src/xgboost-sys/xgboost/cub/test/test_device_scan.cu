#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2016, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of DeviceScan utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <typeinfo>

#include <cub/util_allocator.cuh>
#include <cub/iterator/constant_input_iterator.cuh>
#include <cub/iterator/discard_output_iterator.cuh>
#include <cub/device/device_scan.cuh>

#include <thrust/device_ptr.h>
#include <thrust/scan.h>

#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose           = false;
int                     g_timing_iterations = 0;
int                     g_repeat            = 0;
double                  g_device_giga_bandwidth;
CachingDeviceAllocator  g_allocator(true);

// Dispatch types
enum Backend
{
    CUB,        // CUB method
    THRUST,     // Thrust method
    CDP,        // GPU-based (dynamic parallelism) dispatch to CUB method
};


/**
 * \brief WrapperFunctor (for precluding test-specialized dispatch to *Sum variants)
 */
template<typename OpT>
struct WrapperFunctor
{
    OpT op;

    WrapperFunctor(OpT op) : op(op) {}

    template <typename T>
    __host__ __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return op(a, b);
    }
};


//---------------------------------------------------------------------
// Dispatch to different CUB DeviceScan entrypoints
//---------------------------------------------------------------------

/**
 * Dispatch to exclusive scan entrypoint
 */
template <typename IsPrimitiveT, typename InputIteratorT, typename OutputIteratorT, typename ScanOpT, typename InitialValueT, typename OffsetT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t Dispatch(
    Int2Type<CUB>       dispatch_to,
    IsPrimitiveT        is_primitive,
    int                 timing_timing_iterations,
    size_t              *d_temp_storage_bytes,
    hipError_t         *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    InputIteratorT      d_in,
    OutputIteratorT     d_out,
    ScanOpT             scan_op,
    InitialValueT       initial_value,
    OffsetT             num_items,
    hipStream_t        stream,
    bool                debug_synchronous)
{
    hipError_t error = hipSuccess;
    for (int i = 0; i < timing_timing_iterations; ++i)
    {
        error = DeviceScan::ExclusiveScan(d_temp_storage, temp_storage_bytes, d_in, d_out, scan_op, initial_value, num_items, stream, debug_synchronous);
    }
    return error;
}


/**
 * Dispatch to exclusive sum entrypoint
 */
template <typename InputIteratorT, typename OutputIteratorT, typename InitialValueT, typename OffsetT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t Dispatch(
    Int2Type<CUB>       dispatch_to,
    Int2Type<true>      is_primitive,
    int                 timing_timing_iterations,
    size_t              *d_temp_storage_bytes,
    hipError_t         *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    InputIteratorT      d_in,
    OutputIteratorT     d_out,
    Sum                 scan_op,
    InitialValueT       initial_value,
    OffsetT             num_items,
    hipStream_t        stream,
    bool                debug_synchronous)
{
    hipError_t error = hipSuccess;
    for (int i = 0; i < timing_timing_iterations; ++i)
    {
        error = DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, stream, debug_synchronous);
    }
    return error;
}


/**
 * Dispatch to inclusive scan entrypoint
 */
template <typename IsPrimitiveT, typename InputIteratorT, typename OutputIteratorT, typename ScanOpT, typename OffsetT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t Dispatch(
    Int2Type<CUB>       dispatch_to,
    IsPrimitiveT        is_primitive,
    int                 timing_timing_iterations,
    size_t              *d_temp_storage_bytes,
    hipError_t         *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    InputIteratorT      d_in,
    OutputIteratorT     d_out,
    ScanOpT             scan_op,
    NullType            initial_value,
    OffsetT             num_items,
    hipStream_t        stream,
    bool                debug_synchronous)
{
    hipError_t error = hipSuccess;
    for (int i = 0; i < timing_timing_iterations; ++i)
    {
        error = DeviceScan::InclusiveScan(d_temp_storage, temp_storage_bytes, d_in, d_out, scan_op, num_items, stream, debug_synchronous);
    }
    return error;
}


/**
 * Dispatch to inclusive sum entrypoint
 */
template <typename InputIteratorT, typename OutputIteratorT, typename OffsetT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t Dispatch(
    Int2Type<CUB>       dispatch_to,
    Int2Type<true>      is_primitive,
    int                 timing_timing_iterations,
    size_t              *d_temp_storage_bytes,
    hipError_t         *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    InputIteratorT      d_in,
    OutputIteratorT     d_out,
    Sum                 scan_op,
    NullType            initial_value,
    OffsetT             num_items,
    hipStream_t        stream,
    bool                debug_synchronous)
{
    hipError_t error = hipSuccess;
    for (int i = 0; i < timing_timing_iterations; ++i)
    {
        error = DeviceScan::InclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items, stream, debug_synchronous);
    }
    return error;
}

//---------------------------------------------------------------------
// Dispatch to different Thrust entrypoints
//---------------------------------------------------------------------

/**
 * Dispatch to exclusive scan entrypoint
 */
template <typename IsPrimitiveT, typename InputIteratorT, typename OutputIteratorT, typename ScanOpT, typename InitialValueT, typename OffsetT>
hipError_t Dispatch(
    Int2Type<THRUST>    dispatch_to,
    IsPrimitiveT        is_primitive,
    int                 timing_timing_iterations,
    size_t              *d_temp_storage_bytes,
    hipError_t         *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    InputIteratorT      d_in,
    OutputIteratorT     d_out,
    ScanOpT             scan_op,
    InitialValueT       initial_value,
    OffsetT             num_items,
    hipStream_t        stream,
    bool                debug_synchronous)
{
    // The input value type
    typedef typename std::iterator_traits<InputIteratorT>::value_type InputT;

    // The output value type
    typedef typename If<(Equals<typename std::iterator_traits<OutputIteratorT>::value_type, void>::VALUE),  // OutputT =  (if output iterator's value type is void) ?
        typename std::iterator_traits<InputIteratorT>::value_type,                                          // ... then the input iterator's value type,
        typename std::iterator_traits<OutputIteratorT>::value_type>::Type OutputT;                          // ... else the output iterator's value type

    if (d_temp_storage == 0)
    {
        temp_storage_bytes = 1;
    }
    else
    {
        thrust::device_ptr<InputT> d_in_wrapper(d_in);
        thrust::device_ptr<OutputT> d_out_wrapper(d_out);
        for (int i = 0; i < timing_timing_iterations; ++i)
        {
            thrust::exclusive_scan(d_in_wrapper, d_in_wrapper + num_items, d_out_wrapper, initial_value, scan_op);
        }
    }

    return hipSuccess;
}


/**
 * Dispatch to exclusive sum entrypoint
 */
template <typename InputIteratorT, typename OutputIteratorT, typename InitialValueT, typename OffsetT>
hipError_t Dispatch(
    Int2Type<THRUST>    dispatch_to,
    Int2Type<true>      is_primitive,
    int                 timing_timing_iterations,
    size_t              *d_temp_storage_bytes,
    hipError_t         *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    InputIteratorT      d_in,
    OutputIteratorT     d_out,
    Sum                 scan_op,
    InitialValueT       initial_value,
    OffsetT             num_items,
    hipStream_t        stream,
    bool                debug_synchronous)
{
    // The input value type
    typedef typename std::iterator_traits<InputIteratorT>::value_type InputT;

    // The output value type
    typedef typename If<(Equals<typename std::iterator_traits<OutputIteratorT>::value_type, void>::VALUE),  // OutputT =  (if output iterator's value type is void) ?
        typename std::iterator_traits<InputIteratorT>::value_type,                                          // ... then the input iterator's value type,
        typename std::iterator_traits<OutputIteratorT>::value_type>::Type OutputT;                          // ... else the output iterator's value type

    if (d_temp_storage == 0)
    {
        temp_storage_bytes = 1;
    }
    else
    {
        thrust::device_ptr<InputT> d_in_wrapper(d_in);
        thrust::device_ptr<OutputT> d_out_wrapper(d_out);
        for (int i = 0; i < timing_timing_iterations; ++i)
        {
            thrust::exclusive_scan(d_in_wrapper, d_in_wrapper + num_items, d_out_wrapper);
        }
    }

    return hipSuccess;
}


/**
 * Dispatch to inclusive scan entrypoint
 */
template <typename IsPrimitiveT, typename InputIteratorT, typename OutputIteratorT, typename ScanOpT, typename OffsetT>
hipError_t Dispatch(
    Int2Type<THRUST>    dispatch_to,
    IsPrimitiveT        is_primitive,
    int                 timing_timing_iterations,
    size_t              *d_temp_storage_bytes,
    hipError_t         *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    InputIteratorT      d_in,
    OutputIteratorT     d_out,
    ScanOpT             scan_op,
    NullType            initial_value,
    OffsetT             num_items,
    hipStream_t        stream,
    bool                debug_synchronous)
{
    // The input value type
    typedef typename std::iterator_traits<InputIteratorT>::value_type InputT;

    // The output value type
    typedef typename If<(Equals<typename std::iterator_traits<OutputIteratorT>::value_type, void>::VALUE),  // OutputT =  (if output iterator's value type is void) ?
        typename std::iterator_traits<InputIteratorT>::value_type,                                          // ... then the input iterator's value type,
        typename std::iterator_traits<OutputIteratorT>::value_type>::Type OutputT;                          // ... else the output iterator's value type

    if (d_temp_storage == 0)
    {
        temp_storage_bytes = 1;
    }
    else
    {
        thrust::device_ptr<InputT> d_in_wrapper(d_in);
        thrust::device_ptr<OutputT> d_out_wrapper(d_out);
        for (int i = 0; i < timing_timing_iterations; ++i)
        {
            thrust::inclusive_scan(d_in_wrapper, d_in_wrapper + num_items, d_out_wrapper, scan_op);
        }
    }

    return hipSuccess;
}


/**
 * Dispatch to inclusive sum entrypoint
 */
template <typename InputIteratorT, typename OutputIteratorT, typename OffsetT>
hipError_t Dispatch(
    Int2Type<THRUST>    dispatch_to,
    Int2Type<true>      is_primitive,
    int                 timing_timing_iterations,
    size_t              *d_temp_storage_bytes,
    hipError_t         *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    InputIteratorT      d_in,
    OutputIteratorT     d_out,
    Sum                 scan_op,
    NullType            initial_value,
    OffsetT             num_items,
    hipStream_t        stream,
    bool                debug_synchronous)
{
    // The input value type
    typedef typename std::iterator_traits<InputIteratorT>::value_type InputT;

    // The output value type
    typedef typename If<(Equals<typename std::iterator_traits<OutputIteratorT>::value_type, void>::VALUE),  // OutputT =  (if output iterator's value type is void) ?
        typename std::iterator_traits<InputIteratorT>::value_type,                                          // ... then the input iterator's value type,
        typename std::iterator_traits<OutputIteratorT>::value_type>::Type OutputT;                          // ... else the output iterator's value type

    if (d_temp_storage == 0)
    {
        temp_storage_bytes = 1;
    }
    else
    {
        thrust::device_ptr<InputT> d_in_wrapper(d_in);
        thrust::device_ptr<OutputT> d_out_wrapper(d_out);
        for (int i = 0; i < timing_timing_iterations; ++i)
        {
            thrust::inclusive_scan(d_in_wrapper, d_in_wrapper + num_items, d_out_wrapper);
        }
    }

    return hipSuccess;
}



//---------------------------------------------------------------------
// CUDA Nested Parallelism Test Kernel
//---------------------------------------------------------------------

/**
 * Simple wrapper kernel to invoke DeviceScan
 */
template <typename IsPrimitiveT, typename InputIteratorT, typename OutputIteratorT, typename ScanOpT, typename InitialValueT, typename OffsetT>
__global__ void CnpDispatchKernel(
    IsPrimitiveT        is_primitive,
    int                 timing_timing_iterations,
    size_t              *d_temp_storage_bytes,
    hipError_t         *d_cdp_error,

    void*               d_temp_storage,
    size_t              temp_storage_bytes,
    InputIteratorT      d_in,
    OutputIteratorT     d_out,
    ScanOpT             scan_op,
    InitialValueT       initial_value,
    OffsetT             num_items,
    bool                debug_synchronous)
{
#ifndef CUB_CDP
    *d_cdp_error = hipErrorNotSupported;
#else
    *d_cdp_error = Dispatch(
        Int2Type<CUB>(),
        is_primitive,
        timing_timing_iterations,
        d_temp_storage_bytes,
        d_cdp_error,
        d_temp_storage,
        temp_storage_bytes,
        d_in,
        d_out,
        scan_op,
        initial_value,
        num_items,
        0,
        debug_synchronous);

    *d_temp_storage_bytes = temp_storage_bytes;
#endif
}


/**
 * Dispatch to CDP kernel
 */
template <typename IsPrimitiveT, typename InputIteratorT, typename OutputIteratorT, typename ScanOpT, typename InitialValueT, typename OffsetT>
hipError_t Dispatch(
    Int2Type<CDP>       dispatch_to,
    IsPrimitiveT        is_primitive,
    int                 timing_timing_iterations,
    size_t              *d_temp_storage_bytes,
    hipError_t         *d_cdp_error,

    void*               d_temp_storage,
    size_t&             temp_storage_bytes,
    InputIteratorT      d_in,
    OutputIteratorT     d_out,
    ScanOpT             scan_op,
    InitialValueT       initial_value,
    OffsetT             num_items,
    hipStream_t        stream,
    bool                debug_synchronous)
{
    // Invoke kernel to invoke device-side dispatch
    CnpDispatchKernel<<<1,1>>>(
        is_primitive,
        timing_timing_iterations,
        d_temp_storage_bytes,
        d_cdp_error,
        d_temp_storage,
        temp_storage_bytes,
        d_in,
        d_out,
        scan_op,
        initial_value,
        num_items,
        debug_synchronous);

    // Copy out temp_storage_bytes
    CubDebugExit(hipMemcpy(&temp_storage_bytes, d_temp_storage_bytes, sizeof(size_t) * 1, hipMemcpyDeviceToHost));

    // Copy out error
    hipError_t retval;
    CubDebugExit(hipMemcpy(&retval, d_cdp_error, sizeof(hipError_t) * 1, hipMemcpyDeviceToHost));
    return retval;
}


//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------


/**
 * Initialize problem
 */
template <typename T>
void Initialize(
    GenMode      gen_mode,
    T            *h_in,
    int          num_items)
{
    for (int i = 0; i < num_items; ++i)
    {
        InitValue(gen_mode, h_in[i], i);
    }

    if (g_verbose)
    {
        printf("Input:\n");
        DisplayResults(h_in, num_items);
        printf("\n\n");
    }
}

/**
 * Solve exclusive-scan problem
 */
template <
    typename        InputIteratorT,
    typename        OutputT,
    typename        ScanOpT>
void Solve(
    InputIteratorT  h_in,
    OutputT         *h_reference,
    int             num_items,
    ScanOpT         scan_op,
    OutputT         initial_value)
{
    if (num_items > 0)
    {
        OutputT val         = h_in[0];
        h_reference[0]      = initial_value;
        OutputT inclusive   = scan_op(initial_value, val);

        for (int i = 1; i < num_items; ++i)
        {
            val = h_in[i];
            h_reference[i] = inclusive;
            inclusive = scan_op(inclusive, val);
        }
    }
}


/**
 * Solve inclusive-scan problem
 */
template <
    typename        InputIteratorT,
    typename        OutputT,
    typename        ScanOpT>
void Solve(
    InputIteratorT  h_in,
    OutputT         *h_reference,
    int             num_items,
    ScanOpT         scan_op,
    NullType)
{
    if (num_items > 0)
    {
        OutputT inclusive   = h_in[0];
        h_reference[0]      = inclusive;

        for (int i = 1; i < num_items; ++i)
        {
            OutputT val = h_in[i];
            inclusive = scan_op(inclusive, val);
            h_reference[i] = inclusive;
        }
    }
}


/**
 * Test DeviceScan for a given problem input
 */
template <
    Backend             BACKEND,
    typename            DeviceInputIteratorT,
    typename            OutputT,
    typename            ScanOpT,
    typename            InitialValueT>
void Test(
    DeviceInputIteratorT    d_in,
    OutputT                 *h_reference,
    int                     num_items,
    ScanOpT                 scan_op,
    InitialValueT           initial_value)
{
    typedef typename std::iterator_traits<DeviceInputIteratorT>::value_type InputT;

    // Allocate device output array
    OutputT *d_out = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(OutputT) * num_items));

    // Allocate CDP device arrays
    size_t          *d_temp_storage_bytes = NULL;
    hipError_t     *d_cdp_error = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_temp_storage_bytes,  sizeof(size_t) * 1));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_cdp_error,   sizeof(hipError_t) * 1));

    // Allocate temporary storage
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;
    CubDebugExit(Dispatch(
        Int2Type<BACKEND>(),
        Int2Type<Traits<OutputT>::PRIMITIVE>(),
        1,
        d_temp_storage_bytes,
        d_cdp_error,
        d_temp_storage,
        temp_storage_bytes,
        d_in,
        d_out,
        scan_op,
        initial_value,
        num_items,
        0,
        true));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Clear device output array
    CubDebugExit(hipMemset(d_out, 0, sizeof(OutputT) * num_items));

    // Run warmup/correctness iteration
    CubDebugExit(Dispatch(
        Int2Type<BACKEND>(),
        Int2Type<Traits<OutputT>::PRIMITIVE>(),
        1,
        d_temp_storage_bytes,
        d_cdp_error,
        d_temp_storage,
        temp_storage_bytes,
        d_in,
        d_out,
        scan_op,
        initial_value,
        num_items,
        0,
        true));

    // Check for correctness (and display results, if specified)
    int compare = CompareDeviceResults(h_reference, d_out, num_items, true, g_verbose);
    printf("\t%s", compare ? "FAIL" : "PASS");

    // Flush any stdout/stderr
    fflush(stdout);
    fflush(stderr);

    // Performance
    GpuTimer gpu_timer;
    gpu_timer.Start();
    CubDebugExit(Dispatch(Int2Type<BACKEND>(),
        Int2Type<Traits<OutputT>::PRIMITIVE>(),
        g_timing_iterations,
        d_temp_storage_bytes,
        d_cdp_error,
        d_temp_storage,
        temp_storage_bytes,
        d_in,
        d_out,
        scan_op,
        initial_value,
        num_items,
        0,
        false));
    gpu_timer.Stop();
    float elapsed_millis = gpu_timer.ElapsedMillis();

    // Display performance
    if (g_timing_iterations > 0)
    {
        float avg_millis = elapsed_millis / g_timing_iterations;
        float giga_rate = float(num_items) / avg_millis / 1000.0f / 1000.0f;
        float giga_bandwidth = giga_rate * (sizeof(InputT) + sizeof(OutputT));
        printf(", %.3f avg ms, %.3f billion items/s, %.3f logical GB/s, %.1f%% peak", avg_millis, giga_rate, giga_bandwidth, giga_bandwidth / g_device_giga_bandwidth * 100.0);
    }

    printf("\n\n");

    // Cleanup
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_temp_storage_bytes) CubDebugExit(g_allocator.DeviceFree(d_temp_storage_bytes));
    if (d_cdp_error) CubDebugExit(g_allocator.DeviceFree(d_cdp_error));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

    // Correctness asserts
    AssertEquals(0, compare);
}


/**
 * Test DeviceScan on pointer type
 */
template <
    Backend         BACKEND,
    typename        InputT,
    typename        OutputT,
    typename        ScanOpT,
    typename        InitialValueT>
void TestPointer(
    int             num_items,
    GenMode         gen_mode,
    ScanOpT         scan_op,
    InitialValueT   initial_value)
{
    printf("\nPointer %s %s hipcub::DeviceScan::%s %d items, %s->%s (%d->%d bytes) , gen-mode %s\n",
        (BACKEND == CDP) ? "CDP CUB" : (BACKEND == THRUST) ? "Thrust" : "CUB",
        (Equals<InitialValueT, NullType>::VALUE) ? "Inclusive" : "Exclusive",
        (Equals<ScanOpT, Sum>::VALUE) ? "Sum" : "Scan",
        num_items,
        typeid(InputT).name(), typeid(OutputT).name(), (int) sizeof(InputT), (int) sizeof(OutputT),
        (gen_mode == RANDOM) ? "RANDOM" : (gen_mode == INTEGER_SEED) ? "SEQUENTIAL" : "HOMOGENOUS");
    fflush(stdout);

    // Allocate host arrays
    InputT*     h_in        = new InputT[num_items];
    OutputT*    h_reference = new OutputT[num_items];

    // Initialize problem and solution
    Initialize(gen_mode, h_in, num_items);
    Solve(h_in, h_reference, num_items, scan_op, initial_value);

    // Allocate problem device arrays
    InputT *d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(InputT) * num_items));

    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(InputT) * num_items, hipMemcpyHostToDevice));

    // Run Test
    Test<BACKEND>(d_in, h_reference, num_items, scan_op, initial_value);

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_reference) delete[] h_reference;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
}


/**
 * Test DeviceScan on iterator type
 */
template <
    Backend         BACKEND,
    typename        InputT,
    typename        OutputT,
    typename        ScanOpT,
    typename        InitialValueT>
void TestIterator(
    int             num_items,
    ScanOpT         scan_op,
    InitialValueT   initial_value)
{
    printf("\nIterator %s %s hipcub::DeviceScan::%s %d items, %s->%s (%d->%d bytes)\n",
        (BACKEND == CDP) ? "CDP CUB" : (BACKEND == THRUST) ? "Thrust" : "CUB",
        (Equals<InitialValueT, NullType>::VALUE) ? "Inclusive" : "Exclusive",
        (Equals<ScanOpT, Sum>::VALUE) ? "Sum" : "Scan",
        num_items,
        typeid(InputT).name(), typeid(OutputT).name(), (int) sizeof(InputT), (int) sizeof(OutputT));
    fflush(stdout);

    // Use a constant iterator as the input
    InputT val = InputT();
    ConstantInputIterator<InputT, int> h_in(val);

    // Allocate host arrays
    OutputT*  h_reference = new OutputT[num_items];

    // Initialize problem and solution
    Solve(h_in, h_reference, num_items, scan_op, initial_value);

    // Run Test
    Test<BACKEND>(h_in, h_reference, num_items, scan_op, initial_value);

    // Cleanup
    if (h_reference) delete[] h_reference;
}


/**
 * Test different gen modes
 */
template <
    Backend         BACKEND,
    typename        InputT,
    typename        OutputT,
    typename        ScanOpT,
    typename        InitialValueT>
void Test(
    int             num_items,
    ScanOpT         scan_op,
    InitialValueT   initial_value)
{
    TestPointer<BACKEND, InputT, OutputT>(  num_items, UNIFORM, scan_op, initial_value);
    TestPointer<BACKEND, InputT, OutputT>(  num_items, RANDOM,  scan_op, initial_value);
    TestIterator<BACKEND, InputT, OutputT>( num_items, scan_op, initial_value);
}


/**
 * Test different dispatch
 */
template <
    typename        InputT,
    typename        OutputT,
    typename        ScanOpT,
    typename        InitialValueT>
void Test(
    int             num_items,
    ScanOpT         scan_op,
    InitialValueT   initial_value)
{
    Test<CUB, InputT, OutputT>(num_items, scan_op, initial_value);
#ifdef CUB_CDP
    Test<CDP, InputT, OutputT>(num_items, scan_op, initial_value);
#endif
}


/**
 * Test different operators
 */
template <typename InputT, typename OutputT>
void TestOp(
    int             num_items,
    OutputT         identity,
    OutputT         initial_value)
{
    // Exclusive (use identity as initial value because it will dispatch to *Sum variants that don't take initial values)
    Test<InputT, OutputT>(num_items, hipcub::Sum(), identity);
    Test<InputT, OutputT>(num_items, hipcub::Max(), identity);

    // Exclusive (non-specialized, so we can test initial-value)
    Test<InputT, OutputT>(num_items, WrapperFunctor<hipcub::Sum>(hipcub::Sum()), initial_value);
    Test<InputT, OutputT>(num_items, WrapperFunctor<hipcub::Max>(hipcub::Max()), initial_value);

    // Inclusive (no initial value)
    Test<InputT, OutputT>(num_items, hipcub::Sum(), NullType());
    Test<InputT, OutputT>(num_items, hipcub::Max(), NullType());
}


/**
 * Test different input sizes
 */
template <
    typename InputT,
    typename OutputT>
void TestSize(
    int     num_items,
    OutputT identity,
    OutputT initial_value)
{
    if (num_items < 0)
    {
        TestOp<InputT>(0,        identity, initial_value);
        TestOp<InputT>(1,        identity, initial_value);
        TestOp<InputT>(100,      identity, initial_value);
        TestOp<InputT>(10000,    identity, initial_value);
        TestOp<InputT>(1000000,  identity, initial_value);

        // Randomly select problem size between 1:10,000,000
        unsigned int max_int = (unsigned int) -1;
        for (int i = 0; i < 10; ++i)
        {
            unsigned int num_items;
            RandomBits(num_items);
            num_items = (unsigned int) ((double(num_items) * double(10000000)) / double(max_int));
            num_items = CUB_MAX(1, num_items);
            TestOp<InputT>(num_items,  identity, initial_value);
        }
    }
    else
    {
        TestOp<InputT>(num_items, identity, initial_value);
    }
}



//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    int num_items = -1;

    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("n", num_items);
    args.GetCmdLineArgument("i", g_timing_iterations);
    args.GetCmdLineArgument("repeat", g_repeat);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--n=<input items> "
            "[--i=<timing iterations> "
            "[--device=<device-id>] "
            "[--repeat=<repetitions of entire test suite>]"
            "[--v] "
            "[--cdp]"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());
    g_device_giga_bandwidth = args.device_giga_bandwidth;
    printf("\n");

#ifdef QUICKER_TEST

    // Compile/run basic CUB test
    if (num_items < 0) num_items = 32000000;

    TestPointer<CUB, char, int>(         num_items    , UNIFORM, Sum(), (int) (0));
    TestPointer<CUB, int, int>(         num_items    , UNIFORM, Sum(), (int) (0));

#elif defined(QUICK_TEST)

    // Get device ordinal
    int device_ordinal;
    CubDebugExit(hipGetDevice(&device_ordinal));

    // Get device SM version
    int sm_version;
    CubDebugExit(SmVersion(sm_version, device_ordinal));

    // Compile/run quick tests
    if (num_items < 0) num_items = 32000000;

    TestPointer<CUB, char, char>(        num_items * ((sm_version <= 130) ? 1 : 4), UNIFORM, Sum(), char(0));
    TestPointer<THRUST, char, char>(     num_items * ((sm_version <= 130) ? 1 : 4), UNIFORM, Sum(), char(0));

    printf("----------------------------\n");
    TestPointer<CUB, short, short>(       num_items * ((sm_version <= 130) ? 1 : 2), UNIFORM, Sum(), short(0));
    TestPointer<THRUST, short, short>(    num_items * ((sm_version <= 130) ? 1 : 2), UNIFORM, Sum(), short(0));

    printf("----------------------------\n");
    TestPointer<CUB, int, int>(         num_items    , UNIFORM, Sum(), (int) (0));
    TestPointer<THRUST, int, int>(      num_items    , UNIFORM, Sum(), (int) (0));

    printf("----------------------------\n");
    TestPointer<CUB, long long, long long>(   num_items / 2, UNIFORM, Sum(), (long long) (0));
    TestPointer<THRUST, long long, long long>(num_items / 2, UNIFORM, Sum(), (long long) (0));

    printf("----------------------------\n");
    TestPointer<CUB, TestBar, TestBar>(     num_items / 4, UNIFORM, Sum(), TestBar());
    TestPointer<THRUST, TestBar, TestBar>(  num_items / 4, UNIFORM, Sum(), TestBar());

#else

    // Compile/run thorough tests
    for (int i = 0; i <= g_repeat; ++i)
    {
        // Test different input+output data types
        TestSize<unsigned char>(num_items,      (int) 0, (int) 99);

        // Test same intput+output data types
        TestSize<unsigned char>(num_items,      (unsigned char) 0,      (unsigned char) 99);
        TestSize<char>(num_items,               (char) 0,               (char) 99);
        TestSize<unsigned short>(num_items,     (unsigned short) 0,     (unsigned short)99);
        TestSize<unsigned int>(num_items,       (unsigned int) 0,       (unsigned int) 99);
        TestSize<unsigned long long>(num_items, (unsigned long long) 0, (unsigned long long) 99);

        TestSize<uchar2>(num_items,     make_uchar2(0, 0),              make_uchar2(17, 21));
        TestSize<char2>(num_items,      make_char2(0, 0),               make_char2(17, 21));
        TestSize<ushort2>(num_items,    make_ushort2(0, 0),             make_ushort2(17, 21));
        TestSize<uint2>(num_items,      make_uint2(0, 0),               make_uint2(17, 21));
        TestSize<ulonglong2>(num_items, make_ulonglong2(0, 0),          make_ulonglong2(17, 21));
        TestSize<uchar4>(num_items,     make_uchar4(0, 0, 0, 0),        make_uchar4(17, 21, 32, 85));
        TestSize<char4>(num_items,      make_char4(0, 0, 0, 0),         make_char4(17, 21, 32, 85));

        TestSize<ushort4>(num_items,    make_ushort4(0, 0, 0, 0),       make_ushort4(17, 21, 32, 85));
        TestSize<uint4>(num_items,      make_uint4(0, 0, 0, 0),         make_uint4(17, 21, 32, 85));
        TestSize<ulonglong4>(num_items, make_ulonglong4(0, 0, 0, 0),    make_ulonglong4(17, 21, 32, 85));

        TestSize<TestFoo>(num_items,
            TestFoo::MakeTestFoo(0, 0, 0, 0),
            TestFoo::MakeTestFoo(1ll << 63, 1 << 31, short(1 << 15), char(1 << 7)));

        TestSize<TestBar>(num_items,
            TestBar(0, 0),
            TestBar(1ll << 63, 1 << 31));
    }

#endif

    return 0;
}



