#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2016, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of DeviceReduce::ReduceByKey utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <typeinfo>

#include <cub/util_allocator.cuh>
#include <cub/iterator/constant_input_iterator.cuh>
#include <cub/device/device_reduce.cuh>
#include <cub/device/device_run_length_encode.cuh>
#include <cub/thread/thread_operators.cuh>

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/iterator/constant_iterator.h>

#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose           = false;
int                     g_timing_iterations = 0;
int                     g_repeat            = 0;
CachingDeviceAllocator  g_allocator(true);

// Dispatch types
enum Backend
{
    CUB,        // CUB method
    THRUST,     // Thrust method
    CDP,        // GPU-based (dynamic parallelism) dispatch to CUB method
};


//---------------------------------------------------------------------
// Dispatch to different CUB entrypoints
//---------------------------------------------------------------------

/**
 * Dispatch to reduce-by-key entrypoint
 */
template <
    typename                    KeyInputIteratorT,
    typename                    KeyOutputIteratorT,
    typename                    ValueInputIteratorT,
    typename                    ValueOutputIteratorT,
    typename                    NumRunsIteratorT,
    typename                    EqualityOpT,
    typename                    ReductionOpT,
    typename                    OffsetT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t Dispatch(
    Int2Type<CUB>               dispatch_to,
    int                         timing_timing_iterations,
    size_t                      *d_temp_storage_bytes,
    hipError_t                 *d_cdp_error,

    void                        *d_temp_storage,
    size_t                      &temp_storage_bytes,
    KeyInputIteratorT           d_keys_in,
    KeyOutputIteratorT          d_keys_out,
    ValueInputIteratorT         d_values_in,
    ValueOutputIteratorT        d_values_out,
    NumRunsIteratorT            d_num_runs,
    EqualityOpT                  equality_op,
    ReductionOpT                 reduction_op,
    OffsetT                     num_items,
    hipStream_t                stream,
    bool                        debug_synchronous)
{
    hipError_t error = hipSuccess;
    for (int i = 0; i < timing_timing_iterations; ++i)
    {
        error = DeviceReduce::ReduceByKey(
            d_temp_storage,
            temp_storage_bytes,
            d_keys_in,
            d_keys_out,
            d_values_in,
            d_values_out,
            d_num_runs,
            reduction_op,
            num_items,
            stream,
            debug_synchronous);
    }
    return error;
}


//---------------------------------------------------------------------
// Dispatch to different Thrust entrypoints
//---------------------------------------------------------------------

/**
 * Dispatch to reduce-by-key entrypoint
 */
template <
    typename                    KeyInputIteratorT,
    typename                    KeyOutputIteratorT,
    typename                    ValueInputIteratorT,
    typename                    ValueOutputIteratorT,
    typename                    NumRunsIteratorT,
    typename                    EqualityOpT,
    typename                    ReductionOpT,
    typename                    OffsetT>
hipError_t Dispatch(
    Int2Type<THRUST>            dispatch_to,
    int                         timing_timing_iterations,
    size_t                      *d_temp_storage_bytes,
    hipError_t                 *d_cdp_error,

    void                        *d_temp_storage,
    size_t                      &temp_storage_bytes,
    KeyInputIteratorT           d_keys_in,
    KeyOutputIteratorT          d_keys_out,
    ValueInputIteratorT         d_values_in,
    ValueOutputIteratorT        d_values_out,
    NumRunsIteratorT            d_num_runs,
    EqualityOpT                 equality_op,
    ReductionOpT                reduction_op,
    OffsetT                     num_items,
    hipStream_t                stream,
    bool                        debug_synchronous)
{
    // The input keys type
    typedef typename std::iterator_traits<KeyInputIteratorT>::value_type KeyInputT;

    // The output keys type
    typedef typename If<(Equals<typename std::iterator_traits<KeyOutputIteratorT>::value_type, void>::VALUE),   // OutputT =  (if output iterator's value type is void) ?
        typename std::iterator_traits<KeyInputIteratorT>::value_type,                                           // ... then the input iterator's value type,
        typename std::iterator_traits<KeyOutputIteratorT>::value_type>::Type KeyOutputT;                        // ... else the output iterator's value type

    // The input values type
    typedef typename std::iterator_traits<ValueInputIteratorT>::value_type ValueInputT;

    // The output values type
    typedef typename If<(Equals<typename std::iterator_traits<ValueOutputIteratorT>::value_type, void>::VALUE), // OutputT =  (if output iterator's value type is void) ?
        typename std::iterator_traits<ValueInputIteratorT>::value_type,                                         // ... then the input iterator's value type,
        typename std::iterator_traits<ValueOutputIteratorT>::value_type>::Type ValueOuputT;                     // ... else the output iterator's value type

    if (d_temp_storage == 0)
    {
        temp_storage_bytes = 1;
    }
    else
    {
        thrust::device_ptr<KeyInputT> d_keys_in_wrapper(d_keys_in);
        thrust::device_ptr<KeyOutputT> d_keys_out_wrapper(d_keys_out);

        thrust::device_ptr<ValueInputT> d_values_in_wrapper(d_values_in);
        thrust::device_ptr<ValueOuputT> d_values_out_wrapper(d_values_out);

        thrust::pair<thrust::device_ptr<KeyOutputT>, thrust::device_ptr<ValueOuputT> > d_out_ends;

        for (int i = 0; i < timing_timing_iterations; ++i)
        {
            d_out_ends = thrust::reduce_by_key(
                d_keys_in_wrapper,
                d_keys_in_wrapper + num_items,
                d_values_in_wrapper,
                d_keys_out_wrapper,
                d_values_out_wrapper);
        }

        OffsetT num_segments = d_out_ends.first - d_keys_out_wrapper;
        CubDebugExit(hipMemcpy(d_num_runs, &num_segments, sizeof(OffsetT), hipMemcpyHostToDevice));

    }

    return hipSuccess;
}



//---------------------------------------------------------------------
// CUDA Nested Parallelism Test Kernel
//---------------------------------------------------------------------

/**
 * Simple wrapper kernel to invoke DeviceSelect
 */
template <
    typename                    KeyInputIteratorT,
    typename                    KeyOutputIteratorT,
    typename                    ValueInputIteratorT,
    typename                    ValueOutputIteratorT,
    typename                    NumRunsIteratorT,
    typename                    EqualityOpT,
    typename                    ReductionOpT,
    typename                    OffsetT>
__global__ void CnpDispatchKernel(
    int                         timing_timing_iterations,
    size_t                      *d_temp_storage_bytes,
    hipError_t                 *d_cdp_error,

    void                        *d_temp_storage,
    size_t                      temp_storage_bytes,
    KeyInputIteratorT           d_keys_in,
    KeyOutputIteratorT          d_keys_out,
    ValueInputIteratorT         d_values_in,
    ValueOutputIteratorT        d_values_out,
    NumRunsIteratorT            d_num_runs,
    EqualityOpT                 equality_op,
    ReductionOpT                reduction_op,
    OffsetT                     num_items,
    hipStream_t                stream,
    bool                        debug_synchronous)
{

#ifndef CUB_CDP
    *d_cdp_error = hipErrorNotSupported;
#else
    *d_cdp_error = Dispatch(Int2Type<CUB>(), timing_timing_iterations, d_temp_storage_bytes, d_cdp_error,
        d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in, d_values_out, d_num_runs, equality_op, reduction_op, num_items, 0, debug_synchronous);

    *d_temp_storage_bytes = temp_storage_bytes;
#endif
}


/**
 * Dispatch to CDP kernel
 */
template <
    typename                    KeyInputIteratorT,
    typename                    KeyOutputIteratorT,
    typename                    ValueInputIteratorT,
    typename                    ValueOutputIteratorT,
    typename                    NumRunsIteratorT,
    typename                    EqualityOpT,
    typename                    ReductionOpT,
    typename                    OffsetT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t Dispatch(
    Int2Type<CDP>               dispatch_to,
    int                         timing_timing_iterations,
    size_t                      *d_temp_storage_bytes,
    hipError_t                 *d_cdp_error,

    void                        *d_temp_storage,
    size_t                      &temp_storage_bytes,
    KeyInputIteratorT           d_keys_in,
    KeyOutputIteratorT          d_keys_out,
    ValueInputIteratorT         d_values_in,
    ValueOutputIteratorT        d_values_out,
    NumRunsIteratorT            d_num_runs,
    EqualityOpT                 equality_op,
    ReductionOpT                reduction_op,
    OffsetT                     num_items,
    hipStream_t                stream,
    bool                        debug_synchronous)
{
    // Invoke kernel to invoke device-side dispatch
    CnpDispatchKernel<<<1,1>>>(timing_timing_iterations, d_temp_storage_bytes, d_cdp_error,
        d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in, d_values_out, d_num_runs, equality_op, reduction_op, num_items, 0, debug_synchronous);

    // Copy out temp_storage_bytes
    CubDebugExit(hipMemcpy(&temp_storage_bytes, d_temp_storage_bytes, sizeof(size_t) * 1, hipMemcpyDeviceToHost));

    // Copy out error
    hipError_t retval;
    CubDebugExit(hipMemcpy(&retval, d_cdp_error, sizeof(hipError_t) * 1, hipMemcpyDeviceToHost));
    return retval;
}



//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------


/**
 * Initialize problem
 */
template <typename T>
void Initialize(
    int         entropy_reduction,
    T           *h_in,
    int         num_items,
    int         max_segment)
{
    unsigned int max_int = (unsigned int) -1;

    int key = 0;
    int i = 0;
    while (i < num_items)
    {
        // Select number of repeating occurrences

        int repeat;

        if (max_segment < 0)
        {
            repeat = num_items;
        }
        else if (max_segment < 2)
        {
            repeat = 1;
        }
        else
        {
            RandomBits(repeat, entropy_reduction);
            repeat = (int) ((double(repeat) * double(max_segment)) / double(max_int));
            repeat = CUB_MAX(1, repeat);
        }

        int j = i;
        while (j < CUB_MIN(i + repeat, num_items))
        {
            InitValue(INTEGER_SEED, h_in[j], key);
            j++;
        }

        i = j;
        key++;
    }

    if (g_verbose)
    {
        printf("Input:\n");
        DisplayResults(h_in, num_items);
        printf("\n\n");
    }
}


/**
 * Solve problem.  Returns total number of segments identified
 */
template <
    typename        KeyInputIteratorT,
    typename        ValueInputIteratorT,
    typename        KeyT,
    typename        ValueT,
    typename        EqualityOpT,
    typename        ReductionOpT>
int Solve(
    KeyInputIteratorT       h_keys_in,
    KeyT                    *h_keys_reference,
    ValueInputIteratorT     h_values_in,
    ValueT                  *h_values_reference,
    EqualityOpT             equality_op,
    ReductionOpT            reduction_op,
    int                     num_items)
{
    // First item
    KeyT previous        = h_keys_in[0];
    ValueT aggregate     = h_values_in[0];
    int num_segments    = 0;

    // Subsequent items
    for (int i = 1; i < num_items; ++i)
    {
        if (!equality_op(previous, h_keys_in[i]))
        {
            h_keys_reference[num_segments] = previous;
            h_values_reference[num_segments] = aggregate;
            num_segments++;
            aggregate = h_values_in[i];
        }
        else
        {
            aggregate = reduction_op(aggregate, h_values_in[i]);
        }
        previous = h_keys_in[i];
    }

    h_keys_reference[num_segments] = previous;
    h_values_reference[num_segments] = aggregate;
    num_segments++;

    return num_segments;
}



/**
 * Test DeviceSelect for a given problem input
 */
template <
    Backend             BACKEND,
    typename            DeviceKeyInputIteratorT,
    typename            DeviceValueInputIteratorT,
    typename            KeyT,
    typename            ValueT,
    typename            EqualityOpT,
    typename            ReductionOpT>
void Test(
    DeviceKeyInputIteratorT     d_keys_in,
    DeviceValueInputIteratorT   d_values_in,
    KeyT*                       h_keys_reference,
    ValueT*                     h_values_reference,
    EqualityOpT                 equality_op,
    ReductionOpT                reduction_op,
    int                         num_segments,
    int                         num_items)
{
    // Allocate device output arrays and number of segments
    KeyT*   d_keys_out             = NULL;
    ValueT* d_values_out           = NULL;
    int*    d_num_runs         = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys_out, sizeof(KeyT) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values_out, sizeof(ValueT) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_num_runs, sizeof(int)));

    // Allocate CDP device arrays
    size_t          *d_temp_storage_bytes = NULL;
    hipError_t     *d_cdp_error = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_temp_storage_bytes,  sizeof(size_t) * 1));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_cdp_error,           sizeof(hipError_t) * 1));

    // Allocate temporary storage
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;
    CubDebugExit(Dispatch(Int2Type<BACKEND>(), 1, d_temp_storage_bytes, d_cdp_error, d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in, d_values_out, d_num_runs, equality_op, reduction_op, num_items, 0, true));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Clear device output arrays
    CubDebugExit(hipMemset(d_keys_out, 0, sizeof(KeyT) * num_items));
    CubDebugExit(hipMemset(d_values_out, 0, sizeof(ValueT) * num_items));
    CubDebugExit(hipMemset(d_num_runs, 0, sizeof(int)));

    // Run warmup/correctness iteration
    CubDebugExit(Dispatch(Int2Type<BACKEND>(), 1, d_temp_storage_bytes, d_cdp_error, d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in, d_values_out, d_num_runs, equality_op, reduction_op, num_items, 0, true));

    // Check for correctness (and display results, if specified)
    int compare1 = CompareDeviceResults(h_keys_reference, d_keys_out, num_segments, true, g_verbose);
    printf("\t Keys %s ", compare1 ? "FAIL" : "PASS");

    int compare2 = CompareDeviceResults(h_values_reference, d_values_out, num_segments, true, g_verbose);
    printf("\t Values %s ", compare2 ? "FAIL" : "PASS");

    int compare3 = CompareDeviceResults(&num_segments, d_num_runs, 1, true, g_verbose);
    printf("\t Count %s ", compare3 ? "FAIL" : "PASS");

    // Flush any stdout/stderr
    fflush(stdout);
    fflush(stderr);

    // Performance
    GpuTimer gpu_timer;
    gpu_timer.Start();
    CubDebugExit(Dispatch(Int2Type<BACKEND>(), g_timing_iterations, d_temp_storage_bytes, d_cdp_error, d_temp_storage, temp_storage_bytes, d_keys_in, d_keys_out, d_values_in, d_values_out, d_num_runs, equality_op, reduction_op, num_items, 0, false));
    gpu_timer.Stop();
    float elapsed_millis = gpu_timer.ElapsedMillis();

    // Display performance
    if (g_timing_iterations > 0)
    {
        float   avg_millis  = elapsed_millis / g_timing_iterations;
        float   giga_rate   = float(num_items) / avg_millis / 1000.0f / 1000.0f;
        int     bytes_moved = ((num_items + num_segments) * sizeof(KeyT)) + ((num_items + num_segments) * sizeof(ValueT));
        float   giga_bandwidth  = float(bytes_moved) / avg_millis / 1000.0f / 1000.0f;
        printf(", %.3f avg ms, %.3f billion items/s, %.3f logical GB/s", avg_millis, giga_rate, giga_bandwidth);
    }
    printf("\n\n");

    // Flush any stdout/stderr
    fflush(stdout);
    fflush(stderr);

    // Cleanup
    if (d_keys_out) CubDebugExit(g_allocator.DeviceFree(d_keys_out));
    if (d_values_out) CubDebugExit(g_allocator.DeviceFree(d_values_out));
    if (d_num_runs) CubDebugExit(g_allocator.DeviceFree(d_num_runs));
    if (d_temp_storage_bytes) CubDebugExit(g_allocator.DeviceFree(d_temp_storage_bytes));
    if (d_cdp_error) CubDebugExit(g_allocator.DeviceFree(d_cdp_error));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

    // Correctness asserts
    AssertEquals(0, compare1 | compare2 | compare3);
}


/**
 * Test DeviceSelect on pointer type
 */
template <
    Backend         BACKEND,
    typename        KeyT,
    typename        ValueT,
    typename        ReductionOpT>
void TestPointer(
    int             num_items,
    int             entropy_reduction,
    int             max_segment,
    ReductionOpT    reduction_op)
{
    // Allocate host arrays
    KeyT* h_keys_in        = new KeyT[num_items];
    KeyT* h_keys_reference = new KeyT[num_items];

    ValueT* h_values_in        = new ValueT[num_items];
    ValueT* h_values_reference = new ValueT[num_items];

    for (int i = 0; i < num_items; ++i)
        InitValue(INTEGER_SEED, h_values_in[i], 1);

    // Initialize problem and solution
    Equality equality_op;
    Initialize(entropy_reduction, h_keys_in, num_items, max_segment);
    int num_segments = Solve(h_keys_in, h_keys_reference, h_values_in, h_values_reference, equality_op, reduction_op, num_items);

    printf("\nPointer %s hipcub::DeviceReduce::ReduceByKey %s reduction of %d items, %d segments (avg run length %.3f), {%s,%s} key value pairs, max_segment %d, entropy_reduction %d\n",
        (BACKEND == CDP) ? "CDP CUB" : (BACKEND == THRUST) ? "Thrust" : "CUB",
        (Equals<ReductionOpT, Sum>::VALUE) ? "Sum" : "Max",
        num_items, num_segments, float(num_items) / num_segments,
        typeid(KeyT).name(), typeid(ValueT).name(),
        max_segment, entropy_reduction);
    fflush(stdout);

    // Allocate problem device arrays
    KeyT     *d_keys_in = NULL;
    ValueT   *d_values_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys_in, sizeof(KeyT) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values_in, sizeof(ValueT) * num_items));

    // Initialize device input
    CubDebugExit(hipMemcpy(d_keys_in, h_keys_in, sizeof(KeyT) * num_items, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_values_in, h_values_in, sizeof(ValueT) * num_items, hipMemcpyHostToDevice));

    // Run Test
    Test<BACKEND>(d_keys_in, d_values_in, h_keys_reference, h_values_reference, equality_op, reduction_op, num_segments, num_items);

    // Cleanup
    if (h_keys_in) delete[] h_keys_in;
    if (h_values_in) delete[] h_values_in;
    if (h_keys_reference) delete[] h_keys_reference;
    if (h_values_reference) delete[] h_values_reference;
    if (d_keys_in) CubDebugExit(g_allocator.DeviceFree(d_keys_in));
    if (d_values_in) CubDebugExit(g_allocator.DeviceFree(d_values_in));
}


/**
 * Test on iterator type
 */
template <
    Backend         BACKEND,
    typename        KeyT,
    typename        ValueT,
    typename        ReductionOpT>
void TestIterator(
    int             num_items,
    int             entropy_reduction,
    int             max_segment,
    ReductionOpT    reduction_op)
{
    // Allocate host arrays
    KeyT* h_keys_in        = new KeyT[num_items];
    KeyT* h_keys_reference = new KeyT[num_items];

    ValueT one_val;
    InitValue(INTEGER_SEED, one_val, 1);
    ConstantInputIterator<ValueT, int> h_values_in(one_val);
    ValueT* h_values_reference = new ValueT[num_items];

    // Initialize problem and solution
    Equality equality_op;
    Initialize(entropy_reduction, h_keys_in, num_items, max_segment);
    int num_segments = Solve(h_keys_in, h_keys_reference, h_values_in, h_values_reference, equality_op, reduction_op, num_items);

    printf("\nIterator %s hipcub::DeviceReduce::ReduceByKey %s reduction of %d items, %d segments (avg run length %.3f), {%s,%s} key value pairs, max_segment %d, entropy_reduction %d\n",
        (BACKEND == CDP) ? "CDP CUB" : (BACKEND == THRUST) ? "Thrust" : "CUB",
        (Equals<ReductionOpT, Sum>::VALUE) ? "Sum" : "Max",
        num_items, num_segments, float(num_items) / num_segments,
        typeid(KeyT).name(), typeid(ValueT).name(),
        max_segment, entropy_reduction);
    fflush(stdout);

    // Allocate problem device arrays
    KeyT     *d_keys_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys_in, sizeof(KeyT) * num_items));

    // Initialize device input
    CubDebugExit(hipMemcpy(d_keys_in, h_keys_in, sizeof(KeyT) * num_items, hipMemcpyHostToDevice));

    // Run Test
    Test<BACKEND>(d_keys_in, h_values_in, h_keys_reference, h_values_reference, equality_op, reduction_op, num_segments, num_items);

    // Cleanup
    if (h_keys_in) delete[] h_keys_in;
    if (h_keys_reference) delete[] h_keys_reference;
    if (h_values_reference) delete[] h_values_reference;
    if (d_keys_in) CubDebugExit(g_allocator.DeviceFree(d_keys_in));
}


/**
 * Test different gen modes
 */
template <
    Backend         BACKEND,
    typename        KeyT,
    typename        ValueT,
    typename        ReductionOpT>
void Test(
    int             num_items,
    ReductionOpT    reduction_op,
    int             max_segment)
{
    // 0 key-bit entropy reduction rounds
    TestPointer<BACKEND, KeyT, ValueT>(num_items, 0, max_segment, reduction_op);

    if (max_segment > 1)
    {
        // 2 key-bit entropy reduction rounds
        TestPointer<BACKEND, KeyT, ValueT>(num_items, 2, max_segment, reduction_op);

        // 7 key-bit entropy reduction rounds
        TestPointer<BACKEND, KeyT, ValueT>(num_items, 7, max_segment, reduction_op);
    }
}


/**
 * Test different avg segment lengths modes
 */
template <
    Backend         BACKEND,
    typename        KeyT,
    typename        ValueT,
    typename        ReductionOpT>
void Test(
    int             num_items,
    ReductionOpT    reduction_op)
{
    Test<BACKEND, KeyT, ValueT>(num_items, reduction_op, -1);
    Test<BACKEND, KeyT, ValueT>(num_items, reduction_op, 1);

    // Evaluate different max-segment lengths
    for (int max_segment = 3; max_segment < CUB_MIN(num_items, (unsigned short) -1); max_segment *= 11)
    {
        Test<BACKEND, KeyT, ValueT>(num_items, reduction_op, max_segment);
    }
}



/**
 * Test different dispatch
 */
template <
    typename        KeyT,
    typename        ValueT,
    typename        ReductionOpT>
void TestDispatch(
    int             num_items,
    ReductionOpT    reduction_op)
{
    Test<CUB, KeyT, ValueT>(num_items, reduction_op);
#ifdef CUB_CDP
    Test<CDP, KeyT, ValueT>(num_items, reduction_op);
#endif
}


/**
 * Test different input sizes
 */
template <
    typename        KeyT,
    typename        ValueT,
    typename        ReductionOpT>
void TestSize(
    int             num_items,
    ReductionOpT    reduction_op)
{
    if (num_items < 0)
    {
        TestDispatch<KeyT, ValueT>(1,        reduction_op);
        TestDispatch<KeyT, ValueT>(100,      reduction_op);
        TestDispatch<KeyT, ValueT>(10000,    reduction_op);
        TestDispatch<KeyT, ValueT>(1000000,  reduction_op);
    }
    else
    {
        TestDispatch<KeyT, ValueT>(num_items, reduction_op);
    }

}


template <
    typename        KeyT,
    typename        ValueT>
void TestOp(
    int             num_items)
{
    TestSize<KeyT, ValueT>(num_items, hipcub::Sum());
    TestSize<KeyT, ValueT>(num_items, hipcub::Max());
}



//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    int num_items           = -1;
    int entropy_reduction   = 0;
    int maxseg              = 1000;

    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("n", num_items);
    args.GetCmdLineArgument("i", g_timing_iterations);
    args.GetCmdLineArgument("repeat", g_repeat);
    args.GetCmdLineArgument("maxseg", maxseg);
    args.GetCmdLineArgument("entropy", entropy_reduction);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--n=<input items> "
            "[--i=<timing iterations> "
            "[--device=<device-id>] "
            "[--maxseg=<max segment length>]"
            "[--entropy=<segment length bit entropy reduction rounds>]"
            "[--repeat=<repetitions of entire test suite>]"
            "[--v] "
            "[--cdp]"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());
    printf("\n");

    // Get ptx version
    int ptx_version;
    CubDebugExit(PtxVersion(ptx_version));

#ifdef QUICKER_TEST

    // Compile/run basic CUB test
    if (num_items < 0) num_items = 32000000;

    TestPointer<CUB, int, double>(num_items, entropy_reduction, maxseg, hipcub::Sum());
    TestPointer<CUB, int, int>(num_items, entropy_reduction, maxseg, hipcub::Sum());
    TestIterator<CUB, int, int>(num_items, entropy_reduction, maxseg, hipcub::Sum());

#elif defined(QUICK_TEST)

    // Compile/run quick tests
    if (num_items < 0) num_items = 32000000;

    printf("---- RLE int ---- \n");
    TestIterator<CUB, int, int>(num_items, entropy_reduction, maxseg, hipcub::Sum());

    printf("---- RLE long long ---- \n");
    TestIterator<CUB, long long, int>(num_items, entropy_reduction, maxseg, hipcub::Sum());

    printf("---- int ---- \n");
    TestPointer<CUB, int, int>(num_items, entropy_reduction, maxseg, hipcub::Sum());
    TestPointer<THRUST, int, int>(num_items, entropy_reduction, maxseg, hipcub::Sum());

    printf("---- float ---- \n");
    TestPointer<CUB, int, float>(num_items, entropy_reduction, maxseg, hipcub::Sum());
    TestPointer<THRUST, int, float>(num_items, entropy_reduction, maxseg, hipcub::Sum());

    if (ptx_version > 120)                          // Don't check doubles on PTX120 or below because they're down-converted
    {
        printf("---- double ---- \n");
        TestPointer<CUB, int, double>(num_items, entropy_reduction, maxseg, hipcub::Sum());
        TestPointer<THRUST, int, double>(num_items, entropy_reduction, maxseg, hipcub::Sum());
    }

#else

    // Compile/run thorough tests
    for (int i = 0; i <= g_repeat; ++i)
    {

        // Test different input types
        TestOp<int, char>(num_items);
        TestOp<int, short>(num_items);
        TestOp<int, int>(num_items);
        TestOp<int, long>(num_items);
        TestOp<int, long long>(num_items);
        TestOp<int, float>(num_items);
        if (ptx_version > 120)                          // Don't check doubles on PTX120 or below because they're down-converted
            TestOp<int, double>(num_items);

        TestOp<int, uchar2>(num_items);
        TestOp<int, uint2>(num_items);
        TestOp<int, uint3>(num_items);
        TestOp<int, uint4>(num_items);
        TestOp<int, ulonglong4>(num_items);
        TestOp<int, TestFoo>(num_items);
        TestOp<int, TestBar>(num_items);

        TestOp<char, int>(num_items);
        TestOp<long long, int>(num_items);
        TestOp<TestFoo, int>(num_items);
        TestOp<TestBar, int>(num_items);

    }

#endif

    return 0;
}



