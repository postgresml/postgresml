#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2016, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of WarpScan utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <typeinfo>

#include <cub/warp/warp_scan.cuh>
#include <cub/util_allocator.cuh>

#include "test_util.h"

using namespace hipcub;

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose       = false;
int                     g_repeat        = 0;
CachingDeviceAllocator  g_allocator(true);


/**
 * Primitive variant to test
 */
enum TestMode
{
    BASIC,
    AGGREGATE,
};



/**
 * \brief WrapperFunctor (for precluding test-specialized dispatch to *Sum variants)
 */
template<typename OpT>
struct WrapperFunctor
{
    OpT op;

    WrapperFunctor(OpT op) : op(op) {}

    template <typename T>
    __host__ __device__ __forceinline__ T operator()(const T &a, const T &b) const
    {
        return op(a, b);
    }
};

//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------

/// Exclusive scan basic
template <typename WarpScanT, typename T, typename ScanOpT, typename IsPrimitiveT>
__device__ __forceinline__ void DeviceTest(
    WarpScanT                       &warp_scan,
    T                               &data,
    T                               &initial_value,
    ScanOpT                         &scan_op,
    T                               &aggregate,
    Int2Type<BASIC>                 test_mode,
    IsPrimitiveT                    is_primitive)
{
    // Test basic warp scan
    warp_scan.ExclusiveScan(data, data, initial_value, scan_op);
}

/// Exclusive scan aggregate
template <
    typename    WarpScanT,
    typename    T,
    typename    ScanOpT,
    typename    IsPrimitiveT>
__device__ __forceinline__ void DeviceTest(
    WarpScanT                       &warp_scan,
    T                               &data,
    T                               &initial_value,
    ScanOpT                         &scan_op,
    T                               &aggregate,
    Int2Type<AGGREGATE>             test_mode,
    IsPrimitiveT                    is_primitive)
{
    // Test with cumulative aggregate
    warp_scan.ExclusiveScan(data, data, initial_value, scan_op, aggregate);
}


/// Exclusive sum basic
template <
    typename    WarpScanT,
    typename    T>
__device__ __forceinline__ void DeviceTest(
    WarpScanT                       &warp_scan,
    T                               &data,
    T                               &initial_value,
    Sum                             &scan_op,
    T                               &aggregate,
    Int2Type<BASIC>                 test_mode,
    Int2Type<true>                  is_primitive)
{
    // Test basic warp scan
    warp_scan.ExclusiveSum(data, data);
}


/// Exclusive sum aggregate
template <
    typename    WarpScanT,
    typename    T>
__device__ __forceinline__ void DeviceTest(
    WarpScanT                       &warp_scan,
    T                               &data,
    T                               &initial_value,
    Sum                             &scan_op,
    T                               &aggregate,
    Int2Type<AGGREGATE>             test_mode,
    Int2Type<true>                  is_primitive)
{
    // Test with cumulative aggregate
    warp_scan.ExclusiveSum(data, data, aggregate);
}


/// Inclusive scan basic
template <
    typename    WarpScanT,
    typename    T,
    typename    ScanOpT,
    typename    IsPrimitiveT>
__device__ __forceinline__ void DeviceTest(
    WarpScanT                       &warp_scan,
    T                               &data,
    NullType                        &initial_value,
    ScanOpT                         &scan_op,
    T                               &aggregate,
    Int2Type<BASIC>                 test_mode,
    IsPrimitiveT                    is_primitive)
{
    // Test basic warp scan
    warp_scan.InclusiveScan(data, data, scan_op);
}

/// Inclusive scan aggregate
template <
    typename    WarpScanT,
    typename    T,
    typename    ScanOpT,
    typename    IsPrimitiveT>
__device__ __forceinline__ void DeviceTest(
    WarpScanT                       &warp_scan,
    T                               &data,
    NullType                        &initial_value,
    ScanOpT                         &scan_op,
    T                               &aggregate,
    Int2Type<AGGREGATE>             test_mode,
    IsPrimitiveT                    is_primitive)
{
    // Test with cumulative aggregate
    warp_scan.InclusiveScan(data, data, scan_op, aggregate);
}

/// Inclusive sum basic
template <
    typename    WarpScanT,
    typename    T,
    typename    InitialValueT>
__device__ __forceinline__ void DeviceTest(
    WarpScanT                       &warp_scan,
    T                               &data,
    NullType                        &initial_value,
    Sum                             &scan_op,
    T                               &aggregate,
    Int2Type<BASIC>                 test_mode,
    Int2Type<true>                  is_primitive)
{
    // Test basic warp scan
    warp_scan.InclusiveSum(data, data);
}

/// Inclusive sum aggregate
template <
    typename    WarpScanT,
    typename    T,
    typename    InitialValueT>
__device__ __forceinline__ void DeviceTest(
    WarpScanT                       &warp_scan,
    T                               &data,
    NullType                        &initial_value,
    Sum                             &scan_op,
    T                               &aggregate,
    Int2Type<AGGREGATE>             test_mode,
    Int2Type<true>                  is_primitive)
{
    // Test with cumulative aggregate
    warp_scan.InclusiveSum(data, data, aggregate);
}


/**
 * WarpScan test kernel
 */
template <
    int         LOGICAL_WARP_THREADS,
    TestMode    TEST_MODE,
    typename    T,
    typename    ScanOpT,
    typename    InitialValueT>
__global__ void WarpScanKernel(
    T               *d_in,
    T               *d_out,
    T               *d_aggregate,
    ScanOpT         scan_op,
    InitialValueT   initial_value,
    clock_t         *d_elapsed)
{
    // Cooperative warp-scan utility type (1 warp)
    typedef WarpScan<T, LOGICAL_WARP_THREADS> WarpScanT;

    // Allocate temp storage in shared memory
    __shared__ typename WarpScanT::TempStorage temp_storage;

    // Per-thread tile data
    T data = d_in[threadIdx.x];

    // Start cycle timer
    __threadfence_block();      // workaround to prevent clock hoisting
    clock_t start = clock();
    __threadfence_block();      // workaround to prevent clock hoisting

    T aggregate;

    // Test scan
    WarpScanT warp_scan(temp_storage);
    DeviceTest(
        warp_scan,
        data,
        initial_value,
        scan_op,
        aggregate,
        Int2Type<TEST_MODE>(),
        Int2Type<Traits<T>::PRIMITIVE>());

    // Stop cycle timer
    __threadfence_block();      // workaround to prevent clock hoisting
    clock_t stop = clock();
    __threadfence_block();      // workaround to prevent clock hoisting

    // Store data
    d_out[threadIdx.x] = data;

    if (TEST_MODE != BASIC)
    {
        // Store aggregate
        d_aggregate[threadIdx.x] = aggregate;
    }

    // Store time
    if (threadIdx.x == 0)
    {
        *d_elapsed = (start > stop) ? start - stop : stop - start;
    }
}


//---------------------------------------------------------------------
// Host utility subroutines
//---------------------------------------------------------------------

/**
 * Initialize exclusive-scan problem (and solution)
 */
template <
    typename        T,
    typename        ScanOpT>
T Initialize(
    GenMode         gen_mode,
    T               *h_in,
    T               *h_reference,
    int             num_items,
    ScanOpT         scan_op,
    T               initial_value)
{
    InitValue(gen_mode, h_in[0], 0);

    T block_aggregate   = h_in[0];
    h_reference[0]      = initial_value;
    T inclusive         = scan_op(initial_value, h_in[0]);

    for (int i = 1; i < num_items; ++i)
    {
        InitValue(gen_mode, h_in[i], i);
        h_reference[i] = inclusive;
        inclusive = scan_op(inclusive, h_in[i]);
        block_aggregate = scan_op(block_aggregate, h_in[i]);
    }

    return block_aggregate;
}


/**
 * Initialize inclusive-scan problem (and solution)
 */
template <
    typename    T,
    typename    ScanOpT>
T Initialize(
    GenMode     gen_mode,
    T           *h_in,
    T           *h_reference,
    int         num_items,
    ScanOpT     scan_op,
    NullType)
{
    InitValue(gen_mode, h_in[0], 0);

    T block_aggregate   = h_in[0];
    T inclusive         = h_in[0];
    h_reference[0]      = inclusive;

    for (int i = 1; i < num_items; ++i)
    {
        InitValue(gen_mode, h_in[i], i);
        inclusive = scan_op(inclusive, h_in[i]);
        block_aggregate = scan_op(block_aggregate, h_in[i]);
        h_reference[i] = inclusive;
    }

    return block_aggregate;
}


/**
 * Test warp scan
 */
template <
    int             LOGICAL_WARP_THREADS,
    TestMode        TEST_MODE,
    typename        T,
    typename        ScanOpT,
    typename        InitialValueT>        // NullType implies inclusive-scan, otherwise inclusive scan
void Test(
    GenMode         gen_mode,
    ScanOpT         scan_op,
    InitialValueT   initial_value)
{
    // Allocate host arrays
    T *h_in = new T[LOGICAL_WARP_THREADS];
    T *h_reference = new T[LOGICAL_WARP_THREADS];
    T *h_aggregate = new T[LOGICAL_WARP_THREADS];

    // Initialize problem
    T aggregate = Initialize(
        gen_mode,
        h_in,
        h_reference,
        LOGICAL_WARP_THREADS,
        scan_op,
        initial_value);

    if (g_verbose)
    {
        printf("Input: \n");
        DisplayResults(h_in, LOGICAL_WARP_THREADS);
        printf("\n");
    }

    for (int i = 0; i < LOGICAL_WARP_THREADS; ++i)
    {
        h_aggregate[i] = aggregate;
    }

    // Initialize/clear device arrays
    T *d_in = NULL;
    T *d_out = NULL;
    T *d_aggregate = NULL;
    clock_t *d_elapsed = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(T) * LOGICAL_WARP_THREADS));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(T) * (LOGICAL_WARP_THREADS + 1)));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_aggregate, sizeof(T) * LOGICAL_WARP_THREADS));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_elapsed, sizeof(clock_t)));
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * LOGICAL_WARP_THREADS, hipMemcpyHostToDevice));
    CubDebugExit(hipMemset(d_out, 0, sizeof(T) * (LOGICAL_WARP_THREADS + 1)));
    CubDebugExit(hipMemset(d_aggregate, 0, sizeof(T) * LOGICAL_WARP_THREADS));

    // Run kernel
    printf("Test-mode %d (%s), gen-mode %d (%s), %s warpscan, %d warp threads, %s (%d bytes) elements:\n",
        TEST_MODE, typeid(TEST_MODE).name(),
        gen_mode, typeid(gen_mode).name(),
        (Equals<InitialValueT, NullType>::VALUE) ? "Inclusive" : "Exclusive",
        LOGICAL_WARP_THREADS,
        typeid(T).name(),
        (int) sizeof(T));
    fflush(stdout);

    // Run aggregate/prefix kernel
    WarpScanKernel<LOGICAL_WARP_THREADS, TEST_MODE><<<1, LOGICAL_WARP_THREADS>>>(
        d_in,
        d_out,
        d_aggregate,
        scan_op,
        initial_value,
        d_elapsed);

    printf("\tElapsed clocks: ");
    DisplayDeviceResults(d_elapsed, 1);

    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());

    // Copy out and display results
    printf("\tScan results: ");
    int compare = CompareDeviceResults(h_reference, d_out, LOGICAL_WARP_THREADS, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Copy out and display aggregate
    if (TEST_MODE == AGGREGATE)
    {
        printf("\tScan aggregate: ");
        compare = CompareDeviceResults(h_aggregate, d_aggregate, LOGICAL_WARP_THREADS, g_verbose, g_verbose);
        printf("%s\n", compare ? "FAIL" : "PASS");
        AssertEquals(0, compare);
    }

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_reference) delete[] h_reference;
    if (h_aggregate) delete[] h_aggregate;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_aggregate) CubDebugExit(g_allocator.DeviceFree(d_aggregate));
    if (d_elapsed) CubDebugExit(g_allocator.DeviceFree(d_elapsed));
}


/**
 * Run battery of tests for different primitive variants
 */
template <
    int         LOGICAL_WARP_THREADS,
    typename    ScanOpT,
    typename    T>
void Test(
    GenMode     gen_mode,
    ScanOpT     scan_op,
    T           initial_value)
{
    // Exclusive
    Test<LOGICAL_WARP_THREADS, BASIC, T>(gen_mode, scan_op, T());
    Test<LOGICAL_WARP_THREADS, AGGREGATE, T>(gen_mode, scan_op, T());

    // Exclusive (non-specialized, so we can use initial-value)
    Test<LOGICAL_WARP_THREADS, BASIC, T>(gen_mode, WrapperFunctor<ScanOpT>(scan_op), initial_value);
    Test<LOGICAL_WARP_THREADS, AGGREGATE, T>(gen_mode, WrapperFunctor<ScanOpT>(scan_op), initial_value);

    // Inclusive
    Test<LOGICAL_WARP_THREADS, BASIC, T>(gen_mode, scan_op, NullType());
    Test<LOGICAL_WARP_THREADS, AGGREGATE, T>(gen_mode, scan_op, NullType());
}


/**
 * Run battery of tests for different data types and scan ops
 */
template <int LOGICAL_WARP_THREADS>
void Test(GenMode gen_mode)
{
    // Get device ordinal
    int device_ordinal;
    CubDebugExit(hipGetDevice(&device_ordinal));

    // Get ptx version
    int ptx_version;
    CubDebugExit(PtxVersion(ptx_version));

    // primitive
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), (char) 99);
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), (short) 99);
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), (int) 99);
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), (long) 99);
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), (long long) 99);
    if (gen_mode != RANDOM) {
        // Only test numerically stable inputs
        Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), (float) 99);
        if (ptx_version > 100)
            Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), (double) 99);
    }

    // primitive (alternative scan op)
    Test<LOGICAL_WARP_THREADS>(gen_mode, Max(), (unsigned char) 99);
    Test<LOGICAL_WARP_THREADS>(gen_mode, Max(), (unsigned short) 99);
    Test<LOGICAL_WARP_THREADS>(gen_mode, Max(), (unsigned int) 99);
    Test<LOGICAL_WARP_THREADS>(gen_mode, Max(), (unsigned long long) 99);

    // vec-2
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), make_uchar2(17, 21));
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), make_ushort2(17, 21));
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), make_uint2(17, 21));
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), make_ulong2(17, 21));
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), make_ulonglong2(17, 21));
    if (gen_mode != RANDOM) {
        // Only test numerically stable inputs
        Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), make_float2(17, 21));
        if (ptx_version > 100)
            Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), make_double2(17, 21));
    }

    // vec-4
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), make_char4(17, 21, 32, 85));
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), make_short4(17, 21, 32, 85));
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), make_int4(17, 21, 32, 85));
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), make_long4(17, 21, 32, 85));
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), make_longlong4(17, 21, 32, 85));
    if (gen_mode != RANDOM) {
        // Only test numerically stable inputs
        Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), make_float4(17, 21, 32, 85));
        if (ptx_version > 100)
            Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), make_double4(17, 21, 32, 85));
    }

    // complex
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), TestFoo::MakeTestFoo(17, 21, 32, 85));
    Test<LOGICAL_WARP_THREADS>(gen_mode, Sum(), TestBar(17, 21));

}


/**
 * Run battery of tests for different problem generation options
 */
template <int LOGICAL_WARP_THREADS>
void Test()
{
    Test<LOGICAL_WARP_THREADS>(UNIFORM);
    Test<LOGICAL_WARP_THREADS>(INTEGER_SEED);
    Test<LOGICAL_WARP_THREADS>(RANDOM);
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("repeat", g_repeat);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--repeat=<repetitions of entire test suite>]"
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

#ifdef QUICK_TEST

    // Compile/run quick tests
    Test<32, AGGREGATE, int>(UNIFORM, Sum(), (int) 0);
    Test<32, AGGREGATE, float>(UNIFORM, Sum(), (float) 0);
    Test<32, AGGREGATE, long long>(UNIFORM, Sum(), (long long) 0);
    Test<32, AGGREGATE, double>(UNIFORM, Sum(), (double) 0);

    typedef KeyValuePair<int, float> T;
    hipcub::Sum sum_op;
    Test<32, AGGREGATE, T>(UNIFORM, ReduceBySegmentOp<hipcub::Sum>(sum_op), T());

#else

    // Compile/run thorough tests
    for (int i = 0; i <= g_repeat; ++i)
    {
        // Test logical warp sizes
        Test<32>();
        Test<16>();
        Test<9>();
        Test<7>();
    }

#endif

    return 0;
}




