#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2016, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Simple demonstration of hipcub::BlockScan
 *
 * To compile using the command line:
 *   nvcc -arch=sm_XX example_block_scan.cu -I../.. -lcudart -O3
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console (define before including cub.h)
#define HIPCUB_STDERR

#include <stdio.h>
#include <iostream>

#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_scan.cuh>

#include "../../test/test_util.h"

using namespace hipcub;

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

/// Verbose output
bool g_verbose = false;

/// Timing iterations
int g_timing_iterations = 100;

/// Default grid size
int g_grid_size = 1;



//---------------------------------------------------------------------
// Kernels
//---------------------------------------------------------------------

/**
 * Simple kernel for performing a block-wide exclusive prefix sum over integers
 */
template <
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    BlockScanAlgorithm      ALGORITHM>
__global__ void BlockPrefixSumKernel(
    int         *d_in,          // Tile of input
    int         *d_out,         // Tile of output
    clock_t     *d_elapsed)     // Elapsed cycle count of block scan
{
    // Specialize BlockLoad type for our thread block (uses warp-striped loads for coalescing, then transposes in shared memory to a blocked arrangement)
    typedef BlockLoad<int, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_LOAD_WARP_TRANSPOSE> BlockLoadT;

    // Specialize BlockStore type for our thread block (uses warp-striped loads for coalescing, then transposes in shared memory to a blocked arrangement)
    typedef BlockStore<int, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_STORE_WARP_TRANSPOSE> BlockStoreT;

    // Specialize BlockScan type for our thread block
    typedef BlockScan<int, BLOCK_THREADS, ALGORITHM> BlockScanT;

    // Shared memory
    __shared__ union
    {
        typename BlockLoadT::TempStorage    load;
        typename BlockStoreT::TempStorage   store;
        typename BlockScanT::TempStorage    scan;
    } temp_storage;

    // Per-thread tile data
    int data[ITEMS_PER_THREAD];

    // Load items into a blocked arrangement
    BlockLoadT(temp_storage.load).Load(d_in, data);

    // Barrier for smem reuse
    __syncthreads();

    // Start cycle timer
    clock_t start = clock();

    // Compute exclusive prefix sum
    int aggregate;
    BlockScanT(temp_storage.scan).ExclusiveSum(data, data, aggregate);

    // Stop cycle timer
    clock_t stop = clock();

    // Barrier for smem reuse
    __syncthreads();

    // Store items from a blocked arrangement
    BlockStoreT(temp_storage.store).Store(d_out, data);

    // Store aggregate and elapsed clocks
    if (threadIdx.x == 0)
    {
        *d_elapsed = (start > stop) ? start - stop : stop - start;
        d_out[BLOCK_THREADS * ITEMS_PER_THREAD] = aggregate;
    }
}



//---------------------------------------------------------------------
// Host utilities
//---------------------------------------------------------------------

/**
 * Initialize exclusive prefix sum problem (and solution).
 * Returns the aggregate
 */
int Initialize(
    int *h_in,
    int *h_reference,
    int num_items)
{
    int inclusive = 0;

    for (int i = 0; i < num_items; ++i)
    {
        h_in[i] = i % 17;

        h_reference[i] = inclusive;
        inclusive += h_in[i];
    }

    return inclusive;
}


/**
 * Test thread block scan
 */
template <
    int                 BLOCK_THREADS,
    int                 ITEMS_PER_THREAD,
    BlockScanAlgorithm  ALGORITHM>
void Test()
{
    const int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

    // Allocate host arrays
    int *h_in           = new int[TILE_SIZE];
    int *h_reference    = new int[TILE_SIZE];
    int *h_gpu          = new int[TILE_SIZE + 1];

    // Initialize problem and reference output on host
    int h_aggregate = Initialize(h_in, h_reference, TILE_SIZE);

    // Initialize device arrays
    int *d_in           = NULL;
    int *d_out          = NULL;
    clock_t *d_elapsed  = NULL;
    hipMalloc((void**)&d_in,          sizeof(int) * TILE_SIZE);
    hipMalloc((void**)&d_out,         sizeof(int) * (TILE_SIZE + 1));
    hipMalloc((void**)&d_elapsed,     sizeof(clock_t));

    // Display input problem data
    if (g_verbose)
    {
        printf("Input data: ");
        for (int i = 0; i < TILE_SIZE; i++)
            printf("%d, ", h_in[i]);
        printf("\n\n");
    }

    // Kernel props
    int max_sm_occupancy;
    CubDebugExit(MaxSmOccupancy(max_sm_occupancy, BlockPrefixSumKernel<BLOCK_THREADS, ITEMS_PER_THREAD, ALGORITHM>, BLOCK_THREADS));

    // Copy problem to device
    hipMemcpy(d_in, h_in, sizeof(int) * TILE_SIZE, hipMemcpyHostToDevice);

    printf("BlockScan algorithm %s on %d items (%d timing iterations, %d blocks, %d threads, %d items per thread, %d SM occupancy):\n",
        (ALGORITHM == BLOCK_SCAN_RAKING) ? "BLOCK_SCAN_RAKING" : (ALGORITHM == BLOCK_SCAN_RAKING_MEMOIZE) ? "BLOCK_SCAN_RAKING_MEMOIZE" : "BLOCK_SCAN_WARP_SCANS",
        TILE_SIZE, g_timing_iterations, g_grid_size, BLOCK_THREADS, ITEMS_PER_THREAD, max_sm_occupancy);

    // Run aggregate/prefix kernel
    BlockPrefixSumKernel<BLOCK_THREADS, ITEMS_PER_THREAD, ALGORITHM><<<g_grid_size, BLOCK_THREADS>>>(
        d_in,
        d_out,
        d_elapsed);

    // Check results
    printf("\tOutput items: ");
    int compare = CompareDeviceResults(h_reference, d_out, TILE_SIZE, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Check total aggregate
    printf("\tAggregate: ");
    compare = CompareDeviceResults(&h_aggregate, d_out + TILE_SIZE, 1, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Run this several times and average the performance results
    GpuTimer    timer;
    float       elapsed_millis          = 0.0;
    clock_t     elapsed_clocks          = 0;

    for (int i = 0; i < g_timing_iterations; ++i)
    {
        // Copy problem to device
        hipMemcpy(d_in, h_in, sizeof(int) * TILE_SIZE, hipMemcpyHostToDevice);

        timer.Start();

        // Run aggregate/prefix kernel
        BlockPrefixSumKernel<BLOCK_THREADS, ITEMS_PER_THREAD, ALGORITHM><<<g_grid_size, BLOCK_THREADS>>>(
            d_in,
            d_out,
            d_elapsed);

        timer.Stop();
        elapsed_millis += timer.ElapsedMillis();

        // Copy clocks from device
        clock_t clocks;
        CubDebugExit(hipMemcpy(&clocks, d_elapsed, sizeof(clock_t), hipMemcpyDeviceToHost));
        elapsed_clocks += clocks;

    }

    // Check for kernel errors and STDIO from the kernel, if any
    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());

    // Display timing results
    float avg_millis            = elapsed_millis / g_timing_iterations;
    float avg_items_per_sec     = float(TILE_SIZE * g_grid_size) / avg_millis / 1000.0f;
    float avg_clocks            = float(elapsed_clocks) / g_timing_iterations;
    float avg_clocks_per_item   = avg_clocks / TILE_SIZE;

    printf("\tAverage BlockScan::Sum clocks: %.3f\n", avg_clocks);
    printf("\tAverage BlockScan::Sum clocks per item: %.3f\n", avg_clocks_per_item);
    printf("\tAverage kernel millis: %.4f\n", avg_millis);
    printf("\tAverage million items / sec: %.4f\n", avg_items_per_sec);

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_reference) delete[] h_reference;
    if (h_gpu) delete[] h_gpu;
    if (d_in) hipFree(d_in);
    if (d_out) hipFree(d_out);
    if (d_elapsed) hipFree(d_elapsed);
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("i", g_timing_iterations);
    args.GetCmdLineArgument("grid-size", g_grid_size);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--i=<timing iterations (default:%d)>]"
            "[--grid-size=<grid size (default:%d)>]"
            "[--v] "
            "\n", argv[0], g_timing_iterations, g_grid_size);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Run tests
    Test<1024, 1, BLOCK_SCAN_RAKING>();
    Test<512, 2, BLOCK_SCAN_RAKING>();
    Test<256, 4, BLOCK_SCAN_RAKING>();
    Test<128, 8, BLOCK_SCAN_RAKING>();
    Test<64, 16, BLOCK_SCAN_RAKING>();
    Test<32, 32, BLOCK_SCAN_RAKING>();

    printf("-------------\n");

    Test<1024, 1, BLOCK_SCAN_RAKING_MEMOIZE>();
    Test<512, 2, BLOCK_SCAN_RAKING_MEMOIZE>();
    Test<256, 4, BLOCK_SCAN_RAKING_MEMOIZE>();
    Test<128, 8, BLOCK_SCAN_RAKING_MEMOIZE>();
    Test<64, 16, BLOCK_SCAN_RAKING_MEMOIZE>();
    Test<32, 32, BLOCK_SCAN_RAKING_MEMOIZE>();

    printf("-------------\n");

    Test<1024, 1, BLOCK_SCAN_WARP_SCANS>();
    Test<512, 2, BLOCK_SCAN_WARP_SCANS>();
    Test<256, 4, BLOCK_SCAN_WARP_SCANS>();
    Test<128, 8, BLOCK_SCAN_WARP_SCANS>();
    Test<64, 16, BLOCK_SCAN_WARP_SCANS>();
    Test<32, 32, BLOCK_SCAN_WARP_SCANS>();


    return 0;
}

