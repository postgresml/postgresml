#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2016, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Simple example of sorting a sequence of keys and values (each pair is a
 * randomly-selected int32 paired with its original offset in the unsorted sequence), and then
 * isolating all maximal, non-trivial (having length > 1) "runs" of duplicates.
 *
 * To compile using the command line:
 *   nvcc -arch=sm_XX example_device_sort_find_non_trivial_runs.cu -I../.. -lcudart -O3
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <algorithm>

#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>
#include <cub/device/device_run_length_encode.cuh>

#include "../../test/test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose = false;  // Whether to display input/output to console
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory


//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------

/**
 * Simple key-value pairing for using std::sort on key-value pairs.
 */
template <typename Key, typename Value>
struct Pair
{
    Key     key;
    Value   value;

    bool operator<(const Pair &b) const
    {
        return (key < b.key);
    }
};


/**
 * Pair ostream operator
 */
template <typename Key, typename Value>
std::ostream& operator<<(std::ostream& os, const Pair<Key, Value>& val)
{
    os << '<' << val.key << ',' << val.value << '>';
    return os;
}


/**
 * Initialize problem
 */
template <typename Key, typename Value>
void Initialize(
    Key    *h_keys,
    Value  *h_values,
    int    num_items,
    int    max_key)
{
    float scale = float(max_key) / float(UINT_MAX);
    for (int i = 0; i < num_items; ++i)
    {
        Key sample;
        RandomBits(sample);
        h_keys[i] = (max_key == -1) ? i : (Key) (scale * sample);
        h_values[i] = i;
    }

    if (g_verbose)
    {
        printf("Keys:\n");
        DisplayResults(h_keys, num_items);
        printf("\n\n");

        printf("Values:\n");
        DisplayResults(h_values, num_items);
        printf("\n\n");
    }
}


/**
 * Solve sorted non-trivial subrange problem.  Returns the number
 * of non-trivial runs found.
 */
template <typename Key, typename Value>
int Solve(
    Key     *h_keys,
    Value   *h_values,
    int     num_items,
    int     *h_offsets_reference,
    int     *h_lengths_reference)
{
    // Sort

    Pair<Key, Value> *h_pairs = new Pair<Key, Value>[num_items];
    for (int i = 0; i < num_items; ++i)
    {
        h_pairs[i].key    = h_keys[i];
        h_pairs[i].value  = h_values[i];
    }

    std::stable_sort(h_pairs, h_pairs + num_items);

    if (g_verbose)
    {
        printf("Sorted pairs:\n");
        DisplayResults(h_pairs, num_items);
        printf("\n\n");
    }

    // Find non-trivial runs

    Key     previous        = h_pairs[0].key;
    int     length          = 1;
    int     num_runs        = 0;
    int     run_begin       = 0;

    for (int i = 1; i < num_items; ++i)
    {
        if (previous != h_pairs[i].key)
        {
            if (length > 1)
            {
                h_offsets_reference[num_runs]     = run_begin;
                h_lengths_reference[num_runs]     = length;
                num_runs++;
            }
            length = 1;
            run_begin = i;
        }
        else
        {
            length++;
        }
        previous = h_pairs[i].key;
    }

    if (length > 1)
    {
        h_offsets_reference[num_runs]   = run_begin;
        h_lengths_reference[num_runs]   = length;
        num_runs++;
    }

    delete[] h_pairs;

    return num_runs;
}


//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    typedef unsigned int    Key;
    typedef int             Value;

    int timing_iterations   = 0;
    int num_items           = 40;
    Key max_key             = 20;       // Max item

    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("n", num_items);
    args.GetCmdLineArgument("maxkey", max_key);
    args.GetCmdLineArgument("i", timing_iterations);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--i=<timing iterations> "
            "[--n=<input items, default 40> "
            "[--maxkey=<max key, default 20 (use -1 to test only unique keys)>]"
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Allocate host arrays (problem and reference solution)

    Key     *h_keys                 = new Key[num_items];
    Value   *h_values               = new Value[num_items];
    int     *h_offsets_reference    = new int[num_items];
    int     *h_lengths_reference    = new int[num_items];

    // Initialize key-value pairs and compute reference solution (sort them, and identify non-trivial runs)
    printf("Computing reference solution on CPU for %d items (max key %d)\n", num_items, max_key);
    fflush(stdout);

    Initialize(h_keys, h_values, num_items, max_key);
    int num_runs = Solve(h_keys, h_values, num_items, h_offsets_reference, h_lengths_reference);

    printf("%d non-trivial runs\n", num_runs);
    fflush(stdout);

    // Repeat for performance timing
    GpuTimer gpu_timer;
    GpuTimer gpu_rle_timer;
    float elapsed_millis = 0.0;
    float elapsed_rle_millis = 0.0;
    for (int i = 0; i <= timing_iterations; ++i)
    {

        // Allocate and initialize device arrays for sorting
        DoubleBuffer<Key>       d_keys;
        DoubleBuffer<Value>     d_values;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[0], sizeof(Key) * num_items));
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_keys.d_buffers[1], sizeof(Key) * num_items));
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values.d_buffers[0], sizeof(Value) * num_items));
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values.d_buffers[1], sizeof(Value) * num_items));

        CubDebugExit(hipMemcpy(d_keys.d_buffers[d_keys.selector], h_keys, sizeof(float) * num_items, hipMemcpyHostToDevice));
        CubDebugExit(hipMemcpy(d_values.d_buffers[d_values.selector], h_values, sizeof(int) * num_items, hipMemcpyHostToDevice));

        // Start timer
        gpu_timer.Start();

        // Allocate temporary storage for sorting
        size_t  temp_storage_bytes  = 0;
        void    *d_temp_storage     = NULL;
        CubDebugExit(DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, num_items));
        CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

        // Do the sort
        CubDebugExit(DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, num_items));

        // Free unused buffers and sorting temporary storage
        if (d_keys.d_buffers[d_keys.selector ^ 1]) CubDebugExit(g_allocator.DeviceFree(d_keys.d_buffers[d_keys.selector ^ 1]));
        if (d_values.d_buffers[d_values.selector ^ 1]) CubDebugExit(g_allocator.DeviceFree(d_values.d_buffers[d_values.selector ^ 1]));
        if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

        // Start timer
        gpu_rle_timer.Start();

        // Allocate device arrays for enumerating non-trivial runs
        int     *d_offests_out   = NULL;
        int     *d_lengths_out   = NULL;
        int     *d_num_runs      = NULL;
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_offests_out, sizeof(int) * num_items));
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_lengths_out, sizeof(int) * num_items));
        CubDebugExit(g_allocator.DeviceAllocate((void**)&d_num_runs, sizeof(int) * 1));

        // Allocate temporary storage for isolating non-trivial runs
        d_temp_storage = NULL;
        CubDebugExit(DeviceRunLengthEncode::NonTrivialRuns(
            d_temp_storage,
            temp_storage_bytes,
            d_keys.d_buffers[d_keys.selector],
            d_offests_out,
            d_lengths_out,
            d_num_runs,
            num_items));
        CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

        // Do the isolation
        CubDebugExit(DeviceRunLengthEncode::NonTrivialRuns(
            d_temp_storage,
            temp_storage_bytes,
            d_keys.d_buffers[d_keys.selector],
            d_offests_out,
            d_lengths_out,
            d_num_runs,
            num_items));

        // Free keys buffer
        if (d_keys.d_buffers[d_keys.selector]) CubDebugExit(g_allocator.DeviceFree(d_keys.d_buffers[d_keys.selector]));

        //
        // Hypothetically do stuff with the original key-indices corresponding to non-trivial runs of identical keys
        //

        // Stop sort timer
        gpu_timer.Stop();
        gpu_rle_timer.Stop();

        if (i == 0)
        {
            // First iteration is a warmup: // Check for correctness (and display results, if specified)

            printf("\nRUN OFFSETS: \n");
            int compare = CompareDeviceResults(h_offsets_reference, d_offests_out, num_runs, true, g_verbose);
            printf("\t\t %s ", compare ? "FAIL" : "PASS");

            printf("\nRUN LENGTHS: \n");
            compare |= CompareDeviceResults(h_lengths_reference, d_lengths_out, num_runs, true, g_verbose);
            printf("\t\t %s ", compare ? "FAIL" : "PASS");

            printf("\nNUM RUNS: \n");
            compare |= CompareDeviceResults(&num_runs, d_num_runs, 1, true, g_verbose);
            printf("\t\t %s ", compare ? "FAIL" : "PASS");

            AssertEquals(0, compare);
        }
        else
        {
            elapsed_millis += gpu_timer.ElapsedMillis();
            elapsed_rle_millis += gpu_rle_timer.ElapsedMillis();
        }

        // GPU cleanup

        if (d_values.d_buffers[d_values.selector]) CubDebugExit(g_allocator.DeviceFree(d_values.d_buffers[d_values.selector]));
        if (d_offests_out) CubDebugExit(g_allocator.DeviceFree(d_offests_out));
        if (d_lengths_out) CubDebugExit(g_allocator.DeviceFree(d_lengths_out));
        if (d_num_runs) CubDebugExit(g_allocator.DeviceFree(d_num_runs));
        if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
    }

    // Host cleanup
    if (h_keys) delete[] h_keys;
    if (h_values) delete[] h_values;
    if (h_offsets_reference) delete[] h_offsets_reference;
    if (h_lengths_reference) delete[] h_lengths_reference;

    printf("\n\n");

    if (timing_iterations > 0)
    {
        printf("%d timing iterations, average time to sort and isolate non-trivial duplicates: %.3f ms (%.3f ms spent in RLE isolation)\n",
            timing_iterations,
            elapsed_millis / timing_iterations,
            elapsed_rle_millis / timing_iterations);
    }

    return 0;
}



