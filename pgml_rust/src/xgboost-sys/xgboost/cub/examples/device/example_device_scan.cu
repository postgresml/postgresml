/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2016, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Simple example of DeviceScan::ExclusiveSum().
 *
 * Computes an exclusive sum of int keys.
 *
 * To compile using the command line:
 *   nvcc -arch=sm_XX example_device_scan.cu -I../.. -lcudart -O3
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>

#include <cub/util_allocator.cuh>
#include <cub/device/device_scan.cuh>

#include "../../test/test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose = false;  // Whether to display input/output to console
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory


//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------


/**
 * Initialize problem
 */
void Initialize(
    int        *h_in,
    int          num_items)
{
    for (int i = 0; i < num_items; ++i)
        h_in[i] = i;

    if (g_verbose)
    {
        printf("Input:\n");
        DisplayResults(h_in, num_items);
        printf("\n\n");
    }
}

/**
 * Solve exclusive-scan problem
 */
int Solve(
    int           *h_in,
    int           *h_reference,
    int             num_items)
{
    int inclusive = 0;
    int aggregate = 0;

    for (int i = 0; i < num_items; ++i)
    {
        h_reference[i] = inclusive;
        inclusive += h_in[i];
        aggregate += h_in[i];
    }

    return aggregate;
}



//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    int num_items = 150;

    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("n", num_items);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--n=<input items> "
            "[--device=<device-id>] "
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    printf("hipcub::DeviceScan::ExclusiveSum %d items (%d-byte elements)\n",
        num_items, (int) sizeof(int));
    fflush(stdout);

    // Allocate host arrays
    int*  h_in = new int[num_items];
    int*  h_reference = new int[num_items];

    // Initialize problem and solution
    Initialize(h_in, num_items);
    Solve(h_in, h_reference, num_items);

    // Allocate problem device arrays
    int *d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(int) * num_items));

    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(int) * num_items, hipMemcpyHostToDevice));

    // Allocate device output array
    int *d_out = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(int) * num_items));

    // Allocate temporary storage
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;
    CubDebugExit(DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Run
    CubDebugExit(DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, d_in, d_out, num_items));

    // Check for correctness (and display results, if specified)
    int compare = CompareDeviceResults(h_reference, d_out, num_items, true, g_verbose);
    printf("\t%s", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_reference) delete[] h_reference;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

    printf("\n\n");

    return 0;
}



