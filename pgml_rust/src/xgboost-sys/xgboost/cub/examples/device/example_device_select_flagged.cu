#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2016, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Simple example of DeviceSelect::Flagged().
 *
 * Selects flagged items from from a sequence of int keys using a
 * corresponding sequence of unsigned char flags.
 *
 * To compile using the command line:
 *   nvcc -arch=sm_XX example_device_select_flagged.cu -I../.. -lcudart -O3
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>

#include <cub/util_allocator.cuh>
#include <cub/device/device_select.cuh>

#include "../../test/test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose = false;  // Whether to display input/output to console
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory


//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------


/**
 * Initialize problem, setting flags at distances of random length
 * chosen from [1..max_segment]
 */
void Initialize(
    int             *h_in,
    unsigned char   *h_flags,
    int             num_items,
    int             max_segment)
{
    unsigned short max_short = (unsigned short) -1;

    int key = 0;
    int i = 0;
    while (i < num_items)
    {
        // Select number of repeating occurrences
        unsigned short repeat;
        RandomBits(repeat);
        repeat = (unsigned short) ((float(repeat) * (float(max_segment) / float(max_short))));
        repeat = CUB_MAX(1, repeat);

        int j = i;
        while (j < CUB_MIN(i + repeat, num_items))
        {
            h_flags[j] = 0;
            h_in[j] = key;
            j++;
        }

        h_flags[i] = 1;
        i = j;
        key++;
    }

    if (g_verbose)
    {
        printf("Input:\n");
        DisplayResults(h_in, num_items);
        printf("Flags:\n");
        DisplayResults(h_flags, num_items);
        printf("\n\n");
    }
}


/**
 * Solve unique problem
 */
int Solve(
    int             *h_in,
    unsigned char   *h_flags,
    int             *h_reference,
    int             num_items)
{
    int num_selected = 0;
    for (int i = 0; i < num_items; ++i)
    {
        if (h_flags[i])
        {
            h_reference[num_selected] = h_in[i];
            num_selected++;
        }
        else
        {
            h_reference[num_items - (i - num_selected) - 1] = h_in[i];
        }
    }

    return num_selected;
}


//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    int num_items           = 150;
    int max_segment         = 40;       // Maximum segment length

    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("n", num_items);
    args.GetCmdLineArgument("maxseg", max_segment);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--n=<input items> "
            "[--device=<device-id>] "
            "[--maxseg=<max segment length>] "
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Allocate host arrays
    int             *h_in        = new int[num_items];
    int             *h_reference = new int[num_items];
    unsigned char   *h_flags     = new unsigned char[num_items];

    // Initialize problem and solution
    Initialize(h_in, h_flags, num_items, max_segment);
    int num_selected = Solve(h_in, h_flags, h_reference, num_items);

    printf("hipcub::DeviceSelect::Flagged %d items, %d selected (avg distance %d), %d-byte elements\n",
        num_items, num_selected, (num_selected > 0) ? num_items / num_selected : 0, (int) sizeof(int));
    fflush(stdout);

    // Allocate problem device arrays
    int             *d_in = NULL;
    unsigned char   *d_flags = NULL;

    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(int) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_flags, sizeof(unsigned char) * num_items));

    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(int) * num_items, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_flags, h_flags, sizeof(unsigned char) * num_items, hipMemcpyHostToDevice));

    // Allocate device output array and num selected
    int     *d_out            = NULL;
    int     *d_num_selected_out   = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(int) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_num_selected_out, sizeof(int)));

    // Allocate temporary storage
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;
    CubDebugExit(DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_in, d_flags, d_out, d_num_selected_out, num_items));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Run
    CubDebugExit(DeviceSelect::Flagged(d_temp_storage, temp_storage_bytes, d_in, d_flags, d_out, d_num_selected_out, num_items));

    // Check for correctness (and display results, if specified)
    int compare = CompareDeviceResults(h_reference, d_out, num_selected, true, g_verbose);
    printf("\t Data %s ", compare ? "FAIL" : "PASS");
    compare |= CompareDeviceResults(&num_selected, d_num_selected_out, 1, true, g_verbose);
    printf("\t Count %s ", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_reference) delete[] h_reference;
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_num_selected_out) CubDebugExit(g_allocator.DeviceFree(d_num_selected_out));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_flags) CubDebugExit(g_allocator.DeviceFree(d_flags));

    printf("\n\n");

    return 0;
}



