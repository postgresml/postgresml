#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <GPUTreeShap/gpu_treeshap.h>
#include <algorithm>
#include <iostream>
#include <limits>
#include <string>
#include <vector>
#include "../GPUTreeShap/gpu_treeshap.h"

// Minimal decision tree implementation that stores sample weights of training
// data at each node. The sample weight roughly corresponds to the "amount" of
// data that arrives in this node. Using this we can estimate the probability of
// an instance taking the left or right branch if its feature is unknown.
class DecisionTree {
 public:
  struct Node {
    int parent;
    int left_child;
    int right_child;
    int feature_idx;
    float split_condition;
    float leaf_value;
    float sample_weight;
    bool IsLeaf() const { return left_child == -1 && right_child == -1; }
    bool IsRoot() const { return parent == -1; }
  };
  std::vector<Node> nodes;
  explicit DecisionTree(float total_weight) {
    nodes.push_back({-1, -1, -1, -1, 0.0, 0.0, total_weight});
  }
  void AddSplit(int node_idx, int feature_idx, float split_condition,
                float left_sample_weight, float right_sample_weight,
                float left_leaf_value, float right_leaf_value) {
    nodes[node_idx].split_condition = split_condition;
    nodes[node_idx].feature_idx = feature_idx;
    int left_idx = nodes.size();
    nodes[node_idx].left_child = left_idx;
    nodes.push_back(
        {node_idx, -1, -1, -1, 0.0, left_leaf_value, left_sample_weight});
    int right_idx = nodes.size();
    nodes[node_idx].right_child = right_idx;
    nodes.push_back(
        {node_idx, -1, -1, -1, 0.0, right_leaf_value, right_sample_weight});
  }
};

void RecursivePrint(std::ostream& os, const DecisionTree& dt, int node_idx,
                    int depth) {
  if (node_idx == -1) return;
  DecisionTree::Node node = dt.nodes[node_idx];

  for (int i = 0; i < depth; i++) {
    os << "\t";
  }
  os << node_idx << ":";
  if (node.IsLeaf()) {
    os << "leaf=" << node.leaf_value;
  } else {
    os << "[f" << node.feature_idx << "<" << node.split_condition << "]";
  }
  os << ",cover=" << node.sample_weight;
  os << "\n";
  RecursivePrint(os, dt, node.left_child, depth + 1);
  RecursivePrint(os, dt, node.right_child, depth + 1);
}

std::ostream& operator<<(std::ostream& os, const DecisionTree& dt) {
  RecursivePrint(os, dt, 0, 0);
  return os;
}

// Define a custom split condition implementing EvaluateSplit and Merge
struct MySplitCondition {
  MySplitCondition() = default;
  MySplitCondition(float feature_lower_bound, float feature_upper_bound)
      : feature_lower_bound(feature_lower_bound),
        feature_upper_bound(feature_upper_bound) {
    assert(feature_lower_bound <= feature_upper_bound);
  }

  /*! Feature values >= lower and < upper flow down this path. */
  float feature_lower_bound;
  float feature_upper_bound;

  // Does this instance flow down this path?
  __host__ __device__ bool EvaluateSplit(float x) const {
    return x >= feature_lower_bound && x < feature_upper_bound;
  }

  // Combine two split conditions on the same feature
  __host__ __device__ void Merge(
      const MySplitCondition& other) {  // Combine duplicate features
    feature_lower_bound = max(feature_lower_bound, other.feature_lower_bound);
    feature_upper_bound = min(feature_upper_bound, other.feature_upper_bound);
  }
};

std::vector<gpu_treeshap::PathElement<MySplitCondition>> ExtractPaths(
    const DecisionTree& dt) {
  std::vector<gpu_treeshap::PathElement<MySplitCondition>> paths;
  size_t path_idx = 0;
  // Find leaf nodes
  // Work backwards from leaf to root, order does not matter
  // It's also possible to work from root to leaf
  for (int i = 0; i < static_cast<int>(dt.nodes.size()); i++) {
    if (dt.nodes[i].IsLeaf()) {
      auto child = dt.nodes[i];
      float v = child.leaf_value;
      int child_idx = i;
      const float inf = std::numeric_limits<float>::infinity();
      while (!child.IsRoot()) {
        auto parent = dt.nodes[child.parent];
        float zero_fraction = child.sample_weight / parent.sample_weight;
        // Encode the range of feature values that flow down this path
        bool is_left_path = parent.left_child == child_idx;
        float lower_bound = is_left_path ? -inf : parent.split_condition;
        float upper_bound = is_left_path ? parent.split_condition : inf;
        paths.push_back({path_idx,
                         parent.feature_idx,
                         0,
                         {lower_bound, upper_bound},
                         zero_fraction,
                         v});
        child_idx = child.parent;
        child = parent;
      }
      // Root node has feature -1
      paths.push_back({path_idx, -1, 0, {-inf, inf}, 1.0, v});
      path_idx++;
    }
  }
  return paths;
}

std::ostream& operator<<(
    std::ostream& os,
    const std::vector<gpu_treeshap::PathElement<MySplitCondition>>& paths) {
  std::vector<gpu_treeshap::PathElement<MySplitCondition>> tmp(paths);
  std::sort(tmp.begin(), tmp.end(),
            [&](const gpu_treeshap::PathElement<MySplitCondition>& a,
                const gpu_treeshap::PathElement<MySplitCondition>& b) {
              if (a.path_idx < b.path_idx) return true;
              if (b.path_idx < a.path_idx) return false;

              if (a.feature_idx < b.feature_idx) return true;
              if (b.feature_idx < a.feature_idx) return false;
              return false;
            });

  for (auto i = 0ull; i < tmp.size(); i++) {
    auto e = tmp[i];
    if (i == 0 || e.path_idx != tmp[i - 1].path_idx) {
      os << "path_idx:" << e.path_idx << ", leaf value:" << e.v;
      os << "\n";
    }
    os << " (feature:" << e.feature_idx << ", pz:" << e.zero_fraction << ", ["
       << e.split_condition.feature_lower_bound << "<=x<"
       << e.split_condition.feature_upper_bound << "])";
    os << "\n";
  }
  return os;
}

class DenseDatasetWrapper {
  const float* data;
  int num_rows;
  int num_cols;

 public:
  DenseDatasetWrapper() = default;
  DenseDatasetWrapper(const float* data, int num_rows, int num_cols)
      : data(data), num_rows(num_rows), num_cols(num_cols) {}
  __device__ float GetElement(size_t row_idx, size_t col_idx) const {
    return data[row_idx * num_cols + col_idx];
  }
  __host__ __device__ size_t NumRows() const { return num_rows; }
  __host__ __device__ size_t NumCols() const { return num_cols; }
};

int main() {
  // Create a very basic decision tree
  DecisionTree tree(5.0);
  tree.AddSplit(0, 0, 0.5, 2.0, 3.0, -1.0, 0.0);
  tree.AddSplit(2, 1, 0.5, 1.0, 2.0, -1.0, 0.0);

  tree.AddSplit(4, 2, 0.5, 1.0, 1.0, 1.0, 0.5);

  // Visualise it
  std::cout << "Decision tree:\n";
  std::cout << tree;

  auto paths = ExtractPaths(tree);

  // Visualise unique paths
  std::cout << "Extracted paths:\n";
  std::cout << paths;

  // Create a dataset with two rows in row major format
  thrust::device_vector<float> data(3 * 2);
  // First row
  data[0] = 1.0;
  data[1] = 1.0;
  data[2] = 0.0;
  // Second row
  data[3] = 1.0;
  data[4] = 1.0;
  data[5] = 1.0;
  DenseDatasetWrapper X(data.data().get(), 2, 3);
  thrust::device_vector<float> phis((X.NumCols() + 1) * X.NumRows());
  gpu_treeshap::GPUTreeShap(X, paths.begin(), paths.end(), 1, phis.begin(),
                            phis.end());

  // Print the resulting feature contributions
  std::cout << "\n";
  for (auto i = 0ull; i < X.NumRows(); i++) {
    std::cout << "Row " << i << " contributions:\n";
    for (auto j = 0ull; j < X.NumCols(); j++) {
      std::cout << "f" << j << ":" << phis[i * (X.NumCols() + 1) + j] << " ";
    }
    std::cout << "bias"
              << ":" << phis[i * (X.NumCols() + 1) + X.NumCols()];
    std::cout << "\n";
  }
}
