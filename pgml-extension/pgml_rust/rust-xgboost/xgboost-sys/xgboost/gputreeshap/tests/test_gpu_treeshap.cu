#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2020, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <GPUTreeShap/gpu_treeshap.h>
#include <hip/hip_cooperative_groups.h>
#include <limits>
#include <numeric>
#include <random>
#include <vector>
#include "gtest/gtest.h"
#include "tests/test_utils.h"
#include "../GPUTreeShap/gpu_treeshap.h"

using namespace gpu_treeshap;  // NOLINT

class ParameterisedModelTest
    : public ::testing::TestWithParam<
          std::tuple<size_t, size_t, size_t, size_t, size_t>> {
 protected:
  ParameterisedModelTest() {
    size_t max_depth, num_paths;
    std::tie(num_rows, num_features, num_groups, max_depth, num_paths) =
        GetParam();
    model = GenerateEnsembleModel(num_groups, max_depth, num_features,
                                  num_paths, 78);
    test_data = TestDataset(num_rows, num_features, 22);
    margin = Predict(model, test_data, num_groups);

    X = test_data.GetDeviceWrapper();

    phis.resize(X.NumRows() * (X.NumCols() + 1) * (X.NumCols() + 1) *
                num_groups);
  }
  std::vector<PathElement<XgboostSplitCondition>> model;
  TestDataset test_data;
  DenseDatasetWrapper X;
  std::vector<float> margin;
  thrust::device_vector<float> phis;
  size_t num_groups;
  size_t num_rows;
  size_t num_features;
};

TEST_P(ParameterisedModelTest, ShapSum) {
  GPUTreeShap(X, model.begin(), model.end(), num_groups, phis.begin(),
              phis.end());
  thrust::host_vector<float> result(phis);
  std::vector<float> tmp(result.begin(), result.end());
  std::vector<float> sum(num_rows * num_groups);
  for (auto i = 0ull; i < num_rows; i++) {
    for (auto j = 0ull; j < num_features + 1; j++) {
      for (auto group = 0ull; group < num_groups; group++) {
        size_t result_index = IndexPhi(i, num_groups, group, num_features, j);
        sum[i * num_groups + group] += result[result_index];
      }
    }
  }
  for (auto i = 0ull; i < sum.size(); i++) {
    ASSERT_NEAR(sum[i], margin[i], 1e-3);
  }
}

TEST_P(ParameterisedModelTest, ShapInteractionsSum) {
  thrust::device_vector<float> phis_interactions(
      X.NumRows() * (X.NumCols() + 1) * (X.NumCols() + 1) * num_groups);
  GPUTreeShap(X, model.begin(), model.end(), num_groups, phis.begin(),
              phis.end());
  GPUTreeShapInteractions(X, model.begin(), model.end(), num_groups,
                          phis_interactions.begin(), phis_interactions.end());
  thrust::host_vector<float> interactions_result(phis_interactions);
  std::vector<float> sum(phis.size());
  for (auto row_idx = 0ull; row_idx < num_rows; row_idx++) {
    for (auto group = 0ull; group < num_groups; group++) {
      for (auto i = 0ull; i < num_features + 1; i++) {
        for (auto j = 0ull; j < num_features + 1; j++) {
          size_t result_index = IndexPhiInteractions(row_idx, num_groups, group,
                                                     num_features, i, j);
          sum[IndexPhi(row_idx, num_groups, group, num_features, i)] +=
              interactions_result[result_index];
        }
      }
    }
  }

  thrust::host_vector<float> phis_host(phis);
  for (auto i = 0ull; i < sum.size(); i++) {
    ASSERT_NEAR(sum[i], phis_host[i], 1e-3);
  }
}

TEST_P(ParameterisedModelTest, ShapTaylorInteractionsSum) {
  GPUTreeShapTaylorInteractions(X, model.begin(), model.end(), num_groups,
                                phis.begin(), phis.end());
  thrust::host_vector<float> interactions_result(phis);
  std::vector<float> sum(margin.size());
  for (auto row_idx = 0ull; row_idx < num_rows; row_idx++) {
    for (auto group = 0ull; group < num_groups; group++) {
      for (auto i = 0ull; i < num_features + 1; i++) {
        for (auto j = 0ull; j < num_features + 1; j++) {
          size_t result_index = IndexPhiInteractions(row_idx, num_groups, group,
                                                     num_features, i, j);
          sum[row_idx * num_groups + group] +=
              interactions_result[result_index];
        }
      }
    }
  }

  for (auto i = 0ull; i < sum.size(); i++) {
    ASSERT_NEAR(sum[i], margin[i], 1e-3);
  }
}

TEST_P(ParameterisedModelTest, ShapSumInterventional) {
  auto r_test_data = TestDataset(400, num_features, 10);
  auto R = r_test_data.GetDeviceWrapper();
  GPUTreeShapInterventional(X, R, model.begin(), model.end(), num_groups,
                            phis.begin(), phis.end());
  thrust::host_vector<float> result(phis);
  std::vector<float> tmp(result.begin(), result.end());
  std::vector<float> sum(num_rows * num_groups);
  for (auto i = 0ull; i < num_rows; i++) {
    for (auto j = 0ull; j < num_features + 1; j++) {
      for (auto group = 0ull; group < num_groups; group++) {
        size_t result_index = IndexPhi(i, num_groups, group, num_features, j);
        sum[i * num_groups + group] += result[result_index];
      }
    }
  }
  for (auto i = 0ull; i < sum.size(); i++) {
    ASSERT_NEAR(sum[i], margin[i], 1e-3);
  }
}

std::string PrintTestName(
    const testing::TestParamInfo<ParameterisedModelTest::ParamType>& info) {
  std::string name = "nrow" + std::to_string(std::get<0>(info.param)) + "_";
  name += "nfeat" + std::to_string(std::get<1>(info.param)) + "_";
  name += "ngroup" + std::to_string(std::get<2>(info.param)) + "_";
  name += "mdepth" + std::to_string(std::get<3>(info.param)) + "_";
  name += "npaths" + std::to_string(std::get<4>(info.param));
  return name;
}

// Generate a bunch of random models and check the shap results sum up to the
// predictions
size_t test_num_rows[] = {1, 10, 100, 1000};
size_t test_num_features[] = {1, 5, 8, 31};
size_t test_num_groups[] = {1, 5};
size_t test_max_depth[] = {1, 8, 20};
size_t test_num_paths[] = {1, 10};
INSTANTIATE_TEST_CASE_P(ShapInstantiation, ParameterisedModelTest,
                        testing::Combine(testing::ValuesIn(test_num_rows),
                                         testing::ValuesIn(test_num_features),
                                         testing::ValuesIn(test_num_groups),
                                         testing::ValuesIn(test_max_depth),
                                         testing::ValuesIn(test_num_paths)),
                        PrintTestName);

#define EXPECT_THROW_CONTAINS_MESSAGE(stmt, etype, whatstring)             \
  EXPECT_THROW(try { stmt; } catch (const etype& ex) {                     \
    EXPECT_NE(std::string(ex.what()).find(whatstring), std::string::npos); \
    throw;                                                                 \
  },                                                                       \
               etype)

class APITest : public ::testing::Test {
 protected:
  APITest() {
    const float inf = std::numeric_limits<float>::infinity();
    model = {
        {0, -1, 0, {-inf, inf, false}, 1.0f, 2.0f},
        {0, 0, 0, {0.5f, inf, false}, 0.25f, 2.0f},
        {0, 1, 0, {0.5f, inf, false}, 0.5f, 2.0f},
        {0, 2, 0, {0.5f, inf, false}, 0.6f, 2.0f},
        {0, 3, 0, {0.5f, inf, false}, 1.0f, 2.0f},
    };
    data = std::vector<float>({1.0f, 1.0f, 0.0f, 1.0f, 1.0f, 1.0f, 0.0f, 1.0f});
    X = DenseDatasetWrapper(data.data().get(), 2, 4);
    phis.resize((X.NumRows() * (X.NumCols() + 1) * (X.NumCols() + 1)));
  }
  template <typename ExceptionT>
  void ExpectAPIThrow(std::string message) {
    EXPECT_THROW_CONTAINS_MESSAGE(
        GPUTreeShap(X, model.begin(), model.end(), 1, phis.begin(), phis.end()),
        ExceptionT, message);
    EXPECT_THROW_CONTAINS_MESSAGE(
        GPUTreeShapInteractions(X, model.begin(), model.end(), 1, phis.begin(),
                                phis.end()),
        ExceptionT, message);
    EXPECT_THROW_CONTAINS_MESSAGE(
        GPUTreeShapTaylorInteractions(X, model.begin(), model.end(), 1,
                                      phis.begin(), phis.end()),
        ExceptionT, message);
  }

  thrust::device_vector<float> data;
  std::vector<PathElement<XgboostSplitCondition>> model;
  DenseDatasetWrapper X;
  thrust::device_vector<float> phis;
};

TEST_F(APITest, PathTooLong) {
  model.resize(33);
  model[0] = {0, -1, 0, {0, 0, 0}, 0, 0};
  for (size_t i = 1; i < model.size(); i++) {
    model[i] = {0, static_cast<int64_t>(i), 0, {0, 0, 0}, 0, 0};
  }
  ExpectAPIThrow<std::invalid_argument>("Tree depth must be <= 32");
}

TEST_F(APITest, PathVIncorrect) {
  model = {{0, -1, 0, {0.0f, 0.0f, false}, 0.0, 1.0f},
           {0, 0, 0, {0.0f, 0.0f, false}, 0.0f, 0.5f}};

  ExpectAPIThrow<std::invalid_argument>(
      "Leaf value v should be the same across a single path");
}

TEST_F(APITest, PhisIncorrectLength) {
  phis.resize(1);
  ExpectAPIThrow<std::invalid_argument>("phis_out must be at least of size");
}

// Test a simple tree and compare output to xgb shap values
// 0:[f0<0.5] yes=1,no=2,missing=1,gain=1.63333321,cover=5
//  1:leaf=-1,cover=2
//  2:[f1<0.5] yes=3,no=4,missing=3,gain=2.04166675,cover=3
//    3:leaf=-1,cover=1
//    4:[f2<0.5] yes=5,no=6,missing=5,gain=0.125,cover=2
//      5:leaf=1,cover=1
//      6:leaf=0.5,cover=1
TEST(GPUTreeShap, BasicPaths) {
  const float inf = std::numeric_limits<float>::infinity();
  std::vector<PathElement<XgboostSplitCondition>> path{
      {0, -1, 0, {-inf, inf, false}, 1.0f, 0.5f},
      {0, 0, 0, {0.5f, inf, false}, 0.6f, 0.5f},
      {0, 1, 0, {0.5f, inf, false}, 2.0f / 3, 0.5f},
      {0, 2, 0, {0.5f, inf, false}, 0.5f, 0.5f},
      {1, -1, 0, {-inf, 0.0f, false}, 1.0f, 1.0f},
      {1, 0, 0, {0.5f, inf, false}, 0.6f, 1.0f},
      {1, 1, 0, {0.5f, inf, false}, 2.0f / 3, 1.0f},
      {1, 2, 0, {-inf, 0.5f, false}, 0.5f, 1.0f},
      {2, -1, 0, {-inf, 0.0f, false}, 1.0f, -1},
      {2, 0, 0, {0.5f, inf, false}, 0.6f, -1.0f},
      {2, 1, 0, {-inf, 0.5f, false}, 1.0f / 3, -1.0f},
      {3, -1, 0, {-inf, 0.0f, false}, 1.0f, -1.0f},
      {3, 0, 0, {-inf, 0.5f, false}, 0.4f, -1.0f}};
  thrust::device_vector<float> data =
      std::vector<float>({1.0f, 1.0f, 0.0f, 1.0f, 0.0f, 0.0f});
  DenseDatasetWrapper X(data.data().get(), 2, 3);
  size_t num_trees = 1;
  thrust::device_vector<float> phis(X.NumRows() * (X.NumCols() + 1));
  GPUTreeShap(X, path.begin(), path.end(), 1, phis.begin(), phis.end());
  thrust::host_vector<float> result(phis);
  // First instance
  EXPECT_NEAR(result[0], 0.6277778f * num_trees, 1e-5);
  EXPECT_NEAR(result[1], 0.5027776f * num_trees, 1e-5);
  EXPECT_NEAR(result[2], 0.1694444f * num_trees, 1e-5);
  EXPECT_NEAR(result[3], -0.3f * num_trees, 1e-5);
  // Second instance
  EXPECT_NEAR(result[4], 0.24444449f * num_trees, 1e-5);
  EXPECT_NEAR(result[5], -1.005555f * num_trees, 1e-5);
  EXPECT_NEAR(result[6], 0.0611111f * num_trees, 1e-5);
  EXPECT_NEAR(result[7], -0.3f * num_trees, 1e-5);
}

TEST(GPUTreeShap, BasicPathsInteractions) {
  const float inf = std::numeric_limits<float>::infinity();
  std::vector<PathElement<XgboostSplitCondition>> path{
      {0, -1, 0, {-inf, inf, false}, 1.0f, 0.5f},
      {0, 0, 0, {0.5f, inf, false}, 0.6f, 0.5f},
      {0, 1, 0, {0.5f, inf, false}, 2.0f / 3, 0.5f},
      {0, 2, 0, {0.5f, inf, false}, 0.5f, 0.5f},
      {1, -1, 0, {-inf, 0.0f, false}, 1.0f, 1.0f},
      {1, 0, 0, {0.5f, inf, false}, 0.6f, 1.0f},
      {1, 1, 0, {0.5f, inf, false}, 2.0f / 3, 1.0f},
      {1, 2, 0, {-inf, 0.5f, false}, 0.5f, 1.0f},
      {2, -1, 0, {-inf, 0.0f, false}, 1.0f, -1},
      {2, 0, 0, {0.5f, inf, false}, 0.6f, -1.0f},
      {2, 1, 0, {-inf, 0.5f, false}, 1.0f / 3, -1.0f},
      {3, -1, 0, {-inf, 0.0f, false}, 1.0f, -1.0f},
      {3, 0, 0, {-inf, 0.5f, false}, 0.4f, -1.0f}};
  thrust::device_vector<float> data =
      std::vector<float>({1.0f, 1.0f, 0.0f, 1.0f, 1.0f, 1.0f});
  DenseDatasetWrapper X(data.data().get(), 2, 3);
  thrust::device_vector<float> phis(X.NumRows() * (X.NumCols() + 1) *
                                    (X.NumCols() + 1));
  GPUTreeShapInteractions(X, path.begin(), path.end(), 1, phis.begin(),
                          phis.end());
  std::vector<float> result(phis.begin(), phis.end());
  std::vector<float> expected_result = {
      0.46111116,  0.125,       0.04166666,  0.,          0.125,
      0.34444442,  0.03333333,  0.,          0.04166666,  0.03333335,
      0.09444444,  0.,          0.,          0.,          0.,
      -0.3,        0.47222224,  0.1083333,   -0.04166666, 0.,
      0.10833332,  0.35555553,  -0.03333333, 0.,          -0.04166666,
      -0.03333332, -0.09444447, 0.,          0.,          0.,
      0.,          -0.3};
  for (auto i = 0ull; i < result.size(); i++) {
    EXPECT_NEAR(result[i], expected_result[i], 1e-5);
  }
}

// Test a tree with features occurring multiple times in a path
TEST(GPUTreeShap, BasicPathsWithDuplicates) {
  const float inf = std::numeric_limits<float>::infinity();
  std::vector<PathElement<XgboostSplitCondition>> path{
      {0, -1, 0, {-inf, 0.0f, false}, 1.0f, 3.0f},
      {0, 0, 0, {0.5f, inf, false}, 2.0f / 3, 3.0f},
      {0, 0, 0, {1.5f, inf, false}, 0.5f, 3.0f},
      {0, 0, 0, {2.5f, inf, false}, 0.5f, 3.0f},
      {1, -1, 0, {-inf, 0.0f, false}, 1.0f, 2.0f},
      {1, 0, 0, {0.5f, inf, false}, 2.0f / 3.0f, 2.0f},
      {1, 0, 0, {1.5f, inf, false}, 0.5f, 2.0f},
      {1, 0, 0, {-inf, 2.5f, false}, 0.5f, 2.0f},
      {2, -1, 0, {-inf, 0.0f, false}, 1.0f, 1.0f},
      {2, 0, 0, {0.5f, inf, false}, 2.0f / 3.0f, 1.0f},
      {2, 0, 0, {-inf, 1.5f, false}, 0.5f, 1.0f},
      {3, -1, 0, {-inf, 0.0f, false}, 1.0f, -1.0f},
      {3, 0, 0, {-inf, 0.5f, false}, 1.0f / 3, -1.0f}};
  thrust::device_vector<float> data = std::vector<float>({2.0f});
  DenseDatasetWrapper X(data.data().get(), 1, 1);
  size_t num_trees = 1;
  thrust::device_vector<float> phis(X.NumRows() * (X.NumCols() + 1));
  GPUTreeShap(X, path.begin(), path.end(), 1, phis.begin(), phis.end());
  thrust::host_vector<float> result(phis);
  // First instance
  EXPECT_FLOAT_EQ(result[0], 1.1666666f * num_trees);
  EXPECT_FLOAT_EQ(result[1], 0.83333337f * num_trees);
}

__device__ bool FloatApproximatelyEqual(float a, float b) {
  const float kEps = 1e-5;
  return fabs(a - b) < kEps;
}

// Expose pweight for testing
class TestGroupPath : public detail::GroupPath {
 public:
  __device__ TestGroupPath(const detail::ContiguousGroup& g,
                           float zero_fraction, float one_fraction)
      : detail::GroupPath(g, zero_fraction, one_fraction) {}
  using detail::GroupPath::pweight_;
  using detail::GroupPath::unique_depth_;
};

template <typename DatasetT, typename SplitConditionT>
__global__ void TestExtendKernel(
    DatasetT X, size_t num_path_elements,
    const PathElement<SplitConditionT>* path_elements) {
  cooperative_groups::thread_block block =
      cooperative_groups::this_thread_block();
  auto group =
      cooperative_groups::tiled_partition<32, cooperative_groups::thread_block>(
          block);
  bool thread_active = threadIdx.x < num_path_elements;
  uint32_t mask = __ballot_sync(FULL_MASK, thread_active);
  if (!thread_active) return;

  // Test first training instance
  cooperative_groups::coalesced_group active_group =
      cooperative_groups::coalesced_threads();
  PathElement<SplitConditionT> e = path_elements[active_group.thread_rank()];
  float one_fraction =
      e.split_condition.EvaluateSplit(X.GetElement(0, e.feature_idx));
  float zero_fraction = e.zero_fraction;
  auto labelled_group = detail::active_labeled_partition(mask, 0);
  TestGroupPath path(labelled_group, zero_fraction, one_fraction);
  path.Extend();
  assert(path.unique_depth_ == 1);
  if (active_group.thread_rank() == 0) {
    assert(FloatApproximatelyEqual(path.pweight_, 0.3f));
  } else if (active_group.thread_rank() == 1) {
    assert(FloatApproximatelyEqual(path.pweight_, 0.5f));
  } else {
    assert(FloatApproximatelyEqual(path.pweight_, 0.0f));
  }

  path.Extend();
  assert(path.unique_depth_ == 2);
  if (active_group.thread_rank() == 0) {
    assert(FloatApproximatelyEqual(path.pweight_, 0.133333f));
  } else if (active_group.thread_rank() == 1) {
    assert(FloatApproximatelyEqual(path.pweight_, 0.21111f));
  } else if (active_group.thread_rank() == 2) {
    assert(FloatApproximatelyEqual(path.pweight_, 0.33333f));
  } else {
    assert(FloatApproximatelyEqual(path.pweight_, 0.0f));
  }

  path.Extend();
  assert(path.unique_depth_ == 3);
  if (active_group.thread_rank() == 0) {
    assert(FloatApproximatelyEqual(path.pweight_, 0.05f));
  } else if (active_group.thread_rank() == 1) {
    assert(FloatApproximatelyEqual(path.pweight_, 0.086111f));
  } else if (active_group.thread_rank() == 2) {
    assert(FloatApproximatelyEqual(path.pweight_, 0.147222f));
  } else if (active_group.thread_rank() == 3) {
    assert(FloatApproximatelyEqual(path.pweight_, 0.25f));
  } else {
    assert(FloatApproximatelyEqual(path.pweight_, 0.0f));
  }

  float unwound_sum = path.UnwoundPathSum();

  if (active_group.thread_rank() == 1) {
    assert(FloatApproximatelyEqual(unwound_sum, 0.63888f));
  } else if (active_group.thread_rank() == 2) {
    assert(FloatApproximatelyEqual(unwound_sum, 0.61666f));
  } else if (active_group.thread_rank() == 3) {
    assert(FloatApproximatelyEqual(unwound_sum, 0.67777f));
  } else if (active_group.thread_rank() > 3) {
    assert(FloatApproximatelyEqual(unwound_sum, 0.0f));
  }

  // Test second training instance
  one_fraction =
      e.split_condition.EvaluateSplit(X.GetElement(1, e.feature_idx));
  TestGroupPath path2(labelled_group, zero_fraction, one_fraction);
  path2.Extend();
  assert(path2.unique_depth_ == 1);
  if (active_group.thread_rank() == 0) {
    assert(FloatApproximatelyEqual(path2.pweight_, 0.3f));
  } else if (active_group.thread_rank() == 1) {
    assert(FloatApproximatelyEqual(path2.pweight_, 0.5f));
  } else {
    assert(FloatApproximatelyEqual(path2.pweight_, 0.0f));
  }

  path2.Extend();
  assert(path2.unique_depth_ == 2);
  if (active_group.thread_rank() == 0) {
    assert(FloatApproximatelyEqual(path2.pweight_, 0.133333f));
  } else if (active_group.thread_rank() == 1) {
    assert(FloatApproximatelyEqual(path2.pweight_, 0.11111f));
  } else if (active_group.thread_rank() == 2) {
    assert(FloatApproximatelyEqual(path2.pweight_, 0.0f));
  } else {
    assert(FloatApproximatelyEqual(path2.pweight_, 0.0f));
  }

  path2.Extend();
  assert(path2.unique_depth_ == 3);
  if (active_group.thread_rank() == 0) {
    assert(FloatApproximatelyEqual(path2.pweight_, 0.05f));
  } else if (active_group.thread_rank() == 1) {
    assert(FloatApproximatelyEqual(path2.pweight_, 0.06111f));
  } else if (active_group.thread_rank() == 2) {
    assert(FloatApproximatelyEqual(path2.pweight_, 0.05555f));
  } else if (active_group.thread_rank() == 3) {
    assert(FloatApproximatelyEqual(path2.pweight_, 0.0f));
  } else {
    assert(FloatApproximatelyEqual(path2.pweight_, 0.0f));
  }

  unwound_sum = path2.UnwoundPathSum();

  if (active_group.thread_rank() == 1) {
    assert(FloatApproximatelyEqual(unwound_sum, 0.22222f));
  } else if (active_group.thread_rank() == 2) {
    assert(FloatApproximatelyEqual(unwound_sum, 0.61666f));
  } else if (active_group.thread_rank() == 3) {
    assert(FloatApproximatelyEqual(unwound_sum, 0.244444f));
  } else if (active_group.thread_rank() > 3) {
    assert(FloatApproximatelyEqual(unwound_sum, 0.0f));
  }
}

TEST(GPUTreeShap, Extend) {
  const float inf = std::numeric_limits<float>::infinity();
  std::vector<PathElement<XgboostSplitCondition>> path{
      {0, -1, 0, {-inf, 0.0f, false}, 1.0f, 1.0f},
      {0, 0, 0, {0.5f, inf, false}, 3.0f / 5, 1.0f},
      {0, 1, 0, {0.5f, inf, false}, 2.0f / 3, 1.0f},
      {0, 2, 0, {-inf, 0.5f, false}, 1.0f / 2, 1.0f}};
  thrust::device_vector<PathElement<XgboostSplitCondition>> device_path(path);
  thrust::device_vector<float> data =
      std::vector<float>({1.0f, 1.0f, 0.0f, 1.0f, 0.0f, 0.0f});
  DenseDatasetWrapper X(data.data().get(), 2, 3);
  TestExtendKernel<<<1, 32>>>(X, 4, device_path.data().get());
}
template <typename DatasetT, typename SplitConditionT>
__global__ void TestExtendMultipleKernel(
    DatasetT X, size_t n_first, size_t n_second,
    const PathElement<SplitConditionT>* path_elements) {
  cooperative_groups::thread_block block =
      cooperative_groups::this_thread_block();
  auto warp =
      cooperative_groups::tiled_partition<32, cooperative_groups::thread_block>(
          block);
  bool thread_active = threadIdx.x < n_first + n_second;
  uint32_t mask = __ballot_sync(FULL_MASK, thread_active);
  if (!thread_active) return;
  cooperative_groups::coalesced_group active_group =
      cooperative_groups::coalesced_threads();
  int label = warp.thread_rank() >= n_first;
  auto labeled_group = detail::active_labeled_partition(mask, label);
  PathElement<SplitConditionT> e = path_elements[warp.thread_rank()];

  // Test first training instance
  float one_fraction =
      e.split_condition.EvaluateSplit(X.GetElement(0, e.feature_idx));
  float zero_fraction = e.zero_fraction;
  TestGroupPath path(labeled_group, zero_fraction, one_fraction);
  assert(path.unique_depth_ == 0);
  if (labeled_group.thread_rank() == 0) {
    assert(FloatApproximatelyEqual(path.pweight_, 1.0f));
  } else {
    assert(FloatApproximatelyEqual(path.pweight_, 0.0f));
  }

  path.Extend();
  assert(path.unique_depth_ == 1);
  if (labeled_group.thread_rank() == 0) {
    assert(FloatApproximatelyEqual(path.pweight_, 0.3f));
  } else if (labeled_group.thread_rank() == 1) {
    assert(FloatApproximatelyEqual(path.pweight_, 0.5f));
  } else {
    assert(FloatApproximatelyEqual(path.pweight_, 0.0f));
  }

  path.Extend();
  assert(path.unique_depth_ == 2);
  if (labeled_group.thread_rank() == 0) {
    assert(FloatApproximatelyEqual(path.pweight_, 0.133333f));
  } else if (labeled_group.thread_rank() == 1) {
    assert(FloatApproximatelyEqual(path.pweight_, 0.21111f));
  } else if (labeled_group.thread_rank() == 2) {
    assert(FloatApproximatelyEqual(path.pweight_, 0.33333f));
  } else {
    assert(FloatApproximatelyEqual(path.pweight_, 0.0f));
  }

  // Extend the first group only
  if (label == 0) {
    path.Extend();
    assert(path.unique_depth_ == 3);
    if (labeled_group.thread_rank() == 0) {
      assert(FloatApproximatelyEqual(path.pweight_, 0.05f));
    } else if (labeled_group.thread_rank() == 1) {
      assert(FloatApproximatelyEqual(path.pweight_, 0.086111f));
    } else if (labeled_group.thread_rank() == 2) {
      assert(FloatApproximatelyEqual(path.pweight_, 0.147222f));
    } else if (labeled_group.thread_rank() == 3) {
      assert(FloatApproximatelyEqual(path.pweight_, 0.25f));
    } else {
      assert(FloatApproximatelyEqual(path.pweight_, 0.0f));
    }
  } else {
    assert(path.unique_depth_ == 2);
    if (labeled_group.thread_rank() == 0) {
      assert(FloatApproximatelyEqual(path.pweight_, 0.133333f));
    } else if (labeled_group.thread_rank() == 1) {
      assert(FloatApproximatelyEqual(path.pweight_, 0.21111f));
    } else if (labeled_group.thread_rank() == 2) {
      assert(FloatApproximatelyEqual(path.pweight_, 0.33333f));
    } else {
      assert(FloatApproximatelyEqual(path.pweight_, 0.0f));
    }
  }
  if (label == 0) {
    float unwound_sum = path.UnwoundPathSum();

    if (labeled_group.thread_rank() == 1) {
      assert(FloatApproximatelyEqual(unwound_sum, 0.63888f));
    } else if (labeled_group.thread_rank() == 2) {
      assert(FloatApproximatelyEqual(unwound_sum, 0.61666f));
    } else if (labeled_group.thread_rank() == 3) {
      assert(FloatApproximatelyEqual(unwound_sum, 0.67777f));
    } else if (labeled_group.thread_rank() > 3) {
      assert(FloatApproximatelyEqual(unwound_sum, 0.0f));
    }
  }
}

TEST(GPUTreeShap, ExtendMultiplePaths) {
  const float inf = std::numeric_limits<float>::infinity();
  std::vector<PathElement<XgboostSplitCondition>> path{
      {0, -1, 0, {-inf, 0.0f, false}, 1.0f, 1.0f},
      {0, 0, 0, {0.5f, inf, false}, 3.0f / 5, 1.0f},
      {0, 1, 0, {0.5f, inf, false}, 2.0f / 3, 1.0f},
      {0, 2, 0, {-inf, 0.5f, false}, 1.0f / 2, 1.0f}};
  // Add the first three elements again
  path.emplace_back(path[0]);
  path.emplace_back(path[1]);
  path.emplace_back(path[2]);

  thrust::device_vector<PathElement<XgboostSplitCondition>> device_path(path);
  thrust::device_vector<float> data =
      std::vector<float>({1.0f, 1.0f, 0.0f, 1.0f, 0.0f, 0.0f});
  DenseDatasetWrapper X(data.data().get(), 2, 3);
  TestExtendMultipleKernel<<<1, 32>>>(X, 4, 3, device_path.data().get());
}

__global__ void TestActiveLabeledPartition() {
  cooperative_groups::thread_block block =
      cooperative_groups::this_thread_block();
  auto warp =
      cooperative_groups::tiled_partition<32, cooperative_groups::thread_block>(
          block);
  int label = warp.thread_rank() < 5 ? 3 : 6;
  auto labelled_partition = detail::active_labeled_partition(FULL_MASK, label);

  if (label == 3) {
    assert(labelled_partition.size() == 5);
    assert(labelled_partition.thread_rank() == warp.thread_rank());
  } else if (label == 6) {
    assert(labelled_partition.size() == 32 - 5);
    assert(labelled_partition.thread_rank() == warp.thread_rank() - 5);
  }

  bool odd = warp.thread_rank() % 2 == 1;
  uint32_t odd_mask = __ballot_sync(FULL_MASK, odd);
  uint32_t even_mask = __ballot_sync(FULL_MASK, !odd);
  if (odd) {
    auto labelled_partition2 =
        detail::active_labeled_partition(odd_mask, label);
    if (label == 3) {
      assert(labelled_partition2.size() == 2);
      assert(labelled_partition2.thread_rank() == warp.thread_rank() / 2);
    } else if (label == 6) {
      assert(labelled_partition2.size() == 14);
      assert(labelled_partition2.thread_rank() == (warp.thread_rank() / 2) - 2);
    }
  } else {
    auto labelled_partition2 =
        detail::active_labeled_partition(even_mask, label);
    if (label == 3) {
      assert(labelled_partition2.size() == 3);
      assert(labelled_partition2.thread_rank() == warp.thread_rank() / 2);
    } else if (label == 6) {
      assert(labelled_partition2.size() == 13);
      assert(labelled_partition2.thread_rank() == (warp.thread_rank() / 2) - 3);
    }
  }
}

TEST(GPUTreeShap, ActiveLabeledPartition) {
  TestActiveLabeledPartition<<<1, 32>>>();
  EXPECT_EQ(hipDeviceSynchronize(), 0);
}

TEST(GPUTreeShap, BFDBinPacking) {
  thrust::device_vector<int> counts(3);
  counts[0] = 2;
  counts[1] = 2;
  counts[2] = 1;
  auto bin_packing = detail::BFDBinPacking(counts, 3);
  EXPECT_EQ(bin_packing[0], 0u);
  EXPECT_EQ(bin_packing[1], 1u);
  EXPECT_EQ(bin_packing[2], 0u);

  counts.clear();
  counts.resize(12);
  counts[0] = 3;
  counts[1] = 3;
  counts[2] = 3;
  counts[3] = 3;
  counts[4] = 3;
  counts[5] = 3;
  counts[6] = 2;
  counts[7] = 2;
  counts[8] = 2;
  counts[9] = 2;
  counts[10] = 2;
  counts[11] = 2;
  bin_packing = detail::BFDBinPacking(counts, 10);
  EXPECT_EQ(bin_packing[0], 0u);
  EXPECT_EQ(bin_packing[1], 0u);
  EXPECT_EQ(bin_packing[2], 0u);
  EXPECT_EQ(bin_packing[3], 1u);
  EXPECT_EQ(bin_packing[4], 1u);
  EXPECT_EQ(bin_packing[5], 1u);
  EXPECT_EQ(bin_packing[6], 2u);
  EXPECT_EQ(bin_packing[7], 2u);
  EXPECT_EQ(bin_packing[8], 2u);
  EXPECT_EQ(bin_packing[9], 2u);
  EXPECT_EQ(bin_packing[10], 2u);
  EXPECT_EQ(bin_packing[11], 3u);
}

TEST(GPUTreeShap, NFBinPacking) {
  thrust::device_vector<int> counts(4);
  counts[0] = 3;
  counts[1] = 3;
  counts[2] = 1;
  counts[3] = 2;
  auto bin_packing = detail::NFBinPacking(counts, 5);
  EXPECT_EQ(bin_packing[0], 0u);
  EXPECT_EQ(bin_packing[1], 1u);
  EXPECT_EQ(bin_packing[2], 1u);
  EXPECT_EQ(bin_packing[3], 2u);
}

TEST(GPUTreeShap, FFDBinPacking) {
  thrust::device_vector<int> counts(5);
  counts[0] = 3;
  counts[1] = 2;
  counts[2] = 3;
  counts[3] = 4;
  counts[4] = 1;
  auto bin_packing = detail::FFDBinPacking(counts, 5);
  EXPECT_EQ(bin_packing[0], 1u);
  EXPECT_EQ(bin_packing[1], 1u);
  EXPECT_EQ(bin_packing[2], 2u);
  EXPECT_EQ(bin_packing[3], 0u);
  EXPECT_EQ(bin_packing[4], 0u);
}

__global__ void TestContiguousGroup() {
  int label = threadIdx.x > 2 && threadIdx.x < 6 ? 1 : threadIdx.x >= 6 ? 2 : 0;

  auto group = detail::active_labeled_partition(FULL_MASK, label);

  if (label == 1) {
    assert(group.size() == 3);
    assert(group.thread_rank() == threadIdx.x - 3);
    int up = group.shfl_up(threadIdx.x, 1);
    if (group.thread_rank() > 0) {
      assert(up == threadIdx.x - 1);
    }
    assert(group.shfl(threadIdx.x, 2) == 5);
  }
}

TEST(GPUTreeShap, ContiguousGroup) {
  TestContiguousGroup<<<1, 32>>>();
  EXPECT_EQ(hipDeviceSynchronize(), 0);
}

class DeterminismTest : public ::testing::Test {
 protected:
  DeterminismTest() {
    size_t num_rows = 100;
    size_t num_features = 100;
    num_groups = 1;
    size_t max_depth = 10;
    size_t num_paths = 1000;
    samples = 100;
    model = GenerateEnsembleModel(num_groups, max_depth, num_features,
                                  num_paths, 78);
    test_data = TestDataset(num_rows, num_features, 22, 1e-15);

    X = test_data.GetDeviceWrapper();

    reference_phis.resize(X.NumRows() * (X.NumCols() + 1) * (X.NumCols() + 1) *
                          num_groups);
  }

  std::vector<PathElement<XgboostSplitCondition>> model;
  TestDataset test_data;
  DenseDatasetWrapper X;
  size_t samples;
  size_t num_groups;
  thrust::device_vector<float> reference_phis;
};

TEST_F(DeterminismTest, GPUTreeShap) {
  GPUTreeShap(X, model.begin(), model.end(), num_groups, reference_phis.begin(),
              reference_phis.end());

  for (auto i = 0ull; i < samples; i++) {
    thrust::device_vector<float> phis(reference_phis.size());
    GPUTreeShap(X, model.begin(), model.end(), num_groups, phis.begin(),
                phis.end());
    ASSERT_TRUE(thrust::equal(reference_phis.begin(), reference_phis.end(),
                              phis.begin()));
  }
}

TEST_F(DeterminismTest, GPUTreeShapInteractions) {
  GPUTreeShapInteractions(X, model.begin(), model.end(), num_groups,
                          reference_phis.begin(), reference_phis.end());

  for (auto i = 0ull; i < samples; i++) {
    thrust::device_vector<float> phis(reference_phis.size());
    GPUTreeShapInteractions(X, model.begin(), model.end(), num_groups,
                            phis.begin(), phis.end());
    ASSERT_TRUE(thrust::equal(reference_phis.begin(), reference_phis.end(),
                              phis.begin()));
  }
}

TEST_F(DeterminismTest, GPUTreeShapTaylorInteractions) {
  GPUTreeShapTaylorInteractions(X, model.begin(), model.end(), num_groups,
                                reference_phis.begin(), reference_phis.end());

  for (auto i = 0ull; i < samples; i++) {
    thrust::device_vector<float> phis(reference_phis.size());
    GPUTreeShapTaylorInteractions(X, model.begin(), model.end(), num_groups,
                                  phis.begin(), phis.end());
    ASSERT_TRUE(thrust::equal(reference_phis.begin(), reference_phis.end(),
                              phis.begin()));
  }
}

// Example from page 10 section 4.1
// Dhamdhere, Kedar, Ashish Agarwal, and Mukund Sundararajan. "The Shapley
// Taylor Interaction Index." arXiv preprint arXiv:1902.05622 (2019).
TEST(GPUTreeShap, TaylorInteractionsPaperExample) {
  const float inf = std::numeric_limits<float>::infinity();
  float c = 3.0f;
  std::vector<PathElement<XgboostSplitCondition>> path{
      {0, -1, 0, {-inf, inf, false}, 1.0f, 1.0f},
      {0, 0, 0, {0.5f, inf, false}, 0.0f, 1.0f},
      {1, -1, 0, {-inf, inf, false}, 1.0f, 1.0f},
      {1, 1, 0, {0.5f, inf, false}, 0.0f, 1.0f},
      {2, -1, 0, {-inf, inf, false}, 1.0f, 1.0f},
      {2, 2, 0, {0.5f, inf, false}, 0.0f, 1.0f},
      {3, -1, 0, {-inf, inf, false}, 1.0f, c},
      {3, 0, 0, {0.5f, inf, false}, 0.0f, c},
      {3, 1, 0, {0.5f, inf, false}, 0.0f, c},
      {3, 2, 0, {0.5f, inf, false}, 0.0f, c},
  };
  thrust::device_vector<float> data = std::vector<float>({1.0f, 1.0f, 1.0f});
  DenseDatasetWrapper X(data.data().get(), 1, 3);
  thrust::device_vector<float> interaction_phis(
      X.NumRows() * (X.NumCols() + 1) * (X.NumCols() + 1));
  GPUTreeShapTaylorInteractions(X, path.begin(), path.end(), 1,
                                interaction_phis.begin(),
                                interaction_phis.end());

  std::vector<float> interactions_result(interaction_phis.begin(),
                                         interaction_phis.end());
  std::vector<float> expected_result = {1.0, 0.5, 0.5, 0.0, 0.5, 1.0, 0.5, 0.0,
                                        0.5, 0.5, 1.0, 0.0, 0.0, 0.0, 0.0, 0.0};

  ASSERT_EQ(interaction_phis, expected_result);
}

TEST(GPUTreeShap, TaylorInteractionsBasic) {
  const float inf = std::numeric_limits<float>::infinity();
  std::vector<PathElement<XgboostSplitCondition>> path{
      {0, -1, 0, {-inf, inf, false}, 1.0f, 2.0f},
      {0, 0, 0, {0.5f, inf, false}, 0.25f, 2.0f},
      {0, 1, 0, {0.5f, inf, false}, 0.5f, 2.0f},
      {0, 2, 0, {0.5f, inf, false}, 0.6f, 2.0f},
      {0, 3, 0, {0.5f, inf, false}, 1.0f, 2.0f},
  };
  thrust::device_vector<float> data =
      std::vector<float>({1.0f, 1.0f, 1.0f, 1.0f});
  DenseDatasetWrapper X(data.data().get(), 1, 4);
  thrust::device_vector<float> interaction_phis(
      X.NumRows() * (X.NumCols() + 1) * (X.NumCols() + 1));
  GPUTreeShapTaylorInteractions(X, path.begin(), path.end(), 1,
                                interaction_phis.begin(),
                                interaction_phis.end());

  thrust::host_vector<float> interactions_result(interaction_phis);
  float sum =
      std::accumulate(interaction_phis.begin(), interaction_phis.end(), 0.0f);

  ASSERT_FLOAT_EQ(sum, 2.0f);
}


TEST(GPUTreeShap, GetWCoefficients) {
  EXPECT_DOUBLE_EQ(detail::W(0, 1), 1.0);
  EXPECT_DOUBLE_EQ(detail::W(0, 2), 0.5);
  EXPECT_DOUBLE_EQ(detail::W(1, 2), 0.5);
  EXPECT_DOUBLE_EQ(detail::W(0, 3), 2.0 / 6);
  EXPECT_DOUBLE_EQ(detail::W(1, 3), 1.0 / 6);
  EXPECT_DOUBLE_EQ(detail::W(2, 3), 2.0 / 6);
  EXPECT_DOUBLE_EQ(detail::W(0, 4), 6.0 / 24);
  EXPECT_DOUBLE_EQ(detail::W(1, 4), 2.0 / 24);
  EXPECT_DOUBLE_EQ(detail::W(2, 4), 2.0 / 24);
  EXPECT_DOUBLE_EQ(detail::W(3, 4), 6.0 / 24);
}

TEST(GPUTreeShap, InterventionalBasic) {
  const float inf = std::numeric_limits<float>::infinity();
  std::vector<PathElement<XgboostSplitCondition>> path{
      {0, -1, 0, {-inf, inf, false}, 1.0f, 8.0f},
      {0, 0, 0, {5.0f, inf, false}, 0.0f, 8.0f},
      {0, 1, 0, {5.0f, inf, false}, 0.0f, 8.0f},
      {0, 0, 0, {5.0f, inf, false}, 0.0f, 8.0f},
      {1, -1, 0, {-inf, inf, false}, 1.0f, 6.0f},
      {1, 0, 0, {5.0f, inf, false}, 0.0f, 6.0f},
      {1, 1, 0, {-inf, 5.0f, false}, 0.0f, 6.0f},
      {1, 2, 0, {-5.0f, inf, false}, 0.0f, 6.0f},
      {2, -1, 0, {-inf, inf, false}, 1.0f, 5.0f},
      {2, 0, 0, {5.0f, inf, false}, 0.0f, 5.0f},
      {2, 1, 0, {-inf, 5.0f, false}, 0.0f, 5.0f},
      {2, 2, 0, {-inf, -5.0f, false}, 0.0f, 5.0f},
  };
  thrust::device_vector<float> X_data =
      std::vector<float>({10.0f, 0.0f, 10.0f});
  thrust::device_vector<float> R_data =
      std::vector<float>({10.0f, 10.0f, -10.0f, 10.0f, 10.0f, 10.0f});
  DenseDatasetWrapper X(X_data.data().get(), 1, 3);
  DenseDatasetWrapper R(R_data.data().get(), 2, 3);
  thrust::device_vector<float> phis(X.NumRows() * (X.NumCols() + 1));
  GPUTreeShapInterventional(X, R, path.begin(), path.end(), 1,
                            phis.begin(), phis.end());

  std::vector<float> result(phis.begin(), phis.end());
  ASSERT_FLOAT_EQ(result[0], 0.0f);
  ASSERT_FLOAT_EQ(result[1], -2.25f);
  ASSERT_FLOAT_EQ(result[2], 0.25f);
  ASSERT_FLOAT_EQ(result[3], 8.0f);
}
