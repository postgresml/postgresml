#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Evaluates different tuning configurations of DeviceReduce.
 *
 * The best way to use this program:
 * (1) Find the best all-around single-block tune for a given arch.
 *     For example, 100 samples [1 ..512], 100 timing iterations per config per sample:
 *         ./bin/tune_device_reduce_sm200_nvvm_5.0_abi_i386 --i=100 --s=100 --n=512 --single --device=0
 * (2) Update the single tune in device_reduce.cuh
 * (3) Find the best all-around multi-block tune for a given arch.
 *     For example, 100 samples [single-block tile-size ..  50,331,648], 100 timing iterations per config per sample:
 *         ./bin/tune_device_reduce_sm200_nvvm_5.0_abi_i386 --i=100 --s=100 --device=0
 * (4) Update the multi-block tune in device_reduce.cuh
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <vector>
#include <algorithm>
#include <stdio.h>
#include <hipcub/hipcub.hpp>
#include "../test/test_util.h"

using namespace hipcub;
using namespace std;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

#ifndef TUNE_ARCH
#define TUNE_ARCH 100
#endif

int     g_max_items         = 48 * 1024 * 1024;
int     g_samples           = 100;
int     g_timing_iterations        = 2;
bool    g_verbose           = false;
bool    g_single            = false;
bool    g_verify            = true;
CachingDeviceAllocator  g_allocator;


//---------------------------------------------------------------------
// Host utility subroutines
//---------------------------------------------------------------------

/**
 * Initialize problem
 */
template <typename T>
void Initialize(
    GenMode         gen_mode,
    T               *h_in,
    int             num_items)
{
    for (int i = 0; i < num_items; ++i)
    {
        InitValue(gen_mode, h_in[i], i);
    }
}

/**
 * Sequential reduction
 */
template <typename T, typename ReductionOp>
T Reduce(
    T               *h_in,
    ReductionOp     reduction_op,
    int             num_items)
{
    T retval = h_in[0];
    for (int i = 1; i < num_items; ++i)
        retval = reduction_op(retval, h_in[i]);

    return retval;
}



//---------------------------------------------------------------------
// Full tile test generation
//---------------------------------------------------------------------



/**
 * Wrapper structure for generating and running different tuning configurations
 */
template <
    typename T,
    typename OffsetT,
    typename ReductionOp>
struct Schmoo
{
    //---------------------------------------------------------------------
    // Types
    //---------------------------------------------------------------------

    /// Pairing of kernel function pointer and corresponding dispatch params
    template <typename KernelPtr>
    struct DispatchTuple
    {
        KernelPtr                           kernel_ptr;
        DeviceReduce::KernelDispachParams   params;

        float                               avg_throughput;
        float                               best_avg_throughput;
        OffsetT                              best_size;
        float                               hmean_speedup;


        DispatchTuple() :
            kernel_ptr(0),
            params(DeviceReduce::KernelDispachParams()),
            avg_throughput(0.0),
            best_avg_throughput(0.0),
            hmean_speedup(0.0),
            best_size(0)
        {}
    };

    /**
     * Comparison operator for DispatchTuple.avg_throughput
     */
    template <typename Tuple>
    static bool MinSpeedup(const Tuple &a, const Tuple &b)
    {
        float delta = a.hmean_speedup - b.hmean_speedup;

        return ((delta < 0.02) && (delta > -0.02)) ?
            (a.best_avg_throughput < b.best_avg_throughput) :       // Negligible average performance differences: defer to best performance
            (a.hmean_speedup < b.hmean_speedup);
    }



    /// Multi-block reduction kernel type and dispatch tuple type
    typedef void (*MultiBlockDeviceReduceKernelPtr)(T*, T*, OffsetT, GridEvenShare<OffsetT>, GridQueue<OffsetT>, ReductionOp);
    typedef DispatchTuple<MultiBlockDeviceReduceKernelPtr> MultiDispatchTuple;

    /// Single-block reduction kernel type and dispatch tuple type
    typedef void (*SingleBlockDeviceReduceKernelPtr)(T*, T*, OffsetT, ReductionOp);
    typedef DispatchTuple<SingleBlockDeviceReduceKernelPtr> SingleDispatchTuple;


    //---------------------------------------------------------------------
    // Fields
    //---------------------------------------------------------------------

    vector<MultiDispatchTuple> multi_kernels;       // List of generated multi-block kernels
    vector<SingleDispatchTuple> single_kernels;     // List of generated single-block kernels


    //---------------------------------------------------------------------
    // Kernel enumeration methods
    //---------------------------------------------------------------------

    /**
     * Must have smem that fits in the SM
     * Must have vector load length that divides items per thread
     */
    template <typename TilesReducePolicy, typename ReductionOp>
    struct SmemSize
    {
        enum
        {
            BYTES = sizeof(typename BlockReduceTiles<TilesReducePolicy, T*, OffsetT, ReductionOp>::TempStorage),
            IS_OK = ((BYTES < ArchProps<TUNE_ARCH>::SMEM_BYTES) &&
                     (TilesReducePolicy::ITEMS_PER_THREAD % TilesReducePolicy::VECTOR_LOAD_LENGTH == 0))
        };
    };


    /**
     * Specialization that allows kernel generation with the specified TilesReducePolicy
     */
    template <
        typename    TilesReducePolicy,
        bool        IsOk = SmemSize<TilesReducePolicy, ReductionOp>::IS_OK>
    struct Ok
    {
        /// Enumerate multi-block kernel and add to the list
        template <typename KernelsVector>
        static void GenerateMulti(
            KernelsVector &multi_kernels,
            int subscription_factor)
        {
            MultiDispatchTuple tuple;
            tuple.params.template Init<TilesReducePolicy>(subscription_factor);
            tuple.kernel_ptr = ReducePrivatizedKernel<TilesReducePolicy, T*, T*, OffsetT, ReductionOp>;
            multi_kernels.push_back(tuple);
        }


        /// Enumerate single-block kernel and add to the list
        template <typename KernelsVector>
        static void GenerateSingle(KernelsVector &single_kernels)
        {
            SingleDispatchTuple tuple;
            tuple.params.template Init<TilesReducePolicy>();
            tuple.kernel_ptr = ReduceSingleKernel<TilesReducePolicy, T*, T*, OffsetT, ReductionOp>;
            single_kernels.push_back(tuple);
        }
    };

    /**
     * Specialization that rejects kernel generation with the specified TilesReducePolicy
     */
    template <typename TilesReducePolicy>
    struct Ok<TilesReducePolicy, false>
    {
        template <typename KernelsVector>
        static void GenerateMulti(KernelsVector &multi_kernels, int subscription_factor) {}

        template <typename KernelsVector>
        static void GenerateSingle(KernelsVector &single_kernels) {}
    };


    /// Enumerate block-scheduling variations
    template <
        int                     BLOCK_THREADS,
        int                     ITEMS_PER_THREAD,
        int                     VECTOR_LOAD_LENGTH,
        BlockReduceAlgorithm    BLOCK_ALGORITHM,
        CacheLoadModifier      LOAD_MODIFIER>
    void Enumerate()
    {
        // Multi-block kernels
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, LOAD_MODIFIER, GRID_MAPPING_RAKE> >::GenerateMulti(multi_kernels, 1);
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, LOAD_MODIFIER, GRID_MAPPING_RAKE> >::GenerateMulti(multi_kernels, 2);
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, LOAD_MODIFIER, GRID_MAPPING_RAKE> >::GenerateMulti(multi_kernels, 4);
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, LOAD_MODIFIER, GRID_MAPPING_RAKE> >::GenerateMulti(multi_kernels, 8);
#if TUNE_ARCH >= 200
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, LOAD_MODIFIER, GRID_MAPPING_DYNAMIC> >::GenerateMulti(multi_kernels, 1);
#endif

        // Single-block kernels
        Ok<BlockReduceTilesPolicy<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, LOAD_MODIFIER, GRID_MAPPING_RAKE> >::GenerateSingle(single_kernels);
    }


    /// Enumerate load modifier variations
    template <
        int                     BLOCK_THREADS,
        int                     ITEMS_PER_THREAD,
        int                     VECTOR_LOAD_LENGTH,
        BlockReduceAlgorithm    BLOCK_ALGORITHM>
    void Enumerate()
    {
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, LOAD_DEFAULT>();
#if TUNE_ARCH >= 350
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_ALGORITHM, LOAD_LDG>();
#endif
    }


    /// Enumerate block algorithms
    template <
        int BLOCK_THREADS,
        int ITEMS_PER_THREAD,
        int VECTOR_LOAD_LENGTH>
    void Enumerate()
    {
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_REDUCE_RAKING>();
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, VECTOR_LOAD_LENGTH, BLOCK_REDUCE_WARP_REDUCTIONS>();
    }


    /// Enumerate vectorization variations
    template <
        int BLOCK_THREADS,
        int ITEMS_PER_THREAD>
    void Enumerate()
    {
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, 1>();
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, 2>();
        Enumerate<BLOCK_THREADS, ITEMS_PER_THREAD, 4>();
    }


    /// Enumerate thread-granularity variations
    template <int BLOCK_THREADS>
    void Enumerate()
    {
        Enumerate<BLOCK_THREADS, 7>();
        Enumerate<BLOCK_THREADS, 8>();
        Enumerate<BLOCK_THREADS, 9>();

        Enumerate<BLOCK_THREADS, 11>();
        Enumerate<BLOCK_THREADS, 12>();
        Enumerate<BLOCK_THREADS, 13>();

        Enumerate<BLOCK_THREADS, 15>();
        Enumerate<BLOCK_THREADS, 16>();
        Enumerate<BLOCK_THREADS, 17>();

        Enumerate<BLOCK_THREADS, 19>();
        Enumerate<BLOCK_THREADS, 20>();
        Enumerate<BLOCK_THREADS, 21>();

        Enumerate<BLOCK_THREADS, 23>();
        Enumerate<BLOCK_THREADS, 24>();
        Enumerate<BLOCK_THREADS, 25>();
    }


    /// Enumerate block size variations
    void Enumerate()
    {
        printf("\nEnumerating kernels\n"); fflush(stdout);

        Enumerate<32>();
        Enumerate<64>();
        Enumerate<96>();
        Enumerate<128>();
        Enumerate<160>();
        Enumerate<192>();
        Enumerate<256>();
        Enumerate<512>();
    }


    //---------------------------------------------------------------------
    // Test methods
    //---------------------------------------------------------------------

    /**
     * Test a configuration
     */
    void TestConfiguration(
        MultiDispatchTuple      &multi_dispatch,
        SingleDispatchTuple     &single_dispatch,
        T*                      d_in,
        T*                      d_out,
        T*                      h_reference,
        OffsetT                  num_items,
        ReductionOp             reduction_op)
    {
        // Clear output
        if (g_verify) CubDebugExit(hipMemset(d_out, 0, sizeof(T)));

        // Allocate temporary storage
        void            *d_temp_storage = NULL;
        size_t          temp_storage_bytes = 0;
        CubDebugExit(DeviceReduce::Dispatch(
            d_temp_storage,
            temp_storage_bytes,
            multi_dispatch.kernel_ptr,
            single_dispatch.kernel_ptr,
            FillAndResetDrainKernel<OffsetT>,
            multi_dispatch.params,
            single_dispatch.params,
            d_in,
            d_out,
            num_items,
            reduction_op));
        CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

        // Warmup/correctness iteration
        CubDebugExit(DeviceReduce::Dispatch(
            d_temp_storage,
            temp_storage_bytes,
            multi_dispatch.kernel_ptr,
            single_dispatch.kernel_ptr,
            FillAndResetDrainKernel<OffsetT>,
            multi_dispatch.params,
            single_dispatch.params,
            d_in,
            d_out,
            num_items,
            reduction_op));

        if (g_verify) CubDebugExit(hipDeviceSynchronize());

        // Copy out and display results
        int compare = (g_verify) ?
            CompareDeviceResults(h_reference, d_out, 1, true, false) :
            0;

        // Performance
        GpuTimer gpu_timer;
        float elapsed_millis = 0.0;
        for (int i = 0; i < g_timing_iterations; i++)
        {
            gpu_timer.Start();

            CubDebugExit(DeviceReduce::Dispatch(
                d_temp_storage,
                temp_storage_bytes,
                multi_dispatch.kernel_ptr,
                single_dispatch.kernel_ptr,
                FillAndResetDrainKernel<OffsetT>,
                multi_dispatch.params,
                single_dispatch.params,
                d_in,
                d_out,
                num_items,
                reduction_op));

            gpu_timer.Stop();
            elapsed_millis += gpu_timer.ElapsedMillis();
        }

        // Mooch
        CubDebugExit(hipDeviceSynchronize());

        float avg_elapsed = elapsed_millis / g_timing_iterations;
        float avg_throughput = float(num_items) / avg_elapsed / 1000.0 / 1000.0;
        float avg_bandwidth = avg_throughput * sizeof(T);

        multi_dispatch.avg_throughput = CUB_MAX(avg_throughput, multi_dispatch.avg_throughput);
        if (avg_throughput > multi_dispatch.best_avg_throughput)
        {
            multi_dispatch.best_avg_throughput = avg_throughput;
            multi_dispatch.best_size = num_items;
        }

        single_dispatch.avg_throughput = CUB_MAX(avg_throughput, single_dispatch.avg_throughput);
        if (avg_throughput > single_dispatch.best_avg_throughput)
        {
            single_dispatch.best_avg_throughput = avg_throughput;
            single_dispatch.best_size = num_items;
        }

        if (g_verbose)
        {
            printf("\t%.2f GB/s, multi_dispatch( ", avg_bandwidth);
            multi_dispatch.params.Print();
            printf(" ), single_dispatch( ");
            single_dispatch.params.Print();
            printf(" )\n");
            fflush(stdout);
        }

        AssertEquals(0, compare);

        // Cleanup temporaries
        if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));
    }


    /**
     * Evaluate multi-block configurations
     */
    void TestMulti(
        T*                      h_in,
        T*                      d_in,
        T*                      d_out,
        ReductionOp             reduction_op)
    {
        // Simple single kernel tuple for use with multi kernel sweep
        typedef typename DeviceReduce::TunedPolicies<T, OffsetT, TUNE_ARCH>::SinglePolicy SimpleSinglePolicy;
        SingleDispatchTuple simple_single_tuple;
        simple_single_tuple.params.template Init<SimpleSinglePolicy>();
        simple_single_tuple.kernel_ptr = ReduceSingleKernel<SimpleSinglePolicy, T*, T*, OffsetT, ReductionOp>;

        double max_exponent      = log2(double(g_max_items));
        double min_exponent      = log2(double(simple_single_tuple.params.tile_size));
        unsigned int max_int     = (unsigned int) -1;

        for (int sample = 0; sample < g_samples; ++sample)
        {
            printf("\nMulti-block sample %d, ", sample);

            int num_items;
            if (sample == 0)
            {
                // First sample: use max items
                num_items = g_max_items;
                printf("num_items: %d", num_items); fflush(stdout);
            }
            else
            {
                // Sample a problem size from [2^g_min_exponent, g_max_items].  First 2/3 of the samples are log-distributed, the other 1/3 are uniformly-distributed.
                unsigned int bits;
                RandomBits(bits);
                double scale = double(bits) / max_int;

                if (sample < g_samples / 2)
                {
                    // log bias
                    double exponent = ((max_exponent - min_exponent) * scale) + min_exponent;
                    num_items = pow(2.0, exponent);
                    num_items = CUB_MIN(num_items, g_max_items);
                    printf("num_items: %d (2^%.2f)", num_items, exponent); fflush(stdout);
                }
                else
                {
                    // uniform bias
                    num_items = CUB_MAX(pow(2.0, min_exponent), scale * g_max_items);
                    num_items = CUB_MIN(num_items, g_max_items);
                    printf("num_items: %d (%.2f * %d)", num_items, scale, g_max_items); fflush(stdout);
                }
            }
            if (g_verbose)
                printf("\n");
            else
                printf(", ");

            // Compute reference
            T h_reference = Reduce(h_in, reduction_op, num_items);

            // Run test on each multi-kernel configuration
            float best_avg_throughput = 0.0;
            for (int j = 0; j < multi_kernels.size(); ++j)
            {
                multi_kernels[j].avg_throughput = 0.0;

                TestConfiguration(multi_kernels[j], simple_single_tuple, d_in, d_out, &h_reference, num_items, reduction_op);

                best_avg_throughput = CUB_MAX(best_avg_throughput, multi_kernels[j].avg_throughput);
            }

            // Print best throughput for this problem size
            printf("Best: %.2fe9 items/s (%.2f GB/s)\n", best_avg_throughput, best_avg_throughput * sizeof(T));

            // Accumulate speedup (inverse for harmonic mean)
            for (int j = 0; j < multi_kernels.size(); ++j)
                multi_kernels[j].hmean_speedup += best_avg_throughput / multi_kernels[j].avg_throughput;
        }

        // Find max overall throughput and compute hmean speedups
        float overall_max_throughput = 0.0;
        for (int j = 0; j < multi_kernels.size(); ++j)
        {
            overall_max_throughput = CUB_MAX(overall_max_throughput, multi_kernels[j].best_avg_throughput);
            multi_kernels[j].hmean_speedup = float(g_samples) / multi_kernels[j].hmean_speedup;
        }

        // Sort by cumulative speedup
        sort(multi_kernels.begin(), multi_kernels.end(), MinSpeedup<MultiDispatchTuple>);

        // Print ranked multi configurations
        printf("\nRanked multi_kernels:\n");
        for (int j = 0; j < multi_kernels.size(); ++j)
        {
            printf("\t (%d) params( ", multi_kernels.size() - j);
            multi_kernels[j].params.Print();
            printf(" ) hmean speedup: %.3f, best throughput %.2f @ %d elements (%.2f GB/s, %.2f%%)\n",
                multi_kernels[j].hmean_speedup,
                multi_kernels[j].best_avg_throughput,
                (int) multi_kernels[j].best_size,
                multi_kernels[j].best_avg_throughput * sizeof(T),
                multi_kernels[j].best_avg_throughput / overall_max_throughput);
        }

        printf("\nMax multi-block throughput %.2f (%.2f GB/s)\n", overall_max_throughput, overall_max_throughput * sizeof(T));
    }


    /**
     * Evaluate single-block configurations
     */
    void TestSingle(
        T*                      h_in,
        T*                      d_in,
        T*                      d_out,
        ReductionOp             reduction_op)
     {
        // Construct a NULL-ptr multi-kernel tuple that forces a single-kernel pass
        MultiDispatchTuple multi_tuple;

        double max_exponent     = log2(double(g_max_items));
        unsigned int max_int    = (unsigned int) -1;

        for (int sample = 0; sample < g_samples; ++sample)
        {
            printf("\nSingle-block sample %d, ", sample);

            int num_items;
            if (sample == 0)
            {
                // First sample: use max items
                num_items = g_max_items;
                printf("num_items: %d", num_items); fflush(stdout);
            }
            else
            {
                // Sample a problem size from [2, g_max_items], log-distributed
                unsigned int bits;
                RandomBits(bits);
                double scale = double(bits) / max_int;
                double exponent = ((max_exponent - 1) * scale) + 1;
                num_items = pow(2.0, exponent);
                printf("num_items: %d (2^%.2f)", num_items, exponent); fflush(stdout);
            }

            if (g_verbose)
                printf("\n");
            else
                printf(", ");

            // Compute reference
            T h_reference = Reduce(h_in, reduction_op, num_items);

            // Run test on each single-kernel configuration (pick first multi-config to use, which shouldn't be
            float best_avg_throughput = 0.0;
            for (int j = 0; j < single_kernels.size(); ++j)
            {
                single_kernels[j].avg_throughput = 0.0;

                TestConfiguration(multi_tuple, single_kernels[j], d_in, d_out, &h_reference, num_items, reduction_op);

                best_avg_throughput = CUB_MAX(best_avg_throughput, single_kernels[j].avg_throughput);
            }

            // Print best throughput for this problem size
            printf("Best: %.2fe9 items/s (%.2f GB/s)\n", best_avg_throughput, best_avg_throughput * sizeof(T));

            // Accumulate speedup (inverse for harmonic mean)
            for (int j = 0; j < single_kernels.size(); ++j)
                single_kernels[j].hmean_speedup += best_avg_throughput / single_kernels[j].avg_throughput;
        }

        // Find max overall throughput and compute hmean speedups
        float overall_max_throughput = 0.0;
        for (int j = 0; j < single_kernels.size(); ++j)
        {
            overall_max_throughput = CUB_MAX(overall_max_throughput, single_kernels[j].best_avg_throughput);
            single_kernels[j].hmean_speedup = float(g_samples) / single_kernels[j].hmean_speedup;
        }

        // Sort by cumulative speedup
        sort(single_kernels.begin(), single_kernels.end(), MinSpeedup<SingleDispatchTuple>);

        // Print ranked single configurations
        printf("\nRanked single_kernels:\n");
        for (int j = 0; j < single_kernels.size(); ++j)
        {
            printf("\t (%d) params( ", single_kernels.size() - j);
            single_kernels[j].params.Print();
            printf(" ) hmean speedup: %.3f, best throughput %.2f @ %d elements (%.2f GB/s, %.2f%%)\n",
                single_kernels[j].hmean_speedup,
                single_kernels[j].best_avg_throughput,
                (int) single_kernels[j].best_size,
                single_kernels[j].best_avg_throughput * sizeof(T),
                single_kernels[j].best_avg_throughput / overall_max_throughput);
        }

        printf("\nMax single-block throughput %.2f (%.2f GB/s)\n", overall_max_throughput, overall_max_throughput * sizeof(T));
    }

};



//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    args.GetCmdLineArgument("n", g_max_items);
    args.GetCmdLineArgument("s", g_samples);
    args.GetCmdLineArgument("i", g_timing_iterations);
    g_verbose = args.CheckCmdLineFlag("v");
    g_single = args.CheckCmdLineFlag("single");
    g_verify = !args.CheckCmdLineFlag("noverify");

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--n=<max items>]"
            "[--s=<samples>]"
            "[--i=<timing iterations>]"
            "[--single]"
            "[--v]"
            "[--noverify]"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

#if (TUNE_SIZE == 1)
    typedef unsigned char T;
#elif (TUNE_SIZE == 2)
    typedef unsigned short T;
#elif (TUNE_SIZE == 4)
    typedef unsigned int T;
#elif (TUNE_SIZE == 8)
    typedef unsigned long long T;
#else
    // Default
    typedef unsigned int T;
#endif

    typedef unsigned int OffsetT;
    Sum reduction_op;

    // Enumerate kernels
    Schmoo<T, OffsetT, Sum > schmoo;
    schmoo.Enumerate();

    // Allocate host arrays
    T *h_in = new T[g_max_items];

    // Initialize problem
    Initialize(UNIFORM, h_in, g_max_items);

    // Initialize device arrays
    T *d_in = NULL;
    T *d_out = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(T) * g_max_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(T) * 1));
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * g_max_items, hipMemcpyHostToDevice));

    // Test kernels
    if (g_single)
        schmoo.TestSingle(h_in, d_in, d_out, reduction_op);
    else
        schmoo.TestMulti(h_in, d_in, d_out, reduction_op);

    // Cleanup
    if (h_in) delete[] h_in;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));

    return 0;
}



