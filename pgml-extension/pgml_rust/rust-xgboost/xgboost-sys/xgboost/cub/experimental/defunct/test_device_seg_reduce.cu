#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * An implementation of segmented reduction using a load-balanced parallelization
 * strategy based on the MergePath decision path.
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <iterator>
#include <vector>
#include <string>
#include <algorithm>
#include <stdio.h>

#include <hipcub/hipcub.hpp>

#include "test_util.h"

using namespace hipcub;
using namespace std;


/******************************************************************************
 * Globals, constants, and typedefs
 ******************************************************************************/

bool                    g_verbose           = false;
int                     g_timing_iterations = 1;
CachingDeviceAllocator  g_allocator(true);


/******************************************************************************
 * Utility routines
 ******************************************************************************/


/**
 * An pair of index offsets
 */
template <typename OffsetT>
struct IndexPair
{
    OffsetT a_idx;
    OffsetT b_idx;
};


/**
 * Computes the begin offsets into A and B for the specified
 * location (diagonal) along the merge decision path
 */
template <
    int                 BLOCK_THREADS,
    typename            IteratorA,
    typename            IteratorB,
    typename            OffsetT>
__device__ __forceinline__ void ParallelMergePathSearch(
    OffsetT             diagonal,
    IteratorA           a,
    IteratorB           b,
    IndexPair<OffsetT>  begin,          // Begin offsets into a and b
    IndexPair<OffsetT>  end,            // End offsets into a and b
    IndexPair<OffsetT>  &intersection)  // [out] Intersection offsets into a and b
{
    OffsetT a_split_min = CUB_MAX(diagonal - end.b_idx, begin.a_idx);
    OffsetT a_split_max = CUB_MIN(diagonal, end.a_idx);

    while (a_split_min < a_split_max)
    {
        OffsetT a_distance       = a_split_max - a_split_min;
        OffsetT a_slice          = (a_distance + BLOCK_THREADS - 1) >> Log2<BLOCK_THREADS>::VALUE;
        OffsetT a_split_pivot    = CUB_MIN(a_split_min + (threadIdx.x * a_slice), end.a_idx - 1);

        int move_up = (a[a_split_pivot] <= b[diagonal - a_split_pivot - 1]);
        int num_up = __syncthreads_count(move_up);
/*
        _HipcubLog("a_split_min(%d), a_split_max(%d) a_distance(%d), a_slice(%d), a_split_pivot(%d), move_up(%d), num_up(%d), a_begin(%d), a_end(%d)\n",
            a_split_min, a_split_max, a_distance, a_slice, a_split_pivot, move_up, num_up, a_begin, a_end);
*/
        a_split_max = CUB_MIN(num_up * a_slice, end.a_idx);
        a_split_min = CUB_MAX(a_split_max - a_slice, begin.a_idx) + 1;
    }

    intersection.a_idx = CUB_MIN(a_split_min, end.a_idx);
    intersection.b_idx = CUB_MIN(diagonal - a_split_min, end.b_idx);
}

/**
 * Computes the begin offsets into A and B for the specified
 * location (diagonal) along the merge decision path
 */
template <
    typename            IteratorA,
    typename            IteratorB,
    typename            OffsetT>
__device__ __forceinline__ void MergePathSearch(
    OffsetT             diagonal,
    IteratorA           a,
    IteratorB           b,
    IndexPair<OffsetT>  begin,          // Begin offsets into a and b
    IndexPair<OffsetT>  end,            // End offsets into a and b
    IndexPair<OffsetT>  &intersection)  // [out] Intersection offsets into a and b
{
    OffsetT split_min = CUB_MAX(diagonal - end.b_idx, begin.a_idx);
    OffsetT split_max = CUB_MIN(diagonal, end.a_idx);

    while (split_min < split_max)
    {
        OffsetT split_pivot = (split_min + split_max) >> 1;
        if (a[split_pivot] <= b[diagonal - split_pivot - 1])
        {
            // Move candidate split range up A, down B
            split_min = split_pivot + 1;
        }
        else
        {
            // Move candidate split range up B, down A
            split_max = split_pivot;
        }
    }

    intersection.a_idx = CUB_MIN(split_min, end.a_idx);
    intersection.b_idx = CUB_MIN(diagonal - split_min, end.b_idx);
}


/******************************************************************************
 * Tuning policy types
 ******************************************************************************/

/**
 * Parameterizable tuning policy type for BlockSegReduceRegion
 */
template <
    int                     _BLOCK_THREADS,             ///< Threads per thread block
    int                     _ITEMS_PER_THREAD,          ///< Items per thread (per tile of input)
    bool                    _USE_SMEM_SEGMENT_CACHE,    ///< Whether or not to cache incoming segment offsets in shared memory before reducing each tile
    bool                    _USE_SMEM_VALUE_CACHE,      ///< Whether or not to cache incoming values in shared memory before reducing each tile
    CacheLoadModifier       _LOAD_MODIFIER_SEGMENTS,    ///< Cache load modifier for reading segment offsets
    CacheLoadModifier       _LOAD_MODIFIER_VALUES,      ///< Cache load modifier for reading values
    BlockReduceAlgorithm    _REDUCE_ALGORITHM,          ///< The BlockReduce algorithm to use
    BlockScanAlgorithm      _SCAN_ALGORITHM>            ///< The BlockScan algorithm to use
struct BlockSegReduceRegionPolicy
{
    enum
    {
        BLOCK_THREADS           = _BLOCK_THREADS,               ///< Threads per thread block
        ITEMS_PER_THREAD        = _ITEMS_PER_THREAD,            ///< Items per thread (per tile of input)
        USE_SMEM_SEGMENT_CACHE  = _USE_SMEM_SEGMENT_CACHE,      ///< Whether or not to cache incoming segment offsets in shared memory before reducing each tile
        USE_SMEM_VALUE_CACHE    = _USE_SMEM_VALUE_CACHE,        ///< Whether or not to cache incoming upcoming values in shared memory before reducing each tile
    };

    static const CacheLoadModifier      LOAD_MODIFIER_SEGMENTS  = _LOAD_MODIFIER_SEGMENTS;  ///< Cache load modifier for reading segment offsets
    static const CacheLoadModifier      LOAD_MODIFIER_VALUES    = _LOAD_MODIFIER_VALUES;    ///< Cache load modifier for reading values
    static const BlockReduceAlgorithm   REDUCE_ALGORITHM        = _REDUCE_ALGORITHM;        ///< The BlockReduce algorithm to use
    static const BlockScanAlgorithm     SCAN_ALGORITHM          = _SCAN_ALGORITHM;          ///< The BlockScan algorithm to use
};


/******************************************************************************
 * Persistent thread block types
 ******************************************************************************/

/**
 * \brief BlockSegReduceTiles implements a stateful abstraction of CUDA thread blocks for participating in device-wide segmented reduction.
 */
template <
    typename BlockSegReduceRegionPolicy,    ///< Parameterized BlockSegReduceRegionPolicy tuning policy
    typename SegmentOffsetIterator,         ///< Random-access input iterator type for reading segment end-offsets
    typename ValueIterator,                 ///< Random-access input iterator type for reading values
    typename OutputIteratorT,               ///< Random-access output iterator type for writing segment reductions
    typename ReductionOp,                   ///< Binary reduction operator type having member <tt>T operator()(const T &a, const T &b)</tt>
    typename OffsetT>                       ///< Signed integer type for global offsets
struct BlockSegReduceRegion
{
    //---------------------------------------------------------------------
    // Types and constants
    //---------------------------------------------------------------------

    // Constants
    enum
    {
        BLOCK_THREADS       = BlockSegReduceRegionPolicy::BLOCK_THREADS,
        ITEMS_PER_THREAD    = BlockSegReduceRegionPolicy::ITEMS_PER_THREAD,
        TILE_ITEMS          = BLOCK_THREADS * ITEMS_PER_THREAD,                     /// Number of work items to be processed per tile

        USE_SMEM_SEGMENT_CACHE  = BlockSegReduceRegionPolicy::USE_SMEM_SEGMENT_CACHE,      ///< Whether or not to cache incoming segment offsets in shared memory before reducing each tile
        USE_SMEM_VALUE_CACHE    = BlockSegReduceRegionPolicy::USE_SMEM_VALUE_CACHE,        ///< Whether or not to cache incoming upcoming values in shared memory before reducing each tile

        SMEM_SEGMENT_CACHE_ITEMS    = USE_SMEM_SEGMENT_CACHE ? TILE_ITEMS : 1,
        SMEM_VALUE_CACHE_ITEMS      = USE_SMEM_VALUE_CACHE ? TILE_ITEMS : 1,
    };

    // Segment offset type
    typedef typename std::iterator_traits<SegmentOffsetIterator>::value_type SegmentOffset;

    // Value type
    typedef typename std::iterator_traits<ValueIterator>::value_type Value;

    // Counting iterator type
    typedef CountingInputIterator<SegmentOffsetT, OffsetT> CountingIterator;

    // Segment offsets iterator wrapper type
    typedef typename If<(IsPointer<SegmentOffsetIterator>::VALUE),
            CacheModifiedInputIterator<BlockSegReduceRegionPolicy::LOAD_MODIFIER_SEGMENTS, SegmentOffsetT, OffsetT>,  // Wrap the native input pointer with CacheModifiedInputIterator
            SegmentOffsetIterator>::Type                                                                            // Directly use the supplied input iterator type
        WrappedSegmentOffsetIterator;

    // Values iterator wrapper type
    typedef typename If<(IsPointer<ValueIterator>::VALUE),
            CacheModifiedInputIterator<BlockSegReduceRegionPolicy::LOAD_MODIFIER_VALUES, Value, OffsetT>,        // Wrap the native input pointer with CacheModifiedInputIterator
            ValueIterator>::Type                                                                                // Directly use the supplied input iterator type
        WrappedValueIterator;

    // Tail flag type for marking segment discontinuities
    typedef int TailFlag;

    // Reduce-by-key data type tuple (segment-ID, value)
    typedef KeyValuePair<OffsetT, Value> KeyValuePair;

    // Index pair data type
    typedef IndexPair<OffsetT> IndexPair;

    // BlockScan scan operator for reduction-by-segment
    typedef ReduceByKeyOp<ReductionOp> ReduceByKeyOp;

    // Stateful BlockScan prefix callback type for managing a running total while scanning consecutive tiles
    typedef RunningBlockPrefixCallbackOp<
            KeyValuePair,
            ReduceByKeyOp>
        RunningPrefixCallbackOp;

    // Parameterized BlockShift type for exchanging index pairs
    typedef BlockShift<
            IndexPair,
            BLOCK_THREADS>
        BlockShift;

    // Parameterized BlockReduce type for block-wide reduction
    typedef BlockReduce<
            Value,
            BLOCK_THREADS,
            BlockSegReduceRegionPolicy::REDUCE_ALGORITHM>
        BlockReduce;

    // Parameterized BlockScan type for block-wide reduce-value-by-key
    typedef BlockScan<
            KeyValuePair,
            BLOCK_THREADS,
            BlockSegReduceRegionPolicy::SCAN_ALGORITHM>
        BlockScan;

    // Shared memory type for this thread block
    struct _TempStorage
    {
        union
        {
            // Smem needed for BlockScan
            typename BlockScan::TempStorage scan;

            // Smem needed for BlockReduce
            typename BlockReduce::TempStorage reduce;

            struct
            {
                // Smem needed for communicating start/end indices between threads for a given work tile
                typename BlockShift::TempStorage shift;

                // Smem needed for caching segment end-offsets
                SegmentOffset cached_segment_end_offsets[SMEM_SEGMENT_CACHE_ITEMS + 1];
            };

            // Smem needed for caching values
            Value cached_values[SMEM_VALUE_CACHE_ITEMS];
        };

        IndexPair block_region_idx[2];      // The starting [0] and ending [1] pairs of segment and value indices for the thread block's region

        // The first partial reduction tuple scattered by this thread block
        KeyValuePair first_tuple;
    };


    // Alias wrapper allowing storage to be unioned
    struct TempStorage : Uninitialized<_TempStorage> {};


    //---------------------------------------------------------------------
    // Thread fields
    //---------------------------------------------------------------------

    _TempStorage                    &temp_storage;          ///< Reference to shared storage
    WrappedSegmentOffsetIterator    d_segment_end_offsets;  ///< A sequence of \p num_segments segment end-offsets
    WrappedValueIterator            d_values;               ///< A sequence of \p num_values data to reduce
    OutputIteratorT                  d_output;               ///< A sequence of \p num_segments segment totals
    CountingIterator                d_value_offsets;        ///< A sequence of \p num_values value-offsets
    IndexPair                       *d_block_idx;
    OffsetT                         num_values;             ///< Total number of values to reduce
    OffsetT                         num_segments;           ///< Number of segments being reduced
    Value                           identity;               ///< Identity value (for zero-length segments)
    ReductionOp                     reduction_op;           ///< Reduction operator
    ReduceByKeyOp                   scan_op;                ///< Reduce-by-key scan operator
    RunningPrefixCallbackOp         prefix_op;              ///< Stateful running total for block-wide prefix scan of partial reduction tuples


    //---------------------------------------------------------------------
    // Operations
    //---------------------------------------------------------------------

    /**
     * Constructor
     */
    __device__ __forceinline__
    BlockSegReduceRegion(
        TempStorage             &temp_storage,          ///< Reference to shared storage
        SegmentOffsetIterator   d_segment_end_offsets,  ///< A sequence of \p num_segments segment end-offsets
        ValueIterator           d_values,               ///< A sequence of \p num_values values
        OutputIteratorT          d_output,               ///< A sequence of \p num_segments segment totals
        IndexPair               *d_block_idx,
        OffsetT                 num_values,             ///< Number of values to reduce
        OffsetT                 num_segments,           ///< Number of segments being reduced
        Value                   identity,               ///< Identity value (for zero-length segments)
        ReductionOp             reduction_op)           ///< Reduction operator
    :
        temp_storage(temp_storage.Alias()),
        d_segment_end_offsets(d_segment_end_offsets),
        d_values(d_values),
        d_value_offsets(0),
        d_output(d_output),
        d_block_idx(d_block_idx),
        num_values(num_values),
        num_segments(num_segments),
        identity(identity),
        reduction_op(reduction_op),
        scan_op(reduction_op),
        prefix_op(scan_op)
    {}


    /**
     * Fast-path single-segment tile reduction.  Perform a
     * simple block-wide reduction and accumulate the result into
     * the running total.
     */
    __device__ __forceinline__ void SingleSegmentTile(
        IndexPair next_tile_idx,
        IndexPair block_idx)
    {
        OffsetT tile_values = next_tile_idx.b_idx - block_idx.b_idx;

        // Load a tile's worth of values (using identity for out-of-bounds items)
        Value values[ITEMS_PER_THREAD];
        LoadDirectStriped<BLOCK_THREADS>(threadIdx.x, d_values + block_idx.b_idx, values, tile_values, identity);

        // Barrier for smem reuse
        __syncthreads();

        // Reduce the tile of values and update the running total in thread-0
        KeyValuePair tile_aggregate;
        tile_aggregate.key      = block_idx.a_idx;
        tile_aggregate.value    = BlockReduce(temp_storage.reduce).Reduce(values, reduction_op);

        if (threadIdx.x == 0)
        {
            prefix_op.running_total = scan_op(prefix_op.running_total, tile_aggregate);
        }
    }

    /**
     * Fast-path empty-segment tile reduction.  Write out a tile of identity
     * values to output.
     */
    __device__ __forceinline__ void EmptySegmentsTile(
        IndexPair next_tile_idx,
        IndexPair block_idx)
    {
        Value segment_reductions[ITEMS_PER_THREAD];

        if (threadIdx.x == 0)
        {
            // The first segment gets the running segment total
            segment_reductions[0] = prefix_op.running_total.value;

            // Update the running prefix
            prefix_op.running_total.value = identity;
            prefix_op.running_total.key = next_tile_idx.a_idx;
        }
        else
        {
            // Remainder of segments in this tile get identity
            segment_reductions[0] = identity;
        }

        // Remainder of segments in this tile get identity
        #pragma unroll
        for (int ITEM = 1; ITEM < ITEMS_PER_THREAD; ++ITEM)
            segment_reductions[ITEM] = identity;

        // Store reductions
        OffsetT tile_segments = next_tile_idx.a_idx - block_idx.a_idx;
        StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_output + block_idx.a_idx, segment_reductions, tile_segments);
    }


    /**
     * Multi-segment tile reduction.
     */
    template <bool FULL_TILE>
    __device__ __forceinline__ void MultiSegmentTile(
        IndexPair block_idx,
        IndexPair thread_idx,
        IndexPair next_thread_idx,
        IndexPair next_tile_idx)
    {
        IndexPair local_thread_idx;
        local_thread_idx.a_idx = thread_idx.a_idx - block_idx.a_idx;
        local_thread_idx.b_idx = thread_idx.b_idx - block_idx.b_idx;

        // Check if first segment end-offset is in range
        bool valid_segment = FULL_TILE || (thread_idx.a_idx < next_thread_idx.a_idx);

        // Check if first value offset is in range
        bool valid_value = FULL_TILE || (thread_idx.b_idx < next_thread_idx.b_idx);

        // Load first segment end-offset
        OffsetT segment_end_offset = (valid_segment) ?
            (USE_SMEM_SEGMENT_CACHE)?
                temp_storage.cached_segment_end_offsets[local_thread_idx.a_idx] :
                d_segment_end_offsets[thread_idx.a_idx] :
            -1;

        OffsetT segment_ids[ITEMS_PER_THREAD];
        OffsetT value_offsets[ITEMS_PER_THREAD];

        KeyValuePair first_partial;
        first_partial.key    = thread_idx.a_idx;
        first_partial.value  = identity;

        // Get segment IDs and gather-offsets for values
        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM)
        {
            segment_ids[ITEM]   = -1;
            value_offsets[ITEM] = -1;

            // Whether or not we slide (a) right along the segment path or (b) down the value path
            if (valid_segment && (!valid_value || (segment_end_offset <= thread_idx.b_idx)))
            {
                // Consume this segment index
                segment_ids[ITEM] = thread_idx.a_idx;
                thread_idx.a_idx++;
                local_thread_idx.a_idx++;

                valid_segment = FULL_TILE || (thread_idx.a_idx < next_thread_idx.a_idx);

                // Read next segment end-offset (if valid)
                if (valid_segment)
                {
                    if (USE_SMEM_SEGMENT_CACHE)
                        segment_end_offset = temp_storage.cached_segment_end_offsets[local_thread_idx.a_idx];
                    else
                        segment_end_offset = d_segment_end_offsets[thread_idx.a_idx];
                }
            }
            else if (valid_value)
            {
                // Consume this value index
                value_offsets[ITEM] = thread_idx.b_idx;
                thread_idx.b_idx++;
                local_thread_idx.b_idx++;

                valid_value = FULL_TILE || (thread_idx.b_idx < next_thread_idx.b_idx);
            }
        }

        // Load values
        Value values[ITEMS_PER_THREAD];

        if (USE_SMEM_VALUE_CACHE)
        {
            // Barrier for smem reuse
            __syncthreads();

            OffsetT tile_values = next_tile_idx.b_idx - block_idx.b_idx;

            // Load a tile's worth of values (using identity for out-of-bounds items)
            LoadDirectStriped<BLOCK_THREADS>(threadIdx.x, d_values + block_idx.b_idx, values, tile_values, identity);

            // Store to shared
            StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, temp_storage.cached_values, values, tile_values);

            // Barrier for smem reuse
            __syncthreads();

            #pragma unroll
            for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM)
            {
                values[ITEM] = (value_offsets[ITEM] == -1) ?
                    identity :
                    temp_storage.cached_values[value_offsets[ITEM] - block_idx.b_idx];
            }
        }
        else
        {
            #pragma unroll
            for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM)
            {
                values[ITEM] = (value_offsets[ITEM] == -1) ?
                    identity :
                    d_values[value_offsets[ITEM]];
            }
        }

        // Reduce within thread segments
        KeyValuePair running_total = first_partial;

        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM)
        {
            if (segment_ids[ITEM] != -1)
            {
                // Consume this segment index
                d_output[segment_ids[ITEM]] = running_total.value;

//                _HipcubLog("Updating segment %d with value %lld\n", segment_ids[ITEM], running_total.value)

                if (first_partial.key == segment_ids[ITEM])
                    first_partial.value = running_total.value;

                running_total.key    = segment_ids[ITEM];
                running_total.value  = identity;
            }

            running_total.value = reduction_op(running_total.value, values[ITEM]);
        }
/*

        // Barrier for smem reuse
        __syncthreads();

        // Use prefix scan to reduce values by segment-id.  The segment-reductions end up in items flagged as segment-tails.
        KeyValuePair block_aggregate;
        BlockScan(temp_storage.scan).InclusiveScan(
            pairs,                          // Scan input
            pairs,                          // Scan output
            scan_op,                        // Scan operator
            block_aggregate,                // Block-wide total (unused)
            prefix_op);                     // Prefix operator for seeding the block-wide scan with the running total
*/

/*
        // Check if first segment end-offset is in range
        bool valid_segment = (thread_idx.a_idx < next_thread_idx.a_idx);

        // Check if first value offset is in range
        bool valid_value = (thread_idx.b_idx < next_thread_idx.b_idx);

        // Load first segment end-offset
        OffsetT segment_end_offset = (valid_segment) ?
            d_segment_end_offsets[thread_idx.a_idx] :
            num_values;                                                     // Out of range (the last segment end-offset is one-past the last value offset)

        // Load first value offset
        OffsetT value_offset = (valid_value) ?
            d_value_offsets[thread_idx.b_idx] :
            num_values;                                                     // Out of range (one-past the last value offset)

        // Assemble segment-demarcating tail flags and partial reduction tuples
        TailFlag        tail_flags[ITEMS_PER_THREAD];
        KeyValuePair    partial_reductions[ITEMS_PER_THREAD];

        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM)
        {
            // Default tuple and flag values
            partial_reductions[ITEM].key    = thread_idx.a_idx;
            partial_reductions[ITEM].value  = identity;
            tail_flags[ITEM]                = 0;

            // Whether or not we slide (a) right along the segment path or (b) down the value path
            if (valid_segment && (!valid_value || (segment_end_offset <= value_offset)))
            {
                // Consume this segment index

                // Set tail flag noting the end of the segment
                tail_flags[ITEM] = 1;

                // Increment segment index
                thread_idx.a_idx++;

                // Read next segment end-offset (if valid)
                if ((valid_segment = (thread_idx.a_idx < next_thread_idx.a_idx)))
                    segment_end_offset = d_segment_end_offsets[thread_idx.a_idx];
            }
            else if (valid_value)
            {
                // Consume this value index

                // Update the tuple's value with the value at this index.
                partial_reductions[ITEM].value = d_values[value_offset];

                // Increment value index
                thread_idx.b_idx++;

                // Read next value offset (if valid)
                if ((valid_value = (thread_idx.b_idx < next_thread_idx.b_idx)))
                    value_offset = d_value_offsets[thread_idx.b_idx];
            }
        }

        // Use prefix scan to reduce values by segment-id.  The segment-reductions end up in items flagged as segment-tails.
        KeyValuePair block_aggregate;
        BlockScan(temp_storage.scan).InclusiveScan(
            partial_reductions,             // Scan input
            partial_reductions,             // Scan output
            scan_op,                        // Scan operator
            block_aggregate,                // Block-wide total (unused)
            prefix_op);                     // Prefix operator for seeding the block-wide scan with the running total

        // The first segment index for this region (hoist?)
        OffsetT first_segment_idx = temp_storage.block_idx.a_idx[0];

        // Scatter an accumulated reduction if it is the head of a valid segment
        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++)
        {
            if (tail_flags[ITEM])
            {
                OffsetT segment_idx = partial_reductions[ITEM].key;
                Value   value       = partial_reductions[ITEM].value;

                // Write value reduction to corresponding segment id
                d_output[segment_idx] = value;

                // Save off the first value product that this thread block will scatter
                if (segment_idx == first_segment_idx)
                {
                    temp_storage.first_tuple.value = value;
                }
            }
        }
*/
    }



    /**
     * Have the thread block process the specified region of the MergePath decision path
     */
    __device__ __forceinline__ void ProcessRegion(
        OffsetT         block_diagonal,
        OffsetT         next_block_diagonal,
        KeyValuePair    &first_tuple,       // [Out] Valid in thread-0
        KeyValuePair    &last_tuple)        // [Out] Valid in thread-0
    {
        // Thread block initialization
        if (threadIdx.x < 2)
        {
            // Retrieve block starting and ending indices
            IndexPair block_idx = {0, 0};
            if (gridDim.x > 1)
            {
                block_idx = d_block_idx[blockIdx.x + threadIdx.x];
            }
            else if (threadIdx.x > 0)
            {
                block_idx.a_idx = num_segments;
                block_idx.b_idx = num_values;
            }

            // Share block starting and ending indices
            temp_storage.block_region_idx[threadIdx.x] = block_idx;

            // Initialize the block's running prefix
            if (threadIdx.x == 0)
            {
                prefix_op.running_total.key    = block_idx.a_idx;
                prefix_op.running_total.value  = identity;

                // Initialize the "first scattered partial reduction tuple" to the prefix tuple (in case we don't actually scatter one)
                temp_storage.first_tuple = prefix_op.running_total;
            }
        }

        // Ensure coherence of region indices
        __syncthreads();

        // Read block's starting indices
        IndexPair block_idx = temp_storage.block_region_idx[0];

        // Have the thread block iterate over the region
        #pragma unroll 1
        while (block_diagonal < next_block_diagonal)
        {
            // Read block's ending indices (hoist?)
            IndexPair next_block_idx = temp_storage.block_region_idx[1];

            // Clamp the per-thread search range to within one work-tile of block's current indices
            IndexPair next_tile_idx;
            next_tile_idx.a_idx = CUB_MIN(next_block_idx.a_idx, block_idx.a_idx + TILE_ITEMS);
            next_tile_idx.b_idx = CUB_MIN(next_block_idx.b_idx, block_idx.b_idx + TILE_ITEMS);

            // Have each thread search for the end-indices of its subranges within the segment and value inputs
            IndexPair next_thread_idx;
            if (USE_SMEM_SEGMENT_CACHE)
            {
                // Search in smem cache
                OffsetT num_segments = next_tile_idx.a_idx - block_idx.a_idx;

                // Load global
                SegmentOffset segment_offsets[ITEMS_PER_THREAD];
                LoadDirectStriped<BLOCK_THREADS>(threadIdx.x, d_segment_end_offsets + block_idx.a_idx, segment_offsets, num_segments, num_values);

                // Store to shared
                StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, temp_storage.cached_segment_end_offsets, segment_offsets);

                __syncthreads();

                OffsetT next_thread_diagonal = block_diagonal + ((threadIdx.x + 1) * ITEMS_PER_THREAD);

                MergePathSearch(
                    next_thread_diagonal,                       // Next thread diagonal
                    temp_storage.cached_segment_end_offsets - block_idx.a_idx,                      // A (segment end-offsets)
                    d_value_offsets,                            // B (value offsets)
                    block_idx,                                  // Start indices into A and B
                    next_tile_idx,                              // End indices into A and B
                    next_thread_idx);                           // [out] diagonal intersection indices into A and B
            }
            else
            {
                // Search in global

                OffsetT next_thread_diagonal = block_diagonal + ((threadIdx.x + 1) * ITEMS_PER_THREAD);

                MergePathSearch(
                    next_thread_diagonal,                       // Next thread diagonal
                    d_segment_end_offsets,                      // A (segment end-offsets)
                    d_value_offsets,                            // B (value offsets)
                    block_idx,                                  // Start indices into A and B
                    next_tile_idx,                              // End indices into A and B
                    next_thread_idx);                           // [out] diagonal intersection indices into A and B
            }

            // Share thread end-indices to get thread begin-indices and tile end-indices
            IndexPair thread_idx;

            BlockShift(temp_storage.shift).Up(
                next_thread_idx,    // Input item
                thread_idx,         // [out] Output item
                block_idx,          // Prefix item to be provided to <em>thread</em><sub>0</sub>
                next_tile_idx);     // [out] Suffix item shifted out by the <em>thread</em><sub><tt>BLOCK_THREADS-1</tt></sub> to be provided to all threads

//            if (block_idx.a_idx == next_tile_idx.a_idx)
//            {
//                // There are no segment end-offsets in this tile.  Perform a
//                // simple block-wide reduction and accumulate the result into
//                // the running total.
//                SingleSegmentTile(next_tile_idx, block_idx);
//            }
//          else if (block_idx.b_idx == next_tile_idx.b_idx)
//            {
//                // There are no values in this tile (only empty segments).
//                EmptySegmentsTile(next_tile_idx.a_idx, block_idx.a_idx);
//            }
//            else
            if ((next_tile_idx.a_idx < num_segments) && (next_tile_idx.b_idx < num_values))
            {
                // Merge the tile's segment and value indices (full tile)
                MultiSegmentTile<true>(block_idx, thread_idx, next_thread_idx, next_tile_idx);
            }
            else
            {
                // Merge the tile's segment and value indices (partially full tile)
                MultiSegmentTile<false>(block_idx, thread_idx, next_thread_idx, next_tile_idx);
            }

            // Advance the block's indices in preparation for the next tile
            block_idx = next_tile_idx;

            // Advance to the next region in the decision path
            block_diagonal += TILE_ITEMS;

            // Barrier for smem reuse
            __syncthreads();
        }

        // Get first and last tuples for the region
        if (threadIdx.x == 0)
        {
            first_tuple = temp_storage.first_tuple;
            last_tuple = prefix_op.running_total;
        }

    }


};








/******************************************************************************
 * Tuning policy types
 ******************************************************************************/

/**
 * Parameterizable tuning policy type for BlockSegReduceRegionByKey
 */
template <
    int                     _BLOCK_THREADS,             ///< Threads per thread block
    int                     _ITEMS_PER_THREAD,          ///< Items per thread (per tile of input)
    BlockLoadAlgorithm      _LOAD_ALGORITHM,            ///< The BlockLoad algorithm to use
    bool                    _LOAD_WARP_TIME_SLICING,    ///< Whether or not only one warp's worth of shared memory should be allocated and time-sliced among block-warps during any load-related data transpositions (versus each warp having its own storage)
    CacheLoadModifier       _LOAD_MODIFIER,             ///< Cache load modifier for reading input elements
    BlockScanAlgorithm      _SCAN_ALGORITHM>            ///< The BlockScan algorithm to use
struct BlockSegReduceRegionByKeyPolicy
{
    enum
    {
        BLOCK_THREADS           = _BLOCK_THREADS,               ///< Threads per thread block
        ITEMS_PER_THREAD        = _ITEMS_PER_THREAD,            ///< Items per thread (per tile of input)
        LOAD_WARP_TIME_SLICING  = _LOAD_WARP_TIME_SLICING,      ///< Whether or not only one warp's worth of shared memory should be allocated and time-sliced among block-warps during any load-related data transpositions (versus each warp having its own storage)    };
    };

    static const BlockLoadAlgorithm     LOAD_ALGORITHM          = _LOAD_ALGORITHM;      ///< The BlockLoad algorithm to use
    static const CacheLoadModifier      LOAD_MODIFIER           = _LOAD_MODIFIER;       ///< Cache load modifier for reading input elements
    static const BlockScanAlgorithm     SCAN_ALGORITHM          = _SCAN_ALGORITHM;      ///< The BlockScan algorithm to use
};


/******************************************************************************
 * Persistent thread block types
 ******************************************************************************/

/**
 * \brief BlockSegReduceRegionByKey implements a stateful abstraction of CUDA thread blocks for participating in device-wide reduce-value-by-key.
 */
template <
    typename    BlockSegReduceRegionByKeyPolicy,        ///< Parameterized BlockSegReduceRegionByKeyPolicy tuning policy
    typename    InputIteratorT,                         ///< Random-access iterator referencing key-value input tuples
    typename    OutputIteratorT,                        ///< Random-access iterator referencing segment output totals
    typename    ReductionOp>                            ///< Binary reduction operator type having member <tt>T operator()(const T &a, const T &b)</tt>
struct BlockSegReduceRegionByKey
{
    //---------------------------------------------------------------------
    // Types and constants
    //---------------------------------------------------------------------

    // Constants
    enum
    {
        BLOCK_THREADS       = BlockSegReduceRegionByKeyPolicy::BLOCK_THREADS,
        ITEMS_PER_THREAD    = BlockSegReduceRegionByKeyPolicy::ITEMS_PER_THREAD,
        TILE_ITEMS          = BLOCK_THREADS * ITEMS_PER_THREAD,
    };

    // KeyValuePair input type
    typedef typename std::iterator_traits<InputIteratorT>::value_type KeyValuePair;

    // Signed integer type for global offsets
    typedef typename KeyValuePair::Key OffsetT;

    // Value type
    typedef typename KeyValuePair::Value Value;

    // Head flag type
    typedef int HeadFlag;

    // Input iterator wrapper type for loading KeyValuePair elements through cache
    typedef CacheModifiedInputIterator<
            BlockSegReduceRegionByKeyPolicy::LOAD_MODIFIER,
            KeyValuePair,
            OffsetT>
        WrappedInputIteratorT;

    // Parameterized BlockLoad type
    typedef BlockLoad<
            WrappedInputIteratorT,
            BLOCK_THREADS,
            ITEMS_PER_THREAD,
            BlockSegReduceRegionByKeyPolicy::LOAD_ALGORITHM,
            BlockSegReduceRegionByKeyPolicy::LOAD_WARP_TIME_SLICING>
        BlockLoad;

    // BlockScan scan operator for reduction-by-segment
    typedef ReduceByKeyOp<ReductionOp> ReduceByKeyOp;

    // Stateful BlockScan prefix callback type for managing a running total while scanning consecutive tiles
    typedef RunningBlockPrefixCallbackOp<
            KeyValuePair,
            ReduceByKeyOp>
        RunningPrefixCallbackOp;

    // Parameterized BlockScan type for block-wide reduce-value-by-key
    typedef BlockScan<
            KeyValuePair,
            BLOCK_THREADS,
            BlockSegReduceRegionByKeyPolicy::SCAN_ALGORITHM>
        BlockScan;

    // Parameterized BlockDiscontinuity type for identifying key discontinuities
    typedef BlockDiscontinuity<
            OffsetT,
            BLOCK_THREADS>
        BlockDiscontinuity;

    // Operator for detecting discontinuities in a list of segment identifiers.
    struct NewSegmentOp
    {
        /// Returns true if row_b is the start of a new row
        __device__ __forceinline__ bool operator()(const OffsetT& b, const OffsetT& a)
        {
            return (a != b);
        }
    };

    // Shared memory type for this thread block
    struct _TempStorage
    {
        union
        {
            typename BlockLoad::TempStorage                 load;           // Smem needed for tile loading
            struct {
                typename BlockScan::TempStorage             scan;           // Smem needed for reduce-value-by-segment scan
                typename BlockDiscontinuity::TempStorage    discontinuity;  // Smem needed for head-flagging
            };
        };
    };

    // Alias wrapper allowing storage to be unioned
    struct TempStorage : Uninitialized<_TempStorage> {};


    //---------------------------------------------------------------------
    // Thread fields
    //---------------------------------------------------------------------

    _TempStorage                &temp_storage;          ///< Reference to shared storage
    WrappedInputIteratorT       d_tuple_partials;       ///< A sequence of partial reduction tuples to scan
    OutputIteratorT              d_output;               ///< A sequence of segment totals
    Value                       identity;               ///< Identity value (for zero-length segments)
    ReduceByKeyOp               scan_op;                ///< Reduce-by-key scan operator
    RunningPrefixCallbackOp     prefix_op;              ///< Stateful running total for block-wide prefix scan of partial reduction tuples


    //---------------------------------------------------------------------
    // Operations
    //---------------------------------------------------------------------

    /**
     * Constructor
     */
    __device__ __forceinline__
    BlockSegReduceRegionByKey(
        TempStorage             &temp_storage,          ///< Reference to shared storage
        InputIteratorT          d_tuple_partials,       ///< A sequence of partial reduction tuples to scan
        OutputIteratorT          d_output,               ///< A sequence of segment totals
        Value                   identity,               ///< Identity value (for zero-length segments)
        ReductionOp             reduction_op)           ///< Reduction operator
    :
        temp_storage(temp_storage.Alias()),
        d_tuple_partials(d_tuple_partials),
        d_output(d_output),
        identity(identity),
        scan_op(reduction_op),
        prefix_op(scan_op)
    {}



    /**
     * Processes a reduce-value-by-key input tile, outputting reductions for each segment
     */
    template <bool FULL_TILE>
    __device__ __forceinline__
    void ProcessTile(
        OffsetT block_offset,
        OffsetT first_segment_idx,
        OffsetT last_segment_idx,
        int guarded_items = TILE_ITEMS)
    {
        KeyValuePair    partial_reductions[ITEMS_PER_THREAD];
        OffsetT         segment_ids[ITEMS_PER_THREAD];
        HeadFlag        head_flags[ITEMS_PER_THREAD];

        // Load a tile of block partials from previous kernel
        if (FULL_TILE)
        {
            // Full tile
            BlockLoad(temp_storage.load).Load(d_tuple_partials + block_offset, partial_reductions);
        }
        else
        {
            KeyValuePair oob_default;
            oob_default.key    = last_segment_idx;       // The last segment ID to be reduced
            oob_default.value  = identity;

            // Partially-full tile
            BlockLoad(temp_storage.load).Load(d_tuple_partials + block_offset, partial_reductions, guarded_items, oob_default);
        }

        // Barrier for shared memory reuse
        __syncthreads();

        // Copy the segment IDs for head-flagging
        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++)
        {
            segment_ids[ITEM] = partial_reductions[ITEM].key;
        }

        // FlagT segment heads by looking for discontinuities
        BlockDiscontinuity(temp_storage.discontinuity).FlagHeads(
            head_flags,                         // [out] Head flags
            segment_ids,                        // Segment ids
            NewSegmentOp(),                     // Functor for detecting start of new rows
            prefix_op.running_total.key);       // Last segment ID from previous tile to compare with first segment ID in this tile

        // Reduce-value-by-segment across partial_reductions using exclusive prefix scan
        KeyValuePair block_aggregate;
        BlockScan(temp_storage.scan).ExclusiveScan(
            partial_reductions,                   // Scan input
            partial_reductions,                   // Scan output
            scan_op,                        // Scan operator
            block_aggregate,                // Block-wide total (unused)
            prefix_op);                     // Prefix operator for seeding the block-wide scan with the running total

        // Scatter an accumulated reduction if it is the head of a valid segment
        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ITEM++)
        {
            if (head_flags[ITEM])
            {
                d_output[partial_reductions[ITEM].key] = partial_reductions[ITEM].value;
            }
        }
    }


    /**
     * Iterate over input tiles belonging to this thread block
     */
    __device__ __forceinline__
    void ProcessRegion(
        OffsetT block_offset,
        OffsetT block_end,
        OffsetT first_segment_idx,
        OffsetT last_segment_idx)
    {
        if (threadIdx.x == 0)
        {
            // Initialize running prefix to the first segment index paired with identity
            prefix_op.running_total.key    = first_segment_idx;
            prefix_op.running_total.value  = identity;
        }

        // Process full tiles
        while (block_offset + TILE_ITEMS <= block_end)
        {
            ProcessTile<true>(block_offset, first_segment_idx, last_segment_idx);
            __syncthreads();

            block_offset += TILE_ITEMS;
        }

        // Process final value tile (if present)
        int guarded_items = block_end - block_offset;
        if (guarded_items)
        {
            ProcessTile<false>(block_offset, first_segment_idx, last_segment_idx, guarded_items);
        }
    }
};



/******************************************************************************
 * Kernel entrypoints
 ******************************************************************************/

/**
 * Segmented reduce region kernel entry point (multi-block).
 */

template <
    typename SegmentOffsetIterator,             ///< Random-access input iterator type for reading segment end-offsets
    typename OffsetT>                           ///< Signed integer type for global offsets
__global__ void SegReducePartitionKernel(
    SegmentOffsetIterator       d_segment_end_offsets,  ///< [in] A sequence of \p num_segments segment end-offsets
    IndexPair<OffsetT>          *d_block_idx,
    int                         num_partition_samples,
    OffsetT                     num_values,             ///< [in] Number of values to reduce
    OffsetT                     num_segments,           ///< [in] Number of segments being reduced
    GridEvenShare<OffsetT>      even_share)             ///< [in] Even-share descriptor for mapping an equal number of tiles onto each thread block
{
    // Segment offset type
    typedef typename std::iterator_traits<SegmentOffsetIterator>::value_type SegmentOffset;

    // Counting iterator type
    typedef CountingInputIterator<SegmentOffsetT, OffsetT> CountingIterator;

    // Cache-modified iterator for segment end-offsets
    CacheModifiedInputIterator<LOAD_LDG, SegmentOffsetT, OffsetT> d_wrapped_segment_end_offsets(d_segment_end_offsets);

    // Counting iterator for value offsets
    CountingIterator d_value_offsets(0);

    // Initialize even-share to tell us where to start and stop our tile-processing
    int partition_id = (blockDim.x * blockIdx.x) + threadIdx.x;
    even_share.Init(partition_id);

    // Search for block starting and ending indices
    IndexPair<OffsetT> start_idx = {0, 0};
    IndexPair<OffsetT> end_idx   = {num_segments, num_values};
    IndexPair<OffsetT> block_idx;

    MergePathSearch(
        even_share.block_offset,            // Next thread diagonal
        d_wrapped_segment_end_offsets,      // A (segment end-offsets)
        d_value_offsets,                    // B (value offsets)
        start_idx,                          // Start indices into A and B
        end_idx,                            // End indices into A and B
        block_idx);                         // [out] diagonal intersection indices into A and B

    // Write output
    if (partition_id < num_partition_samples)
    {
        d_block_idx[partition_id] = block_idx;
    }
}


/**
 * Segmented reduce region kernel entry point (multi-block).
 */
template <
    typename BlockSegReduceRegionPolicy,        ///< Parameterized BlockSegReduceRegionPolicy tuning policy
    typename SegmentOffsetIterator,             ///< Random-access input iterator type for reading segment end-offsets
    typename ValueIterator,                     ///< Random-access input iterator type for reading values
    typename OutputIteratorT,                   ///< Random-access output iterator type for writing segment reductions
    typename ReductionOp,                       ///< Binary reduction operator type having member <tt>T operator()(const T &a, const T &b)</tt>
    typename OffsetT,                           ///< Signed integer type for global offsets
    typename Value>                             ///< Value type
__launch_bounds__ (BlockSegReduceRegionPolicy::BLOCK_THREADS)
__global__ void SegReduceRegionKernel(
    SegmentOffsetIterator       d_segment_end_offsets,  ///< [in] A sequence of \p num_segments segment end-offsets
    ValueIterator               d_values,               ///< [in] A sequence of \p num_values values
    OutputIteratorT              d_output,               ///< [out] A sequence of \p num_segments segment totals
    KeyValuePair<OffsetT, Value> *d_tuple_partials,      ///< [out] A sequence of (gridDim.x * 2) partial reduction tuples
    IndexPair<OffsetT>          *d_block_idx,
    OffsetT                     num_values,             ///< [in] Number of values to reduce
    OffsetT                     num_segments,           ///< [in] Number of segments being reduced
    Value                       identity,               ///< [in] Identity value (for zero-length segments)
    ReductionOp                 reduction_op,           ///< [in] Reduction operator
    GridEvenShare<OffsetT>      even_share)             ///< [in] Even-share descriptor for mapping an equal number of tiles onto each thread block
{
    typedef KeyValuePair<OffsetT, Value> KeyValuePair;

    // Specialize thread block abstraction type for reducing a range of segmented values
    typedef BlockSegReduceRegion<
            BlockSegReduceRegionPolicy,
            SegmentOffsetIterator,
            ValueIterator,
            OutputIteratorT,
            ReductionOp,
            OffsetT>
        BlockSegReduceRegion;

    // Shared memory allocation
    __shared__ typename BlockSegReduceRegion::TempStorage temp_storage;

    // Initialize thread block even-share to tell us where to start and stop our tile-processing
    even_share.BlockInit();

    // Construct persistent thread block
    BlockSegReduceRegion thread_block(
        temp_storage,
        d_segment_end_offsets,
        d_values,
        d_output,
        d_block_idx,
        num_values,
        num_segments,
        identity,
        reduction_op);

    // First and last partial reduction tuples within the range (valid in thread-0)
    KeyValuePair first_tuple, last_tuple;

    // Consume block's region of work
    thread_block.ProcessRegion(
        even_share.block_offset,
        even_share.block_end,
        first_tuple,
        last_tuple);

    if (threadIdx.x == 0)
    {
        if (gridDim.x > 1)
        {
            // Special case where the first segment written and the carry-out are for the same segment
            if (first_tuple.key == last_tuple.key)
            {
                first_tuple.value = identity;
            }

            // Write the first and last partial products from this thread block so
            // that they can be subsequently "fixed up" in the next kernel.
            d_tuple_partials[blockIdx.x * 2]          = first_tuple;
            d_tuple_partials[(blockIdx.x * 2) + 1]    = last_tuple;
        }
    }

}


/**
 * Segmented reduce region kernel entry point (single-block).
 */
template <
    typename    BlockSegReduceRegionByKeyPolicy,        ///< Parameterized BlockSegReduceRegionByKeyPolicy tuning policy
    typename    InputIteratorT,                         ///< Random-access iterator referencing key-value input tuples
    typename    OutputIteratorT,                        ///< Random-access iterator referencing segment output totals
    typename    ReductionOp,                            ///< Binary reduction operator type having member <tt>T operator()(const T &a, const T &b)</tt>
    typename    OffsetT,                                ///< Signed integer type for global offsets
    typename    Value>                                  ///< Value type
__launch_bounds__ (BlockSegReduceRegionByKeyPolicy::BLOCK_THREADS, 1)
__global__ void SegReduceRegionByKeyKernel(
    InputIteratorT          d_tuple_partials,           ///< [in] A sequence of partial reduction tuples
    OutputIteratorT          d_output,                   ///< [out] A sequence of \p num_segments segment totals
    OffsetT                 num_segments,               ///< [in] Number of segments in the \p d_output sequence
    int                     num_tuple_partials,         ///< [in] Number of partial reduction tuples being reduced
    Value                   identity,                   ///< [in] Identity value (for zero-length segments)
    ReductionOp             reduction_op)               ///< [in] Reduction operator
{
    // Specialize thread block abstraction type for reducing a range of values by key
    typedef BlockSegReduceRegionByKey<
            BlockSegReduceRegionByKeyPolicy,
            InputIteratorT,
            OutputIteratorT,
            ReductionOp>
        BlockSegReduceRegionByKey;

    // Shared memory allocation
    __shared__ typename BlockSegReduceRegionByKey::TempStorage temp_storage;

    // Construct persistent thread block
    BlockSegReduceRegionByKey thread_block(
        temp_storage,
        d_tuple_partials,
        d_output,
        identity,
        reduction_op);

    // Process input tiles
    thread_block.ProcessRegion(
        0,                          // Region start
        num_tuple_partials,         // Region end
        0,                          // First segment ID
        num_segments);              // Last segment ID (one-past)
}




/******************************************************************************
 * Dispatch
 ******************************************************************************/

/**
 * Utility class for dispatching the appropriately-tuned kernels for DeviceReduce
 */
template <
    typename ValueIterator,                     ///< Random-access input iterator type for reading values
    typename SegmentOffsetIterator,             ///< Random-access input iterator type for reading segment end-offsets
    typename OutputIteratorT,                   ///< Random-access output iterator type for writing segment reductions
    typename ReductionOp,                       ///< Binary reduction operator type having member <tt>T operator()(const T &a, const T &b)</tt>
    typename OffsetT>                           ///< Signed integer type for global offsets
struct DeviceSegReduceDispatch
{
    // Value type
    typedef typename std::iterator_traits<ValueIterator>::value_type Value;

    // Reduce-by-key data type tuple (segment-ID, value)
    typedef KeyValuePair<OffsetT, Value> KeyValuePair;

    // Index pair data type
    typedef IndexPair<OffsetT>IndexPair;


    /******************************************************************************
     * Tuning policies
     ******************************************************************************/

    /// SM35
    struct Policy350
    {
        // ReduceRegionPolicy
        typedef BlockSegReduceRegionPolicy<
                128,                            ///< Threads per thread block
                6,                              ///< Items per thread (per tile of input)
                true,                           ///< Whether or not to cache incoming segment offsets in shared memory before reducing each tile
                false,                          ///< Whether or not to cache incoming values in shared memory before reducing each tile
                LOAD_DEFAULT,                   ///< Cache load modifier for reading segment offsets
                LOAD_LDG,                       ///< Cache load modifier for reading values
                BLOCK_REDUCE_RAKING,            ///< The BlockReduce algorithm to use
                BLOCK_SCAN_WARP_SCANS>          ///< The BlockScan algorithm to use
            SegReduceRegionPolicy;

        // ReduceRegionByKeyPolicy
        typedef BlockSegReduceRegionByKeyPolicy<
                256,                            ///< Threads per thread block
                9,                             ///< Items per thread (per tile of input)
                BLOCK_LOAD_DIRECT,              ///< The BlockLoad algorithm to use
                false,                          ///< Whether or not only one warp's worth of shared memory should be allocated and time-sliced among block-warps during any load-related data transpositions (versus each warp having its own storage)
                LOAD_LDG,                       ///< Cache load modifier for reading input elements
                BLOCK_SCAN_WARP_SCANS>          ///< The BlockScan algorithm to use
            SegReduceRegionByKeyPolicy;
    };


    /// SM10
    struct Policy100
    {
        // ReduceRegionPolicy
        typedef BlockSegReduceRegionPolicy<
                128,                            ///< Threads per thread block
                3,                              ///< Items per thread (per tile of input)
                false,                          ///< Whether or not to cache incoming segment offsets in shared memory before reducing each tile
                false,                          ///< Whether or not to cache incoming values in shared memory before reducing each tile
                LOAD_DEFAULT,                   ///< Cache load modifier for reading segment offsets
                LOAD_DEFAULT,                   ///< Cache load modifier for reading values
                BLOCK_REDUCE_RAKING,            ///< The BlockReduce algorithm to use
                BLOCK_SCAN_RAKING>              ///< The BlockScan algorithm to use
            SegReduceRegionPolicy;

        // ReduceRegionByKeyPolicy
        typedef BlockSegReduceRegionByKeyPolicy<
                128,                            ///< Threads per thread block
                3,                              ///< Items per thread (per tile of input)
                BLOCK_LOAD_WARP_TRANSPOSE,      ///< The BlockLoad algorithm to use
                false,                          ///< Whether or not only one warp's worth of shared memory should be allocated and time-sliced among block-warps during any load-related data transpositions (versus each warp having its own storage)
                LOAD_DEFAULT,                   ///< Cache load modifier for reading input elements
                BLOCK_SCAN_WARP_SCANS>          ///< The BlockScan algorithm to use
            SegReduceRegionByKeyPolicy;
    };


    /******************************************************************************
     * Tuning policies of current PTX compiler pass
     ******************************************************************************/

#if (HIPCUB_ARCH >= 350)
    typedef Policy350 PtxPolicy;
/*
#elif (HIPCUB_ARCH >= 300)
    typedef Policy300 PtxPolicy;

#elif (HIPCUB_ARCH >= 200)
    typedef Policy200 PtxPolicy;

#elif (HIPCUB_ARCH >= 130)
    typedef Policy130 PtxPolicy;
*/
#else
    typedef Policy100 PtxPolicy;

#endif

    // "Opaque" policies (whose parameterizations aren't reflected in the type signature)
    struct PtxSegReduceRegionPolicy           : PtxPolicy::SegReduceRegionPolicy {};
    struct PtxSegReduceRegionByKeyPolicy      : PtxPolicy::SegReduceRegionByKeyPolicy {};


    /******************************************************************************
     * Utilities
     ******************************************************************************/

    /**
     * Initialize kernel dispatch configurations with the policies corresponding to the PTX assembly we will use
     */
    template <
        typename SegReduceKernelConfig,
        typename SegReduceByKeyKernelConfig>
    __host__ __device__ __forceinline__
    static void InitConfigs(
        int                         ptx_version,
        SegReduceKernelConfig       &seg_reduce_region_config,
        SegReduceByKeyKernelConfig  &seg_reduce_region_by_key_config)
    {
    #if (HIPCUB_ARCH > 0)

        // We're on the device, so initialize the kernel dispatch configurations with the current PTX policy
        seg_reduce_region_config.Init<PtxSegReduceRegionPolicy>();
        seg_reduce_region_by_key_config.Init<PtxSegReduceRegionByKeyPolicy>();

    #else

        // We're on the host, so lookup and initialize the kernel dispatch configurations with the policies that match the device's PTX version
        if (ptx_version >= 350)
        {
            seg_reduce_region_config.template          Init<typename Policy350::SegReduceRegionPolicy>();
            seg_reduce_region_by_key_config.template   Init<typename Policy350::SegReduceRegionByKeyPolicy>();
        }
/*
        else if (ptx_version >= 300)
        {
            seg_reduce_region_config.template          Init<typename Policy300::SegReduceRegionPolicy>();
            seg_reduce_region_by_key_config.template   Init<typename Policy300::SegReduceRegionByKeyPolicy>();
        }
        else if (ptx_version >= 200)
        {
            seg_reduce_region_config.template          Init<typename Policy200::SegReduceRegionPolicy>();
            seg_reduce_region_by_key_config.template   Init<typename Policy200::SegReduceRegionByKeyPolicy>();
        }
        else if (ptx_version >= 130)
        {
            seg_reduce_region_config.template          Init<typename Policy130::SegReduceRegionPolicy>();
            seg_reduce_region_by_key_config.template   Init<typename Policy130::SegReduceRegionByKeyPolicy>();
        }
*/
        else
        {
            seg_reduce_region_config.template          Init<typename Policy100::SegReduceRegionPolicy>();
            seg_reduce_region_by_key_config.template   Init<typename Policy100::SegReduceRegionByKeyPolicy>();
        }

    #endif
    }


    /**
     * SegReduceRegionKernel kernel dispatch configuration
     */
    struct SegReduceKernelConfig
    {
        int                     block_threads;
        int                     items_per_thread;
        bool                    use_smem_segment_cache;
        bool                    use_smem_value_cache;
        CacheLoadModifier       load_modifier_segments;
        CacheLoadModifier       load_modifier_values;
        BlockReduceAlgorithm    reduce_algorithm;
        BlockScanAlgorithm      scan_algorithm;

        template <typename SegReduceRegionPolicy>
        __host__ __device__ __forceinline__
        void Init()
        {
            block_threads               = SegReduceRegionPolicy::BLOCK_THREADS;
            items_per_thread            = SegReduceRegionPolicy::ITEMS_PER_THREAD;
            use_smem_segment_cache      = SegReduceRegionPolicy::USE_SMEM_SEGMENT_CACHE;
            use_smem_value_cache        = SegReduceRegionPolicy::USE_SMEM_VALUE_CACHE;
            load_modifier_segments      = SegReduceRegionPolicy::LOAD_MODIFIER_SEGMENTS;
            load_modifier_values        = SegReduceRegionPolicy::LOAD_MODIFIER_VALUES;
            reduce_algorithm            = SegReduceRegionPolicy::REDUCE_ALGORITHM;
            scan_algorithm              = SegReduceRegionPolicy::SCAN_ALGORITHM;
        }
    };

    /**
     * SegReduceRegionByKeyKernel kernel dispatch configuration
     */
    struct SegReduceByKeyKernelConfig
    {
        int                     block_threads;
        int                     items_per_thread;
        BlockLoadAlgorithm      load_algorithm;
        bool                    load_warp_time_slicing;
        CacheLoadModifier       load_modifier;
        BlockScanAlgorithm      scan_algorithm;

        template <typename SegReduceRegionByKeyPolicy>
        __host__ __device__ __forceinline__
        void Init()
        {
            block_threads               = SegReduceRegionByKeyPolicy::BLOCK_THREADS;
            items_per_thread            = SegReduceRegionByKeyPolicy::ITEMS_PER_THREAD;
            load_algorithm              = SegReduceRegionByKeyPolicy::LOAD_ALGORITHM;
            load_warp_time_slicing      = SegReduceRegionByKeyPolicy::LOAD_WARP_TIME_SLICING;
            load_modifier               = SegReduceRegionByKeyPolicy::LOAD_MODIFIER;
            scan_algorithm              = SegReduceRegionByKeyPolicy::SCAN_ALGORITHM;
        }
    };


    /******************************************************************************
     * Dispatch entrypoints
     ******************************************************************************/

    /**
     * Internal dispatch routine for computing a device-wide segmented reduction.
     */
    template <
        typename                        SegReducePartitionKernelPtr,
        typename                        SegReduceRegionKernelPtr,               ///< Function type of hipcub::SegReduceRegionKernel
        typename                        SegReduceRegionByKeyKernelPtr>          ///< Function type of hipcub::SegReduceRegionByKeyKernel
    __host__ __device__ __forceinline__
    static hipError_t Dispatch(
        void*               d_temp_storage,                        ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
        size_t                          &temp_storage_bytes,                    ///< [in,out] Reference to size in bytes of \p d_temp_storage allocation.
        ValueIterator                   d_values,                               ///< [in] A sequence of \p num_values data to reduce
        SegmentOffsetIterator           d_segment_offsets,                      ///< [in] A sequence of (\p num_segments + 1) segment offsets
        OutputIteratorT                  d_output,                               ///< [out] A sequence of \p num_segments segment totals
        OffsetT                         num_values,                             ///< [in] Total number of values to reduce
        OffsetT                         num_segments,                           ///< [in] Number of segments being reduced
        Value                           identity,                               ///< [in] Identity value (for zero-length segments)
        ReductionOp                     reduction_op,                           ///< [in] Reduction operator
        hipStream_t                    stream,                                 ///< [in] <b>[optional]</b> CUDA stream to launch kernels within.  Default is stream<sub>0</sub>.
        bool                            debug_synchronous,                      ///< [in] <b>[optional]</b> Whether or not to synchronize the stream after every kernel launch to check for errors.  Also causes launch configurations to be printed to the console.  Default is \p false.
        int                             sm_version,                             ///< [in] SM version of target device to use when computing SM occupancy
        SegReducePartitionKernelPtr     seg_reduce_partition_kernel,            ///< [in] Kernel function pointer to parameterization of hipcub::SegReduceRegionKernel
        SegReduceRegionKernelPtr        seg_reduce_region_kernel,               ///< [in] Kernel function pointer to parameterization of hipcub::SegReduceRegionKernel
        SegReduceRegionByKeyKernelPtr   seg_reduce_region_by_key_kernel,        ///< [in] Kernel function pointer to parameterization of hipcub::SegReduceRegionByKeyKernel
        SegReduceKernelConfig           &seg_reduce_region_config,              ///< [in] Dispatch parameters that match the policy that \p seg_reduce_region_kernel was compiled for
        SegReduceByKeyKernelConfig      &seg_reduce_region_by_key_config)       ///< [in] Dispatch parameters that match the policy that \p seg_reduce_region_by_key_kernel was compiled for
    {
#ifndef CUB_RUNTIME_ENABLED

        // Kernel launch not supported from this device
        return HipcubDebug(hipErrorNotSupported );

#else

        hipError_t error = hipSuccess;
        do
        {
            // Dispatch two kernels: (1) a multi-block segmented reduction
            // to reduce regions by block, and (2) a single-block reduce-by-key kernel
            // to "fix up" segments spanning more than one region.

            // Tile size of seg_reduce_region_kernel
            int tile_size = seg_reduce_region_config.block_threads * seg_reduce_region_config.items_per_thread;

            // Get device ordinal
            int device_ordinal;
            if (HipcubDebug(error = hipGetDevice(&device_ordinal))) break;

            // Get SM count
            int sm_count;
            if (HipcubDebug(error = hipDeviceGetAttribute (&sm_count, hipDeviceAttributeMultiprocessorCount, device_ordinal))) break;

            // Get SM occupancy for histogram_region_kernel
            int seg_reduce_region_sm_occupancy;
            if (HipcubDebug(error = MaxSmOccupancy(
                seg_reduce_region_sm_occupancy,
                sm_version,
                seg_reduce_region_kernel,
                seg_reduce_region_config.block_threads))) break;

            // Get device occupancy for histogram_region_kernel
            int seg_reduce_region_occupancy = seg_reduce_region_sm_occupancy * sm_count;

            // Even-share work distribution
            int num_diagonals = num_values + num_segments;                  // Total number of work items
            int subscription_factor = seg_reduce_region_sm_occupancy;       // Amount of CTAs to oversubscribe the device beyond actively-resident (heuristic)
            int max_grid_size = seg_reduce_region_occupancy * subscription_factor;
            GridEvenShare<OffsetT>even_share(
                num_diagonals,
                max_grid_size,
                tile_size);

            // Get grid size for seg_reduce_region_kernel
            int seg_reduce_region_grid_size = even_share.grid_size;

            // Number of "fix-up" reduce-by-key tuples (2 per thread block)
            int num_tuple_partials = seg_reduce_region_grid_size * 2;
            int num_partition_samples = seg_reduce_region_grid_size + 1;

            // Temporary storage allocation requirements
            void* allocations[2] = {};
            size_t allocation_sizes[2] =
            {
                num_tuple_partials * sizeof(KeyValuePair),  // bytes needed for "fix-up" reduce-by-key tuples
                num_partition_samples * sizeof(IndexPair),  // bytes needed block indices
            };

            // Alias the temporary allocations from the single storage blob (or set the necessary size of the blob)
            if (HipcubDebug(error = AliasTemporaries(d_temp_storage, temp_storage_bytes, allocations, allocation_sizes))) break;
            if (d_temp_storage == NULL)
            {
                // Return if the caller is simply requesting the size of the storage allocation
                return hipSuccess;
            }

            // Alias the allocations
            KeyValuePair    *d_tuple_partials   = (KeyValuePair*) allocations[0];           // "fix-up" tuples
            IndexPair       *d_block_idx        = (IndexPair *) allocations[1];             // block starting/ending indices

            // Array of segment end-offsets
            SegmentOffsetIterator d_segment_end_offsets = d_segment_offsets + 1;

            // Grid launch params for seg_reduce_partition_kernel
            int partition_block_size = 32;
            int partition_grid_size = (num_partition_samples + partition_block_size - 1) / partition_block_size;

            // Partition work among multiple thread blocks if necessary
            if (seg_reduce_region_grid_size > 1)
            {
                // Log seg_reduce_partition_kernel configuration
                if (debug_synchronous) _HipcubLog("Invoking seg_reduce_partition_kernel<<<%d, %d, 0, %lld>>>()\n",
                    partition_grid_size, partition_block_size, (long long) stream);

                // Invoke seg_reduce_partition_kernel
                seg_reduce_partition_kernel<<<partition_grid_size, partition_block_size, 0, stream>>>(
                    d_segment_end_offsets,  ///< [in] A sequence of \p num_segments segment end-offsets
                    d_block_idx,
                    num_partition_samples,
                    num_values,             ///< [in] Number of values to reduce
                    num_segments,           ///< [in] Number of segments being reduced
                    even_share);            ///< [in] Even-share descriptor for mapping an equal number of tiles onto each thread block

                // Sync the stream if specified
                if (debug_synchronous && (HipcubDebug(error = SyncStream(stream)))) break;
            }

            // Log seg_reduce_region_kernel configuration
            if (debug_synchronous) _HipcubLog("Invoking seg_reduce_region_kernel<<<%d, %d, 0, %lld>>>(), %d items per thread, %d SM occupancy\n",
                seg_reduce_region_grid_size, seg_reduce_region_config.block_threads, (long long) stream, seg_reduce_region_config.items_per_thread, seg_reduce_region_sm_occupancy);

            // Mooch
            if (HipcubDebug(error = hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte))) break;

            // Invoke seg_reduce_region_kernel
            seg_reduce_region_kernel<<<seg_reduce_region_grid_size, seg_reduce_region_config.block_threads, 0, stream>>>(
                d_segment_end_offsets,
                d_values,
                d_output,
                d_tuple_partials,
                d_block_idx,
                num_values,
                num_segments,
                identity,
                reduction_op,
                even_share);

            // Sync the stream if specified
            if (debug_synchronous && (HipcubDebug(error = SyncStream(stream)))) break;
/*
            // Perform "fix-up" of region partial reductions if grid size is greater than one thread block
            if (seg_reduce_region_grid_size > 1)
            {
                // Log seg_reduce_region_by_key_kernel configuration
                if (debug_synchronous) _HipcubLog("Invoking seg_reduce_region_by_key_kernel<<<%d, %d, 0, %lld>>>(), %d items per thread\n",
                    1, seg_reduce_region_by_key_config.block_threads, (long long) stream, seg_reduce_region_by_key_config.items_per_thread);

                // Invoke seg_reduce_region_by_key_kernel
                seg_reduce_region_by_key_kernel<<<1, seg_reduce_region_by_key_config.block_threads, 0, stream>>>(
                    d_tuple_partials,
                    d_output,
                    num_segments,
                    num_tuple_partials,
                    identity,
                    reduction_op);

                // Sync the stream if specified
                if (debug_synchronous && (HipcubDebug(error = SyncStream(stream)))) break;
            }
*/
        }

        while (0);

        return error;

#endif // CUB_RUNTIME_ENABLED
    }


    /**
     * Internal dispatch routine for computing a device-wide segmented reduction.
     */
    __host__ __device__ __forceinline__
    static hipError_t Dispatch(
        void*               d_temp_storage,                        ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
        size_t                          &temp_storage_bytes,                    ///< [in,out] Reference to size in bytes of \p d_temp_storage allocation.
        ValueIterator                   d_values,                               ///< [in] A sequence of \p num_values data to reduce
        SegmentOffsetIterator           d_segment_offsets,                      ///< [in] A sequence of (\p num_segments + 1) segment offsets
        OutputIteratorT                  d_output,                               ///< [out] A sequence of \p num_segments segment totals
        OffsetT                         num_values,                             ///< [in] Total number of values to reduce
        OffsetT                         num_segments,                           ///< [in] Number of segments being reduced
        Value                           identity,                               ///< [in] Identity value (for zero-length segments)
        ReductionOp                     reduction_op,                           ///< [in] Reduction operator
        hipStream_t                    stream,                                 ///< [in] <b>[optional]</b> CUDA stream to launch kernels within.  Default is stream<sub>0</sub>.
        bool                            debug_synchronous)                      ///< [in] <b>[optional]</b> Whether or not to synchronize the stream after every kernel launch to check for errors.  Also causes launch configurations to be printed to the console.  Default is \p false.
    {
        hipError_t error = hipSuccess;
        do
        {
            // Get PTX version
            int ptx_version = 0;
    #if (HIPCUB_ARCH == 0)
            if (HipcubDebug(error = PtxVersion(ptx_version))) break;
    #else
            ptx_version = HIPCUB_ARCH;
    #endif

            // Get kernel kernel dispatch configurations
            SegReduceKernelConfig seg_reduce_region_config;
            SegReduceByKeyKernelConfig seg_reduce_region_by_key_config;

            InitConfigs(ptx_version, seg_reduce_region_config, seg_reduce_region_by_key_config);

            // Dispatch
            if (HipcubDebug(error = Dispatch(
                d_temp_storage,
                temp_storage_bytes,
                d_values,
                d_segment_offsets,
                d_output,
                num_values,
                num_segments,
                identity,
                reduction_op,
                stream,
                debug_synchronous,
                ptx_version,            // Use PTX version instead of SM version because, as a statically known quantity, this improves device-side launch dramatically but at the risk of imprecise occupancy calculation for mismatches
                SegReducePartitionKernel<SegmentOffsetIterator, OffsetT>,
                SegReduceRegionKernel<PtxSegReduceRegionPolicy, SegmentOffsetIterator, ValueIterator, OutputIteratorT, ReductionOp, OffsetT, Value>,
                SegReduceRegionByKeyKernel<PtxSegReduceRegionByKeyPolicy, KeyValuePair*, OutputIteratorT, ReductionOp, OffsetT, Value>,
                seg_reduce_region_config,
                seg_reduce_region_by_key_config))) break;
        }
        while (0);

        return error;

    }
};




/******************************************************************************
 * DeviceSegReduce
 *****************************************************************************/

/**
 * \brief DeviceSegReduce provides operations for computing a device-wide, parallel segmented reduction across a sequence of data items residing within global memory.
 * \ingroup DeviceModule
 *
 * \par Overview
 * A <a href="http://en.wikipedia.org/wiki/Reduce_(higher-order_function)"><em>reduction</em></a> (or <em>fold</em>)
 * uses a binary combining operator to compute a single aggregate from a list of input elements.
 *
 * \par Usage Considerations
 * \cdp_class{DeviceReduce}
 *
 */
struct DeviceSegReduce
{
    /**
     * \brief Computes a device-wide segmented reduction using the specified binary \p reduction_op functor.
     *
     * \par
     * Does not support non-commutative reduction operators.
     *
     * \devicestorage
     *
     * \cdp
     *
     * \iterator
     *
     * \tparam ValueIterator            <b>[inferred]</b> Random-access input iterator type for reading values
     * \tparam SegmentOffsetIterator    <b>[inferred]</b> Random-access input iterator type for reading segment end-offsets
     * \tparam OutputIteratorT           <b>[inferred]</b> Random-access output iterator type for writing segment reductions
     * \tparam Value                    <b>[inferred]</b> Value type
     * \tparam ReductionOp              <b>[inferred]</b> Binary reduction operator type having member <tt>T operator()(const T &a, const T &b)</tt>
     */
    template <
        typename                ValueIterator,
        typename                SegmentOffsetIterator,
        typename                OutputIteratorT,
        typename                Value,
        typename                ReductionOp>
    __host__ __device__ __forceinline__
    static hipError_t Reduce(
        void*               d_temp_storage,                        ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
        size_t                  &temp_storage_bytes,                    ///< [in,out] Reference to size in bytes of \p d_temp_storage allocation.
        ValueIterator           d_values,                               ///< [in] A sequence of \p num_values data to reduce
        SegmentOffsetIterator   d_segment_offsets,                      ///< [in] A sequence of (\p num_segments + 1) segment offsets
        OutputIteratorT          d_output,                               ///< [out] A sequence of \p num_segments segment totals
        int                     num_values,                             ///< [in] Total number of values to reduce
        int                     num_segments,                           ///< [in] Number of segments being reduced
        Value                   identity,                               ///< [in] Identity value (for zero-length segments)
        ReductionOp             reduction_op,                           ///< [in] Reduction operator
        hipStream_t            stream              = 0,                ///< [in] <b>[optional]</b> CUDA stream to launch kernels within.  Default is stream<sub>0</sub>.
        bool                    debug_synchronous   = false)            ///< [in] <b>[optional]</b> Whether or not to synchronize the stream after every kernel launch to check for errors.  Also causes launch configurations to be printed to the console.  Default is \p false.
    {
        // Signed integer type for global offsets
        typedef int OffsetT;

        typedef DeviceSegReduceDispatch<
                ValueIterator,
                SegmentOffsetIterator,
                OutputIteratorT,
                ReductionOp,
                OffsetT>
            DeviceSegReduceDispatch;

        return DeviceSegReduceDispatch::Dispatch(
            d_temp_storage,
            temp_storage_bytes,
            d_values,
            d_segment_offsets,
            d_output,
            num_values,
            num_segments,
            identity,
            reduction_op,
            stream,
            debug_synchronous);
    }


    /**
     * \brief Computes a device-wide segmented sum using the addition ('+') operator.
     *
     * \par
     * Does not support non-commutative summation.
     *
     * \devicestorage
     *
     * \cdp
     *
     * \iterator
     *
     * \tparam ValueIterator            <b>[inferred]</b> Random-access input iterator type for reading values
     * \tparam SegmentOffsetIterator    <b>[inferred]</b> Random-access input iterator type for reading segment end-offsets
     * \tparam OutputIteratorT           <b>[inferred]</b> Random-access output iterator type for writing segment reductions
     */
    template <
        typename                ValueIterator,
        typename                SegmentOffsetIterator,
        typename                OutputIteratorT>
    __host__ __device__ __forceinline__
    static hipError_t Sum(
        void*               d_temp_storage,                        ///< [in] %Device allocation of temporary storage.  When NULL, the required allocation size is returned in \p temp_storage_bytes and no work is done.
        size_t                  &temp_storage_bytes,                    ///< [in,out] Reference to size in bytes of \p d_temp_storage allocation.
        ValueIterator           d_values,                               ///< [in] A sequence of \p num_values data to reduce
        SegmentOffsetIterator   d_segment_offsets,                      ///< [in] A sequence of (\p num_segments + 1) segment offsets
        OutputIteratorT          d_output,                               ///< [out] A sequence of \p num_segments segment totals
        int                     num_values,                             ///< [in] Total number of values to reduce
        int                     num_segments,                           ///< [in] Number of segments being reduced
        hipStream_t            stream              = 0,                ///< [in] <b>[optional]</b> CUDA stream to launch kernels within.  Default is stream<sub>0</sub>.
        bool                    debug_synchronous   = false)            ///< [in] <b>[optional]</b> Whether or not to synchronize the stream after every kernel launch to check for errors.  Also causes launch configurations to be printed to the console.  Default is \p false.
    {
        // Signed integer type for global offsets
        typedef int OffsetT;

        // Value type
        typedef typename std::iterator_traits<ValueIterator>::value_type Value;

        Value identity = Value();
        hipcub::Sum reduction_op;

        typedef DeviceSegReduceDispatch<
                ValueIterator,
                SegmentOffsetIterator,
                OutputIteratorT,
                hipcub::Sum,
                OffsetT>
            DeviceSegReduceDispatch;

        return DeviceSegReduceDispatch::Dispatch(
            d_temp_storage,
            temp_storage_bytes,
            d_values,
            d_segment_offsets,
            d_output,
            num_values,
            num_segments,
            identity,
            reduction_op,
            stream,
            debug_synchronous);
    }
};




//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------

/**
 * Initialize problem
 */
template <typename OffsetT, typename Value>
void Initialize(
    GenMode         gen_mode,
    Value           *h_values,
    vector<OffsetT> &segment_offsets,
    int             num_values,
    int             avg_segment_size)
{
    // Initialize values
//    if (g_verbose) printf("Values: ");
    for (int i = 0; i < num_values; ++i)
    {
        InitValue(gen_mode, h_values[i], i);
//        if (g_verbose) std::cout << h_values[i] << ", ";
    }
//    if (g_verbose) printf("\n\n");

    // Initialize segment lengths
    const unsigned int  MAX_INTEGER         = -1u;
    const unsigned int  MAX_SEGMENT_LENGTH  = avg_segment_size * 2;
    const double        SCALE_FACTOR        = double(MAX_SEGMENT_LENGTH) / double(MAX_INTEGER);

    segment_offsets.push_back(0);

    OffsetT consumed = 0;
    OffsetT remaining = num_values;
    while (remaining > 0)
    {
        // Randomly sample a 32-bit unsigned int
        unsigned int segment_length;
        RandomBits(segment_length);

        // Scale to maximum segment length
        segment_length = (unsigned int) (double(segment_length) * SCALE_FACTOR);
        segment_length = CUB_MIN(segment_length, remaining);

        consumed += segment_length;
        remaining -= segment_length;

        segment_offsets.push_back(consumed);
    }
}


/**
 * Compute reference answer
 */
template <typename OffsetT, typename Value>
void ComputeReference(
    Value       *h_values,
    OffsetT     *h_segment_offsets,
    Value       *h_reference,
    int         num_segments,
    Value       identity)
{
    if (g_verbose) printf("%d segment reductions: ", num_segments);
    for (int segment = 0; segment < num_segments; ++segment)
    {
        h_reference[segment] = identity;

        for (int i = h_segment_offsets[segment]; i < h_segment_offsets[segment + 1]; ++i)
        {
            h_reference[segment] += h_values[i];
        }
        if (g_verbose) std::cout << h_reference[segment] << ", ";
    }
    if (g_verbose) printf("\n\n");
}


/**
 * Simple test of device
 */
template <
    bool            CDP,
    typename        OffsetT,
    typename        Value,
    typename        ReductionOp>
void Test(
    OffsetT         num_values,
    int             avg_segment_size,
    ReductionOp     reduction_op,
    Value           identity,
    char*           type_string)
{
    Value   *h_values = NULL;
    Value   *h_reference = NULL;
    OffsetT *h_segment_offsets = NULL;

    printf("%d\n", num_values);

    // Initialize problem on host
    h_values = new Value[num_values];
    vector<OffsetT> segment_offsets;
    Initialize(UNIFORM, h_values, segment_offsets, num_values, avg_segment_size);

    // Allocate simple offsets array and copy STL vector into it
    h_segment_offsets = new OffsetT[segment_offsets.size()];
    for (int i = 0; i < segment_offsets.size(); ++i)
        h_segment_offsets[i] = segment_offsets[i];

    OffsetT num_segments = segment_offsets.size() - 1;
    if (g_verbose)
    {
        printf("%d segment offsets: ", num_segments);
        for (int i = 0; i < num_segments; ++i)
            std::cout << h_segment_offsets[i] << "(" << h_segment_offsets[i + 1] - h_segment_offsets[i] << "), ";
        if (g_verbose) std::cout << std::endl << std::endl;
    }

    // Solve problem on host
    h_reference = new Value[num_segments];
    ComputeReference(h_values, h_segment_offsets, h_reference, num_segments, identity);

    printf("\n\n%s hipcub::DeviceSegReduce::%s %d items (%d-byte %s), %d segments (%d-byte offset indices)\n",
        (CDP) ? "CDP device invoked" : "Host-invoked",
        (Equals<ReductionOp, Sum>::VALUE) ? "Sum" : "Reduce",
        num_values, (int) sizeof(Value), type_string,
        num_segments, (int) sizeof(OffsetT));
    fflush(stdout);

    // Allocate and initialize problem on device
    Value   *d_values = NULL;
    OffsetT *d_segment_offsets = NULL;
    Value   *d_output = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_values, sizeof(Value) * num_values));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_segment_offsets, sizeof(OffsetT) * (num_segments + 1)));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_output, sizeof(Value) * num_segments));
    CubDebugExit(hipMemcpy(d_values, h_values, sizeof(Value) * num_values, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(d_segment_offsets, h_segment_offsets, sizeof(OffsetT) * (num_segments + 1), hipMemcpyHostToDevice));

    // Request and allocate temporary storage
    void    *d_temp_storage = NULL;
    size_t  temp_storage_bytes = 0;
    CubDebugExit(DeviceSegReduce::Sum(d_temp_storage, temp_storage_bytes, d_values, d_segment_offsets, d_output, num_values, num_segments, 0, false));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Clear device output
    CubDebugExit(hipMemset(d_output, 0, sizeof(Value) * num_segments));

    // Run warmup/correctness iteration
    CubDebugExit(DeviceSegReduce::Sum(d_temp_storage, temp_storage_bytes, d_values, d_segment_offsets, d_output, num_values, num_segments, 0, true));

    // Check for correctness (and display results, if specified)
    int compare = CompareDeviceResults(h_reference, d_output, num_segments, true, g_verbose);
    printf("\t%s", compare ? "FAIL" : "PASS");

    // Flush any stdout/stderr
    fflush(stdout);
    fflush(stderr);

    // Performance
    GpuTimer gpu_timer;
    gpu_timer.Start();
    for (int i = 0; i < g_timing_iterations; ++i)
    {
        CubDebugExit(DeviceSegReduce::Sum(d_temp_storage, temp_storage_bytes, d_values, d_segment_offsets, d_output, num_values, num_segments, 0, false));
    }
    gpu_timer.Stop();
    float elapsed_millis = gpu_timer.ElapsedMillis();

    // Display performance
    if (g_timing_iterations > 0)
    {
        float avg_millis = elapsed_millis / g_timing_iterations;
        float giga_rate = float(num_values) / avg_millis / 1000.0 / 1000.0;
        float giga_bandwidth = giga_rate *
        printf(", %.3f avg ms, %.3f billion items/s, %.3f logical GB/s", avg_millis, giga_rate, giga_bandwidth);
    }

    // Device cleanup
    if (d_values) CubDebugExit(g_allocator.DeviceFree(d_values));
    if (d_segment_offsets) CubDebugExit(g_allocator.DeviceFree(d_segment_offsets));
    if (d_output) CubDebugExit(g_allocator.DeviceFree(d_output));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

    // Host cleanup
    if (h_values)           delete[] h_values;
    if (h_segment_offsets)  delete[] h_segment_offsets;
    if (h_reference)        delete[] h_reference;
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    int num_values          = 32 * 1024 * 1024;
    int avg_segment_size    = 500;

    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("n", num_values);
    args.GetCmdLineArgument("ss", avg_segment_size);
    args.GetCmdLineArgument("i", g_timing_iterations);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--v] "
            "[--i=<timing iterations>] "
            "[--n=<input samples>]\n"
            "[--ss=<average segment size>]\n"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    Test<false>((int) num_values, avg_segment_size, Sum(), (long long) 0, CUB_TYPE_STRING(long long));

    return 0;
}



