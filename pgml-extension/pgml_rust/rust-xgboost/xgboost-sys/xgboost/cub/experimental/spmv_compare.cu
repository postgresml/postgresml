#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIAeBILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

//---------------------------------------------------------------------
// SpMV comparison tool
//---------------------------------------------------------------------

#include <stdio.h>
#include <map>
#include <vector>
#include <algorithm>
#include <cstdio>
#include <fstream>

#include <hipsparse.h>

#include "sparse_matrix.h"

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <cub/device/device_spmv.cuh>
#include <cub/util_allocator.cuh>
#include <cub/iterator/tex_ref_input_iterator.cuh>
#include <test/test_util.h>

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants, and type declarations
//---------------------------------------------------------------------

bool                    g_quiet     = false;        // Whether to display stats in CSV format
bool                    g_verbose   = false;        // Whether to display output to console
bool                    g_verbose2  = false;        // Whether to display input to console
CachingDeviceAllocator  g_allocator(true);          // Caching allocator for device memory


//---------------------------------------------------------------------
// SpMV verification
//---------------------------------------------------------------------

// Compute reference SpMV y = Ax
template <
    typename ValueT,
    typename OffsetT>
void SpmvGold(
    CsrMatrix<ValueT, OffsetT>&     a,
    ValueT*                         vector_x,
    ValueT*                         vector_y_in,
    ValueT*                         vector_y_out,
    ValueT                          alpha,
    ValueT                          beta)
{
    for (OffsetT row = 0; row < a.num_rows; ++row)
    {
        ValueT partial = beta * vector_y_in[row];
        for (
            OffsetT offset = a.row_offsets[row];
            offset < a.row_offsets[row + 1];
            ++offset)
        {
            partial += alpha * a.values[offset] * vector_x[a.column_indices[offset]];
        }
        vector_y_out[row] = partial;
    }
}


//---------------------------------------------------------------------
// GPU I/O proxy
//---------------------------------------------------------------------

/**
 * Read every matrix nonzero value, read every corresponding vector value
 */
template <
    int         BLOCK_THREADS,
    int         ITEMS_PER_THREAD,
    typename    ValueT,
    typename    OffsetT,
    typename    VectorItr>
__launch_bounds__ (int(BLOCK_THREADS))
__global__ void NonZeroIoKernel(
    SpmvParams<ValueT, OffsetT> params,
    VectorItr                   d_vector_x)
{
    enum
    {
        TILE_ITEMS      = BLOCK_THREADS * ITEMS_PER_THREAD,
    };


    ValueT nonzero = 0.0;

    int tile_idx = blockIdx.x;

    OffsetT block_offset = tile_idx * TILE_ITEMS;

    OffsetT column_indices[ITEMS_PER_THREAD];
    ValueT values[ITEMS_PER_THREAD];

    #pragma unroll
    for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM)
    {
        OffsetT nonzero_idx = block_offset + (ITEM * BLOCK_THREADS) + threadIdx.x;

        OffsetT* ci = params.d_column_indices + nonzero_idx;
        ValueT*a = params.d_values + nonzero_idx;

        column_indices[ITEM]    = (nonzero_idx < params.num_nonzeros) ? *ci : 0;
        values[ITEM]            = (nonzero_idx < params.num_nonzeros) ? *a : 0.0;
    }

    __syncthreads();

    // Read vector
    #pragma unroll
    for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM)
    {
        ValueT vector_value    = ThreadLoad<LOAD_LDG>(params.d_vector_x + column_indices[ITEM]);
        nonzero                += vector_value * values[ITEM];
    }

    __syncthreads();

    if (block_offset < params.num_rows)
    {
        #pragma unroll
        for (int ITEM = 0; ITEM < ITEMS_PER_THREAD; ++ITEM)
        {
            OffsetT row_idx = block_offset + (ITEM * BLOCK_THREADS) + threadIdx.x;
            if (row_idx < params.num_rows)
            {
                OffsetT row_end_offset = ThreadLoad<LOAD_DEFAULT>(params.d_row_end_offsets + row_idx);

                if ((row_end_offset >= 0) && (nonzero == nonzero))
                    params.d_vector_y[row_idx] = nonzero;
            }
        }
    }

}


/**
 * Run GPU I/O proxy
 */
template <
    typename ValueT,
    typename OffsetT>
float TestGpuCsrIoProxy(
    SpmvParams<ValueT, OffsetT>&    params,
    int                             timing_iterations)
{
    enum {
        BLOCK_THREADS       = 128,
        ITEMS_PER_THREAD    = 7,
        TILE_SIZE           = BLOCK_THREADS * ITEMS_PER_THREAD,
    };

//    size_t smem = 1024 * 16;
    size_t smem = 1024 * 0;

    unsigned int nonzero_blocks = (params.num_nonzeros + TILE_SIZE - 1) / TILE_SIZE;
    unsigned int row_blocks = (params.num_rows + TILE_SIZE - 1) / TILE_SIZE;
    unsigned int blocks = std::max(nonzero_blocks, row_blocks);

    typedef TexRefInputIterator<ValueT, 1234, int> TexItr;
    TexItr x_itr;
    CubDebugExit(x_itr.BindTexture(params.d_vector_x));

    // Get device ordinal
    int device_ordinal;
    CubDebugExit(hipGetDevice(&device_ordinal));

    // Get device SM version
    int sm_version;
    CubDebugExit(SmVersion(sm_version, device_ordinal));

    void (*kernel)(SpmvParams<ValueT, OffsetT>, TexItr) = NonZeroIoKernel<BLOCK_THREADS, ITEMS_PER_THREAD>;


    int spmv_sm_occupancy;
    CubDebugExit(MaxSmOccupancy(spmv_sm_occupancy, kernel, BLOCK_THREADS, smem));

    if (!g_quiet)
        printf("NonZeroIoKernel<%d,%d><<<%d, %d>>>, sm occupancy %d\n", BLOCK_THREADS, ITEMS_PER_THREAD, blocks, BLOCK_THREADS, spmv_sm_occupancy);

    // Warmup
    NonZeroIoKernel<BLOCK_THREADS, ITEMS_PER_THREAD><<<blocks, BLOCK_THREADS, smem>>>(params, x_itr);

    // Check for failures
    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(SyncStream(0));

    // Timing
    GpuTimer timer;
    float elapsed_millis = 0.0;
    timer.Start();
    for (int it = 0; it < timing_iterations; ++it)
    {
        NonZeroIoKernel<BLOCK_THREADS, ITEMS_PER_THREAD><<<blocks, BLOCK_THREADS, smem>>>(params, x_itr);
    }
    timer.Stop();
    elapsed_millis += timer.ElapsedMillis();

    CubDebugExit(x_itr.UnbindTexture());

    return elapsed_millis / timing_iterations;
}



//---------------------------------------------------------------------
// cuSparse HybMV
//---------------------------------------------------------------------

/**
 * Run cuSparse HYB SpMV (specialized for fp32)
 */
template <
    typename OffsetT>
float TestCusparseHybmv(
    float*                          vector_y_in,
    float*                          reference_vector_y_out,
    SpmvParams<float, OffsetT>&     params,
    int                             timing_iterations,
    hipsparseHandle_t                cusparse)
{
    CpuTimer cpu_timer;
    cpu_timer.Start();

    // Construct Hyb matrix
    hipsparseMatDescr_t mat_desc;
    hipsparseHybMat_t hyb_desc;
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseCreateMatDescr(&mat_desc));
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseCreateHybMat(&hyb_desc));
    hipsparseStatus_t status = hipsparseScsr2hyb(
        cusparse,
        params.num_rows, params.num_cols,
        mat_desc,
        params.d_values, params.d_row_end_offsets, params.d_column_indices,
        hyb_desc,
        0,
        HIPSPARSE_HYB_PARTITION_AUTO);
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, status);

    hipDeviceSynchronize();
    cpu_timer.Stop();
    float elapsed_millis = cpu_timer.ElapsedMillis();
    printf("HYB setup ms, %.5f, ", elapsed_millis);

    // Reset input/output vector y
    CubDebugExit(hipMemcpy(params.d_vector_y, vector_y_in, sizeof(float) * params.num_rows, hipMemcpyHostToDevice));

    // Warmup
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseShybmv(
        cusparse,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &params.alpha, mat_desc,
        hyb_desc,
        params.d_vector_x, &params.beta, params.d_vector_y));

    if (!g_quiet)
    {
        int compare = CompareDeviceResults(reference_vector_y_out, params.d_vector_y, params.num_rows, true, g_verbose);
        printf("\t%s\n", compare ? "FAIL" : "PASS"); fflush(stdout);
    }

    // Timing
    elapsed_millis    = 0.0;
    GpuTimer timer;

    timer.Start();
    for(int it = 0; it < timing_iterations; ++it)
    {
        AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseShybmv(
            cusparse,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &params.alpha, mat_desc,
            hyb_desc,
            params.d_vector_x, &params.beta, params.d_vector_y));
    }
    timer.Stop();
    elapsed_millis += timer.ElapsedMillis();

    // Cleanup
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseDestroyHybMat(hyb_desc));
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseDestroyMatDescr(mat_desc));

    return elapsed_millis / timing_iterations;
}


/**
 * Run cuSparse HYB SpMV (specialized for fp64)
 */
template <
    typename OffsetT>
float TestCusparseHybmv(
    double*                         vector_y_in,
    double*                         reference_vector_y_out,
    SpmvParams<double, OffsetT>&    params,
    int                             timing_iterations,
    hipsparseHandle_t                cusparse)
{
    CpuTimer cpu_timer;
    cpu_timer.Start();

    // Construct Hyb matrix
    hipsparseMatDescr_t mat_desc;
    hipsparseHybMat_t hyb_desc;
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseCreateMatDescr(&mat_desc));
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseCreateHybMat(&hyb_desc));
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseDcsr2hyb(
        cusparse,
        params.num_rows, params.num_cols,
        mat_desc,
        params.d_values, params.d_row_end_offsets, params.d_column_indices,
        hyb_desc,
        0,
        HIPSPARSE_HYB_PARTITION_AUTO));

    hipDeviceSynchronize();
    cpu_timer.Stop();
    float elapsed_millis = cpu_timer.ElapsedMillis();
    printf("HYB setup ms, %.5f, ", elapsed_millis);

    // Reset input/output vector y
    CubDebugExit(hipMemcpy(params.d_vector_y, vector_y_in, sizeof(float) * params.num_rows, hipMemcpyHostToDevice));

    // Warmup
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseDhybmv(
        cusparse,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &params.alpha, mat_desc,
        hyb_desc,
        params.d_vector_x, &params.beta, params.d_vector_y));

    if (!g_quiet)
    {
        int compare = CompareDeviceResults(reference_vector_y_out, params.d_vector_y, params.num_rows, true, g_verbose);
        printf("\t%s\n", compare ? "FAIL" : "PASS"); fflush(stdout);
    }

    // Timing
    elapsed_millis    = 0.0;
    GpuTimer timer;

    timer.Start();
    for(int it = 0; it < timing_iterations; ++it)
    {
        AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseDhybmv(
            cusparse,
            HIPSPARSE_OPERATION_NON_TRANSPOSE,
            &params.alpha, mat_desc,
            hyb_desc,
            params.d_vector_x, &params.beta, params.d_vector_y));
    }
    timer.Stop();
    elapsed_millis += timer.ElapsedMillis();

    // Cleanup
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseDestroyHybMat(hyb_desc));
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseDestroyMatDescr(mat_desc));

    return elapsed_millis / timing_iterations;
}



//---------------------------------------------------------------------
// cuSparse CsrMV
//---------------------------------------------------------------------

/**
 * Run cuSparse SpMV (specialized for fp32)
 */
template <
    typename OffsetT>
float TestCusparseCsrmv(
    float*                          vector_y_in,
    float*                          reference_vector_y_out,
    SpmvParams<float, OffsetT>&     params,
    int                             timing_iterations,
    hipsparseHandle_t                cusparse)
{
    hipsparseMatDescr_t desc;
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseCreateMatDescr(&desc));

    // Reset input/output vector y
    CubDebugExit(hipMemcpy(params.d_vector_y, vector_y_in, sizeof(float) * params.num_rows, hipMemcpyHostToDevice));

    // Warmup
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseScsrmv(
        cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        params.num_rows, params.num_cols, params.num_nonzeros, &params.alpha, desc,
        params.d_values, params.d_row_end_offsets, params.d_column_indices,
        params.d_vector_x, &params.beta, params.d_vector_y));

    if (!g_quiet)
    {
        int compare = CompareDeviceResults(reference_vector_y_out, params.d_vector_y, params.num_rows, true, g_verbose);
        printf("\t%s\n", compare ? "FAIL" : "PASS"); fflush(stdout);
    }

    // Timing
    float elapsed_millis    = 0.0;
    GpuTimer timer;

    timer.Start();
    for(int it = 0; it < timing_iterations; ++it)
    {
        AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseScsrmv(
            cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            params.num_rows, params.num_cols, params.num_nonzeros, &params.alpha, desc,
            params.d_values, params.d_row_end_offsets, params.d_column_indices,
            params.d_vector_x, &params.beta, params.d_vector_y));
    }
    timer.Stop();
    elapsed_millis += timer.ElapsedMillis();

    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseDestroyMatDescr(desc));
    return elapsed_millis / timing_iterations;
}


/**
 * Run cuSparse SpMV (specialized for fp64)
 */
template <
    typename OffsetT>
float TestCusparseCsrmv(
    double*                         vector_y_in,
    double*                         reference_vector_y_out,
    SpmvParams<double, OffsetT>&    params,
    int                             timing_iterations,
    hipsparseHandle_t                cusparse)
{
    hipsparseMatDescr_t desc;
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseCreateMatDescr(&desc));

    // Reset input/output vector y
    CubDebugExit(hipMemcpy(params.d_vector_y, vector_y_in, sizeof(float) * params.num_rows, hipMemcpyHostToDevice));

    // Warmup
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseDcsrmv(
        cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
        params.num_rows, params.num_cols, params.num_nonzeros, &params.alpha, desc,
        params.d_values, params.d_row_end_offsets, params.d_column_indices,
        params.d_vector_x, &params.beta, params.d_vector_y));

    if (!g_quiet)
    {
        int compare = CompareDeviceResults(reference_vector_y_out, params.d_vector_y, params.num_rows, true, g_verbose);
        printf("\t%s\n", compare ? "FAIL" : "PASS"); fflush(stdout);
    }

    // Timing
    float elapsed_millis = 0.0;
    GpuTimer timer;
    timer.Start();
    for(int it = 0; it < timing_iterations; ++it)
    {
        AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseDcsrmv(
            cusparse, HIPSPARSE_OPERATION_NON_TRANSPOSE,
            params.num_rows, params.num_cols, params.num_nonzeros, &params.alpha, desc,
            params.d_values, params.d_row_end_offsets, params.d_column_indices,
            params.d_vector_x, &params.beta, params.d_vector_y));

    }
    timer.Stop();
    elapsed_millis += timer.ElapsedMillis();

    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseDestroyMatDescr(desc));
    return elapsed_millis / timing_iterations;
}

//---------------------------------------------------------------------
// GPU Merge-based SpMV
//---------------------------------------------------------------------

/**
 * Run CUB SpMV
 */
template <
    typename ValueT,
    typename OffsetT>
float TestGpuMergeCsrmv(
    ValueT*                         vector_y_in,
    ValueT*                         reference_vector_y_out,
    SpmvParams<ValueT, OffsetT>&    params,
    int                             timing_iterations)
{
    // Allocate temporary storage
    size_t temp_storage_bytes = 0;
    void *d_temp_storage = NULL;

    // Get amount of temporary storage needed
    CubDebugExit(DeviceSpmv::CsrMV(
        d_temp_storage, temp_storage_bytes,
        params.d_values, params.d_row_end_offsets, params.d_column_indices,
        params.d_vector_x, params.d_vector_y,
        params.num_rows, params.num_cols, params.num_nonzeros,
// params.alpha, params.beta,
        (hipStream_t) 0, false));

    // Allocate
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Reset input/output vector y
    CubDebugExit(hipMemcpy(params.d_vector_y, vector_y_in, sizeof(ValueT) * params.num_rows, hipMemcpyHostToDevice));

    // Warmup
    CubDebugExit(DeviceSpmv::CsrMV(
        d_temp_storage, temp_storage_bytes,
        params.d_values, params.d_row_end_offsets, params.d_column_indices,
        params.d_vector_x, params.d_vector_y,
        params.num_rows, params.num_cols, params.num_nonzeros, 
// params.alpha, params.beta,
        (hipStream_t) 0, !g_quiet));

    if (!g_quiet)
    {
        int compare = CompareDeviceResults(reference_vector_y_out, params.d_vector_y, params.num_rows, true, g_verbose);
        printf("\t%s\n", compare ? "FAIL" : "PASS"); fflush(stdout);
    }

    // Timing
    GpuTimer timer;
    float elapsed_millis = 0.0;

    timer.Start();
    for(int it = 0; it < timing_iterations; ++it)
    {
        CubDebugExit(DeviceSpmv::CsrMV(
            d_temp_storage, temp_storage_bytes,
            params.d_values, params.d_row_end_offsets, params.d_column_indices,
            params.d_vector_x, params.d_vector_y,
            params.num_rows, params.num_cols, params.num_nonzeros, 
// params.alpha, params.beta,
            (hipStream_t) 0, false));
    }
    timer.Stop();
    elapsed_millis += timer.ElapsedMillis();

    return elapsed_millis / timing_iterations;
}

//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------

/**
 * Display perf
 */
template <typename ValueT, typename OffsetT>
void DisplayPerf(
    float                           device_giga_bandwidth,
    double                          avg_millis,
    CsrMatrix<ValueT, OffsetT>&     csr_matrix)
{
    double nz_throughput, effective_bandwidth;
    size_t total_bytes = (csr_matrix.num_nonzeros * (sizeof(ValueT) * 2 + sizeof(OffsetT))) +
        (csr_matrix.num_rows) * (sizeof(OffsetT) + sizeof(ValueT));

    nz_throughput       = double(csr_matrix.num_nonzeros) / avg_millis / 1.0e6;
    effective_bandwidth = double(total_bytes) / avg_millis / 1.0e6;

    if (!g_quiet)
        printf("fp%d: %.4f avg ms, %.5f gflops, %.3lf effective GB/s (%.2f%% peak)\n",
            sizeof(ValueT) * 8,
            avg_millis,
            2 * nz_throughput,
            effective_bandwidth,
            effective_bandwidth / device_giga_bandwidth * 100);
    else
        printf("%.5f, %.6f, %.3lf, %.2f%%, ",
            avg_millis,
            2 * nz_throughput,
            effective_bandwidth,
            effective_bandwidth / device_giga_bandwidth * 100);

    fflush(stdout);
}



/**
 * Run tests
 */
template <
    typename ValueT,
    typename OffsetT>
void RunTest(
    bool                        rcm_relabel,
    ValueT                      alpha,
    ValueT                      beta,
    CooMatrix<ValueT, OffsetT>& coo_matrix,
    int                         timing_iterations,
    CommandLineArgs&            args)
{
    // Adaptive timing iterations: run 16 billion nonzeros through
    if (timing_iterations == -1)
        timing_iterations = std::min(50000ull, std::max(100ull, ((16ull << 30) / coo_matrix.num_nonzeros)));

    if (!g_quiet)
        printf("\t%d timing iterations\n", timing_iterations);

    // Convert to CSR
    CsrMatrix<ValueT, OffsetT> csr_matrix;
    csr_matrix.FromCoo(coo_matrix);
    if (!args.CheckCmdLineFlag("csrmv"))
        coo_matrix.Clear();

    // Relabel
    if (rcm_relabel)
    {
        if (!g_quiet)
        {
            csr_matrix.Stats().Display();
            printf("\n");
            csr_matrix.DisplayHistogram();
            printf("\n");
            if (g_verbose2)
                csr_matrix.Display();
            printf("\n");
        }

        RcmRelabel(csr_matrix, !g_quiet);

        if (!g_quiet) printf("\n");
    }

    // Display matrix info
    csr_matrix.Stats().Display(!g_quiet);
    if (!g_quiet)
    {
        printf("\n");
        csr_matrix.DisplayHistogram();
        printf("\n");
        if (g_verbose2)
            csr_matrix.Display();
        printf("\n");
    }
    fflush(stdout);

    // Allocate input and output vectors
    ValueT* vector_x        = new ValueT[csr_matrix.num_cols];
    ValueT* vector_y_in     = new ValueT[csr_matrix.num_rows];
    ValueT* vector_y_out    = new ValueT[csr_matrix.num_rows];

    for (int col = 0; col < csr_matrix.num_cols; ++col)
        vector_x[col] = 1.0;

    for (int row = 0; row < csr_matrix.num_rows; ++row)
        vector_y_in[row] = 1.0;

    // Compute reference answer
    SpmvGold(csr_matrix, vector_x, vector_y_in, vector_y_out, alpha, beta);

    float avg_millis;

    if (g_quiet) {
        printf("%s, %s, ", args.deviceProp.name, (sizeof(ValueT) > 4) ? "fp64" : "fp32"); fflush(stdout);
    }

    // Get GPU device bandwidth (GB/s)
    float device_giga_bandwidth = args.device_giga_bandwidth;

    // Allocate and initialize GPU problem
    SpmvParams<ValueT, OffsetT> params;

    CubDebugExit(g_allocator.DeviceAllocate((void **) &params.d_values,          sizeof(ValueT) * csr_matrix.num_nonzeros));
    CubDebugExit(g_allocator.DeviceAllocate((void **) &params.d_row_end_offsets, sizeof(OffsetT) * (csr_matrix.num_rows + 1)));
    CubDebugExit(g_allocator.DeviceAllocate((void **) &params.d_column_indices,  sizeof(OffsetT) * csr_matrix.num_nonzeros));
    CubDebugExit(g_allocator.DeviceAllocate((void **) &params.d_vector_x,        sizeof(ValueT) * csr_matrix.num_cols));
    CubDebugExit(g_allocator.DeviceAllocate((void **) &params.d_vector_y,        sizeof(ValueT) * csr_matrix.num_rows));
    params.num_rows         = csr_matrix.num_rows;
    params.num_cols         = csr_matrix.num_cols;
    params.num_nonzeros     = csr_matrix.num_nonzeros;
    params.alpha            = alpha;
    params.beta             = beta;

    CubDebugExit(hipMemcpy(params.d_values,            csr_matrix.values,          sizeof(ValueT) * csr_matrix.num_nonzeros, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(params.d_row_end_offsets,   csr_matrix.row_offsets,     sizeof(OffsetT) * (csr_matrix.num_rows + 1), hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(params.d_column_indices,    csr_matrix.column_indices,  sizeof(OffsetT) * csr_matrix.num_nonzeros, hipMemcpyHostToDevice));
    CubDebugExit(hipMemcpy(params.d_vector_x,          vector_x,                   sizeof(ValueT) * csr_matrix.num_cols, hipMemcpyHostToDevice));

    if (!g_quiet) printf("\n\n");
    printf("GPU CSR I/O Prox, "); fflush(stdout);
    avg_millis = TestGpuCsrIoProxy(params, timing_iterations);
    DisplayPerf(device_giga_bandwidth, avg_millis, csr_matrix);

    if (args.CheckCmdLineFlag("csrmv"))
    {
        if (!g_quiet) printf("\n\n");
        printf("CUB, "); fflush(stdout);
        avg_millis = TestGpuMergeCsrmv(vector_y_in, vector_y_out, params, timing_iterations);
        DisplayPerf(device_giga_bandwidth, avg_millis, csr_matrix);
    }

    // Initialize cuSparse
    hipsparseHandle_t cusparse;
    AssertEquals(HIPSPARSE_STATUS_SUCCESS, hipsparseCreate(&cusparse));

    if (args.CheckCmdLineFlag("csrmv"))
    {
        if (!g_quiet) printf("\n\n");
        printf("Cusparse CsrMV, "); fflush(stdout);
        avg_millis = TestCusparseCsrmv(vector_y_in, vector_y_out, params, timing_iterations, cusparse);
        DisplayPerf(device_giga_bandwidth, avg_millis, csr_matrix);
    }

    if (args.CheckCmdLineFlag("hybmv"))
    {
        if (!g_quiet) printf("\n\n");
        printf("Cusparse HybMV, "); fflush(stdout);

        avg_millis = TestCusparseHybmv(vector_y_in, vector_y_out, params, timing_iterations, cusparse);
        DisplayPerf(device_giga_bandwidth, avg_millis, csr_matrix);
    }


    // Cleanup
    if (params.d_values)            CubDebugExit(g_allocator.DeviceFree(params.d_values));
    if (params.d_row_end_offsets)   CubDebugExit(g_allocator.DeviceFree(params.d_row_end_offsets));
    if (params.d_column_indices)    CubDebugExit(g_allocator.DeviceFree(params.d_column_indices));
    if (params.d_vector_x)          CubDebugExit(g_allocator.DeviceFree(params.d_vector_x));
    if (params.d_vector_y)          CubDebugExit(g_allocator.DeviceFree(params.d_vector_y));

    if (vector_x)                   delete[] vector_x;
    if (vector_y_in)                delete[] vector_y_in;
    if (vector_y_out)               delete[] vector_y_out;
}

/**
 * Run tests
 */
template <
    typename ValueT,
    typename OffsetT>
void RunTests(
    bool                rcm_relabel,
    ValueT              alpha,
    ValueT              beta,
    const std::string&  mtx_filename,
    int                 grid2d,
    int                 grid3d,
    int                 wheel,
    int                 dense,
    int                 timing_iterations,
    CommandLineArgs&    args)
{
    // Initialize matrix in COO form
    CooMatrix<ValueT, OffsetT> coo_matrix;

    if (!mtx_filename.empty())
    {
        // Parse matrix market file
        printf("%s, ", mtx_filename.c_str()); fflush(stdout);
        coo_matrix.InitMarket(mtx_filename, 1.0, !g_quiet);

        if ((coo_matrix.num_rows == 1) || (coo_matrix.num_cols == 1) || (coo_matrix.num_nonzeros == 1))
        {
            if (!g_quiet) printf("Trivial dataset\n");
            exit(0);
        }
    }
    else if (grid2d > 0)
    {
        // Generate 2D lattice
        printf("grid2d_%d, ", grid2d); fflush(stdout);
        coo_matrix.InitGrid2d(grid2d, false);
    }
    else if (grid3d > 0)
    {
        // Generate 3D lattice
        printf("grid3d_%d, ", grid3d); fflush(stdout);
        coo_matrix.InitGrid3d(grid3d, false);
    }
    else if (wheel > 0)
    {
        // Generate wheel graph
        printf("wheel_%d, ", grid2d); fflush(stdout);
        coo_matrix.InitWheel(wheel);
    }
    else if (dense > 0)
    {
        // Generate dense graph
        OffsetT size = 1 << 24; // 16M nnz
        args.GetCmdLineArgument("size", size);

        OffsetT rows = size / dense;
        printf("dense_%d_x_%d, ", rows, dense); fflush(stdout);
        coo_matrix.InitDense(rows, dense);
    }
    else
    {
        fprintf(stderr, "No graph type specified.\n");
        exit(1);
    }

    RunTest(
        rcm_relabel,
        alpha,
        beta,
        coo_matrix,
        timing_iterations,
        args);
}



/**
 * Main
 */
int main(int argc, char **argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    if (args.CheckCmdLineFlag("help"))
    {
        printf(
            "%s "
            "[--csrmv | --hybmv | --bsrmv ] "
            "[--device=<device-id>] "
            "[--quiet] "
            "[--v] "
            "[--i=<timing iterations>] "
            "[--fp64] "
            "[--rcm] "
            "[--alpha=<alpha scalar (default: 1.0)>] "
            "[--beta=<beta scalar (default: 0.0)>] "
            "\n\t"
                "--mtx=<matrix market file> "
            "\n\t"
                "--dense=<cols>"
            "\n\t"
                "--grid2d=<width>"
            "\n\t"
                "--grid3d=<width>"
            "\n\t"
                "--wheel=<spokes>"
            "\n", argv[0]);
        exit(0);
    }

    bool                fp64;
    bool                rcm_relabel;
    std::string         mtx_filename;
    int                 grid2d              = -1;
    int                 grid3d              = -1;
    int                 wheel               = -1;
    int                 dense               = -1;
    int                 timing_iterations   = -1;
    float               alpha               = 1.0;
    float               beta                = 0.0;

    g_verbose = args.CheckCmdLineFlag("v");
    g_verbose2 = args.CheckCmdLineFlag("v2");
    g_quiet = args.CheckCmdLineFlag("quiet");
    fp64 = args.CheckCmdLineFlag("fp64");
    rcm_relabel = args.CheckCmdLineFlag("rcm");
    args.GetCmdLineArgument("i", timing_iterations);
    args.GetCmdLineArgument("mtx", mtx_filename);
    args.GetCmdLineArgument("grid2d", grid2d);
    args.GetCmdLineArgument("grid3d", grid3d);
    args.GetCmdLineArgument("wheel", wheel);
    args.GetCmdLineArgument("dense", dense);
    args.GetCmdLineArgument("alpha", alpha);
    args.GetCmdLineArgument("beta", beta);

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Run test(s)
    if (fp64)
    {
        RunTests<double, int>(rcm_relabel, alpha, beta, mtx_filename, grid2d, grid3d, wheel, dense, timing_iterations, args);
    }
    else
    {
        RunTests<float, int>(rcm_relabel, alpha, beta, mtx_filename, grid2d, grid3d, wheel, dense, timing_iterations, args);
    }

    CubDebugExit(hipDeviceSynchronize());
    printf("\n");

    return 0;
}
