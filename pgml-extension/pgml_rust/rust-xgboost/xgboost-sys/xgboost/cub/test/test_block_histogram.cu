#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of BlockHistogram utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <limits>
#include <string>
#include <typeinfo>

#include <cub/block/block_histogram.cuh>
#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/util_allocator.cuh>

#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose           = false;
int                     g_timing_iterations = 0;
int                     g_repeat            = 0;
CachingDeviceAllocator  g_allocator(true);


//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------

/**
 * BlockHistogram test kernel.
 */
template <
    int                     BINS,
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    BlockHistogramAlgorithm ALGORITHM,
    typename                T,
    typename                HistoCounter>
__global__ void BlockHistogramKernel(
    T                       *d_samples,
    HistoCounter            *d_histogram)
{
    // Parameterize BlockHistogram type for our thread block
    typedef BlockHistogram<T, BLOCK_THREADS, ITEMS_PER_THREAD, BINS, ALGORITHM> BlockHistogram;

    // Allocate temp storage in shared memory
    __shared__ typename BlockHistogram::TempStorage temp_storage;

    // Per-thread tile data
    T data[ITEMS_PER_THREAD];
    LoadDirectStriped<BLOCK_THREADS>(threadIdx.x, d_samples, data);

    // Test histo (writing directly to histogram buffer in global)
    BlockHistogram(temp_storage).Histogram(data, d_histogram);
}


/**
 * Initialize problem (and solution)
 */
template <
    int             BINS,
    typename        SampleT>
void Initialize(
    GenMode         gen_mode,
    SampleT         *h_samples,
    int             *h_histograms_linear,
    int             num_samples)
{
    // Init bins
    for (int bin = 0; bin < BINS; ++bin)
    {
        h_histograms_linear[bin] = 0;
    }

    if (g_verbose) printf("Samples: \n");

    // Initialize interleaved channel samples and histogram them correspondingly
    for (int i = 0; i < num_samples; ++i)
    {
        InitValue(gen_mode, h_samples[i], i);
        h_samples[i] %= BINS;

        if (g_verbose) std::cout << CoutCast(h_samples[i]) << ", ";

        h_histograms_linear[h_samples[i]]++;
    }

    if (g_verbose) printf("\n\n");
}


/**
 * Test BlockHistogram
 */
template <
    typename                    SampleT,
    int                         BINS,
    int                         BLOCK_THREADS,
    int                         ITEMS_PER_THREAD,
    BlockHistogramAlgorithm     ALGORITHM>
void Test(
    GenMode                     gen_mode)
{
    int num_samples = BLOCK_THREADS * ITEMS_PER_THREAD;

    printf("hipcub::BlockHistogram %s %d %s samples (%dB), %d bins, %d threads, gen-mode %s\n",
        (ALGORITHM == BLOCK_HISTO_SORT) ? "BLOCK_HISTO_SORT" : "BLOCK_HISTO_ATOMIC",
        num_samples,
        typeid(SampleT).name(),
        (int) sizeof(SampleT),
        BINS,
        BLOCK_THREADS,
        (gen_mode == RANDOM) ? "RANDOM" : (gen_mode == INTEGER_SEED) ? "SEQUENTIAL" : "HOMOGENOUS");
    fflush(stdout);

    // Allocate host arrays
    SampleT         *h_samples          = new SampleT[num_samples];
    int   *h_reference = new int[BINS];

    // Initialize problem
    Initialize<BINS>(gen_mode, h_samples, h_reference, num_samples);

    // Allocate problem device arrays
    SampleT         *d_samples = NULL;
    int             *d_histogram = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_samples,             sizeof(SampleT) * num_samples));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_histogram,   sizeof(int) * BINS));

    // Initialize/clear device arrays
    CubDebugExit(hipMemcpy(d_samples, h_samples, sizeof(SampleT) * num_samples, hipMemcpyHostToDevice));
    CubDebugExit(hipMemset(d_histogram, 0, sizeof(int) * BINS));

    // Run kernel
    BlockHistogramKernel<BINS, BLOCK_THREADS, ITEMS_PER_THREAD, ALGORITHM><<<1, BLOCK_THREADS>>>(
        d_samples,
        d_histogram);

    // Check for correctness (and display results, if specified)
    int compare = CompareDeviceResults((int*) h_reference, d_histogram, BINS, g_verbose, g_verbose);
    printf("\t%s\n\n", compare ? "FAIL" : "PASS");

    // Flush any stdout/stderr
    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());
    fflush(stdout);
    fflush(stderr);

    // Cleanup
    if (h_samples) delete[] h_samples;
    if (h_reference) delete[] h_reference;
    if (d_samples) CubDebugExit(g_allocator.DeviceFree(d_samples));
    if (d_histogram) CubDebugExit(g_allocator.DeviceFree(d_histogram));

    // Correctness asserts
    AssertEquals(0, compare);
}


/**
 * Test different sample distributions
 */
template <
    typename                    SampleT,
    int                         BINS,
    int                         BLOCK_THREADS,
    int                         ITEMS_PER_THREAD,
    BlockHistogramAlgorithm     ALGORITHM>
void Test()
{
    Test<SampleT, BINS, BLOCK_THREADS, ITEMS_PER_THREAD, ALGORITHM>(UNIFORM);
    Test<SampleT, BINS, BLOCK_THREADS, ITEMS_PER_THREAD, ALGORITHM>(INTEGER_SEED);
    Test<SampleT, BINS, BLOCK_THREADS, ITEMS_PER_THREAD, ALGORITHM>(RANDOM);
}


/**
 * Test different ALGORITHM
 */
template <
    typename                    SampleT,
    int                         BINS,
    int                         BLOCK_THREADS,
    int                         ITEMS_PER_THREAD>
void Test()
{
    Test<SampleT, BINS, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_HISTO_SORT>();
    Test<SampleT, BINS, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_HISTO_ATOMIC>();
}


/**
 * Test different ITEMS_PER_THREAD
 */
template <
    typename                    SampleT,
    int                         BINS,
    int                         BLOCK_THREADS>
void Test()
{
    Test<SampleT, BINS, BLOCK_THREADS, 1>();
    Test<SampleT, BINS, BLOCK_THREADS, 5>();
}


/**
 * Test different BLOCK_THREADS
 */
template <
    typename                    SampleT,
    int                         BINS>
void Test()
{
    Test<SampleT, BINS, 32>();
    Test<SampleT, BINS, 96>();
    Test<SampleT, BINS, 128>();
}





//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("repeat", g_repeat);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--n=<total input samples across all channels> "
            "[--device=<device-id>] "
            "[--repeat=<repetitions of entire test suite>]"
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

#ifdef QUICK_TEST

    // Compile/run quick tests
    Test<unsigned char, 256, 128, 4, BLOCK_HISTO_SORT>(RANDOM);
    Test<unsigned char, 256, 128, 4, BLOCK_HISTO_ATOMIC>(RANDOM);

#else

    // Compile/run thorough tests
    for (int i = 0; i <= g_repeat; ++i)
    {
        Test<unsigned char, 32>();
        Test<unsigned char, 256>();
        Test<unsigned short, 1024>();
    }

#endif

    return 0;
}



