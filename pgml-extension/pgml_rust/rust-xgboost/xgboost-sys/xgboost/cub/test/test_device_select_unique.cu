#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of DeviceSelect::Unique utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>
#include <typeinfo>

#include <thrust/device_ptr.h>
#include <thrust/unique.h>

#include <cub/util_allocator.cuh>
#include <cub/iterator/counting_input_iterator.cuh>
#include <cub/device/device_select.cuh>

#include <thrust/device_ptr.h>
#include <thrust/unique.h>

#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose               = false;
int                     g_timing_iterations     = 0;
int                     g_repeat                = 0;
float                   g_device_giga_bandwidth;
CachingDeviceAllocator  g_allocator(true);

// Dispatch types
enum Backend
{
    CUB,        // CUB method
    THRUST,     // Thrust method
    CDP,        // GPU-based (dynamic parallelism) dispatch to CUB method
};


//---------------------------------------------------------------------
// Dispatch to different CUB DeviceSelect entrypoints
//---------------------------------------------------------------------


/**
 * Dispatch to unique entrypoint
 */
template <typename InputIteratorT, typename OutputIteratorT, typename NumSelectedIteratorT, typename OffsetT>
HIPCUB_RUNTIME_FUNCTION __forceinline__
hipError_t Dispatch(
    Int2Type<CUB>               /*dispatch_to*/,
    int                         timing_timing_iterations,
    size_t                      */*d_temp_storage_bytes*/,
    hipError_t                 */*d_cdp_error*/,

    void*               d_temp_storage,
    size_t                      &temp_storage_bytes,
    InputIteratorT              d_in,
    OutputIteratorT              d_out,
    NumSelectedIteratorT         d_num_selected_out,
    OffsetT                     num_items,
    hipStream_t                stream,
    bool                        debug_synchronous)
{
    hipError_t error = hipSuccess;
    for (int i = 0; i < timing_timing_iterations; ++i)
    {
        error = DeviceSelect::Unique(d_temp_storage, temp_storage_bytes, d_in, d_out, d_num_selected_out, num_items, stream, debug_synchronous);
    }
    return error;
}


//---------------------------------------------------------------------
// Dispatch to different Thrust entrypoints
//---------------------------------------------------------------------


/**
 * Dispatch to unique entrypoint
 */
template <typename InputIteratorT, typename OutputIteratorT, typename NumSelectedIteratorT, typename OffsetT>
__host__ __forceinline__
hipError_t Dispatch(
    Int2Type<THRUST>            /*dispatch_to*/,
    int                         timing_timing_iterations,
    size_t                      */*d_temp_storage_bytes*/,
    hipError_t                 */*d_cdp_error*/,

    void                        *d_temp_storage,
    size_t                      &temp_storage_bytes,
    InputIteratorT              d_in,
    OutputIteratorT             d_out,
    NumSelectedIteratorT        d_num_selected_out,
    OffsetT                     num_items,
    hipStream_t                /*stream*/,
    bool                        /*debug_synchronous*/)
{
    // The input value type
    typedef typename std::iterator_traits<InputIteratorT>::value_type InputT;

    // The output value type
    typedef typename If<(Equals<typename std::iterator_traits<OutputIteratorT>::value_type, void>::VALUE),  // OutputT =  (if output iterator's value type is void) ?
        typename std::iterator_traits<InputIteratorT>::value_type,                                          // ... then the input iterator's value type,
        typename std::iterator_traits<OutputIteratorT>::value_type>::Type OutputT;                          // ... else the output iterator's value type

    if (d_temp_storage == 0)
    {
        temp_storage_bytes = 1;
    }
    else
    {
        thrust::device_ptr<OutputT> d_out_wrapper_end;
        thrust::device_ptr<InputT> d_in_wrapper(d_in);
        thrust::device_ptr<OutputT> d_out_wrapper(d_out);
        for (int i = 0; i < timing_timing_iterations; ++i)
        {
            d_out_wrapper_end = thrust::unique_copy(d_in_wrapper, d_in_wrapper + num_items, d_out_wrapper);
        }

        OffsetT num_selected = OffsetT(d_out_wrapper_end - d_out_wrapper);
        CubDebugExit(hipMemcpy(d_num_selected_out, &num_selected, sizeof(OffsetT), hipMemcpyHostToDevice));

    }

    return hipSuccess;
}



//---------------------------------------------------------------------
// CUDA Nested Parallelism Test Kernel
//---------------------------------------------------------------------

/**
 * Simple wrapper kernel to invoke DeviceSelect
 */
template <typename InputIteratorT, typename OutputIteratorT, typename NumSelectedIteratorT, typename OffsetT>
__global__ void CnpDispatchKernel(
    int                         timing_timing_iterations,
    size_t                      *d_temp_storage_bytes,
    hipError_t                 *d_cdp_error,

    void*               d_temp_storage,
    size_t                      temp_storage_bytes,
    InputIteratorT              d_in,
    OutputIteratorT              d_out,
    NumSelectedIteratorT         d_num_selected_out,
    OffsetT                     num_items,
    bool                        debug_synchronous)
{

#ifndef CUB_CDP
    (void)timing_timing_iterations;
    (void)d_temp_storage_bytes;
    (void)d_cdp_error;
    (void)d_temp_storage;
    (void)temp_storage_bytes;
    (void)d_in;
    (void)d_out;
    (void)d_num_selected_out;
    (void)num_items;
    (void)debug_synchronous;
    *d_cdp_error = hipErrorNotSupported;
#else
    *d_cdp_error = Dispatch(Int2Type<CUB>(), timing_timing_iterations, d_temp_storage_bytes, d_cdp_error,
        d_temp_storage, temp_storage_bytes, d_in, d_out, d_num_selected_out, num_items, 0, debug_synchronous);
    *d_temp_storage_bytes = temp_storage_bytes;
#endif
}


/**
 * Dispatch to CDP kernel
 */
template <typename InputIteratorT, typename OutputIteratorT, typename NumSelectedIteratorT, typename OffsetT>
hipError_t Dispatch(
    Int2Type<CDP>               dispatch_to,
    int                         timing_timing_iterations,
    size_t                      *d_temp_storage_bytes,
    hipError_t                 *d_cdp_error,

    void*               d_temp_storage,
    size_t                      &temp_storage_bytes,
    InputIteratorT              d_in,
    OutputIteratorT              d_out,
    NumSelectedIteratorT         d_num_selected_out,
    OffsetT                     num_items,
    hipStream_t                stream,
    bool                        debug_synchronous)
{
    // Invoke kernel to invoke device-side dispatch
    CnpDispatchKernel<<<1,1>>>(timing_timing_iterations, d_temp_storage_bytes, d_cdp_error,
        d_temp_storage, temp_storage_bytes, d_in, d_out, d_num_selected_out, num_items, debug_synchronous);

    // Copy out temp_storage_bytes
    CubDebugExit(hipMemcpy(&temp_storage_bytes, d_temp_storage_bytes, sizeof(size_t) * 1, hipMemcpyDeviceToHost));

    // Copy out error
    hipError_t retval;
    CubDebugExit(hipMemcpy(&retval, d_cdp_error, sizeof(hipError_t) * 1, hipMemcpyDeviceToHost));
    return retval;
}



//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------


/**
 * Initialize problem
 */
template <typename T>
void Initialize(
    int         entropy_reduction,
    T           *h_in,
    int         num_items,
    int         max_segment)
{
    unsigned int max_int = (unsigned int) -1;

    int key = 0;
    int i = 0;
    while (i < num_items)
    {
        // Select number of repeating occurrences for the current run
        int repeat;
        if (max_segment < 0)
        {
            repeat = num_items;
        }
        else if (max_segment < 2)
        {
            repeat = 1;
        }
        else
        {
            RandomBits(repeat, entropy_reduction);
            repeat = (int) ((double(repeat) * double(max_segment)) / double(max_int));
            repeat = CUB_MAX(1, repeat);
        }

        int j = i;
        while (j < CUB_MIN(i + repeat, num_items))
        {
            InitValue(INTEGER_SEED, h_in[j], key);
            j++;
        }

        i = j;
        key++;
    }

    if (g_verbose)
    {
        printf("Input:\n");
        DisplayResults(h_in, num_items);
        printf("\n\n");
    }
}


/**
 * Solve unique problem
 */
template <
    typename        InputIteratorT,
    typename        T>
int Solve(
    InputIteratorT  h_in,
    T               *h_reference,
    int             num_items)
{
    int num_selected = 0;
    if (num_items > 0)
    {
        h_reference[num_selected] = h_in[0];
        num_selected++;
    }

    for (int i = 1; i < num_items; ++i)
    {
        if (h_in[i] != h_in[i - 1])
        {
            h_reference[num_selected] = h_in[i];
            num_selected++;
        }
    }

    return num_selected;
}



/**
 * Test DeviceSelect for a given problem input
 */
template <
    Backend             BACKEND,
    typename            DeviceInputIteratorT,
    typename            T>
void Test(
    DeviceInputIteratorT d_in,
    T                   *h_reference,
    int                 num_selected,
    int                 num_items)
{
    // Allocate device output array and num selected
    T       *d_out            = NULL;
    int     *d_num_selected_out   = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(T) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_num_selected_out, sizeof(int)));

    // Allocate CDP device arrays
    size_t          *d_temp_storage_bytes = NULL;
    hipError_t     *d_cdp_error = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_temp_storage_bytes,  sizeof(size_t) * 1));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_cdp_error,           sizeof(hipError_t) * 1));

    // Allocate temporary storage
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;
    CubDebugExit(Dispatch(Int2Type<BACKEND>(), 1, d_temp_storage_bytes, d_cdp_error, d_temp_storage, temp_storage_bytes, d_in, d_out, d_num_selected_out, num_items, 0, true));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Clear device output array
    CubDebugExit(hipMemset(d_out, 0, sizeof(T) * num_items));
    CubDebugExit(hipMemset(d_num_selected_out, 0, sizeof(int)));

    // Run warmup/correctness iteration
    CubDebugExit(Dispatch(Int2Type<BACKEND>(), 1, d_temp_storage_bytes, d_cdp_error, d_temp_storage, temp_storage_bytes, d_in, d_out, d_num_selected_out, num_items, 0, true));

    // Check for correctness (and display results, if specified)
    int compare1 = CompareDeviceResults(h_reference, d_out, num_selected, true, g_verbose);
    printf("\t Data %s ", compare1 ? "FAIL" : "PASS");

    int compare2 = CompareDeviceResults(&num_selected, d_num_selected_out, 1, true, g_verbose);
    printf("\t Count %s ", compare2 ? "FAIL" : "PASS");

    // Flush any stdout/stderr
    fflush(stdout);
    fflush(stderr);

    // Performance
    GpuTimer gpu_timer;
    gpu_timer.Start();
    CubDebugExit(Dispatch(Int2Type<BACKEND>(), g_timing_iterations, d_temp_storage_bytes, d_cdp_error, d_temp_storage, temp_storage_bytes, d_in, d_out, d_num_selected_out, num_items, 0, false));
    gpu_timer.Stop();
    float elapsed_millis = gpu_timer.ElapsedMillis();

    // Display performance
    if (g_timing_iterations > 0)
    {
        float avg_millis        = elapsed_millis / g_timing_iterations;
        float giga_rate         = float(num_items) / avg_millis / 1000.0f / 1000.0f;
        float giga_bandwidth    = float((num_items + num_selected) * sizeof(T)) / avg_millis / 1000.0f / 1000.0f;
        printf(", %.3f avg ms, %.3f billion items/s, %.3f logical GB/s, %.1f%% peak", avg_millis, giga_rate, giga_bandwidth, giga_bandwidth / g_device_giga_bandwidth * 100.0);
    }
    printf("\n\n");

    // Flush any stdout/stderr
    fflush(stdout);
    fflush(stderr);

    // Cleanup
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_num_selected_out) CubDebugExit(g_allocator.DeviceFree(d_num_selected_out));
    if (d_temp_storage_bytes) CubDebugExit(g_allocator.DeviceFree(d_temp_storage_bytes));
    if (d_cdp_error) CubDebugExit(g_allocator.DeviceFree(d_cdp_error));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

    // Correctness asserts
    AssertEquals(0, compare1 | compare2);
}


/**
 * Test DeviceSelect on pointer type
 */
template <
    Backend         BACKEND,
    typename        T>
void TestPointer(
    int             num_items,
    int             entropy_reduction,
    int             max_segment)
{
    // Allocate host arrays
    T*  h_in        = new T[num_items];
    T*  h_reference = new T[num_items];

    // Initialize problem and solution
    Initialize(entropy_reduction, h_in, num_items, max_segment);
    int num_selected = Solve(h_in, h_reference, num_items);

    printf("\nPointer %s hipcub::DeviceSelect::Unique %d items, %d selected (avg run length %.3f), %s %d-byte elements, entropy_reduction %d\n",
        (BACKEND == CDP) ? "CDP CUB" : (BACKEND == THRUST) ? "Thrust" : "CUB",
        num_items, num_selected, float(num_items) / num_selected,
        typeid(T).name(),
        (int) sizeof(T),
        entropy_reduction);
    fflush(stdout);

    // Allocate problem device arrays
    T *d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(T) * num_items));

    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * num_items, hipMemcpyHostToDevice));

    // Run Test
    Test<BACKEND>(d_in, h_reference, num_selected, num_items);

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_reference) delete[] h_reference;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
}


/**
 * Test DeviceSelect on iterator type
 */
template <
    Backend         BACKEND,
    typename        T>
void TestIterator(
    int             num_items)
{
    // Use a counting iterator as the input
    CountingInputIterator<T, int> h_in(0);

    // Allocate host arrays
    T*  h_reference = new T[num_items];

    // Initialize problem and solution
    int num_selected = Solve(h_in, h_reference, num_items);

    printf("\nIterator %s hipcub::DeviceSelect::Unique %d items, %d selected (avg run length %.3f), %s %d-byte elements\n",
        (BACKEND == CDP) ? "CDP CUB" : (BACKEND == THRUST) ? "Thrust" : "CUB",
        num_items, num_selected, float(num_items) / num_selected,
        typeid(T).name(),
        (int) sizeof(T));
    fflush(stdout);

    // Run Test
    Test<BACKEND>(h_in, h_reference, num_selected, num_items);

    // Cleanup
    if (h_reference) delete[] h_reference;
}


/**
 * Test different gen modes
 */
template <
    Backend         BACKEND,
    typename        T>
void Test(
    int             num_items)
{
    for (int max_segment = 1; ((max_segment > 0) && (max_segment < num_items)); max_segment *= 11)
    {
        TestPointer<BACKEND, T>(num_items, 0, max_segment);
        TestPointer<BACKEND, T>(num_items, 2, max_segment);
        TestPointer<BACKEND, T>(num_items, 7, max_segment);
    }
}


/**
 * Test different dispatch
 */
template <
    typename        T>
void TestOp(
    int             num_items)
{
    Test<CUB, T>(num_items);
#ifdef CUB_CDP
    Test<CDP, T>(num_items);
#endif
}


/**
 * Test different input sizes
 */
template <typename T>
void Test(
    int             num_items)
{
    if (num_items < 0)
    {
        TestOp<T>(0);
        TestOp<T>(1);
        TestOp<T>(100);
        TestOp<T>(10000);
        TestOp<T>(1000000);
    }
    else
    {
        TestOp<T>(num_items);
    }
}



//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    int num_items           = -1;
    int entropy_reduction   = 0;
    int maxseg              = 1000;

    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("n", num_items);
    args.GetCmdLineArgument("i", g_timing_iterations);
    args.GetCmdLineArgument("repeat", g_repeat);
    args.GetCmdLineArgument("maxseg", maxseg);
    args.GetCmdLineArgument("entropy", entropy_reduction);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--n=<input items> "
            "[--i=<timing iterations> "
            "[--device=<device-id>] "
            "[--maxseg=<max segment length>]"
            "[--entropy=<segment length bit entropy reduction rounds>]"
            "[--repeat=<repetitions of entire test suite>]"
            "[--v] "
            "[--cdp]"
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());
    g_device_giga_bandwidth = args.device_giga_bandwidth;
    printf("\n");

#ifdef QUICKER_TEST

    // Compile/run basic CUB test
    if (num_items < 0) num_items = 32000000;
    TestPointer<CUB, int>(         num_items,                                 entropy_reduction, maxseg);

#elif defined(QUICK_TEST)

    // Get device ordinal
    int device_ordinal;
    CubDebugExit(hipGetDevice(&device_ordinal));

    // Get device SM version
    int sm_version;
    CubDebugExit(SmVersion(sm_version, device_ordinal));

    // Compile/run quick tests
    if (num_items < 0) num_items = 32000000;

    printf("-- Iterator ----------------------------\n");
    TestIterator<CUB, int>(        num_items);

    printf("----------------------------\n");
    TestPointer<CUB, char>(        num_items * ((sm_version <= 130) ? 1 : 4), entropy_reduction, maxseg);
    TestPointer<THRUST, char>(     num_items * ((sm_version <= 130) ? 1 : 4), entropy_reduction, maxseg);

    printf("----------------------------\n");
    TestPointer<CUB, short>(       num_items * ((sm_version <= 130) ? 1 : 2), entropy_reduction, maxseg);
    TestPointer<THRUST, short>(    num_items * ((sm_version <= 130) ? 1 : 2), entropy_reduction, maxseg);

    printf("----------------------------\n");
    TestPointer<CUB, int>(         num_items,                                 entropy_reduction, maxseg);
    TestPointer<THRUST, int>(      num_items,                                 entropy_reduction, maxseg);

    printf("----------------------------\n");
    TestPointer<CUB, long long>(   num_items / 2,                             entropy_reduction, maxseg);
    TestPointer<THRUST, long long>(num_items / 2,                             entropy_reduction, maxseg);

    printf("----------------------------\n");
    TestPointer<CUB, TestFoo>(     num_items / 4,                             entropy_reduction, maxseg);
    TestPointer<THRUST, TestFoo>(  num_items / 4,                             entropy_reduction, maxseg);

#else

    // Compile/run thorough tests
    for (int i = 0; i <= g_repeat; ++i)
    {
        // Test different input types
        Test<unsigned char>(num_items);
        Test<unsigned short>(num_items);
        Test<unsigned int>(num_items);
        Test<unsigned long long>(num_items);

        Test<uchar2>(num_items);
        Test<ushort2>(num_items);
        Test<uint2>(num_items);
        Test<ulonglong2>(num_items);

        Test<uchar4>(num_items);
        Test<ushort4>(num_items);
        Test<uint4>(num_items);
        Test<ulonglong4>(num_items);

        Test<TestFoo>(num_items);
        Test<TestBar>(num_items);
    }

#endif

    return 0;
}



