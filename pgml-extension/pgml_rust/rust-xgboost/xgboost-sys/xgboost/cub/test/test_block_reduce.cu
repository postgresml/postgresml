#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Test of BlockReduce utilities
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>

#include <hip/hip_runtime_api.h>
#include <typeinfo>

#include <cub/block/block_reduce.cuh>
#include <cub/block/block_load.cuh>
#include <cub/util_ptx.cuh>
#include <cub/util_allocator.cuh>
#include <cub/util_debug.cuh>

#include "test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose       = false;
int                     g_repeat        = 0;
CachingDeviceAllocator  g_allocator(true);



//---------------------------------------------------------------------
// Test kernels
//---------------------------------------------------------------------


/// Generic reduction (full, 1)
template <typename BlockReduceT, typename T, typename ReductionOp>
__device__ __forceinline__ T DeviceTest(
    BlockReduceT &block_reduce, T (&data)[1], ReductionOp &reduction_op)
{
    return block_reduce.Reduce(data[0], reduction_op);
}

/// Generic reduction (full, ITEMS_PER_THREAD)
template <typename BlockReduceT, typename T, int ITEMS_PER_THREAD, typename ReductionOp>
__device__ __forceinline__ T DeviceTest(
    BlockReduceT &block_reduce, T (&data)[ITEMS_PER_THREAD], ReductionOp &reduction_op)
{
    return block_reduce.Reduce(data, reduction_op);
}

/// Generic reduction (partial, 1)
template <typename BlockReduceT, typename T, typename ReductionOp>
__device__ __forceinline__ T DeviceTest(
    BlockReduceT &block_reduce, T &data, ReductionOp &reduction_op, int valid_threads)
{
    return block_reduce.Reduce(data, reduction_op, valid_threads);
}

/// Sum reduction (full, 1)
template <typename BlockReduceT, typename T>
__device__ __forceinline__ T DeviceTest(
    BlockReduceT &block_reduce, T (&data)[1], Sum &reduction_op)
{
    return block_reduce.Sum(data[0]);
}

/// Sum reduction (full, ITEMS_PER_THREAD)
template <typename BlockReduceT, typename T, int ITEMS_PER_THREAD>
__device__ __forceinline__ T DeviceTest(
    BlockReduceT &block_reduce, T (&data)[ITEMS_PER_THREAD], Sum &reduction_op)
{
    return block_reduce.Sum(data);
}

/// Sum reduction (partial, 1)
template <typename BlockReduceT, typename T>
__device__ __forceinline__ T DeviceTest(
    BlockReduceT &block_reduce, T &data, Sum &reduction_op, int valid_threads)
{
    return block_reduce.Sum(data, valid_threads);
}


/**
 * Test full-tile reduction kernel (where num_items is an even
 * multiple of BLOCK_THREADS)
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_DIM_X,
    int                     BLOCK_DIM_Y,
    int                     BLOCK_DIM_Z,
    int                     ITEMS_PER_THREAD,
    typename                T,
    typename                ReductionOp>
__launch_bounds__ (BLOCK_DIM_X * BLOCK_DIM_Y * BLOCK_DIM_Z)
__global__ void FullTileReduceKernel(
    T                       *d_in,
    T                       *d_out,
    ReductionOp             reduction_op,
    int                     tiles,
    clock_t                 *d_elapsed)
{
    const int BLOCK_THREADS     = BLOCK_DIM_X * BLOCK_DIM_Y * BLOCK_DIM_Z;
    const int TILE_SIZE         = BLOCK_THREADS * ITEMS_PER_THREAD;

    // Cooperative thread block reduction utility type (returns aggregate in thread 0)
    typedef BlockReduce<T, BLOCK_DIM_X, ALGORITHM, BLOCK_DIM_Y, BLOCK_DIM_Z> BlockReduceT;

    // Allocate temp storage in shared memory
    __shared__ typename BlockReduceT::TempStorage temp_storage;

    int linear_tid = RowMajorTid(BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z);

    // Per-thread tile data
    T data[ITEMS_PER_THREAD];

    // Load first tile of data
    int block_offset = 0;

    if (block_offset < TILE_SIZE * tiles)
    {
        LoadDirectBlocked(linear_tid, d_in + block_offset, data);
        block_offset += TILE_SIZE;

        // Start cycle timer
        clock_t start = clock();

        // Cooperative reduce first tile
        BlockReduceT block_reduce(temp_storage) ;
        T block_aggregate = DeviceTest(block_reduce, data, reduction_op);

        // Stop cycle timer
 #if HIPCUB_ARCH == 100
        // Bug: recording stop clock causes mis-write of running prefix value
        clock_t stop = 0;
#else
        clock_t stop = clock();
#endif // HIPCUB_ARCH == 100
        clock_t elapsed = (start > stop) ? start - stop : stop - start;

        // Loop over input tiles
        while (block_offset < TILE_SIZE * tiles)
        {
            // TestBarrier between thread block reductions
            __syncthreads();
    
            // Load tile of data
            LoadDirectBlocked(linear_tid, d_in + block_offset, data);
            block_offset += TILE_SIZE;

            // Start cycle timer
            clock_t start = clock();

            // Cooperatively reduce the tile's aggregate
            BlockReduceT block_reduce(temp_storage) ;
            T tile_aggregate = DeviceTest(block_reduce, data, reduction_op);

            // Stop cycle timer
#if HIPCUB_ARCH == 100
            // Bug: recording stop clock causes mis-write of running prefix value
            clock_t stop = 0;
#else
            clock_t stop = clock();
#endif // HIPCUB_ARCH == 100
            elapsed += (start > stop) ? start - stop : stop - start;

            // Reduce thread block aggregate
            block_aggregate = reduction_op(block_aggregate, tile_aggregate);
        }

        // Store data
        if (linear_tid == 0)
        {
            d_out[0] = block_aggregate;
            *d_elapsed = elapsed;
        }
    }
}



/**
 * Test partial-tile reduction kernel (where num_items < BLOCK_THREADS)
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_DIM_X,
    int                     BLOCK_DIM_Y,
    int                     BLOCK_DIM_Z,
    typename                T,
    typename                ReductionOp>
__launch_bounds__ (BLOCK_DIM_X * BLOCK_DIM_Y * BLOCK_DIM_Z)
__global__ void PartialTileReduceKernel(
    T                       *d_in,
    T                       *d_out,
    int                     num_items,
    ReductionOp             reduction_op,
    clock_t                 *d_elapsed)
{
    // Cooperative thread block reduction utility type (returns aggregate only in thread-0)
    typedef BlockReduce<T, BLOCK_DIM_X, ALGORITHM, BLOCK_DIM_Y, BLOCK_DIM_Z> BlockReduceT;

    // Allocate temp storage in shared memory
    __shared__ typename BlockReduceT::TempStorage temp_storage;

    int linear_tid = RowMajorTid(BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z);

    // Per-thread tile data
    T partial;

    // Load partial tile data
    if (linear_tid < num_items)
    {
        partial = d_in[linear_tid];
    }

    // Start cycle timer
    clock_t start = clock();

    // Cooperatively reduce the tile's aggregate
    BlockReduceT block_reduce(temp_storage) ;
    T tile_aggregate = DeviceTest(block_reduce, partial, reduction_op, num_items);

    // Stop cycle timer
#if HIPCUB_ARCH == 100
    // Bug: recording stop clock causes mis-write of running prefix value
    clock_t stop = 0;
#else
    clock_t stop = clock();
#endif // HIPCUB_ARCH == 100

    clock_t elapsed = (start > stop) ? start - stop : stop - start;

    // Store data
    if (linear_tid == 0)
    {
        d_out[0] = tile_aggregate;
        *d_elapsed = elapsed;
    }
}


//---------------------------------------------------------------------
// Host utility subroutines
//---------------------------------------------------------------------

/**
 * Initialize problem (and solution)
 */
template <
    typename    T,
    typename    ReductionOp>
void Initialize(
    GenMode     gen_mode,
    T           *h_in,
    T           h_reference[1],
    ReductionOp reduction_op,
    int         num_items)
{
    for (int i = 0; i < num_items; ++i)
    {
        InitValue(gen_mode, h_in[i], i);
        if (i == 0)
            h_reference[0] = h_in[0];
        else
            h_reference[0] = reduction_op(h_reference[0], h_in[i]);
    }

    if (g_verbose)
    {
        printf("Input:\n");
        DisplayResults(h_in, num_items);
        printf("\n");
    }
}


//---------------------------------------------------------------------
// Full tile test generation
//---------------------------------------------------------------------


/**
 * Test full-tile reduction.  (Specialized for sufficient resources)
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_DIM_X,
    int                     BLOCK_DIM_Y,
    int                     BLOCK_DIM_Z,
    int                     ITEMS_PER_THREAD,
    typename                T,
    typename                ReductionOp>
void TestFullTile(
    GenMode                 gen_mode,
    int                     tiles,
    ReductionOp             reduction_op,
    Int2Type<true>          /*sufficient_resources*/)
{
    const int BLOCK_THREADS     = BLOCK_DIM_X * BLOCK_DIM_Y * BLOCK_DIM_Z;
    const int TILE_SIZE         = BLOCK_THREADS * ITEMS_PER_THREAD;

    int num_items = TILE_SIZE * tiles;

    // Allocate host arrays
    T *h_in = new T[num_items];
    T h_reference[1];

    // Initialize problem
    Initialize(gen_mode, h_in, h_reference, reduction_op, num_items);

    // Initialize/clear device arrays
    T       *d_in = NULL;
    T       *d_out = NULL;
    clock_t *d_elapsed = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_elapsed, sizeof(unsigned long long)));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(T) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(T) * 1));
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * num_items, hipMemcpyHostToDevice));
    CubDebugExit(hipMemset(d_out, 0, sizeof(T) * 1));

    // Test multi-tile (unguarded)
    printf("TestFullTile %s, %s, gen-mode %d, num_items(%d), BLOCK_THREADS(%d) (%d,%d,%d), ITEMS_PER_THREAD(%d), tiles(%d), %s (%d bytes) elements:\n",
        Equals<ReductionOp, Sum>::VALUE ? "Sum" : "Max",
        (ALGORITHM == BLOCK_REDUCE_RAKING) ? "BLOCK_REDUCE_RAKING" : (ALGORITHM == BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY) ? "BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY" : "BLOCK_REDUCE_WARP_REDUCTIONS",
        gen_mode,
        num_items,
        BLOCK_THREADS, BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z,
        ITEMS_PER_THREAD,
        tiles,
        typeid(T).name(),
        (int) sizeof(T));
    fflush(stdout);

    dim3 block_dims(BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z);
    FullTileReduceKernel<ALGORITHM, BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z, ITEMS_PER_THREAD><<<1, block_dims>>>(
        d_in,
        d_out,
        reduction_op,
        tiles,
        d_elapsed);

    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());

    // Copy out and display results
    printf("\tReduction results: ");
    int compare = CompareDeviceResults(h_reference, d_out, 1, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    printf("\tElapsed clocks: ");
    DisplayDeviceResults(d_elapsed, 1);

    // Cleanup
    if (h_in) delete[] h_in;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_elapsed) CubDebugExit(g_allocator.DeviceFree(d_elapsed));
}


/**
 * Test full-tile reduction.  (Specialized for insufficient resources)
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_DIM_X,
    int                     BLOCK_DIM_Y,
    int                     BLOCK_DIM_Z,
    int                     ITEMS_PER_THREAD,
    typename                T,
    typename                ReductionOp>
void TestFullTile(
    GenMode                 gen_mode,
    int                     tiles,
    ReductionOp             reduction_op,
    Int2Type<false>         sufficient_resources)
{}


/**
 * Test full-tile reduction.
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_DIM_X,
    int                     BLOCK_DIM_Y,
    int                     BLOCK_DIM_Z,
    int                     ITEMS_PER_THREAD,
    typename                T,
    typename                ReductionOp>
void TestFullTile(
    GenMode                 gen_mode,
    int                     tiles,
    ReductionOp             reduction_op)
{
    // Check size of smem storage for the target arch to make sure it will fit
    typedef BlockReduce<T, BLOCK_DIM_X, ALGORITHM, BLOCK_DIM_Y, BLOCK_DIM_Z, TEST_ARCH> BlockReduceT;

    enum 
    {
#if defined(SM100) || defined(SM110) || defined(SM130)
        sufficient_smem       = (sizeof(typename BlockReduceT::TempStorage) <= 16 * 1024),
        sufficient_threads    = ((BLOCK_DIM_X * BLOCK_DIM_Y * BLOCK_DIM_Z) <= 512),
#else
        sufficient_smem       = (sizeof(typename BlockReduceT::TempStorage) <= 48 * 1024),
        sufficient_threads    = ((BLOCK_DIM_X * BLOCK_DIM_Y * BLOCK_DIM_Z) <= 1024),
#endif
    };

    TestFullTile<ALGORITHM, BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z, ITEMS_PER_THREAD, T>(gen_mode, tiles, reduction_op, Int2Type<sufficient_smem && sufficient_threads>());
}


/**
 * Run battery of tests for different thread block dimensions
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_THREADS,
    int                     ITEMS_PER_THREAD,
    typename                T,
    typename                ReductionOp>
void TestFullTile(
    GenMode                 gen_mode,
    int                     tiles,
    ReductionOp             reduction_op)
{
    TestFullTile<ALGORITHM, BLOCK_THREADS, 1, 1, ITEMS_PER_THREAD, T>(gen_mode, tiles, reduction_op);
    TestFullTile<ALGORITHM, BLOCK_THREADS, 2, 2, ITEMS_PER_THREAD, T>(gen_mode, tiles, reduction_op);
}

/**
 * Run battery of tests for different thread items
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_THREADS,
    typename                T,
    typename                ReductionOp>
void TestFullTile(
    GenMode                 gen_mode,
    int                     tiles,
    ReductionOp             reduction_op)
{
    TestFullTile<ALGORITHM, BLOCK_THREADS, 1, T>(gen_mode, tiles, reduction_op);
    TestFullTile<ALGORITHM, BLOCK_THREADS, 4, T>(gen_mode, tiles, reduction_op);
}


/**
 * Run battery of full-tile tests for different numbers of tiles
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_THREADS,
    typename                T,
    typename                ReductionOp>
void TestFullTile(
    GenMode                 gen_mode,
    ReductionOp             reduction_op)
{
    for (int tiles = 1; tiles < 3; tiles++)
    {
        TestFullTile<ALGORITHM, BLOCK_THREADS, T>(gen_mode, tiles, reduction_op);
    }
}


//---------------------------------------------------------------------
// Partial-tile test generation
//---------------------------------------------------------------------

/**
 * Test partial-tile reduction.  (Specialized for sufficient resources)
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_DIM_X,
    int                     BLOCK_DIM_Y,
    int                     BLOCK_DIM_Z,
    typename                T,
    typename                ReductionOp>
void TestPartialTile(
    GenMode                 gen_mode,
    int                     num_items,
    ReductionOp             reduction_op,
    Int2Type<true>          /*sufficient_resources*/)
{
    const int BLOCK_THREADS     = BLOCK_DIM_X * BLOCK_DIM_Y * BLOCK_DIM_Z;
    const int TILE_SIZE         = BLOCK_THREADS;

    // Allocate host arrays
    T *h_in = new T[num_items];
    T h_reference[1];

    // Initialize problem
    Initialize(gen_mode, h_in, h_reference, reduction_op, num_items);

    // Initialize/clear device arrays
    T       *d_in = NULL;
    T       *d_out = NULL;
    clock_t *d_elapsed = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_elapsed, sizeof(unsigned long long)));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(T) * TILE_SIZE));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(T) * 1));
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(T) * num_items, hipMemcpyHostToDevice));
    CubDebugExit(hipMemset(d_out, 0, sizeof(T) * 1));

    printf("TestPartialTile %s, gen-mode %d, num_items(%d), BLOCK_THREADS(%d) (%d,%d,%d), %s (%d bytes) elements:\n",
        (ALGORITHM == BLOCK_REDUCE_RAKING) ? "BLOCK_REDUCE_RAKING" : (ALGORITHM == BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY) ? "BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY" : "BLOCK_REDUCE_WARP_REDUCTIONS",
        gen_mode,
        num_items,
        BLOCK_THREADS, BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z,
        typeid(T).name(),
        (int) sizeof(T));
    fflush(stdout);

    dim3 block_dims(BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z);
    PartialTileReduceKernel<ALGORITHM, BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z><<<1, block_dims>>>(
        d_in,
        d_out,
        num_items,
        reduction_op,
        d_elapsed);

    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());

    // Copy out and display results
    printf("\tReduction results: ");
    int compare = CompareDeviceResults(h_reference, d_out, 1, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    printf("\tElapsed clocks: ");
    DisplayDeviceResults(d_elapsed, 1);

    // Cleanup
    if (h_in) delete[] h_in;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_elapsed) CubDebugExit(g_allocator.DeviceFree(d_elapsed));
}



/**
 * Test partial-tile reduction (specialized for insufficient resources)
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_DIM_X,
    int                     BLOCK_DIM_Y,
    int                     BLOCK_DIM_Z,
    typename                T,
    typename                ReductionOp>
void TestPartialTile(
    GenMode                 gen_mode,
    int                     num_items,
    ReductionOp             reduction_op,
    Int2Type<false>         sufficient_resources)
{}


/**
 *  Run battery of partial-tile tests for different numbers of effective threads and thread dimensions
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_DIM_X,
    int                     BLOCK_DIM_Y,
    int                     BLOCK_DIM_Z,
    typename                T,
    typename                ReductionOp>
void TestPartialTile(
    GenMode                 gen_mode,
    int                     num_items,
    ReductionOp             reduction_op)
{
    // Check size of smem storage for the target arch to make sure it will fit
    typedef BlockReduce<T, BLOCK_DIM_X, ALGORITHM, BLOCK_DIM_Y, BLOCK_DIM_Z, TEST_ARCH> BlockReduceT;

    enum 
    {
#if defined(SM100) || defined(SM110) || defined(SM130)
        sufficient_smem       = sizeof(typename BlockReduceT::TempStorage)  <= 16 * 1024,
        sufficient_threads    = (BLOCK_DIM_X * BLOCK_DIM_Y * BLOCK_DIM_Z)   <= 512,
#else
        sufficient_smem       = sizeof(typename BlockReduceT::TempStorage)  <= 48 * 1024,
        sufficient_threads    = (BLOCK_DIM_X * BLOCK_DIM_Y * BLOCK_DIM_Z)   <= 1024,
#endif
    };

    TestPartialTile<ALGORITHM, BLOCK_DIM_X, BLOCK_DIM_Y, BLOCK_DIM_Z, T>(gen_mode, num_items, reduction_op, Int2Type<sufficient_smem && sufficient_threads>());
}



/**
 *  Run battery of partial-tile tests for different numbers of effective threads and thread dimensions
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_THREADS,
    typename                T,
    typename                ReductionOp>
void TestPartialTile(
    GenMode                 gen_mode,
    ReductionOp             reduction_op)
{
    for (
        int num_items = 1;
        num_items < BLOCK_THREADS;
        num_items += CUB_MAX(1, BLOCK_THREADS / 5))
    {
        TestPartialTile<ALGORITHM, BLOCK_THREADS, 1, 1, T>(gen_mode, num_items, reduction_op);
        TestPartialTile<ALGORITHM, BLOCK_THREADS, 2, 2, T>(gen_mode, num_items, reduction_op);
    }
}



//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Run battery of full-tile tests for different gen modes
 */
template <
    BlockReduceAlgorithm    ALGORITHM,
    int                     BLOCK_THREADS,
    typename                T,
    typename                ReductionOp>
void Test(
    ReductionOp             reduction_op)
{
    TestFullTile<ALGORITHM, BLOCK_THREADS, T>(UNIFORM, reduction_op);
    TestPartialTile<ALGORITHM, BLOCK_THREADS, T>(UNIFORM, reduction_op);

    TestFullTile<ALGORITHM, BLOCK_THREADS, T>(INTEGER_SEED, reduction_op);
    TestPartialTile<ALGORITHM, BLOCK_THREADS, T>(INTEGER_SEED, reduction_op);

    if (Traits<T>::CATEGORY != FLOATING_POINT)
    {
        // Don't test randomly-generated floats b/c of stability
        TestFullTile<ALGORITHM, BLOCK_THREADS, T>(RANDOM, reduction_op);
        TestPartialTile<ALGORITHM, BLOCK_THREADS, T>(RANDOM, reduction_op);
    }
}


/**
 * Run battery of tests for different block-reduction algorithmic variants
 */
template <
    int             BLOCK_THREADS,
    typename        T,
    typename        ReductionOp>
void Test(
    ReductionOp     reduction_op)
{
  (void)reduction_op;
#ifdef TEST_RAKING
    Test<BLOCK_REDUCE_RAKING, BLOCK_THREADS, T>(reduction_op);
    Test<BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY, BLOCK_THREADS, T>(reduction_op);
#endif
#ifdef TEST_WARP_REDUCTIONS
    Test<BLOCK_REDUCE_WARP_REDUCTIONS, BLOCK_THREADS, T>(reduction_op);
#endif
}


/**
 * Run battery of tests for different block sizes
 */
template <
    typename        T,
    typename        ReductionOp>
void Test(
    ReductionOp     reduction_op)
{
    Test<7,   T>(reduction_op);
    Test<32,  T>(reduction_op);
    Test<63,  T>(reduction_op);
    Test<97,  T>(reduction_op);
    Test<128, T>(reduction_op);
    Test<238, T>(reduction_op);
}


/**
 * Run battery of tests for different block sizes
 */
template <typename T>
void Test()
{
    Test<T>(Sum());
    Test<T>(Max());
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("repeat", g_repeat);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--repeat=<repetitions of entire test suite>]"
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Get ptx version
    int ptx_version = 0;
    CubDebugExit(PtxVersion(ptx_version));

#ifdef QUICK_TEST

    // Compile/run quick tests


    printf("\n full tile ------------------------\n\n");

    TestFullTile<BLOCK_REDUCE_RAKING,                   128, 1, 1, 4, int>(RANDOM, 1, Sum());
    TestFullTile<BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,  128, 1, 1, 4, int>(RANDOM, 1, Sum());
    TestFullTile<BLOCK_REDUCE_WARP_REDUCTIONS,          128, 1, 1, 4, int>(RANDOM, 1, Sum());

    TestFullTile<BLOCK_REDUCE_RAKING,                   128, 1, 1, 1, int>(RANDOM, 1, Sum());
    TestFullTile<BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,  128, 1, 1, 1, int>(RANDOM, 1, Sum());
    TestFullTile<BLOCK_REDUCE_WARP_REDUCTIONS,          128, 1, 1, 1, int>(RANDOM, 1, Sum());

    printf("\n partial tile ------------------------\n\n");

    TestPartialTile<BLOCK_REDUCE_RAKING,                   128, 1, 1, int>(RANDOM, 7, Sum());
    TestPartialTile<BLOCK_REDUCE_RAKING_COMMUTATIVE_ONLY,  128, 1, 1, int>(RANDOM, 7, Sum());
    TestPartialTile<BLOCK_REDUCE_WARP_REDUCTIONS,          128, 1, 1, int>(RANDOM, 7, Sum());

#else

    // Compile/run thorough tests
    for (int i = 0; i <= g_repeat; ++i)
    {
        // primitives
        Test<char>();
        Test<short>();
        Test<int>();
        Test<long long>();
        if (ptx_version > 120)                          // Don't check doubles on PTX120 or below because they're down-converted
            Test<double>();

        Test<float>();

        // vector types
        Test<char2>();
        Test<short2>();
        Test<int2>();
        Test<longlong2>();

        Test<char4>();
        Test<short4>();
        Test<int4>();
        Test<longlong4>();

        // Complex types
        Test<TestFoo>();
        Test<TestBar>();
    }

#endif

    return 0;
}


