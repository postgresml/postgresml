#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Simple demonstration of hipcub::BlockRadixSort
 *
 * To compile using the command line:
 *   nvcc -arch=sm_XX example_block_radix_sort.cu -I../.. -lcudart -O3
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console (define before including cub.h)
#define HIPCUB_STDERR

#include <stdio.h>
#include <iostream>
#include <algorithm>

#include <cub/block/block_load.cuh>
#include <cub/block/block_store.cuh>
#include <cub/block/block_radix_sort.cuh>

#include "../../test/test_util.h"

using namespace hipcub;

//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

/// Verbose output
bool g_verbose = false;

/// Timing iterations
int g_timing_iterations = 100;

/// Default grid size
int g_grid_size = 1;

/// Uniform key samples
bool g_uniform_keys;


//---------------------------------------------------------------------
// Kernels
//---------------------------------------------------------------------

/**
 * Simple kernel for performing a block-wide sorting over integers
 */
template <
    typename    Key,
    int         BLOCK_THREADS,
    int         ITEMS_PER_THREAD>
__launch_bounds__ (BLOCK_THREADS)
__global__ void BlockSortKernel(
    Key         *d_in,          // Tile of input
    Key         *d_out,         // Tile of output
    clock_t     *d_elapsed)     // Elapsed cycle count of block scan
{
    enum { TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD };

    // Specialize BlockLoad type for our thread block (uses warp-striped loads for coalescing, then transposes in shared memory to a blocked arrangement)
    typedef BlockLoad<Key, BLOCK_THREADS, ITEMS_PER_THREAD, BLOCK_LOAD_WARP_TRANSPOSE> BlockLoadT;

    // Specialize BlockRadixSort type for our thread block
    typedef BlockRadixSort<Key, BLOCK_THREADS, ITEMS_PER_THREAD> BlockRadixSortT;

    // Shared memory
    __shared__ union TempStorage
    {
        typename BlockLoadT::TempStorage        load;
        typename BlockRadixSortT::TempStorage   sort;
    } temp_storage;

    // Per-thread tile items
    Key items[ITEMS_PER_THREAD];

    // Our current block's offset
    int block_offset = blockIdx.x * TILE_SIZE;

    // Load items into a blocked arrangement
    BlockLoadT(temp_storage.load).Load(d_in + block_offset, items);

    // Barrier for smem reuse
    __syncthreads();

    // Start cycle timer
    clock_t start = clock();

    // Sort keys
    BlockRadixSortT(temp_storage.sort).SortBlockedToStriped(items);

    // Stop cycle timer
    clock_t stop = clock();

    // Store output in striped fashion
    StoreDirectStriped<BLOCK_THREADS>(threadIdx.x, d_out + block_offset, items);

    // Store elapsed clocks
    if (threadIdx.x == 0)
    {
        d_elapsed[blockIdx.x] = (start > stop) ? start - stop : stop - start;
    }
}



//---------------------------------------------------------------------
// Host utilities
//---------------------------------------------------------------------


/**
 * Initialize sorting problem (and solution).
 */
template <typename Key>
void Initialize(
    Key *h_in,
    Key *h_reference,
    int num_items,
    int tile_size)
{
    for (int i = 0; i < num_items; ++i)
    {
        if (g_uniform_keys)
        {
            h_in[i] = 0;
        }
        else
        {
            RandomBits(h_in[i]);
        }
        h_reference[i] = h_in[i];
    }

    // Only sort the first tile
    std::sort(h_reference, h_reference + tile_size);
}


/**
 * Test BlockScan
 */
template <
    typename    Key,
    int         BLOCK_THREADS,
    int         ITEMS_PER_THREAD>
void Test()
{
    const int TILE_SIZE = BLOCK_THREADS * ITEMS_PER_THREAD;

    // Allocate host arrays
    Key *h_in               = new Key[TILE_SIZE * g_grid_size];
    Key *h_reference        = new Key[TILE_SIZE * g_grid_size];
    clock_t *h_elapsed      = new clock_t[g_grid_size];

    // Initialize problem and reference output on host
    Initialize(h_in, h_reference, TILE_SIZE * g_grid_size, TILE_SIZE);

    // Initialize device arrays
    Key *d_in       = NULL;
    Key *d_out      = NULL;
    clock_t *d_elapsed  = NULL;
    CubDebugExit(hipMalloc((void**)&d_in,          sizeof(Key) * TILE_SIZE * g_grid_size));
    CubDebugExit(hipMalloc((void**)&d_out,         sizeof(Key) * TILE_SIZE * g_grid_size));
    CubDebugExit(hipMalloc((void**)&d_elapsed,     sizeof(clock_t) * g_grid_size));

    // Display input problem data
    if (g_verbose)
    {
        printf("Input data: ");
        for (int i = 0; i < TILE_SIZE; i++)
            std::cout << h_in[i] << ", ";
        printf("\n\n");
    }

    // Kernel props
    int max_sm_occupancy;
    CubDebugExit(MaxSmOccupancy(max_sm_occupancy, BlockSortKernel<Key, BLOCK_THREADS, ITEMS_PER_THREAD>, BLOCK_THREADS));

    // Copy problem to device
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(Key) * TILE_SIZE * g_grid_size, hipMemcpyHostToDevice));

    printf("BlockRadixSort %d items (%d timing iterations, %d blocks, %d threads, %d items per thread, %d SM occupancy):\n",
        TILE_SIZE * g_grid_size, g_timing_iterations, g_grid_size, BLOCK_THREADS, ITEMS_PER_THREAD, max_sm_occupancy);
    fflush(stdout);

    // Run kernel once to prime caches and check result
    BlockSortKernel<Key, BLOCK_THREADS, ITEMS_PER_THREAD><<<g_grid_size, BLOCK_THREADS>>>(
        d_in,
        d_out,
        d_elapsed);

    // Check for kernel errors and STDIO from the kernel, if any
    CubDebugExit(hipPeekAtLastError());
    CubDebugExit(hipDeviceSynchronize());

    // Check results
    printf("\tOutput items: ");
    int compare = CompareDeviceResults(h_reference, d_out, TILE_SIZE, g_verbose, g_verbose);
    printf("%s\n", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);
    fflush(stdout);

    // Run this several times and average the performance results
    GpuTimer            timer;
    float               elapsed_millis          = 0.0;
    unsigned long long  elapsed_clocks          = 0;

    for (int i = 0; i < g_timing_iterations; ++i)
    {
        timer.Start();

        // Run kernel
        BlockSortKernel<Key, BLOCK_THREADS, ITEMS_PER_THREAD><<<g_grid_size, BLOCK_THREADS>>>(
            d_in,
            d_out,
            d_elapsed);

        timer.Stop();
        elapsed_millis += timer.ElapsedMillis();

        // Copy clocks from device
        CubDebugExit(hipMemcpy(h_elapsed, d_elapsed, sizeof(clock_t) * g_grid_size, hipMemcpyDeviceToHost));
        for (int i = 0; i < g_grid_size; i++)
            elapsed_clocks += h_elapsed[i];
    }

    // Check for kernel errors and STDIO from the kernel, if any
    CubDebugExit(hipDeviceSynchronize());

    // Display timing results
    float avg_millis            = elapsed_millis / g_timing_iterations;
    float avg_items_per_sec     = float(TILE_SIZE * g_grid_size) / avg_millis / 1000.0f;
    double avg_clocks           = double(elapsed_clocks) / g_timing_iterations / g_grid_size;
    double avg_clocks_per_item  = avg_clocks / TILE_SIZE;

    printf("\tAverage BlockRadixSort::SortBlocked clocks: %.3f\n", avg_clocks);
    printf("\tAverage BlockRadixSort::SortBlocked clocks per item: %.3f\n", avg_clocks_per_item);
    printf("\tAverage kernel millis: %.4f\n", avg_millis);
    printf("\tAverage million items / sec: %.4f\n", avg_items_per_sec);
    fflush(stdout);

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_reference) delete[] h_reference;
    if (h_elapsed) delete[] h_elapsed;
    if (d_in) CubDebugExit(hipFree(d_in));
    if (d_out) CubDebugExit(hipFree(d_out));
    if (d_elapsed) CubDebugExit(hipFree(d_elapsed));
}


/**
 * Main
 */
int main(int argc, char** argv)
{
    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    g_uniform_keys = args.CheckCmdLineFlag("uniform");
    args.GetCmdLineArgument("i", g_timing_iterations);
    args.GetCmdLineArgument("grid-size", g_grid_size);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--device=<device-id>] "
            "[--i=<timing iterations (default:%d)>]"
            "[--grid-size=<grid size (default:%d)>]"
            "[--v] "
            "\n", argv[0], g_timing_iterations, g_grid_size);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());
    fflush(stdout);

    // Run tests
    printf("\nuint32:\n"); fflush(stdout);
    Test<unsigned int, 128, 13>();
    printf("\n"); fflush(stdout);

    printf("\nfp32:\n"); fflush(stdout);
    Test<float, 128, 13>();
    printf("\n"); fflush(stdout);

    printf("\nuint8:\n"); fflush(stdout);
    Test<unsigned char, 128, 13>();
    printf("\n"); fflush(stdout);

    return 0;
}

