#include "hip/hip_runtime.h"
/******************************************************************************
 * Copyright (c) 2011, Duane Merrill.  All rights reserved.
 * Copyright (c) 2011-2018, NVIDIA CORPORATION.  All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *     * Redistributions of source code must retain the above copyright
 *       notice, this list of conditions and the following disclaimer.
 *     * Redistributions in binary form must reproduce the above copyright
 *       notice, this list of conditions and the following disclaimer in the
 *       documentation and/or other materials provided with the distribution.
 *     * Neither the name of the NVIDIA CORPORATION nor the
 *       names of its contributors may be used to endorse or promote products
 *       derived from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 * WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 * DISCLAIMED. IN NO EVENT SHALL NVIDIA CORPORATION BE LIABLE FOR ANY
 * DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 * (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 * LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
 * ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 * SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 *
 ******************************************************************************/

/******************************************************************************
 * Simple example of DeviceSelect::If().
 *
 * Selects items from from a sequence of int keys using a
 * section functor (greater-than)
 *
 * To compile using the command line:
 *   nvcc -arch=sm_XX example_device_select_if.cu -I../.. -lcudart -O3
 *
 ******************************************************************************/

// Ensure printing of CUDA runtime errors to console
#define HIPCUB_STDERR

#include <stdio.h>

#include <cub/util_allocator.cuh>
#include <cub/device/device_select.cuh>

#include "../../test/test_util.h"

using namespace hipcub;


//---------------------------------------------------------------------
// Globals, constants and typedefs
//---------------------------------------------------------------------

bool                    g_verbose = false;  // Whether to display input/output to console
CachingDeviceAllocator  g_allocator(true);  // Caching allocator for device memory


/// Selection functor type
struct GreaterThan
{
    int compare;

    __host__ __device__ __forceinline__
    GreaterThan(int compare) : compare(compare) {}

    __host__ __device__ __forceinline__
    bool operator()(const int &a) const {
        return (a > compare);
    }
};


//---------------------------------------------------------------------
// Test generation
//---------------------------------------------------------------------

/**
 * Initialize problem, setting runs of random length chosen from [1..max_segment]
 */
void Initialize(
    int     *h_in,
    int     num_items,
    int     max_segment)
{
    int key = 0;
    int i = 0;
    while (i < num_items)
    {
        // Randomly select number of repeating occurrences uniformly from [1..max_segment]
        unsigned short max_short = (unsigned short) -1;
        unsigned short repeat;
        RandomBits(repeat);
        repeat = (unsigned short) ((float(repeat) * (float(max_segment) / float(max_short))));
        repeat = CUB_MAX(1, repeat);

        int j = i;
        while (j < CUB_MIN(i + repeat, num_items))
        {
            h_in[j] = key;
            j++;
        }

        i = j;
        key++;
    }

    if (g_verbose)
    {
        printf("Input:\n");
        DisplayResults(h_in, num_items);
        printf("\n\n");
    }
}


/**
 * Solve unique problem
 */
template <typename SelectOp>
int Solve(
    int             *h_in,
    SelectOp        select_op,
    int             *h_reference,
    int             num_items)
{
    int num_selected = 0;
    for (int i = 0; i < num_items; ++i)
    {
        if (select_op(h_in[i]))
        {
            h_reference[num_selected] = h_in[i];
            num_selected++;
        }
        else
        {
            h_reference[num_items - (i - num_selected) - 1] = h_in[i];
        }
    }

    return num_selected;
}


//---------------------------------------------------------------------
// Main
//---------------------------------------------------------------------

/**
 * Main
 */
int main(int argc, char** argv)
{
    int num_items           = 150;
    int max_segment         = 40;       // Maximum segment length

    // Initialize command line
    CommandLineArgs args(argc, argv);
    g_verbose = args.CheckCmdLineFlag("v");
    args.GetCmdLineArgument("n", num_items);
    args.GetCmdLineArgument("maxseg", max_segment);

    // Print usage
    if (args.CheckCmdLineFlag("help"))
    {
        printf("%s "
            "[--n=<input items> "
            "[--device=<device-id>] "
            "[--maxseg=<max segment length>]"
            "[--v] "
            "\n", argv[0]);
        exit(0);
    }

    // Initialize device
    CubDebugExit(args.DeviceInit());

    // Allocate host arrays
    int *h_in        = new int[num_items];
    int *h_reference = new int[num_items];

    // Select a pivot index
    unsigned int pivot_index;
    unsigned int max_int = (unsigned int) -1;
    RandomBits(pivot_index);
    pivot_index = (unsigned int) ((float(pivot_index) * (float(num_items - 1) / float(max_int))));
    printf("Pivot idx: %d\n", pivot_index); fflush(stdout);

    // Initialize problem and solution
    Initialize(h_in, num_items, max_segment);
    GreaterThan select_op(h_in[pivot_index]);

    int num_selected = Solve(h_in, select_op, h_reference, num_items);

    printf("hipcub::DeviceSelect::If %d items, %d selected (avg run length %d), %d-byte elements\n",
        num_items, num_selected, (num_selected > 0) ? num_items / num_selected : 0, (int) sizeof(int));
    fflush(stdout);

    // Allocate problem device arrays
    int *d_in = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_in, sizeof(int) * num_items));

    // Initialize device input
    CubDebugExit(hipMemcpy(d_in, h_in, sizeof(int) * num_items, hipMemcpyHostToDevice));

    // Allocate device output array and num selected
    int     *d_out            = NULL;
    int     *d_num_selected_out   = NULL;
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_out, sizeof(int) * num_items));
    CubDebugExit(g_allocator.DeviceAllocate((void**)&d_num_selected_out, sizeof(int)));

    // Allocate temporary storage
    void            *d_temp_storage = NULL;
    size_t          temp_storage_bytes = 0;
    CubDebugExit(DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_in, d_out, d_num_selected_out, num_items, select_op));
    CubDebugExit(g_allocator.DeviceAllocate(&d_temp_storage, temp_storage_bytes));

    // Run
    CubDebugExit(DeviceSelect::If(d_temp_storage, temp_storage_bytes, d_in, d_out, d_num_selected_out, num_items, select_op));

    // Check for correctness (and display results, if specified)
    int compare = CompareDeviceResults(h_reference, d_out, num_selected, true, g_verbose);
    printf("\t Data %s ", compare ? "FAIL" : "PASS");
    compare = compare | CompareDeviceResults(&num_selected, d_num_selected_out, 1, true, g_verbose);
    printf("\t Count %s ", compare ? "FAIL" : "PASS");
    AssertEquals(0, compare);

    // Cleanup
    if (h_in) delete[] h_in;
    if (h_reference) delete[] h_reference;
    if (d_in) CubDebugExit(g_allocator.DeviceFree(d_in));
    if (d_out) CubDebugExit(g_allocator.DeviceFree(d_out));
    if (d_num_selected_out) CubDebugExit(g_allocator.DeviceFree(d_num_selected_out));
    if (d_temp_storage) CubDebugExit(g_allocator.DeviceFree(d_temp_storage));

    printf("\n\n");

    return 0;
}

